#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <aggregation/aggregation_amg_level.h>
#include <profile.h>
#include <matrix_analysis.h>

#ifdef _WIN32
#pragma warning (push)
#pragma warning (disable : 4244 4267 4521)
#endif
#ifdef _WIN32
#pragma warning (pop)
#endif

#include <basic_types.h>
#include <util.h>
#include <fstream>
#include <cutil.h>
#include <multiply.h>
#include <transpose.h>
#include <blas.h>
#include <string>
#include <string.h>
#include <iostream>
#include <algorithm>
#include <amgx_timer.h>

#include <amgx_types/util.h>

#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/transform.h>
#include <thrust/binary_search.h>
#include <thrust/unique.h>
#include <thrust/inner_product.h>

namespace amgx
{

namespace aggregation
{


// ----------------------
// Kernels
// ----------------------

template <typename IndexType, typename ValueType>
__global__
void set_to_one_kernel(IndexType start, IndexType end, IndexType *ind, ValueType *v)
{
    for (int tid = start + blockDim.x * blockIdx.x + threadIdx.x; tid < end; tid += gridDim.x * blockDim.x)
    {
        v[ind[tid]] = types::util<ValueType>::get_one();
    }
}

template <typename IndexType>
__global__
void renumberAggregatesKernel(const IndexType *renumbering, const int interior_offset, const int bdy_offset, IndexType *aggregates, const int num_aggregates, const int n_interior, const int renumbering_size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (tid < num_aggregates)
    {
        IndexType new_agg_id;

        if (renumbering_size == 0)
        {
            new_agg_id = aggregates[tid];
        }
        else
        {
            new_agg_id = renumbering[aggregates[tid]];
        }

        //if (aggregates[tid] > num_aggregates)
        //{
        //printf("ID %d old %d + %d = %d\n", tid, new_agg_id, ((new_agg_id >= n_interior) ? bdy_offset : interior_offset), new_agg_id + ((new_agg_id >= n_interior) ? bdy_offset : interior_offset));
        //}
        new_agg_id +=  ((new_agg_id >= n_interior) ? bdy_offset : interior_offset);
        aggregates[tid] = new_agg_id;
        tid += gridDim.x * blockDim.x;
    }
}

// Kernel to restrict residual using csr_format
template <typename IndexType, typename ValueType>
__global__
void restrictResidualKernel(const IndexType *row_offsets, const IndexType *column_indices, const ValueType *r, ValueType *rr, const int num_aggregates)
{
    int jmin, jmax;

    for (int tid = blockDim.x * blockIdx.x + threadIdx.x; tid < num_aggregates; tid += gridDim.x * blockDim.x)
    {
        ValueType temp(types::util<ValueType>::get_zero());
        jmin = row_offsets[tid];
        jmax = row_offsets[tid + 1];

        for (int j = jmin; j < jmax; j++)
        {
            int j_col = column_indices[j];
            temp = temp + r[j_col];
        }

        rr[tid] = temp;
    }
}

// Kernel to restrict residual using block_dia_csr_format
template <typename IndexType, typename ValueType, int bsize>
__global__
void restrictResidualBlockDiaCsrKernel(const IndexType *row_offsets, const IndexType *column_indices, const ValueType *r, ValueType *rr, const int num_aggregates)
{
    ValueType rr_temp[bsize];
    int offset, jmin, jmax;

    for (int tid = blockDim.x * blockIdx.x + threadIdx.x; tid < num_aggregates; tid += gridDim.x * blockDim.x)
    {
        // Initialize to zero
#pragma unroll
        for (int m = 0; m < bsize; m++)
        {
            rr_temp[m] = types::util<ValueType>::get_zero();
        }

        jmin = row_offsets[tid];
        jmax = row_offsets[tid + 1];

        for (int j = jmin; j < jmax; j++)
        {
            int jcol = column_indices[j];
            offset = jcol * bsize;
#pragma unroll

            for (int m = 0; m < bsize; m++)
            {
                rr_temp[m] = rr_temp[m] + r[offset + m];
            }
        }

        offset = tid * bsize;
#pragma unroll

        for (int m = 0; m < bsize; m++)
        {
            rr[offset + m] = rr_temp[m];
        };
    }
}

// Kernel to prolongate and apply the correction for csr format
template <typename IndexType, typename ValueType>
__global__
void prolongateAndApplyCorrectionKernel(const ValueType alpha, const int num_rows, ValueType *x, const ValueType *e, const IndexType *aggregates, IndexType num_aggregates)
{
    for (int tid = blockDim.x * blockIdx.x + threadIdx.x; tid < num_rows; tid += gridDim.x * blockDim.x)
    {
        IndexType I = aggregates[tid];
        x[tid] = x[tid] + alpha * e[I];
    }
}

// Kernel to prolongate and apply the correction for block-dia-csr format
template <typename IndexType, typename ValueType>
__global__
void prolongateAndApplyCorrectionBlockDiaCsrKernel(const ValueType alpha, const int num_block_rows, ValueType *x, const ValueType *e, const IndexType *aggregates, IndexType num_aggregates, const int bsize)
{
    for (int tid = blockDim.x * blockIdx.x + threadIdx.x; tid < num_block_rows; tid += gridDim.x * blockDim.x)
    {
        IndexType I = aggregates[tid];

        for (int  m = 0; m < bsize; m++)
        {
            x[tid * bsize + m] = x[tid * bsize + m] + alpha * e[I * bsize + m];
        }
    }
}

template <typename IndexType, typename ValueType>
__global__
void prolongateVector(const IndexType *aggregates, const ValueType *in, ValueType *out, IndexType fine_rows, IndexType coarse_rows, int blocksize)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < fine_rows * blocksize )
    {
        int i = tid / blocksize;
        int e = tid % blocksize;
        IndexType I = aggregates[i];
        out[tid] = in[ I * blocksize + e ];
        tid += gridDim.x * blockDim.x;
    }
}

template <typename IndexType, typename ValueType>
__global__
void applyCorrection(ValueType lambda, const ValueType *e, ValueType *x, IndexType numRows )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < numRows )
    {
        x[tid] = x[tid] + lambda * e[tid];
        tid += gridDim.x * blockDim.x;
    }
}

// -------------------------------
//  Methods
// ------------------------------

//  Constructor
template <class T_Config>
Aggregation_AMG_Level_Base<T_Config>::Aggregation_AMG_Level_Base(AMG_Class *amg, ThreadManager *tmng) : AMG_Level<T_Config>(amg, tmng)
{
    m_selector = SelectorFactory<T_Config>::allocate(*(amg->m_cfg), amg->m_cfg_scope);
    m_coarseAGenerator = CoarseAGeneratorFactory<T_Config>::allocate(*(amg->m_cfg), amg->m_cfg_scope);
    m_matrix_halo_exchange = amg->m_cfg->AMG_Config::getParameter<int>("matrix_halo_exchange", amg->m_cfg_scope);
    m_print_aggregation_info = amg->m_cfg->AMG_Config::getParameter<int>("print_aggregation_info", amg->m_cfg_scope) != 0;
    m_error_scaling = amg->m_cfg->AMG_Config::getParameter<int>("error_scaling", amg->m_cfg_scope );
    reuse_scale = amg->m_cfg->AMG_Config::getParameter<int>("reuse_scale", amg->m_cfg_scope );
    scaling_smoother_steps = amg->m_cfg->AMG_Config::getParameter<int>("scaling_smoother_steps", amg->m_cfg_scope );
    scale_counter = 0;
}

//  Destructor
template <class T_Config>
Aggregation_AMG_Level_Base<T_Config>::~Aggregation_AMG_Level_Base()
{
    delete m_selector;
    delete m_coarseAGenerator;
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::transfer_level(AMG_Level<TConfig1> *ref_lvl)
{
    Aggregation_AMG_Level_Base<TConfig1> *ref_agg_lvl = dynamic_cast<Aggregation_AMG_Level_Base<TConfig1>*>(ref_lvl);
    this->scale_counter = ref_agg_lvl->scale_counter;
    this->scale = ref_agg_lvl->scale;
    this->m_R_row_offsets.copy(ref_agg_lvl->m_R_row_offsets);
    this->m_R_column_indices.copy(ref_agg_lvl->m_R_column_indices);
    this->m_aggregates.copy(ref_agg_lvl->m_aggregates);
    this->m_aggregates_fine_idx.copy(ref_agg_lvl->m_aggregates_fine_idx);
    this->m_num_aggregates = ref_agg_lvl->m_num_aggregates;
    this->m_num_all_aggregates = ref_agg_lvl->m_num_all_aggregates;
}


typedef std::pair<int, int> mypair;
bool comparator ( const mypair &l, const mypair &r) { return l.first < r.first; }

// Method to compute R
// General path
// TODO: this could be merged with selector to save some computations
template <typename T_Config>
void Aggregation_AMG_Level_Base<T_Config>::computeRestrictionOperator_common()
{
    m_R_row_offsets.resize(m_num_all_aggregates + 1); //create one more row for the pseudo aggregate
    IVector R_row_indices(m_aggregates);
#if AMGX_ASYNCCPU_PROOF_OF_CONCEPT
    bool use_cpu = m_aggregates.size() < 4096;

    if (use_cpu)
    {
        struct computeRestrictionTask : public task
        {
            Aggregation_AMG_Level_Base<T_Config> *self;
            IVector *R_row_indices;

            void run()
            {
                int N = self->m_aggregates.size();
                IVector_h R_row_indices_host(self->m_aggregates);
                std::vector<mypair> pairs(N);

                for (int i = 0; i < N; i++)
                {
                    pairs[i].first = R_row_indices_host[i];
                    pairs[i].second = i;
                }

                std::stable_sort(pairs.begin(), pairs.end(), comparator);
                IVector_h R_column_indices(self->A->get_num_rows());

                for (int i = 0; i < N; i++)
                {
                    R_column_indices[i] = pairs[i].second;
                    R_row_indices_host[i] = pairs[i].first;
                }

                self->m_R_column_indices = R_column_indices;
                *R_row_indices = R_row_indices_host;
            }
        };
        computeRestrictionTask *t = new computeRestrictionTask();
        t->self = this;
        t->R_row_indices = &R_row_indices;
        t->run();
        delete t;
    }
    else
#endif
    {
        m_R_column_indices.resize(this->A->get_num_rows());
        thrust::sequence(m_R_column_indices.begin(), m_R_column_indices.end());
        cudaCheckError();
        thrust::sort_by_key(R_row_indices.begin(), R_row_indices.end(), m_R_column_indices.begin());
        cudaCheckError();
    }

    thrust::lower_bound(R_row_indices.begin(),
                        R_row_indices.end(),
                        thrust::counting_iterator<typename IVector::value_type>(0),
                        thrust::counting_iterator<typename IVector::value_type>(m_R_row_offsets.size()),
                        m_R_row_offsets.begin());
    cudaCheckError();
}


// two methods below could be merged
// Method to compute R on HOST using csr format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::computeRestrictionOperator_1x1()
{
    this->m_R_row_offsets.resize(this->m_num_all_aggregates + 1);
    this->m_R_column_indices.resize(this->A->get_num_rows());
    this->fillRowOffsetsAndColIndices(this->A->get_num_rows());
}

// Method to compute R on HOST using block dia-csr format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::computeRestrictionOperator_4x4()
{
    this->m_R_row_offsets.resize(this->m_num_all_aggregates + 1);
    this->m_R_column_indices.resize(this->A->get_num_rows());
    this->fillRowOffsetsAndColIndices(this->A->get_num_rows());
}

// Method to create R_row_offsest and R_column_indices array on HOST using csr or block dia-csr format
template <typename T_Config>
void Aggregation_AMG_Level_Base<T_Config>::fillRowOffsetsAndColIndices(const int R_num_cols)
{
    for (int i = 0; i < m_num_all_aggregates + 1; i++)
    {
        m_R_row_offsets[i] = 0;
    }

    // Count number of neighbors for each row
    for (int i = 0; i < R_num_cols; i++)
    {
        int I = m_aggregates[i];
        m_R_row_offsets[I]++;
    }

    m_R_row_offsets[m_num_all_aggregates] = R_num_cols;

    for (int i = m_num_all_aggregates - 1; i >= 0; i--)
    {
        m_R_row_offsets[i] = m_R_row_offsets[i + 1] - m_R_row_offsets[i];
    }

    /* Set column indices. */
    for (int i = 0; i < R_num_cols; i++)
    {
        int I = m_aggregates[i];
        int Ip = m_R_row_offsets[I]++;
        m_R_column_indices[Ip] = i;
    }

    /* Reset r[i] to start of row memory. */
    for (int i = m_num_all_aggregates - 1; i > 0; i--)
    {
        m_R_row_offsets[i] = m_R_row_offsets[i - 1];
    }

    m_R_row_offsets[0] = 0;
}

// Method to compute R on DEVICE using block dia-csr format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::computeRestrictionOperator_4x4()
{
    this->computeRestrictionOperator_common();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::computeRestrictionOperator_1x1()
{
    this->computeRestrictionOperator_common();
}

// Method to restrict Residual on host using csr_matrix format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::restrictResidual_1x1(const VVector &r, VVector &rr)
{
    ValueTypeB temp;

    for (int i = 0; i < this->m_num_aggregates; i++)
    {
        temp = types::util<ValueTypeB>::get_zero();

        for (int j = this->m_R_row_offsets[i]; j < this->m_R_row_offsets[i + 1]; j++)
        {
            int j_col = this->m_R_column_indices[j];
            temp = temp + r[j_col];
        }

        rr[i] = temp;
    }
}

// Method to restrict Residual on host using block_dia_csr_matrix format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::restrictResidual_4x4(const VVector &r, VVector &rr)
{
    IndexType bsize = this->A->get_block_dimy();
    ValueTypeB *temp = new ValueTypeB[bsize];

    for (int i = 0; i < this->m_num_aggregates; i++)
    {
        // Initialize temp to 0
        for (int k = 0; k < bsize; k++)
        {
            temp[k]  =  types::util<ValueTypeB>::get_zero();
        }

        // Add contributions from each fine point
        for (int j = this->m_R_row_offsets[i]; j < this->m_R_row_offsets[i + 1]; j++)
        {
            int j_col = this->m_R_column_indices[j];

            for (int k = 0; k < bsize; k++)
            {
                temp[k] = temp[k] + r[j_col * bsize + k];
            }
        }

        // Store result
        for (int k = 0; k < bsize; k++)
        {
            rr[i * bsize + k] = temp[k];
        }
    }
}

// Method to restrict Residual on device using csr_matrix format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::restrictResidual_1x1(const VVector &r, VVector &rr)
{
    AMGX_CPU_PROFILER("Aggregation_AMG_Level::restrict_residual_1x1 ");
    int block_size = 64;
    int max_threads;;

    if (!this->isConsolidationLevel())
    {
        max_threads = this->m_num_aggregates;
    }
    else
    {
        max_threads = this->m_num_all_aggregates;
    }

    int num_blocks = min( AMGX_GRID_MAX_SIZE, (max_threads - 1) / block_size + 1);
    const IndexType *R_row_offsets_ptr = this->m_R_row_offsets.raw();
    const IndexType *R_column_indices_ptr = this->m_R_column_indices.raw();
    const ValueTypeB *r_ptr = r.raw();
    ValueTypeB *rr_ptr = rr.raw();
    restrictResidualKernel <<< num_blocks, block_size>>>(R_row_offsets_ptr, R_column_indices_ptr, r_ptr, rr_ptr, max_threads);
    cudaCheckError();
}

// Method to restrict Residual on device using block_dia_csr_matrix format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::restrictResidual_4x4(const VVector &r, VVector &rr)
{
    AMGX_CPU_PROFILER("Aggregation_AMG_Level::restrict_residual_4x4 ");
    int block_size = 64;
    int max_threads;

    if (!this->isConsolidationLevel())
    {
        max_threads = this->m_num_aggregates;
    }
    else
    {
        max_threads = this->m_num_all_aggregates;
    };

    const int num_blocks = min( AMGX_GRID_MAX_SIZE, (max_threads + block_size - 1) / block_size);

    const IndexType *R_row_offsets_ptr = this->m_R_row_offsets.raw();

    const IndexType *R_column_indices_ptr = this->m_R_column_indices.raw();

    const ValueTypeB *r_ptr = r.raw();

    ValueTypeB *rr_ptr = rr.raw();

    cudaCheckError();

    switch ( this->getA().get_block_dimy() )
    {
        case 2:
            restrictResidualBlockDiaCsrKernel<IndexType, ValueTypeB, 2> <<< num_blocks, block_size>>>(R_row_offsets_ptr, R_column_indices_ptr, r_ptr, rr_ptr, max_threads);
            break;

        case 3:
            restrictResidualBlockDiaCsrKernel<IndexType, ValueTypeB, 3> <<< num_blocks, block_size>>>(R_row_offsets_ptr, R_column_indices_ptr, r_ptr, rr_ptr, max_threads);
            break;

        case 4:
            restrictResidualBlockDiaCsrKernel<IndexType, ValueTypeB, 4> <<< num_blocks, block_size>>>(R_row_offsets_ptr, R_column_indices_ptr, r_ptr, rr_ptr, max_threads);
            break;

        case 5:
            restrictResidualBlockDiaCsrKernel<IndexType, ValueTypeB, 5> <<< num_blocks, block_size>>>(R_row_offsets_ptr, R_column_indices_ptr, r_ptr, rr_ptr, max_threads);
            break;

        case 8:
            restrictResidualBlockDiaCsrKernel<IndexType, ValueTypeB, 8> <<< num_blocks, block_size>>>(R_row_offsets_ptr, R_column_indices_ptr, r_ptr, rr_ptr, max_threads);
            break;

        case 10:
            restrictResidualBlockDiaCsrKernel<IndexType, ValueTypeB, 10> <<< num_blocks, block_size>>>(R_row_offsets_ptr, R_column_indices_ptr, r_ptr, rr_ptr, max_threads);
            break;

        default:
            FatalError( "Unsupported block size in restrictResidual_4x4!!!", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE );
    }

    cudaCheckError();
}

__inline__ float getAlpha(float &nom, float &denom)
{
    float alpha;

    if (nom * denom <= 0. || std::abs(nom) < std::abs(denom))
    {
        alpha = 1.;
    }
    else if (std::abs(nom) > 2.*std::abs(denom))
    {
        alpha = 2.;
    }
    else
    {
        alpha = nom / denom;
    }

    return alpha;
}

__inline__ double getAlpha(double &nom, double &denom)
{
    double alpha;

    if (nom * denom <= 0. || std::abs(nom) < std::abs(denom))
    {
        alpha = 1.;
    }
    else if (std::abs(nom) > 2.*std::abs(denom))
    {
        alpha = 2.;
    }
    else
    {
        alpha = nom / denom;
    }

    return alpha;
}

__inline__ hipComplex getAlpha(hipComplex &nom, hipComplex &denom)
{
    hipComplex alpha;

    if (types::util<hipComplex>::abs(nom) < types::util<hipComplex>::abs(denom))
    {
        alpha = make_hipComplex(1.f, 0.f);
    }
    else if (types::util<hipComplex>::abs(nom) > 2.*types::util<hipComplex>::abs(denom))
    {
        alpha = make_hipComplex(2.f, 0.f);
    }
    else
    {
        alpha = nom / denom;
    }

    return alpha;
}

__inline__ hipDoubleComplex getAlpha(hipDoubleComplex &nom, hipDoubleComplex &denom)
{
    hipDoubleComplex alpha;

    if (types::util<hipDoubleComplex>::abs(nom) < types::util<hipDoubleComplex>::abs(denom))
    {
        alpha = make_hipDoubleComplex(1., 0.);
    }
    else if (types::util<hipDoubleComplex>::abs(nom) > 2.*types::util<hipDoubleComplex>::abs(denom))
    {
        alpha = make_hipDoubleComplex(2., 0.);
    }
    else
    {
        alpha = nom / denom;
    }

    return alpha;
}

template< class T_Config>
typename T_Config::VecPrec Aggregation_AMG_Level_Base<T_Config>::computeAlpha(const Vector<T_Config> &e, const Vector<T_Config> &bc, const Vector<T_Config> &tmp)
{
    typename T_Config::VecPrec alpha =  types::util<ValueTypeB>::get_one();
    Matrix<TConfig> &Ac = this->getNextLevel( MemorySpace( ) )->getA();
    int size = Ac.get_num_rows();
    VVector v(2,  types::util<ValueTypeB>::get_zero());
    v[0] = thrust::inner_product(e.begin(), e.begin() + size, bc.begin(),  types::util<ValueTypeB>::get_zero());
    v[1] = thrust::inner_product(e.begin(), e.begin() + size, tmp.begin(),  types::util<ValueTypeB>::get_zero());
    cudaCheckError();
    return getAlpha(v[0], v[1]);
}

// Method to prolongate the error on HOST using csr format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec>  >::prolongateAndApplyCorrection_1x1(Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &e, Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &bc, Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &x, Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &tmp)
{
    Matrix<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &A = this->getA();
    Matrix<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &C = this->next_h->getA();

    if ( this->m_error_scaling >= 2 )
    {
        FatalError("error_scaling=2,3 is not implemented on host", AMGX_ERR_NOT_IMPLEMENTED );
    }

    ValueTypeB alpha = types::util<ValueTypeB>::get_one();

    if (this->m_error_scaling)
    {
        multiply(this->next_h->getA(), e, tmp);
        alpha = this->computeAlpha (e, bc, tmp);
    }

    // Apply correction on all (interior and exterior) equations.
    for (int i = 0; i < A.get_num_cols(); i++)
    {
        int I = this->m_aggregates[i];
        x[i] = x[i] + alpha * e[I];
    }
}

// Method to prolongate the error on HOST using block_dia_csr format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::prolongateAndApplyCorrection_4x4(Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &e, Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &bc, Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &x, Vector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> > &tmp)
{
    if (this->A->get_block_dimy() != this->A->get_block_dimx())
    {
        FatalError("Aggregation_AMG_Level not implemented for non square blocks, exiting", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
    }

    if ( this->m_error_scaling >= 2 )
    {
        FatalError("error_scaling=2,3 is not implemented on host", AMGX_ERR_NOT_IMPLEMENTED );
    }

    Matrix<TConfig> &C = this->next_h->getA();
    ValueTypeB alpha = types::util<ValueTypeB>::get_one();

    if (this->m_error_scaling)
    {
        multiply(this->next_h->getA(), e, tmp);
        alpha = this->computeAlpha (e, bc, tmp);
    }

    // Apply correction on all equations.
    for (int i = 0; i < this->A->get_num_rows(); i++)
    {
        int I = this->m_aggregates[i];

        for (int k = 0; k < this->A->get_block_dimy(); k++)
        {
            x[i * this->A->get_block_dimy() + k] =  x[i * this->A->get_block_dimy() + k] + alpha * e[I * this->A->get_block_dimy() + k];
        }
    }
}

// Prolongate the error on DEVICE using csr format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::prolongateAndApplyCorrection_1x1(Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &e, Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &bc, Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &x, Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &tmp)
{
    AMGX_CPU_PROFILER("Aggregation_AMG_Level::prolongate_and_apply_correction_1x1 ");
    ValueTypeB alpha = types::util<ValueTypeB>::get_one();
    const int block_size = 64;
    const int num_blocks = min( AMGX_GRID_MAX_SIZE, (int) ( (this->A->get_num_rows() + block_size - 1) / block_size ) );
    const IndexType *aggregates_ptr = this->m_aggregates.raw();
    ValueTypeB *x_ptr = x.raw();
    const ValueTypeB *e_ptr = e.raw();

    if (this->m_error_scaling)
    {
        FatalError("error_scaling=1 is deprecated", AMGX_ERR_NOT_IMPLEMENTED );
    }

    prolongateAndApplyCorrectionKernel <<< num_blocks, block_size>>>(alpha, (int)this->A->get_num_rows(), x_ptr, e_ptr, aggregates_ptr, this->m_num_aggregates);
    cudaCheckError();
}

// Prolongate the error on DEVICE using block dia-csr format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void Aggregation_AMG_Level<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::prolongateAndApplyCorrection_4x4(Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &ec,
        Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &bf,
        Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &xf,
        Vector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> > &rf)
{
    AMGX_CPU_PROFILER("Aggregation_AMG_Level::prolongate_and_apply_correction_4x4 ");

    if ( this->m_error_scaling >= 2 )
    {
        if ( this->scale_counter > 0 )
        {
            const IndexType *aggregates_ptr = this->m_aggregates.raw();
            ValueTypeB *x_ptr = xf.raw();
            const ValueTypeB *e_ptr = ec.raw();
            const int block_size = 64;
            const int num_blocks = min( AMGX_GRID_MAX_SIZE, (int) ((this->A->get_num_rows() - 1) / block_size + 1));
            prolongateAndApplyCorrectionBlockDiaCsrKernel <<< num_blocks, block_size>>>(this->scale, (int)this->getA().get_num_rows(), x_ptr, e_ptr, aggregates_ptr, this->m_num_aggregates, this->getA().get_block_dimy());
            cudaCheckError();
            this->scale_counter--;
            return;
        }

        bool vanek_scaling = this->m_error_scaling > 3;
        IndexType numRowsCoarse = this->next_d->getA().get_num_rows();
        IndexType numRowsFine = this->A->get_num_rows();
        IndexType blockdim = this->A->get_block_dimx();

        if ( blockdim != this->A->get_block_dimy() )
        {
            FatalError("Unsupported dimension for aggregation amg level", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
        }

        VVector ef( rf.size() );
        VVector Aef( rf.size() );
        ef.set_block_dimy( blockdim );
        Aef.set_block_dimy( blockdim );
        // prolongate e
        const int threads_per_block = 256;
        const int num_block_values = min( AMGX_GRID_MAX_SIZE, (numRowsFine * blockdim - 1) / threads_per_block + 1);
        const hipStream_t stream = nullptr;
        prolongateVector <<< num_block_values, threads_per_block, 0, stream>>>( this->m_aggregates.raw(), ec.raw(), ef.raw(), numRowsFine, numRowsCoarse, blockdim );
        ef.dirtybit = 1;
        hipStreamSynchronize(stream);
        cudaCheckError();
        int preSmooth;

        if ( vanek_scaling )
        {
            preSmooth = this->amg->getNumPostsweeps();
        }
        else
        {
            preSmooth = this->scaling_smoother_steps;
        }

        //smooth error
        this->smoother->setTolerance( 0.0 );
        this->smoother->set_max_iters( preSmooth );

        if ( vanek_scaling )
        {
            thrust::fill( Aef.begin(), Aef.end(), types::util<ValueTypeB>::get_zero() );
            cudaCheckError();
            this->smoother->solve( Aef, ef, false ); //smooth correction with rhs 0
            this->smoother->solve( bf, xf, false ); // smooth x with rhs residual
            //recompute residual
            int offset, size;
            this->getA().getOffsetAndSizeForView(OWNED, &offset, &size);
            axmb( this->getA(), xf, bf, rf, offset, size );
        }
        else
        {
            this->smoother->solve( rf, ef, false ); //smooth correction with rhs residual
        }

        // multiply for lambda computation
        multiply(this->getA(), ef, Aef, OWNED);
        ValueTypeB nominator, denominator;
        int offset = 0, size = 0;
        this->A->getOffsetAndSizeForView(OWNED, &offset, &size);

        if ( this->m_error_scaling == 2 || this->m_error_scaling == 4 )
        {
            // compute lambda=<rf,Aef>/<Aef,Aef>
            nominator = thrust::inner_product( rf.begin(), rf.end(), Aef.begin(), types::util<ValueTypeB>::get_zero() );
            denominator = thrust::inner_product( Aef.begin(), Aef.end(), Aef.begin(), types::util<ValueTypeB>::get_zero() );
            cudaCheckError();
        }

        if ( this->m_error_scaling == 3 || this->m_error_scaling == 5)
        {
            // compute lambda=<rf,ef>/<ef,Aef>
            nominator = thrust::inner_product( rf.begin(), rf.begin() + size * blockdim, ef.begin(), types::util<ValueTypeB>::get_zero() );
            denominator = thrust::inner_product( ef.begin(), ef.begin() + size * blockdim, Aef.begin(), types::util<ValueTypeB>::get_zero() );

            if (!this->A->is_matrix_singleGPU())
            {
                this->A->getManager()->global_reduce_sum(&nominator);
                this->A->getManager()->global_reduce_sum(&denominator);
            }

            cudaCheckError();
        }

        if (types::util<ValueTypeB>::abs(denominator) == 0.0)
        {
            nominator = denominator = types::util<ValueTypeB>::get_one();
        }

        // apply correction x <- x + lambda*e
        const int num_block_fine = min( AMGX_GRID_MAX_SIZE, (numRowsFine * blockdim - 1) / threads_per_block + 1 );
        ValueTypeB alpha = nominator / denominator;

        if ( types::util<ValueTypeB>::abs(alpha) < .3 )
        {
            alpha = (alpha / types::util<ValueTypeB>::abs(alpha)) * .3;    // it was this before: alpha = .3, which is not 100% equal
        }

        if ( types::util<ValueTypeB>::abs(alpha) > 10 )
        {
            alpha = (alpha / types::util<ValueTypeB>::abs(alpha)) * 10.;    // it was this before: alpha = 10., which is not 100% equal
        }

        applyCorrection <<< num_block_fine, threads_per_block, 0, stream>>>( alpha, ef.raw(), xf.raw(), numRowsFine * blockdim );
        cudaCheckError();
        this->scale_counter = this->reuse_scale; //reuse this scale scale_counter times
        this->scale = alpha;
        return;
    }

    ValueTypeB alpha = types::util<ValueTypeB>::get_one();
    const int block_size = 64;
    const int num_blocks = min( AMGX_GRID_MAX_SIZE, (int) ((this->A->get_num_rows() - 1) / block_size + 1));
    const IndexType *aggregates_ptr = this->m_aggregates.raw();
    ValueTypeB *x_ptr = xf.raw();
    const ValueTypeB *e_ptr = ec.raw();

    if (this->m_error_scaling == 1)
    {
        FatalError("error_scaling=1 is deprecated", AMGX_ERR_NOT_IMPLEMENTED );
    }

    prolongateAndApplyCorrectionBlockDiaCsrKernel <<< num_blocks, block_size>>>(alpha, (int)this->A->get_num_rows(), x_ptr, e_ptr, aggregates_ptr, this->m_num_aggregates, this->A->get_block_dimy());
    cudaCheckError();
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config >::prolongateAndApplyCorrection(VVector &e, VVector &bf, VVector &x, VVector &tmp)
{
    Matrix<TConfig> &Ac = this->getNextLevel( MemorySpace( ) )->getA();

    //this is dirty, but error scaling 2 and 3 do not have a specialized version. Instead, the general version sits in the 4x4 function
    if ( this->m_error_scaling >= 2 )
    {
        prolongateAndApplyCorrection_4x4(e, bf, x, tmp);
    }
    else if (this->A->get_block_size() == 1)
    {
        prolongateAndApplyCorrection_1x1(e, bf, x, tmp);
    }
    else if (this->A->get_block_dimx() == this->A->get_block_dimy() )
    {
        prolongateAndApplyCorrection_4x4(e, bf, x, tmp);
    }
    else
    {
        FatalError("Unsupported dimension for aggregation amg level", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
    }

    x.dirtybit = 1;

    if (!this->A->is_matrix_singleGPU() && x.delayed_send == 0)
    {
        if (x.in_transfer & RECEIVING) { this->A->manager->exchange_halo_wait(x, x.tag); }

        this->A->manager->exchange_halo_async(x, x.tag);
    }
}


template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::restrictResidual(VVector &r, VVector &rr)
{
    if (this->A->get_block_size() == 1)
    {
        restrictResidual_1x1(r, rr);
    }
    else if (this->A->get_block_dimx() == this->A->get_block_dimy() )
    {
        restrictResidual_4x4(r, rr);
    }
    else
    {
        FatalError("Unsupported dimension for aggregation amg level", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
    }

    //TODO: check level transfer between host and device for multiGPU
    if (!this->A->is_matrix_singleGPU())
    {
        Matrix<TConfig> &Ac = this->getNextLevel( MemorySpace( ) )->getA();
        rr.dirtybit = 1;

        if (!Ac.is_matrix_singleGPU() && !this->isConsolidationLevel() && rr.delayed_send == 0)
        {
            Matrix<TConfig> &Ac = this->getNextLevel( MemorySpace( ) )->getA(); //TODO problem in memoryspace transfer is here

            if (rr.in_transfer & RECEIVING) { Ac.manager->exchange_halo_wait(rr, rr.tag); }

            Ac.manager->exchange_halo_async(rr, rr.tag);
        }
    }
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::computeRestrictionOperator()
{
    if (this->A->get_block_size() == 1)
    {
        computeRestrictionOperator_1x1();
    }
    else if (this->A->get_block_dimx() == 4 && this->A->get_block_dimy() == 4)
    {
        computeRestrictionOperator_4x4();
    }
    else
    {
        this->computeRestrictionOperator_common();
    }
}

template <typename IndexType>
__global__ void coarse_to_global(IndexType *aggregates, IndexType *aggregates_global, IndexType *renumbering, IndexType num_elements, int64_t offset)
{
    int element = blockIdx.x * blockDim.x + threadIdx.x;

    while (element < num_elements)
    {
        renumbering[aggregates[element]] = aggregates_global[element] + offset; //this won't be a problem, because we are overwriting the same thing
        element += blockDim.x * gridDim.x;
    }
}

template <typename T, typename IndexType>
__global__ void export_matrix_elements(IndexType *row_offsets, IndexType *col_indices, T *values, IndexType *maps, IndexType *renumbering, IndexType *new_row_offsets, IndexType *new_col_indices, T *new_values, IndexType bsize, IndexType size)
{
    int idx = blockIdx.x * blockDim.x / 32 + threadIdx.x / 32;
    int coopIdx = threadIdx.x % 32;

    while (idx < size)
    {
        int row = maps[idx];
        INDEX_TYPE src_base = row_offsets[row];
        INDEX_TYPE dst_base = new_row_offsets[idx];

        for (int m = coopIdx; m < row_offsets[row + 1]*bsize - src_base * bsize; m += 32)
        {
            new_values[dst_base * bsize + m] = values[src_base * bsize + m];
        }

        for (int m = coopIdx; m < row_offsets[row + 1] - src_base; m += 32)
        {
            new_col_indices[dst_base + m] = renumbering[col_indices[src_base + m]];
        }

        idx += gridDim.x * blockDim.x / 32;
    }
}

template <class T>
__global__ void export_matrix_diagonal(T *values, INDEX_TYPE bsize, INDEX_TYPE *maps, T *output, INDEX_TYPE size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while (idx < size)
    {
        int row = maps[idx];
        INDEX_TYPE src_base = row;
        INDEX_TYPE dst_base = idx;

        for (int m = 0; m < bsize; m++)
        {
            output[dst_base * bsize + m] = values[src_base * bsize + m];
        }

        idx += gridDim.x * blockDim.x;
    }
}

__global__ void remove_boundary(INDEX_TYPE *flags, INDEX_TYPE *maps, INDEX_TYPE size)
{
    int element = blockIdx.x * blockDim.x + threadIdx.x;

    while (element < size)
    {
        flags[maps[element]] = 0; //this won't be a problem, because we are overwriting the same thing
        element += blockDim.x * gridDim.x;
    }
}

__global__ void calc_inverse_renumbering(INDEX_TYPE *renum, INDEX_TYPE *irenum, INDEX_TYPE *renum_gbl, INDEX_TYPE base_index, INDEX_TYPE max_element)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < max_element)
    {
        irenum[renum[idx]] = renum_gbl[idx] - base_index;
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void create_halo_mapping(INDEX_TYPE *mapping, INDEX_TYPE *node_list, INDEX_TYPE base_index, INDEX_TYPE map_offset, INDEX_TYPE size)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    while (row < size)
    {
        int idx = node_list[row] - base_index;
        mapping[idx] = map_offset + row;
        row += blockDim.x * gridDim.x;
    }
}

__global__ void map_col_indices_and_count_rowlen(INDEX_TYPE *row_offsets, INDEX_TYPE *col_indices, INDEX_TYPE *row_length, INDEX_TYPE *renumbering, INDEX_TYPE *mapping, INDEX_TYPE *map_offsets, int64_t *index_ranges, INDEX_TYPE part_id, INDEX_TYPE my_id, INDEX_TYPE base_index, INDEX_TYPE my_range, INDEX_TYPE num_neighbors, INDEX_TYPE num_rows)
{
    extern __shared__ volatile int reduction[];
    int row = blockIdx.x * blockDim.x / 4 + threadIdx.x / 4;
    int coopIdx = threadIdx.x % 4;

    while (row < num_rows)
    {
        int valid = 0;

        for (int idx = row_offsets[row] + coopIdx; idx < row_offsets[row + 1]; idx += 4) //this may look horrible, but I expect low branch divergence, because col indices in a row usually belong to the same partition (or at most one more)
        {
            int colIdx = col_indices[idx];
            int part = -2;

            if (colIdx >= index_ranges[2 * part_id] && colIdx < index_ranges[2 * part_id + 1]) //the col index probably belongs to the partition I am working on
            {
                part = part_id;
            }
            else if (colIdx >= base_index && colIdx < base_index + my_range)     //or points back to the owned partition
            {
                part = -1;
            }
            else        //or else it points to a third partition
            {
                for (int i = 0; i < num_neighbors; i++)
                {
                    if (colIdx >= index_ranges[2 * i] && colIdx < index_ranges[2 * i + 1])
                    {
                        part = i;
                    }
                }
            }

            if (part == -2)
            {
                col_indices[idx] = -1;
#ifdef DEBUG
                printf("Column index encountered that does not belong to any of my neighbors!! %d\n", colIdx);
#endif
            }
            else
            {
                if (part == -1)
                {
                    col_indices[idx] = renumbering[colIdx - base_index];
                    valid++;
                }
                else
                {
                    int new_col_idx = mapping[map_offsets[part] + colIdx - index_ranges[2 * part]];

                    if (new_col_idx >= 0)
                    {
                        valid++;
                        col_indices[idx] = new_col_idx;
                    }
                    else
                    {
                        col_indices[idx] = -1;
                    }
                }
            }
        }

        reduction[threadIdx.x] = valid;

        for (int s = 2; s > 0; s >>= 1)
        {
            if (coopIdx < s)
            {
                reduction[threadIdx.x] += reduction[threadIdx.x + s];
            }

            __syncthreads();
        }

        if (coopIdx == 0)
        {
            row_length[row] = reduction[threadIdx.x];
        }

        row += gridDim.x * blockDim.x / 4;
    }
}

__global__ void map_col_indices(INDEX_TYPE *row_offsets, INDEX_TYPE *col_indices, int64_t *halo_ranges, INDEX_TYPE *halo_renumbering, INDEX_TYPE *halo_rows, INDEX_TYPE *global_renumbering, INDEX_TYPE num_neighbors, INDEX_TYPE num_rows, INDEX_TYPE num_rows_processed)
{
    int row = blockIdx.x * blockDim.x / 4 + threadIdx.x / 4;
    int coopIdx = threadIdx.x % 4;

    while (row < num_rows_processed)
    {
        for (int idx = row_offsets[row] + coopIdx; idx < row_offsets[row + 1]; idx += 4)
        {
            int colIdx = col_indices[idx];
            int part = 0;

            if (colIdx < num_rows)
            {
                part = -1;
            }
            else
            {
                colIdx = global_renumbering[colIdx];

                for (int i = 0; i < num_neighbors; i++)
                {
                    if (colIdx >= halo_ranges[2 * i] && colIdx < halo_ranges[2 * i + 1])
                    {
                        part = i;
                        break;
                    }
                }
            }

            if (part == -1)
            {
                col_indices[idx] = colIdx;
            }
            else
            {
                col_indices[idx] = halo_renumbering[halo_rows[part] + colIdx - halo_ranges[2 * part]];
            }
        }

        row += gridDim.x * blockDim.x / 4;
    }
}

template <class T>
__global__ void reorder_whole_matrix(INDEX_TYPE *old_rows, INDEX_TYPE *old_cols, T *old_vals, INDEX_TYPE *rows, INDEX_TYPE *cols, T *vals, INDEX_TYPE bsize, INDEX_TYPE num_rows)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    while (row < num_rows)
    {
        INDEX_TYPE dst_row = row;
        INDEX_TYPE src_base = old_rows[row];
        INDEX_TYPE dst = rows[dst_row];

        for (int i = 0; i < old_rows[row + 1] - src_base; i++)
        {
            INDEX_TYPE colIdx = old_cols[src_base + i];

            if (colIdx >= 0)
            {
                cols[dst] = colIdx;

                for (int j = 0; j < bsize; j++) { vals[dst * bsize + j] = old_vals[(src_base + i) * bsize + j]; }

                dst++;
            }
        }

        row += blockDim.x * gridDim.x;
    }
}

__global__ void calc_gbl_renumbering(INDEX_TYPE *inv_renum, INDEX_TYPE *gbl_renum, INDEX_TYPE size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        gbl_renum[inv_renum[idx]] = idx;
        idx += blockDim.x * gridDim.x;
    }
}

template <typename ValueType>
__global__ void write_diagonals(ValueType *values, INDEX_TYPE *diag, INDEX_TYPE *map, ValueType *output, INDEX_TYPE bsize, INDEX_TYPE size)
{
    int nzPerBlock = blockDim.x / bsize;
    int row = blockIdx.x * nzPerBlock + threadIdx.x / bsize;
    int vecIdx = threadIdx.x % bsize;

    if (threadIdx.x >= (blockDim.x / bsize)*bsize) { return; }

    while (row < size)
    {
        output[row * bsize + vecIdx] = values[diag[map[row]] * bsize + vecIdx];
        row += gridDim.x * nzPerBlock;
    }
}

template <typename ValueType>
__global__ void write_diagonals_back(ValueType *values, INDEX_TYPE *diag, ValueType *source, INDEX_TYPE bsize, INDEX_TYPE size)
{
    int nzPerBlock = blockDim.x / bsize;
    int row = blockIdx.x * nzPerBlock + threadIdx.x / bsize;
    int vecIdx = threadIdx.x % bsize;

    if (threadIdx.x >= (blockDim.x / bsize)*bsize) { return; }

    while (row < size)
    {
        values[diag[row]*bsize + vecIdx] = source[row * bsize + vecIdx];
        row += gridDim.x * nzPerBlock;
    }
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::prepareNextLevelMatrix_full(const Matrix<TConfig> &A, Matrix<TConfig> &Ac)
{
    if (A.is_matrix_singleGPU()) { return; }

    int num_neighbors = A.manager->neighbors.size();

    if (TConfig::memSpace == AMGX_host)
    {
        FatalError("Aggregation AMG Not implemented for host", AMGX_ERR_NOT_IMPLEMENTED);
    }
    else
    {
        int c_size = Ac.get_num_rows();
        int f_size = A.get_num_rows();
        int diag = Ac.hasProps(DIAG);

        if (A.manager->B2L_rings[0].size() > 2) { FatalError("Aggregation_AMG_Level prepareNextLevelMatrix not implemented >1 halo rings", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE); }

        //get coarse -> fine global renumbering
        IVector renumbering(c_size);
        int num_blocks = min(4096, (c_size + 127) / 128);
        coarse_to_global <<< num_blocks, 128>>>(this->m_aggregates.raw(), this->m_aggregates_fine_idx.raw(), renumbering.raw(), f_size, 0);
        cudaCheckError();
        //
        // Step 0 - form halo matrices that are exported to neighbors
        //
        std::vector<Matrix<TConfig> > halo_rows(num_neighbors);
        std::vector<DistributedManager<TConfig> > halo_btl(num_neighbors);

        for (int i = 0; i < num_neighbors; i++ )
        {
            int num_unique = Ac.manager->B2L_rings[i][1];
            //prepare export halo matrices
            halo_btl[i].resize(1, 1);
            halo_btl[i].set_global_id(Ac.manager->global_id());
            halo_btl[i].B2L_maps[0].resize(num_unique);
            halo_btl[i].B2L_rings[0].resize(2);
            halo_btl[i].B2L_rings[0][0] = 0;
            halo_btl[i].B2L_rings[0][1] = num_unique;
            halo_btl[i].set_index_range(A.manager->index_range());
            halo_btl[i].set_base_index(A.manager->base_index());
            //global indices of rows of the halo matrix
            thrust::copy(thrust::make_permutation_iterator( renumbering.begin(), Ac.manager->B2L_maps[i].begin()),
                         thrust::make_permutation_iterator( renumbering.begin(), Ac.manager->B2L_maps[i].begin() + num_unique),
                         halo_btl[i].B2L_maps[0].begin());
            cudaCheckError();
            halo_rows[i].addProps(CSR);

            if (diag) { halo_rows[i].addProps(DIAG); }

            //calculate row length and row_offsets
            halo_rows[i].row_offsets.resize(num_unique + 1);
            thrust::transform(thrust::make_permutation_iterator(Ac.row_offsets.begin() + 1, Ac.manager->B2L_maps[i].begin()),
                              thrust::make_permutation_iterator(Ac.row_offsets.begin() + 1, Ac.manager->B2L_maps[i].end()),
                              thrust::make_permutation_iterator(Ac.row_offsets.begin(), Ac.manager->B2L_maps[i].begin()),
                              halo_rows[i].row_offsets.begin(),
                              thrust::minus<IndexType>());
            cudaCheckError();
            thrust::exclusive_scan(halo_rows[i].row_offsets.begin(), halo_rows[i].row_offsets.end(), halo_rows[i].row_offsets.begin());
            cudaCheckError();
            //resize halo matrix
            IndexType num_nz = halo_rows[i].row_offsets[num_unique];
            halo_rows[i].resize(num_unique, num_unique, num_nz, Ac.get_block_dimy(), Ac.get_block_dimx(), 1);
            //copy relevant rows and renumber their column indices
            num_blocks = min(4096, (num_unique + 127) / 128);
            export_matrix_elements <<< num_blocks, 128>>>(Ac.row_offsets.raw(), Ac.col_indices.raw(), Ac.values.raw(), Ac.manager->B2L_maps[i].raw(), renumbering.raw(), halo_rows[i].row_offsets.raw(), halo_rows[i].col_indices.raw(), halo_rows[i].values.raw(), A.get_block_size(), num_unique);
            cudaCheckError();

            if (diag)
            {
                export_matrix_diagonal <<< num_blocks, 128>>>(Ac.values.raw() + Ac.row_offsets[Ac.get_num_rows()]*Ac.get_block_size(), Ac.get_block_size(), Ac.manager->B2L_maps[i].raw(), halo_rows[i].values.raw() + halo_rows[i].row_offsets[halo_rows[i].get_num_rows()]*Ac.get_block_size(), num_unique);
                cudaCheckError();
            }
        }

        Ac.manager->getComms()->exchange_matrix_halo(halo_rows, halo_btl, Ac);
        //--------------------- renumbering/reordering matrix, integrating halo -----------------------------
        Ac.set_initialized(0);
        //number of owned rows
        c_size = Ac.manager->halo_offsets[0];
        f_size = A.manager->halo_offsets[0];
        num_blocks = min(4096, (c_size + 511) / 512);
        int rings = 1;
        //
        // Step 1 - calculate inverse renumbering (to global indices - base_index)
        //
        Ac.manager->inverse_renumbering.resize(c_size);
        thrust::transform(renumbering.begin(),
                          renumbering.begin() + c_size,
                          thrust::constant_iterator<IndexType>(A.manager->base_index()),
                          Ac.manager->inverse_renumbering.begin(),
                          thrust::minus<IndexType>());
        cudaCheckError();
        //big renumbering table for going from global index to owned local index
        IVector global_to_coarse_local(Ac.manager->index_range());
        thrust::fill(global_to_coarse_local.begin(), global_to_coarse_local.begin() + Ac.manager->index_range(), -1);
        cudaCheckError();
        calc_gbl_renumbering <<< num_blocks, 512>>>(Ac.manager->inverse_renumbering.raw(), global_to_coarse_local.raw(), c_size);
        cudaCheckError();
        Ac.manager->set_num_halo_rows(Ac.manager->halo_offsets[Ac.manager->halo_offsets.size() - 1] - c_size);
        cudaCheckError();
        //
        // Step 2 - create big mapping table of all halo indices we received (this may use a little too much memory sum(fine nodes per neighbor)
        //
        thrust::host_vector<INDEX_TYPE> neighbor_rows(num_neighbors + 1);
        int max_num_rows = 0;

        for (int i = 0; i < num_neighbors; i++)
        {
            neighbor_rows[i] = halo_rows[i].manager->index_range();
            max_num_rows = max_num_rows > halo_rows[i].get_num_rows() ? max_num_rows : halo_rows[i].get_num_rows();
        }

        thrust::exclusive_scan(neighbor_rows.begin(), neighbor_rows.end(), neighbor_rows.begin());
        cudaCheckError();
        int total_rows_of_neighbors = neighbor_rows[num_neighbors];
        IVector halo_mapping(total_rows_of_neighbors);
        thrust::fill(halo_mapping.begin(), halo_mapping.end(), -1);
        cudaCheckError();

        for (int ring = 0; ring < rings; ring++)
        {
            for (int i = 0; i < num_neighbors; i++)
            {
                int size = halo_btl[i].B2L_rings[0][ring + 1] - halo_btl[i].B2L_rings[0][ring];
                int num_blocks = min(4096, (size + 127) / 128);
                create_halo_mapping <<< num_blocks, 128>>>(halo_mapping.raw() + neighbor_rows[i],
                        halo_btl[i].B2L_maps[0].raw() + halo_btl[i].B2L_rings[0][ring],
                        halo_btl[i].base_index(),
                        Ac.manager->halo_offsets[ring * num_neighbors + i], size);
            }
        }

        cudaCheckError();
        //
        // Step 3 - renumber halo matrices and calculate row length (to eventually append to the big matrix)
        //
        INDEX_TYPE owned_nnz = Ac.row_offsets[c_size];
        IVector neighbor_rows_d(num_neighbors + 1);
        thrust::copy(neighbor_rows.begin(), neighbor_rows.end(), neighbor_rows_d.begin());
        cudaCheckError();
        //map column indices of my own matrix (the ones that point outward)
        map_col_indices <<< num_blocks, 512>>>(Ac.row_offsets.raw() + Ac.manager->num_interior_nodes(),
                                               Ac.col_indices.raw(),
                                               Ac.manager->halo_ranges.raw(),
                                               halo_mapping.raw(),
                                               neighbor_rows_d.raw(),
                                               renumbering.raw(),
                                               num_neighbors, c_size, c_size - Ac.manager->num_interior_nodes());
        cudaCheckError();
        IVector temp_row_len(max_num_rows);

        for (int i = 0; i < num_neighbors; i++)
        {
            //map column indices of halo matrices
            int size = halo_rows[i].get_num_rows();
            int num_blocks = min(4096, (size + 127) / 128);
            map_col_indices_and_count_rowlen <<< num_blocks, 128, 128 * sizeof(INDEX_TYPE)>>>(
                halo_rows[i].row_offsets.raw(),
                halo_rows[i].col_indices.raw(),
                temp_row_len.raw(),
                global_to_coarse_local.raw(),
                halo_mapping.raw(),
                neighbor_rows_d.raw(),
                Ac.manager->halo_ranges.raw(),
                i,
                Ac.manager->global_id(),
                Ac.manager->base_index(),
                Ac.manager->index_range(),
                num_neighbors,
                size);

            for (int ring = 0; ring < rings; ring++)
            {
                thrust::copy(temp_row_len.begin() + halo_btl[i].B2L_rings[0][ring], temp_row_len.begin() + halo_btl[i].B2L_rings[0][ring + 1], Ac.row_offsets.begin() + Ac.manager->halo_offsets[ring * num_neighbors + i]);
            }
        }

        cudaCheckError();
        INDEX_TYPE old_nnz = Ac.row_offsets[Ac.row_offsets.size() - 1];
        thrust::exclusive_scan(Ac.row_offsets.begin() + c_size, Ac.row_offsets.end(), Ac.row_offsets.begin() + c_size, owned_nnz);
        cudaCheckError();
        //
        // Step 4 - consolidate column indices and values
        //
        int new_nnz = Ac.row_offsets[Ac.row_offsets.size() - 1];

        Ac.col_indices.resize(new_nnz);
        Ac.values.resize((new_nnz + 1 + diag * (Ac.row_offsets.size() - 2)) * A.get_block_size());

        if (diag)
        {
            MVector diags(c_size * Ac.get_block_size());
            thrust::copy(Ac.values.begin() + old_nnz * Ac.get_block_size(),
                         Ac.values.begin() + old_nnz * Ac.get_block_size() + c_size * Ac.get_block_size(),
                         diags.begin());
            thrust::copy(diags.begin(), diags.begin() + c_size * Ac.get_block_size(),
                         Ac.values.begin() + Ac.row_offsets[Ac.get_num_rows()]*Ac.get_block_size());
            cudaCheckError();
        }

        int cumulative_num_rows = c_size;

        for (int i = 0; i < num_neighbors; i++)
        {
            for (int ring = 0; ring < rings; ring++)
            {
                int num_rows = halo_btl[i].B2L_rings[0][ring + 1] - halo_btl[i].B2L_rings[0][ring];
                int num_blocks = min(4096, (num_rows + 127) / 128);
                reorder_whole_matrix <<< num_blocks, 128>>>(halo_rows[i].row_offsets.raw() + halo_btl[i].B2L_rings[0][ring], halo_rows[i].col_indices.raw(), halo_rows[i].values.raw(), Ac.row_offsets.raw() + Ac.manager->halo_offsets[ring * num_neighbors + i], Ac.col_indices.raw(), Ac.values.raw(), Ac.get_block_size(), num_rows);

                if (diag)
                {
                    thrust::copy(halo_rows[i].values.begin() + (halo_rows[i].row_offsets[halo_rows[i].get_num_rows()] + halo_btl[i].B2L_rings[0][ring])*Ac.get_block_size(),
                                 halo_rows[i].values.begin() + (halo_rows[i].row_offsets[halo_rows[i].get_num_rows()] + halo_btl[i].B2L_rings[0][ring + 1])*Ac.get_block_size(),
                                 Ac.values.begin() + (Ac.row_offsets[Ac.get_num_rows()] + cumulative_num_rows)*Ac.get_block_size());
                    cumulative_num_rows += num_rows;
                }
            }
        }

        cudaCheckError();
        Ac.set_num_cols(Ac.manager->halo_offsets[Ac.manager->halo_offsets.size() - 1]);
        Ac.set_num_rows(Ac.get_num_cols());
        Ac.set_num_nz(new_nnz);
        Ac.delProps(COO);
        Ac.set_initialized(1);
        Ac.computeDiagonal();
    }
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::prepareNextLevelMatrix_diag(const Matrix<TConfig> &A, Matrix<TConfig> &Ac)
{
    if (A.is_matrix_singleGPU()) { return; }

    int num_neighbors = A.manager->neighbors.size();

    if (TConfig::memSpace == AMGX_host)
    {
        FatalError("Aggregation AMG Not implemented for host", AMGX_ERR_NOT_IMPLEMENTED);
    }
    else
    {
        int c_size = Ac.manager->halo_offsets[0];
        int f_size = A.manager->halo_offsets[0];
        int diag = Ac.hasProps(DIAG);
        Ac.manager->inverse_renumbering.resize(c_size);
        //get coarse -> fine renumbering
        int num_blocks = min(4096, (c_size + 127) / 128);
        coarse_to_global <<< num_blocks, 128>>>(this->m_aggregates.raw(), this->m_aggregates_fine_idx.raw(), Ac.manager->inverse_renumbering.raw(), f_size, -1 * A.manager->base_index());
        cudaCheckError();
        Ac.manager->set_num_halo_rows(Ac.manager->halo_offsets[Ac.manager->halo_offsets.size() - 1] - c_size);

        if (!diag) { Ac.computeDiagonal(); }

        Ac.set_initialized(1);
        std::vector<MVector> diagonals(num_neighbors);

        for (int i = 0; i < num_neighbors; i++)
        {
            int size = Ac.manager->B2L_rings[i][Ac.manager->B2L_rings.size() - 1];
            diagonals[i].resize(Ac.get_block_size()*size);
            int num_blocks = min(4096, (size + 127) / 128);
            write_diagonals <<< num_blocks, 128>>>(Ac.values.raw(), Ac.diag.raw(), Ac.manager->B2L_maps[i].raw(), diagonals[i].raw(), Ac.get_block_size(), size);
        }

        cudaCheckError();
        Ac.manager->getComms()->exchange_vectors(diagonals, Ac, this->tag * 100 + 10 + 2);

        for (int i = 0; i < num_neighbors; i++)
        {
            int size = Ac.manager->halo_offsets[i + 1] - Ac.manager->halo_offsets[i];

            if (Ac.hasProps(DIAG)) { thrust::copy(diagonals[i].begin(), diagonals[i].begin() + Ac.get_block_size()*size, Ac.values.begin() + Ac.get_block_size() * (Ac.diagOffset() + Ac.manager->halo_offsets[i])); }
            else
            {
                int num_blocks = min(4096, (size + 127) / 128);
                write_diagonals_back <<< num_blocks, 128>>>(Ac.values.raw(), Ac.diag.raw() + Ac.manager->halo_offsets[i], diagonals[i].raw(), Ac.get_block_size(), size);
            }
        }

        cudaCheckError();
    }
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::prepareNextLevelMatrix_none(const Matrix<TConfig> &A, Matrix<TConfig> &Ac)
{
    if (A.is_matrix_singleGPU()) { return; }

    int num_neighbors = A.manager->neighbors.size();

    if (TConfig::memSpace == AMGX_host)
    {
        FatalError("Aggregation AMG Not implemented for host", AMGX_ERR_NOT_IMPLEMENTED);
    }
    else
    {
        int c_size = Ac.manager->halo_offsets[0];
        int f_size = A.manager->halo_offsets[0];
        int diag = Ac.hasProps(DIAG);
        Ac.manager->inverse_renumbering.resize(c_size);
        //get coarse -> fine renumbering
        int num_blocks = min(4096, (c_size + 127) / 128);
        coarse_to_global <<< num_blocks, 128>>>(this->m_aggregates.raw(), this->m_aggregates_fine_idx.raw(), Ac.manager->inverse_renumbering.raw(), f_size, 0);
        cudaCheckError();
        Ac.manager->set_num_halo_rows(Ac.manager->halo_offsets[Ac.manager->halo_offsets.size() - 1] - c_size);
        Ac.set_initialized(1);

        if (!diag) { Ac.computeDiagonal(); }
    }
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::prepareNextLevelMatrix(const Matrix<TConfig> &A, Matrix<TConfig> &Ac)
{
    if (m_matrix_halo_exchange == 0)
    {
        this->prepareNextLevelMatrix_none(A, Ac);
    }
    else if (m_matrix_halo_exchange == 1)
    {
        this->prepareNextLevelMatrix_diag(A, Ac);
    }
    else if (m_matrix_halo_exchange == 2)
    {
        this->prepareNextLevelMatrix_full(A, Ac);
    }
    else
    {
        FatalError("Invalid Aggregation matrix_halo_exchange parameter", AMGX_ERR_NOT_IMPLEMENTED);
    }
}


__global__ void set_halo_rowlen(INDEX_TYPE *work, INDEX_TYPE *output, INDEX_TYPE  size, INDEX_TYPE diag)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        if (work[idx + 1] - work[idx] > 0)
        {
            output[idx] += work[idx + 1] - work[idx] - (1 - diag);
        }

        idx += blockDim.x * gridDim.x;
    }
}

template <typename T>
__global__ void append_halo_nz(INDEX_TYPE *row_offsets, INDEX_TYPE *new_row_offsets, INDEX_TYPE *col_indices, INDEX_TYPE *new_col_indices, T *values, T *new_values, INDEX_TYPE size, INDEX_TYPE diag, INDEX_TYPE halo_offset, INDEX_TYPE block_size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        int add_diag = !diag;

        if (!diag && new_col_indices[new_row_offsets[idx]] != -1) { add_diag = 0; } //if diag or there is already soimething in the row, then don't add diagonal nonzero (inside diag)

        int append_offset = -1;

        for (int i = new_row_offsets[idx]; i < new_row_offsets[idx + 1]; i++)
        {
            if (new_col_indices[i] == -1) {append_offset = i; break;}
        }

        for (int i = row_offsets[idx]; i < row_offsets[idx + 1]; i++)
        {
            if (diag && i == row_offsets[idx])   //if outside diag and this is the first nonzero in a non-empty row, overwrite diagonal value
            {
                for (int j = 0; j < block_size; j++)
                {
                    new_values[(new_row_offsets[size] + halo_offset + idx)*block_size + j] = values[(row_offsets[size] + halo_offset + idx) * block_size + j];
                }
            }

            int col_idx = col_indices[i];

            if (append_offset == -1 && (col_idx != halo_offset + idx)) {printf("ERROR: append offset is -1 but row has nonzeros in it old %d to %d new %d to %d\n", row_offsets[idx], row_offsets[idx + 1], new_row_offsets[idx], new_row_offsets[idx + 1]); append_offset = 0;}

            if (col_idx != halo_offset + idx || add_diag)
            {
                new_col_indices[append_offset] = col_idx;

                for (int j = 0; j < block_size; j++)
                {
                    new_values[append_offset * block_size + j] = values[i * block_size + j];
                }

                append_offset++;
            }
        }

        idx += blockDim.x * gridDim.x;
    }
}

template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::createCoarseB2LMaps(std::vector<IVector> &in_coarse_B2L_maps)
{
    Matrix<TConfig> &A = this->getA();
    m_num_all_aggregates = m_num_aggregates;
    int num_neighbors = A.manager->neighbors.size();
    IndexType max_b2l = 0;

    for (int i = 0; i < num_neighbors; i++ ) { max_b2l = max_b2l > A.manager->B2L_rings[i][1] ? max_b2l : A.manager->B2L_rings[i][1]; }

    IVector B2L_aggregates(max_b2l);
    IVector indices(max_b2l);

    for (int i = 0; i < num_neighbors; i++ )
    {
        int size = A.manager->B2L_rings[i][1];
        thrust::fill(B2L_aggregates.begin(), B2L_aggregates.begin() + size, 0);
        thrust::sequence(indices.begin(), indices.begin() + size);
        //substitute coarse aggregate indices for fine boundary nodes
        thrust::copy(thrust::make_permutation_iterator(this->m_aggregates.begin(), A.manager->B2L_maps[i].begin()),
                     thrust::make_permutation_iterator(this->m_aggregates.begin(), A.manager->B2L_maps[i].begin() + size),
                     B2L_aggregates.begin());
        //find the unique ones
        thrust::sort_by_key(B2L_aggregates.begin(), B2L_aggregates.begin() + size, indices.begin());
        IndexType num_unique = thrust::unique_by_key(B2L_aggregates.begin(), B2L_aggregates.begin() + size, indices.begin()).first - B2L_aggregates.begin();
        in_coarse_B2L_maps[i].resize(num_unique);
        //sort it back so we have the original ordering
        thrust::sort_by_key(indices.begin(), indices.begin() + num_unique, B2L_aggregates.begin());
        thrust::copy(B2L_aggregates.begin(), B2L_aggregates.begin() + num_unique, in_coarse_B2L_maps[i].begin());
    }

    cudaCheckError();
}


__global__ void populate_coarse_boundary(INDEX_TYPE *flags, INDEX_TYPE *indices, INDEX_TYPE *maps, INDEX_TYPE *output, INDEX_TYPE  size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        output[flags[maps[indices[idx]]]] = maps[indices[idx]];
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void flag_coarse_boundary(INDEX_TYPE *flags, INDEX_TYPE *indices, INDEX_TYPE *maps, INDEX_TYPE  size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        flags[maps[indices[idx]]] = 1;
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void flag_halo_indices(INDEX_TYPE *flags, INDEX_TYPE *indices, INDEX_TYPE  offset, INDEX_TYPE  size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        flags[indices[idx] - offset] = 1;
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void apply_halo_aggregate_indices(INDEX_TYPE *flags, INDEX_TYPE *indices, INDEX_TYPE *output, INDEX_TYPE offset, INDEX_TYPE aggregates_offset, INDEX_TYPE  size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        output[idx] = flags[indices[idx] - offset] + aggregates_offset;
        idx += blockDim.x * gridDim.x;
    }
}

// renumbering the aggregates/communicationg with neighbors
template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::setNeighborAggregates()
{
    Matrix<TConfig> &A = this->getA();
    Matrix<TConfig> &Ac = this->getNextLevel( MemorySpace( ) )->getA();
    m_num_all_aggregates = m_num_aggregates;

    /* WARNING: the matrix reordering always happens inside createRenumbering routine. There are three ways to get to this routine
       1. matrix_upload_all -> uploadMatrix -> initializeUploadReorderAll -> reorder_matrix -> createRenumbering
       2. read_system_distributed -> renumberMatrixOneRing -> reorder_matrix_owned -> createRenumbering
       3. solver_setup -> ... -> AMG_Level::setup -> createCoarseMatrices -> setNeighborAggregates -> createRenumbering
       If you are reading the renumbering from file you might need to add intercept code in if statement below,
       otherwise this routine will exit before calling createRenumbering routine (in case of single or disjoint partitions).
    */
    if (this->getA().is_matrix_singleGPU()) { return; }

    int num_neighbors = A.manager->neighbors.size();

    //
    // Step 0 - set up coarse matrix metadata
    //
    if (Ac.manager == NULL) { Ac.manager = new DistributedManager<T_Config>(); }

    Ac.manager->resize(A.manager->neighbors.size(), 1);
    Ac.manager->A = &Ac;
    int f_size = A.get_num_rows();
    Ac.manager->setComms(A.manager->getComms());
    Ac.manager->set_global_id(A.manager->global_id());
    Ac.manager->neighbors = A.manager->neighbors;
    Ac.manager->set_base_index(A.manager->base_index());
    Ac.manager->halo_ranges = A.manager->halo_ranges;
    Ac.manager->set_index_range(A.manager->index_range());
    //-------------------------------------- Section 1 - renumbering -----------------------------------------------------------
    //
    // Step 1 - calculate coarse level B2L maps - any aggregate that has a fine boundary node, becomes a coarse boundary node
    //
    m_num_all_aggregates = m_num_aggregates;
    int vec_size = m_num_aggregates + 1; //A.manager->num_boundary_nodes()+1;
    IVector B2L_aggregates(vec_size);

    for (int i = 0; i < A.manager->neighbors.size(); i++)
    {
        thrust::fill(B2L_aggregates.begin(), B2L_aggregates.begin() + vec_size, 0);
        int size = A.manager->B2L_rings[i][1];
        int block_size = 128;
        int grid_size = std::min( 4096, ( size + block_size - 1 ) / block_size);
        flag_coarse_boundary <<< grid_size, block_size>>>(B2L_aggregates.raw(), A.manager->B2L_maps[i].raw(), this->m_aggregates.raw(), size);
        thrust::exclusive_scan(B2L_aggregates.begin(), B2L_aggregates.begin() + vec_size, B2L_aggregates.begin());
        (Ac.manager->B2L_maps)[i].resize(B2L_aggregates[vec_size - 1]);
        populate_coarse_boundary <<< grid_size, block_size>>>(B2L_aggregates.raw(), A.manager->B2L_maps[i].raw(), this->m_aggregates.raw(), Ac.manager->B2L_maps[i].raw(), size);
    }

    cudaCheckError();

    for (int i = 0; i < num_neighbors; i++)
    {
        Ac.manager->B2L_rings[i].resize(2);
        Ac.manager->B2L_rings[i][0] = 0;
        Ac.manager->B2L_rings[i][1] = Ac.manager->B2L_maps[i].size();
    }

    DistributedArranger<T_Config> *prep = new DistributedArranger<T_Config>;
    prep->initialize_B2L_maps_offsets(Ac, 1);
    delete prep;
    Ac.set_num_rows(m_num_aggregates);
    IVector renumbering(m_num_aggregates + 1); /* +1 is actually not needed, it will be resized in createRenumbering */
    Ac.manager->createRenumbering(renumbering);
    //
    // Step 2 - renumber aggregates, so boundary nodes will have higher index than interior ones (based on the renumberiong we have been calculating)
    //
    /* WARNING: 1. Thrust scatter and gather routines seem more appropriate here, but they implicitly assume that the input
                and output have certain size correlation, which is not matched by vectors in our case. The only remaining option
                is to use make_permutation as is done below. Example of Thrust scatter and gather calls
                IVector ttt(f_size,-1);
                thrust::scatter(this->m_aggregates.begin(), this->m_aggregates.begin()+f_size, renumbering.begin(), ttt.begin());
                thrust::gather(renumbering.begin(), renumbering.end(), this->m_aggregates.begin(), ttt.begin());
                thrust::copy(ttt.begin(), ttt.end(), this->m_aggregates.begin());

                2. The original thrust composite call is illegal because it uses the same array (m_aggregates) for input and output.
                thrust::copy(thrust::make_permutation_iterator(renumbering.begin(), this->m_aggregates.begin()),
                             thrust::make_permutation_iterator(renumbering.begin(), this->m_aggregates.begin()+f_size),
                             this->m_aggregates.begin());
                Although it somehow still works, it is much safer to use explicit temporary storage for the intermediate result.
    */
    /* WARNING: must save unreordered aggregates for later use before reordering them. */
    IVector unreordered_aggregates(this->m_aggregates);
    /* WARNING: change Thrust call to explicitly use temporary storage for the intermediate result. The earlier version is illegal, but somehow still works. */
    IVector ttt(f_size, -1);
    thrust::copy(thrust::make_permutation_iterator(renumbering.begin(), this->m_aggregates.begin()),
                 thrust::make_permutation_iterator(renumbering.begin(), this->m_aggregates.begin() + f_size),
                 ttt.begin());
    thrust::copy(ttt.begin(), ttt.end(), this->m_aggregates.begin());
    cudaCheckError();

    //we don't need renumbering anymore, it will be identity on the coarse level

    //-------------------------------------- Section 2 - communication -----------------------------------------------------------

    //
    // Step 3 - populate aggregates_fine_idx, which stores for every fine node the original global index of the aggregate (which is lowest global index of nodes aggregated together)
    //

    //
    // These are different when we do /don't do matrix halo exchanges - when we do we need global indices to match nodes,
    // and in this case Ac after computeA will not have the same ordering of halo nodes as after prepareNextLevel_full.
    // However when we do not do matrix halo exchange we are only interested in the ordering of halo nodes on the coarse level,
    // and we can get that by exchanging the (already renumbered) aggregates vector.
    //
    if (m_matrix_halo_exchange == 2)
    {
        //Find original global indices of nodes that have the minimum id in the aggregates.
        thrust::copy(thrust::make_permutation_iterator(A.manager->inverse_renumbering.begin(), this->m_aggregates_fine_idx.begin()),
                     thrust::make_permutation_iterator(A.manager->inverse_renumbering.begin(), this->m_aggregates_fine_idx.begin() + f_size),
                     this->m_aggregates_fine_idx.begin());
        thrust::transform(this->m_aggregates_fine_idx.begin(),
                          this->m_aggregates_fine_idx.begin() + f_size,
                          thrust::constant_iterator<IndexType>(A.manager->base_index()),
                          this->m_aggregates_fine_idx.begin(),
                          thrust::plus<IndexType>());
        //communicate
        this->m_aggregates_fine_idx.set_block_dimx(1);
        this->m_aggregates_fine_idx.set_block_dimy(1);
        m_aggregates_fine_idx.dirtybit = 1;
        A.manager->exchange_halo(m_aggregates_fine_idx, this->tag * 100 + 1 * 10 + 0);
    }
    else
    {
        //communicate
        this->m_aggregates.set_block_dimx(1);
        this->m_aggregates.set_block_dimy(1);
        m_aggregates.dirtybit = 1;
        /* WARNING: you should exchange unreordered aggregates, and append them to your own reordered aggregates, to conform to asusmptions done by distributed_mamanger. */
        //A.manager->exchange_halo(m_aggregates, this->tag*100+1*10+0); //wrong
        A.manager->exchange_halo(unreordered_aggregates, this->tag * 100 + 1 * 10 + 0);
        thrust::copy(unreordered_aggregates.begin() + f_size, unreordered_aggregates.end(), this->m_aggregates.begin() + f_size);
    }

    cudaCheckError();
    //
    // Step 4 - consolidate neighbors' aggregates into own list to be able to perform Galerkin product with the n-ring halo
    //
    IVector &exchanged_aggregates = m_matrix_halo_exchange == 2 ? this->m_aggregates_fine_idx : this->m_aggregates;
    int min_index = thrust::reduce(exchanged_aggregates.begin() + A.manager->halo_offsets[0], exchanged_aggregates.begin() + A.manager->halo_offsets[num_neighbors], (int)0xFFFFFFF, thrust::minimum<int>());
    int max_index = thrust::reduce(exchanged_aggregates.begin() + A.manager->halo_offsets[0], exchanged_aggregates.begin() + A.manager->halo_offsets[num_neighbors], (int)0, thrust::maximum<int>());
    cudaCheckError();
    int s_size = max_index - min_index + 2;
    IVector scratch(s_size);

    for (int i = 0; i < num_neighbors; i++)
    {
        int size = A.manager->halo_offsets[i + 1] - A.manager->halo_offsets[i];
        //Could also use local minimums to perform the same operation. The results are the same.
        //int min_local = thrust::reduce(exchanged_aggregates.begin()+A.manager->halo_offsets[i], exchanged_aggregates.begin()+A.manager->halo_offsets[i+1], (int)0xFFFFFFF, thrust::minimum<int>());
        thrust::fill(scratch.begin(), scratch.begin() + s_size, 0);
        int block_size = 128;
        int grid_size = std::min( 4096, ( size + block_size - 1 ) / block_size);
        flag_halo_indices <<< grid_size, block_size>>>(scratch.raw(), exchanged_aggregates.raw() + A.manager->halo_offsets[i], min_index /*min_local*/, size);
        thrust::exclusive_scan(scratch.begin(), scratch.begin() + s_size, scratch.begin());
        apply_halo_aggregate_indices <<< grid_size, block_size>>>(scratch.raw(), exchanged_aggregates.raw() + A.manager->halo_offsets[i], this->m_aggregates.raw() + A.manager->halo_offsets[i], min_index /*min_local*/, m_num_all_aggregates, size);
        Ac.manager->halo_offsets[i] = m_num_all_aggregates;
        m_num_all_aggregates += scratch[s_size - 1];
    }

    cudaCheckError();
    Ac.manager->halo_offsets[num_neighbors] = m_num_all_aggregates;
}

//TODO: The consolidate and unconsolidate parts could be made more efficient by only sending the
//      nonzero values
template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::consolidateVector(VVector &x)
{
    int my_id = this->getA().manager->global_id();

    if (this->getA().manager->isRootPartition())
    {
        // Here all partitions being consolidated should have same vector size, see TODO above
        INDEX_TYPE num_parts = this->getA().manager->getNumPartsToConsolidate();

        for (int i = 0; i < num_parts; i++)
        {
            int current_part = this->getA().manager->getPartsToConsolidate()[i];

            // Vector has been set to correct size
            if (current_part != my_id)
            {
                //printf("Root partition %d receiving %d -> %d and %d -> %d (total %d)\n", this->getA().manager->global_id(), this->getA().manager->getConsolidationArrayOffsets()[i], this->getA().manager->getConsolidationArrayOffsets()[i+1], this->getA().manager->getConsolidationArrayOffsets()[num_parts+i], this->getA().manager->getConsolidationArrayOffsets()[num_parts+i+1], (int)x.size()/x.get_block_size());
                this->getA().manager->getComms()->recv_vector(x, current_part, 10000 + current_part, x.get_block_size()*this->getA().manager->getConsolidationArrayOffsets()[i], x.get_block_size() * (this->getA().manager->getConsolidationArrayOffsets()[i + 1] - this->getA().manager->getConsolidationArrayOffsets()[i]));
                this->getA().manager->getComms()->recv_vector(x, current_part, 20000 + current_part, x.get_block_size()*this->getA().manager->getConsolidationArrayOffsets()[num_parts + i], x.get_block_size() * (this->getA().manager->getConsolidationArrayOffsets()[num_parts + i + 1] - this->getA().manager->getConsolidationArrayOffsets()[num_parts + i]));
            }
        }
    }
    else
    {
        int my_destination_part = this->getA().manager->getMyDestinationPartition();
        int i_off, i_size, b_off, b_size;
        this->getA().manager->getConsolidationOffsets(&i_off, &i_size, &b_off, &b_size);
        // Here all partitions being consolidated should have same vector size, see TODO above
        this->getA().manager->getComms()->send_vector_async(x, my_destination_part, 10000 + my_id, i_off * x.get_block_size(), i_size * x.get_block_size());
        this->getA().manager->getComms()->send_vector_async(x, my_destination_part, 20000 + my_id, b_off * x.get_block_size(), b_size * x.get_block_size());
    }
}

//TODO: The consolidate and unconsolidate parts could be made more efficient by only sending the
//      nonzero values
template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::unconsolidateVector(VVector &x)
{
    if (this->getA().manager->isRootPartition())
    {
        INDEX_TYPE num_parts = this->getA().manager->getNumPartsToConsolidate();

        for (int i = 0; i < num_parts; i++)
        {
            int current_part = this->getA().manager->getPartsToConsolidate()[i];

            // Vector has been set to correct size
            if (current_part != this->getA().manager->global_id())
            {
                this->getA().manager->getComms()->send_vector_async(x, current_part, 30000 + current_part, x.get_block_size()*this->getA().manager->getConsolidationArrayOffsets()[i], x.get_block_size() * (this->getA().manager->getConsolidationArrayOffsets()[i + 1] - this->getA().manager->getConsolidationArrayOffsets()[i]));
                this->getA().manager->getComms()->send_vector_async(x, current_part, 40000 + current_part, x.get_block_size()*this->getA().manager->getConsolidationArrayOffsets()[num_parts + i], x.get_block_size() * (this->getA().manager->getConsolidationArrayOffsets()[num_parts + i + 1] - this->getA().manager->getConsolidationArrayOffsets()[num_parts + i]));
            }
        }
    }
    else
    {
        int my_destination_part = this->getA().manager->getMyDestinationPartition();
        // Vector x is of unknown size
        int i_off, i_size, b_off, b_size;
        this->getA().manager->getConsolidationOffsets(&i_off, &i_size, &b_off, &b_size);
        this->getA().manager->getComms()->recv_vector(x, my_destination_part, 30000 + this->getA().manager->global_id(), i_off * x.get_block_size(), i_size * x.get_block_size());
        this->getA().manager->getComms()->recv_vector(x, my_destination_part, 40000 + this->getA().manager->global_id(), b_off * x.get_block_size(), b_size * x.get_block_size());
    }
}


template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::createCoarseVertices()
{
    profileSubphaseFindAggregates();
    //Set the aggregates
    this->Profile.tic("setAggregates");
    this->m_selector->setAggregates(this->getA(), this->m_aggregates, this->m_aggregates_fine_idx, this->m_num_aggregates);
    this->Profile.toc("setAggregates");

    if ( this->m_print_aggregation_info )
    {
        this->m_selector->printAggregationInfo( this->m_aggregates, this->m_aggregates_fine_idx, this->m_num_aggregates );
    }

    this->getA().template setParameter< int > ("aggregates_num", this->m_num_aggregates); // ptr to aaggregates
}

//  Creating the next level
template <class T_Config>
void Aggregation_AMG_Level_Base<T_Config>::createCoarseMatrices()
{
    Matrix<TConfig> &A = this->getA();
    Matrix<TConfig> &Ac = this->getNextLevel( MemorySpace( ) )->getA();
    profileSubphaseFindAggregates();
    int num_parts, num_fine_neighbors, my_id;

    if (!A.is_matrix_singleGPU())
    {
        num_parts = A.manager->getComms()->get_num_partitions();
        num_fine_neighbors = A.manager->neighbors.size();
        my_id = A.manager->global_id();
    }
    else
    {
        num_parts = 1;
        num_fine_neighbors = 0;
        my_id = 0;
    }

    if (!A.is_matrix_singleGPU() && this->isConsolidationLevel())
    {
        // ----------------------------------------------------
        // Consolidate multiple fine matrices into one coarse matrix
        // ----------------------------------------------------
        // ----------------
        // Step 1
        // Decide which partitions should be merged together, store in destination_partitions vector
        // ---------------
        IVector_h &destination_part = A.manager->getDestinationPartitions();
        int my_destination_part = A.manager->getMyDestinationPartition();

        if (my_destination_part >= num_parts)
        {
            FatalError("During consolidation, sending data to partition that doesn't exist", AMGX_ERR_NOT_IMPLEMENTED);
        }

        // Create mapping from coarse partition indices (ranks on the coarse consolidated level) to partition indices on the fine level (ranks on the fine level)
        IVector_h coarse_part_to_fine_part = destination_part;
        thrust::sort(coarse_part_to_fine_part.begin(), coarse_part_to_fine_part.end());
        cudaCheckError();
        coarse_part_to_fine_part.erase(thrust::unique(coarse_part_to_fine_part.begin(), coarse_part_to_fine_part.end()), coarse_part_to_fine_part.end());
        cudaCheckError();
        //Then, the number of coarse partitions is simply the size of this vector
        int num_coarse_partitions = coarse_part_to_fine_part.size();
        // Create mapping from fine partition indices to coarse partition indices, with fine partitions that are merging together having the same coarse indices
        IVector_h fine_part_to_coarse_part(num_parts);
        thrust::lower_bound(coarse_part_to_fine_part.begin(), coarse_part_to_fine_part.end(), destination_part.begin(), destination_part.end(), fine_part_to_coarse_part.begin());
        cudaCheckError();
        // Create mapping from this specific partition's neighbors to consolidated coarse neighbors, but using their fine index (aka. destination partition indices for my neighbors)
        IVector_h fine_neigh_to_fine_part;
        A.manager->createNeighToDestPartMap(fine_neigh_to_fine_part, A.manager->neighbors, destination_part, num_fine_neighbors);
        // Create mapping from consolidated coarse neighbors to fine partition indices (even if the current partition is not going to be a root)
        IVector_h coarse_neigh_to_fine_part;
        int num_coarse_neighbors;
        A.manager->createConsolidatedNeighToPartMap(coarse_neigh_to_fine_part, fine_neigh_to_fine_part, my_destination_part, destination_part, num_coarse_neighbors);
        // Create mapping from fine neighbors to coarse neighbors, with fine neighbors this partition is merging with labeled with -1
        IVector_h fine_neigh_to_coarse_neigh;
        A.manager->createNeighToConsNeigh(fine_neigh_to_coarse_neigh, coarse_neigh_to_fine_part, fine_neigh_to_fine_part, my_destination_part, num_fine_neighbors);
        /*
         EXAMPLE
          Take the following partition graph (that describes connections between partitions, vertices are the partitions themselves), this is the same graph that is used in the setup example
          number of partitions num_parts=12
          CSR row_offsets [0 4 8 13 21 25 32 36 41 46 50 57 61]
          CSR col_indices [0 1 3 8
                       0 1 2 3
                       1 2 3 4 5
                       0 1 2 3 4 5 8 10
                       2 4 5 6
                       2 3 4 5 6 7 10
                       4 5 6 7
                       5 6 7 9 10
                       0 3 8 10 11
                       7 9 10 11
                       3 5 7 8 9 10 11
                       8 9 10 11]
          destination_part = [0 0 0 0 4 4 4 4 8 8 8 8]
          coarse_part_to_fine_part = [0 4 8] num_coarse_partitions = 3
          fine_part_to_coarse_part = [0 0 0 0 1 1 1 1 2 2 2 2]
          original neighbor lists correspond to the rows of the matrix, minus the diagonal elements: (part 0)[1 3 8] (part 3)[0 1 2 4 5 8 10] (part 10)[3 5 7 8 9 11]
          fine_neigh_to_fine_part (part 0)[0 0 2] (part 3)[0 0 0 0 1 2 2] (part 10)[0 1 1 2 2 2]
          coarse_neigh_to_fine_part (part 0)[8] (part 3)[4 8] (part 10)[0 4]
          fine_neigh_to_coarse_neigh (part 0)[-1 -1 0] (part 3)[-1 -1 -1 0 0 1 1] (part 10)[0 1 1 -1 -1 -1]
         */
        // --------------------------
        // Step 2
        // Create coarse B2L_maps, by mapping fine B2L maps to coarse indices using this->m_aggregates and eliminating duplicates
        // --------------------------
        std::vector<IVector> coarse_B2L_maps(num_fine_neighbors);
        m_num_all_aggregates = m_num_aggregates;
        int num_neighbors_temp = A.manager->neighbors.size();
        int num_rings = A.manager->B2L_rings[0].size() - 1;

        if (num_rings != 1)
        {
            FatalError("num_rings > 1 not supported in consolidation\n", AMGX_ERR_NOT_IMPLEMENTED);
        }

        IndexType max_b2l = 0;

        for (int i = 0; i < num_neighbors_temp; i++ ) { max_b2l = max_b2l > A.manager->B2L_rings[i][1] ? max_b2l : A.manager->B2L_rings[i][1]; }

        IVector B2L_aggregates(max_b2l);
        IVector indices(max_b2l);

        //TODO: use the algorithm from setNeighborAggregates()
        for (int i = 0; i < num_neighbors_temp; i++ )
        {
            int size = A.manager->B2L_rings[i][1];
            thrust::fill(B2L_aggregates.begin(), B2L_aggregates.begin() + size, 0);
            thrust::sequence(indices.begin(), indices.begin() + size);
            //substitute coarse aggregate indices for fine boundary nodes
            thrust::copy(thrust::make_permutation_iterator(this->m_aggregates.begin(), A.manager->B2L_maps[i].begin()),
                         thrust::make_permutation_iterator(this->m_aggregates.begin(), A.manager->B2L_maps[i].begin() + size),
                         B2L_aggregates.begin());
            //find the unique ones
            thrust::sort_by_key(B2L_aggregates.begin(), B2L_aggregates.begin() + size, indices.begin());
            IndexType num_unique = thrust::unique_by_key(B2L_aggregates.begin(), B2L_aggregates.begin() + size, indices.begin()).first - B2L_aggregates.begin();
            coarse_B2L_maps[i].resize(num_unique);
            //sort it back so we have the original ordering
            thrust::sort_by_key(indices.begin(), indices.begin() + num_unique, B2L_aggregates.begin());
            thrust::copy(B2L_aggregates.begin(), B2L_aggregates.begin() + num_unique, coarse_B2L_maps[i].begin());
        }

        cudaCheckError();
        /*
         * EXAMPLE
         say, partition 3 has the following coarse B2L_maps:
         neighbors [0 1 2 4 5 8 10]
         B2L_maps[0(=0)] = [6 7 8]
         B2L_maps[1(=1)] = [8 9 10]
         B2L_maps[2(=2)] = [10 11 12 13]
         B2L_maps[3(=4)] = [13 14 15]
         B2L_maps[4(=5)] = [15 16 17]
         B2L_maps[5(=8)] = [6 18 19]
         B2L_maps[6(=10)] = [17 20 19]
         */
        // ---------------------------------------------------
        // Step 3
        // create new B2L maps for each merged destination neighbor and drop B2L maps to neighbors we are merging with
        // ---------------------------------------------------
        std::vector<IVector> dest_coarse_B2L_maps;
        A.manager->consolidateB2Lmaps(dest_coarse_B2L_maps, coarse_B2L_maps, fine_neigh_to_coarse_neigh, num_coarse_neighbors, num_fine_neighbors);
        /*
         * EXAMPLE
         Then, merging the coarse B2L maps on partition 3, we get:
         coarse_neigh_to_fine_part [4 8]
         dest_coarse_B2L_maps[0(=4)] = [13 14 15 16 17]
         dest_coarse_B2L_maps[1(=8)] = [6 17 18 19 20]
         */
        // -----------------------
        // Step 4
        // Create interior-boundary renumbering of aggregates according to dest_coarse_B2L_maps
        // -----------------------
        // Now renumber the aggregates with all interior aggregates first, boundary aggregates second
        int num_interior_aggregates; //returned by createAggregatesRenumbering
        int num_boundary_aggregates; //returned by createAggregatesRenumbering
        IVector renumbering; //returned by createAggregatesRenumbering
        // Following calls create renumbering array and modifies B2L_maps
        A.manager->createAggregatesRenumbering(renumbering, dest_coarse_B2L_maps, this->m_num_aggregates, num_coarse_neighbors, num_interior_aggregates, num_boundary_aggregates, num_rings);
        /*
         * EXAMPLE
         Partition 3 will get a renumbering vector of size 21, for the 21 owned agggregates:
         [0 1 2 3 4 5 17 6 7 8 9 10 11 12 13 14 15 16 18 19 20]
         num_interior_aggregates = 12
         num_boundary_aggregates = 9
         */
        // -------------------------------------------------
        // Step 5
        // Determine whether root partition, make list of partitions merged into one
        // ------------------------------------------------
        // Check if I'm root partition and how fine partitions (including myself) are merging into me
        bool is_root_partition = false;
        int num_fine_parts_to_consolidate = 0;
        IVector_h fine_parts_to_consolidate;

        for (int i = 0; i < num_parts; i++)
        {
            if (destination_part[i] == my_id)
            {
                is_root_partition = true;
                num_fine_parts_to_consolidate++;
            }
        }

        fine_parts_to_consolidate.resize(num_fine_parts_to_consolidate);
        int count = 0;

        for (int i = 0; i < num_parts; i++)
        {
            if (destination_part[i] == my_id)
            {
                fine_parts_to_consolidate[count] = i;
                count++;
            }
        }

        //save this information as state, as this will also be required during solve for restriction/prolongation
        A.manager->setIsRootPartition(is_root_partition);
        A.manager->setNumPartsToConsolidate(num_fine_parts_to_consolidate);
        A.manager->setPartsToConsolidate(fine_parts_to_consolidate);
        /*
         * EXAMPLE
         isRootPartition is true for partitions 0,4,8 false for others
         num_fine_parts_to_consolidate = 4 for partitions 0,4,8
         fine_parts_to_consolidate (part 0)[0 1 2 3] (part 4)[4 5 6 7] (part 8)[8 9 10 11]
         */
        // ----------------------
        // Step 6
        // Compute number of interior, boundary and total nodes in the consolidated coarse matrix. Create offsets so that partitions being merged together will have their aggregate indices ordered like this:
        // [num_interior(fine_parts_to_consolidate[0]] num_interior(fine_parts_to_consolidate[1]] ... num_interior(fine_parts_to_consolidate[num_fine_parts_to_consolidate]
        //        num_boundary(fine_parts_to_consolidate[0]] num_boundary(fine_parts_to_consolidate[1]] ... num_boundary(fine_parts_to_consolidate[num_fine_parts_to_consolidate] ]
        // ----------------------
        // Gather to get number of interior/boundary aggregates of neighbors I will merge with
        std::vector<IVector_h> vertex_counts;
        int interior_offset, boundary_offset, total_interior_rows_in_merged, total_boundary_rows_in_merged;
        int total_rows_in_merged;
        //Computes these offsets on the root, sends them back
        A.manager->computeConsolidatedOffsets(my_id, my_destination_part, is_root_partition, num_interior_aggregates, num_boundary_aggregates, vertex_counts, fine_parts_to_consolidate, num_fine_parts_to_consolidate, interior_offset, boundary_offset, total_interior_rows_in_merged, total_boundary_rows_in_merged, total_rows_in_merged, A.manager->getComms());
        //Partitions save these offsets, as it will be required during solve restriction/prolongation
        A.manager->setConsolidationOffsets(interior_offset, num_interior_aggregates, boundary_offset + num_interior_aggregates, num_boundary_aggregates);
        /*
         * EXAMPLE
         For root partition 0, say we have the following interior/boundary counts (note that partition 1 has 0 boundary, as it is only connected to partitions it is merging with)
         part 0 - interior: 10 boundary 3
         part 1 - interior: 18
         part 2 - interior: 10 boundary 16
         part 3 - interior: 12 boundary 9
         interior_offset for partitions 0,1,2,3: 0 10 28 38 (total_interior_rows_in_merged 50)
         boundary_offset for partitions 0,1,2,3: 0 3 3 19 (total_boundary_rows_in_merged 28)
         */
        // ----------------------
        // Step 7
        // Each partition renumbers its aggregates and dest_coarse_B2L_maps using offsets computed in Step 6 and permutation in Step 4
        // ----------------------
        // Kernel to renumber the aggregates
        int block_size = 128;
        int grid_size = std::min( 4096, ( A.manager->halo_offsets[0] + block_size - 1 ) / block_size);
        renumberAggregatesKernel <<< grid_size, block_size >>>(renumbering.raw(), interior_offset, boundary_offset, this->m_aggregates.raw(), A.manager->halo_offsets[0], num_interior_aggregates, renumbering.size());
        cudaCheckError();

        for (int i = 0; i < num_coarse_neighbors; i++)
        {
            thrust::transform(dest_coarse_B2L_maps[i].begin(),
                              dest_coarse_B2L_maps[i].end(),
                              thrust::constant_iterator<IndexType>(boundary_offset),
                              dest_coarse_B2L_maps[i].begin(),
                              thrust::plus<IndexType>());
        }

        cudaCheckError();
        /*
         * EXAMPLE
         Partition 3 had a renumbering vector:
         [0 1 2 3 4 5 17 6 7 8 9 10 11 12 13 14 15 16 18 19 20]
         which is now adjusted to account for the consolidated coarse matrices' indices:
         [38 39 40 41 42 43 74 44 45 46 47 48 49 69 70 71 72 73 75 76 77]
         And the dest_coarse_B2L_maps, which looked like:
         dest_coarse_B2L_maps[0(=4)] = [13 14 15 16 17]
         dest_coarse_B2L_maps[1(=8)] = [6 17 18 19 20]
         is now:
         dest_coarse_B2L_maps[0(=4)] = [69 70 71 72 73]
         dest_coarse_B2L_maps[1(=8)] = [74 73 75 76 77]
         */
        // -------------------------------------------------
        // Step 8
        // Send dest_coarse_B2L_maps to root partitions
        // ------------------------------------------------
        // Each fine partition sends to its root the number of coarse neighbors it has, their ids, and the number of boundary nodes for each coarse neighbor
        IVector_h num_bdy_per_coarse_neigh(num_coarse_neighbors);

        for (int i = 0; i < num_coarse_neighbors; i++)
        {
            num_bdy_per_coarse_neigh[i] = dest_coarse_B2L_maps[i].size();
        }

        IVector_h consolidated_coarse_neigh_to_fine_part; //consolidated list of coarse neighbors for the root partition, using fine partition indices
        int num_consolidated_neighbors = 0;
        std::vector<IVector> consolidated_B2L_maps; //concatenates dest_coarse_B2L_maps received from partitions that are merging into the same root and pointing to the same destination coarse neighbor
        A.manager->consolidateB2LmapsOnRoot(num_consolidated_neighbors, consolidated_B2L_maps, consolidated_coarse_neigh_to_fine_part, dest_coarse_B2L_maps, coarse_neigh_to_fine_part, num_bdy_per_coarse_neigh, fine_parts_to_consolidate, num_fine_parts_to_consolidate, my_id, my_destination_part, is_root_partition, num_coarse_neighbors, A.manager->getComms());
        //
        // Step 9 - figuring out halo aggregate IDs
        //
        //Now we need to update halo aggregate IDs - this is just a halo exchange on this->m_aggregates between partitions
        //that are being merged together, but we need to send other halos to the root to come up with the halo renumbering
        //TODO: separate transactions, send "real halo" to the root nodes (coarse neighbors) immediately
        //Step 9.1: takes care of synchronizing the aggregate IDs between partitions we are merging together and got consistent halo aggregate IDs for neighbor we are not merging with (which are going to be sent to the root in 9.2)
        A.manager->exchange_halo(this->m_aggregates, 6666);
        /*
         * EXAMPLE 2
         This example is independent from the previous ones.
         Say partition 0 and 1 are merging (into 0) partition 0 is neighbors with 1,2,3 and partition 1 is neighbors with 0,3,4
         Partitions 3 and 4 are merging (into partition 3) and partition 2 is not merging with anyone.
         This example details the renumbering of halo indices on partition 0 and partition 1.
         After the exchange halo, we have:
         this->m_aggregates on partition 0:
         [(fine interior nodes) (fine boundary nodes) (fine halo from part 1) (fine halo from part 2) (fine halo from part 3)]
         [(fine interior nodes) (fine boundary nodes) (13 13 15) (12 15 17) (14 16 18)]
         aggregates on partition 1:
         [(fine interior nodes) (fine boundary nodes) (fine halo from part 0) (fine halo from part 3) (fine halo from part 4)]
         [(fine interior nodes) (fine boundary nodes) (14 16 17) (18 19 19) (15 15 17)]
         indices in  (fine halo from part 0) and (fine halo from part 1) actually contain interior aggregate indices (if they are not connected to partitions 2,3 or 4), because the boundary is disappearing there.
         Indices in halo regions contain remote-local indices.

         This example is used throughout consolidateAndRenumberHalos
         */
        //Step 9.2 - 9.5
        IVector_h halo_offsets(num_consolidated_neighbors + 1, 0);
        A.manager->consolidateAndRenumberHalos(this->m_aggregates, A.manager->halo_offsets, halo_offsets, A.manager->neighbors, num_fine_neighbors, consolidated_coarse_neigh_to_fine_part, num_consolidated_neighbors, destination_part, my_destination_part, is_root_partition, fine_parts_to_consolidate, num_fine_parts_to_consolidate, num_parts, my_id, total_rows_in_merged, this->m_num_all_aggregates, A.manager->getComms());

        if (is_root_partition)
        {
            for (int i = 0; i < consolidated_B2L_maps.size(); i++)
            {
                thrust::sort(consolidated_B2L_maps[i].begin(), consolidated_B2L_maps[i].end());
            }

            cudaCheckError();
        }

        // Step 10 do the Galerkin product
        //
        ViewType oldView = this->getA().currentView();
        this->getA().setView(ALL);
        // If we reuse the level we keep the previous restriction operator
        this->Profile.tic("computeR");
        computeRestrictionOperator();
        this->Profile.toc("computeR");
        profileSubphaseComputeCoarseA();
        this->Profile.tic("computeA");
        Ac.copyAuxData(&A);

        if (Ac.manager == NULL)
        {
            Ac.manager = new DistributedManager<TConfig>();
        }

        this->m_coarseAGenerator->computeAOperator(A, Ac, this->m_aggregates, this->m_R_row_offsets, this->m_R_column_indices, this->m_num_all_aggregates);
        Ac.setColsReorderedByColor(false);
        ViewType oldViewC = Ac.currentView();
        Ac.setView(FULL);
        this->Profile.toc("computeA");

        //
        // Step 11, send matrices to root, consolidate
        //

        if (!is_root_partition)
        {
            A.manager->getComms()->send_vector_async(Ac.row_offsets, my_destination_part, 1111);
            A.manager->getComms()->send_vector_async(Ac.col_indices, my_destination_part, 1112);
            A.manager->getComms()->send_vector_async(Ac.values, my_destination_part, 1113);
        }
        else
        {
            int total_num_rows = this->m_num_all_aggregates;
            IVector new_row_offsets(total_num_rows + 1, 0);

            //if diags are inside then we won't be counting those twice when computing halo row length
            if (!Ac.hasProps(DIAG))
            {
                thrust::fill(new_row_offsets.begin() + halo_offsets[0], new_row_offsets.begin() + halo_offsets[num_consolidated_neighbors], 1);
                cudaCheckError();
            }

            std::vector<IVector> recv_row_offsets(num_fine_parts_to_consolidate);
            std::vector<VecInt_t> num_nz(num_fine_parts_to_consolidate);
            IVector *work_row_offsets;
            std::vector<VecInt_t> index_offset_array(2 * num_fine_parts_to_consolidate + 1);
            int interior_offset = 0;
            int boundary_offset = 0;

            for (int i = 0; i < num_fine_parts_to_consolidate; i++)
            {
                boundary_offset += vertex_counts[i][0];
            }

            int max_num_nz = 0;

            for (int i = 0; i < num_fine_parts_to_consolidate; i++)
            {
                int current_part = fine_parts_to_consolidate[i];

                //receive row offsets
                if (current_part != my_id)
                {
                    recv_row_offsets[i].resize(total_num_rows + 1);
                    A.manager->getComms()->recv_vector(recv_row_offsets[i], current_part, 1111);
                    work_row_offsets = &(recv_row_offsets[i]);
                    num_nz[i] = (*work_row_offsets)[work_row_offsets->size() - 1];
                    max_num_nz = max_num_nz > num_nz[i] ? max_num_nz : num_nz[i];
                }
                else
                {
                    work_row_offsets = &(Ac.row_offsets);
                    num_nz[i] = Ac.get_num_nz();
                }

                //Get interior row length
                thrust::transform(work_row_offsets->begin() + interior_offset + 1,
                                  work_row_offsets->begin() + interior_offset + vertex_counts[i][0] + 1,
                                  work_row_offsets->begin() + interior_offset,
                                  new_row_offsets.begin() + interior_offset,
                                  thrust::minus<IndexType>());
                cudaCheckError();
                //Get boundary row length
                thrust::transform(work_row_offsets->begin() + boundary_offset + 1,
                                  work_row_offsets->begin() + boundary_offset + vertex_counts[i][1] + 1,
                                  work_row_offsets->begin() + boundary_offset,
                                  new_row_offsets.begin() + boundary_offset,
                                  thrust::minus<IndexType>());
                cudaCheckError();
                //Increment halo row length by one for every nonzero that is an edge from the halo into this partition
                int size = halo_offsets[num_consolidated_neighbors] - halo_offsets[0];
                const int block_size = 128;
                const int num_blocks = min( AMGX_GRID_MAX_SIZE, (size - 1) / block_size + 1);
                set_halo_rowlen <<< num_blocks, block_size>>>(work_row_offsets->raw() + halo_offsets[0], new_row_offsets.raw() + halo_offsets[0], size, Ac.hasProps(DIAG));
                cudaCheckError();
                index_offset_array[i] = interior_offset;
                index_offset_array[num_fine_parts_to_consolidate + i] = boundary_offset;
                interior_offset += vertex_counts[i][0];
                boundary_offset += vertex_counts[i][1];
                index_offset_array[i + 1] = interior_offset;
                index_offset_array[num_fine_parts_to_consolidate + i + 1] = boundary_offset;
            }

            A.manager->setConsolidationArrayOffsets(index_offset_array);
            //Exclusive scan row length array to get row offsets
            thrust::exclusive_scan(new_row_offsets.begin(), new_row_offsets.end(), new_row_offsets.begin());
            cudaCheckError();
            //Prepare to receive column indices and values
            int num_nz_consolidated = new_row_offsets[new_row_offsets.size() - 1];
            IVector recv_col_indices(max_num_nz);
            IVector new_col_indices(num_nz_consolidated);
            MVector recv_values((max_num_nz + 1 + Ac.hasProps(DIAG) * (halo_offsets[num_consolidated_neighbors] - 1))*Ac.get_block_size());
            MVector new_values((num_nz_consolidated + 1 + Ac.hasProps(DIAG) * (halo_offsets[num_consolidated_neighbors] - 1))*Ac.get_block_size());
            thrust::fill(new_col_indices.begin() + new_row_offsets[halo_offsets[0]], new_col_indices.end(), -1); //Set all the halo col indices to -1

            if (!Ac.hasProps(DIAG)) { thrust::fill(new_values.begin() + num_nz_consolidated * Ac.get_block_size(), new_values.end(), types::util<ValueTypeA>::get_zero()); }

            cudaCheckError();
            IVector *work_col_indices;
            MVector *work_values;
            interior_offset = 0;
            boundary_offset = 0;

            for (int i = 0; i < num_fine_parts_to_consolidate; i++)
            {
                int current_part = fine_parts_to_consolidate[i];
                boundary_offset += vertex_counts[i][0];
            }

            for (int i = 0; i < num_fine_parts_to_consolidate; i++)
            {
                int current_part = fine_parts_to_consolidate[i];

                if (current_part != my_id)
                {
                    A.manager->getComms()->recv_vector(recv_col_indices, current_part, 1112, 0, num_nz[i]);
                    A.manager->getComms()->recv_vector(recv_values, current_part, 1113, 0, (num_nz[i] + 1 + Ac.hasProps(DIAG) * (halo_offsets[num_consolidated_neighbors] - 1))*Ac.get_block_size());
                    work_col_indices = &(recv_col_indices);
                    work_row_offsets = &(recv_row_offsets[i]);
                    work_values = &(recv_values);
                }
                else
                {
                    work_row_offsets = &(Ac.row_offsets);
                    work_col_indices = &(Ac.col_indices);
                    work_values = &(Ac.values);
                }

                //Put interior rows in place
                thrust::copy(work_col_indices->begin() + (*work_row_offsets)[interior_offset],
                             work_col_indices->begin() + (*work_row_offsets)[interior_offset + vertex_counts[i][0]],
                             new_col_indices.begin() + new_row_offsets[interior_offset]);
                cudaCheckError();
                thrust::copy(work_values->begin() + (*work_row_offsets)[interior_offset]*Ac.get_block_size(),
                             work_values->begin() + ((*work_row_offsets)[interior_offset + vertex_counts[i][0]])*Ac.get_block_size(),
                             new_values.begin() + new_row_offsets[interior_offset]*Ac.get_block_size());
                cudaCheckError();
                //Put boundary rows in place
                thrust::copy(work_col_indices->begin() + (*work_row_offsets)[boundary_offset],
                             work_col_indices->begin() + (*work_row_offsets)[boundary_offset + vertex_counts[i][1]],
                             new_col_indices.begin() + new_row_offsets[boundary_offset]);
                cudaCheckError();
                thrust::copy(work_values->begin() + (*work_row_offsets)[boundary_offset]*Ac.get_block_size(),
                             work_values->begin() + ((*work_row_offsets)[boundary_offset + vertex_counts[i][1]])*Ac.get_block_size(),
                             new_values.begin() + new_row_offsets[boundary_offset]*Ac.get_block_size());
                cudaCheckError();
                //Process halo rows (merge)
                int size = halo_offsets[num_consolidated_neighbors] - halo_offsets[0];
                const int block_size = 128;
                const int num_blocks = min( AMGX_GRID_MAX_SIZE, (size - 1) / block_size + 1);
                //TODO: vectorise this kernel, will be inefficient for larger block sizes
                append_halo_nz <<< num_blocks, block_size>>>(work_row_offsets->raw() + halo_offsets[0],
                        new_row_offsets.raw() + halo_offsets[0],
                        work_col_indices->raw(),
                        new_col_indices.raw(),
                        work_values->raw(),
                        new_values.raw(),
                        size, Ac.hasProps(DIAG), halo_offsets[0], Ac.get_block_size());
                cudaCheckError();

                // Diagonals
                if (Ac.hasProps(DIAG))
                {
                    // Diagonal corresponding to interior rows
                    thrust::copy(work_values->begin() + (num_nz[i] + interior_offset)*Ac.get_block_size(),
                                 work_values->begin() + (num_nz[i] + interior_offset + vertex_counts[i][0])*Ac.get_block_size(),
                                 new_values.begin() + (new_row_offsets[halo_offsets[halo_offsets.size() - 1]] + interior_offset)*Ac.get_block_size());
                    // Diagonal corresponding to boundary rows
                    thrust::copy(work_values->begin() + (num_nz[i] + boundary_offset)*Ac.get_block_size(),
                                 work_values->begin() + (num_nz[i] + boundary_offset + vertex_counts[i][1])*Ac.get_block_size(),
                                 new_values.begin() + (new_row_offsets[halo_offsets[halo_offsets.size() - 1]] + boundary_offset)*Ac.get_block_size());
                    cudaCheckError();
                }

                interior_offset += vertex_counts[i][0];
                boundary_offset += vertex_counts[i][1];
            }

            Ac.set_initialized(0);
            Ac.row_offsets = new_row_offsets;
            Ac.col_indices = new_col_indices;
            Ac.values = new_values;
        }

        // Create a new distributed communicator for coarse levels that only contains active partitions
        Ac.manager->setComms(A.manager->getComms()->Clone());
        Ac.manager->getComms()->createSubComm(coarse_part_to_fine_part, is_root_partition);

        //
        // Step 12 - finalizing, bookkeping
        //
        if (is_root_partition)
        {
            int my_consolidated_id = fine_part_to_coarse_part[my_id];

            for (int i = 0; i < num_consolidated_neighbors; i++)
            {
                consolidated_coarse_neigh_to_fine_part[i] = fine_part_to_coarse_part[consolidated_coarse_neigh_to_fine_part[i]];
            }

            Ac.manager->initializeAfterConsolidation(
                my_consolidated_id,
                Ac,
                consolidated_coarse_neigh_to_fine_part,
                total_interior_rows_in_merged,
                total_boundary_rows_in_merged,
                this->m_num_all_aggregates,
                halo_offsets,
                consolidated_B2L_maps,
                1,
                true);
            Ac.manager->B2L_rings.resize(num_consolidated_neighbors + 1);

            for (int i = 0; i < num_consolidated_neighbors; i++)
            {
                Ac.manager->B2L_rings[i].resize(2);
                Ac.manager->B2L_rings[i][0] = 0;
                Ac.manager->B2L_rings[i][1] = consolidated_B2L_maps[i].size();
            }

            Ac.manager->set_initialized(Ac.row_offsets);
            Ac.manager->getComms()->set_neighbors(num_consolidated_neighbors);
            int new_nnz = Ac.row_offsets[Ac.row_offsets.size() - 1];
            Ac.set_num_nz(new_nnz);
            Ac.set_num_cols(Ac.manager->halo_offsets[Ac.manager->halo_offsets.size() - 1]);
            Ac.set_num_rows(Ac.get_num_cols());

            if (A.hasProps(DIAG)) { Ac.addProps(DIAG); }

            Ac.computeDiagonal();
            Ac.set_initialized(1);
        }
        else
        {
            Ac.set_initialized(0);
            // set size of Ac to be zero
            Ac.resize(0, 0, 0, 1);
            Ac.set_initialized(1);
        }

        this->getA().setView(oldView);
        Ac.setView(OWNED);
    }
    else
    {
        /* WARNING: do not recompute prolongation (P) and restriction (R) when you
                    are reusing the level structure (structure_reuse_levels > 0).
                    Notice that in aggregation path, prolongation P is implicit,
                    and is used through the aggregates array. */
        if (this->isReuseLevel() == false)
        {
            this->setNeighborAggregates();
        }

        this->getA().setView(ALL);

        // Compute restriction operator
        // TODO: computing the restriction operator could be merged with the selector to save some work
        // If we reuse the level we keep the previous restriction operator
        if (this->isReuseLevel() == false)
        {
            profileSubphaseComputeRestriction();
            this->Profile.tic("computeR");
            computeRestrictionOperator();
            this->Profile.toc("computeR");
        }

        profileSubphaseComputeCoarseA();
        this->Profile.tic("computeA");
        Ac.set_initialized(0);
        Ac.copyAuxData(&A);
        this->m_coarseAGenerator->computeAOperator(A, Ac, this->m_aggregates, this->m_R_row_offsets, this->m_R_column_indices, this->m_num_all_aggregates);
        Ac.setColsReorderedByColor(false);
        Ac.setView(FULL);
        this->Profile.toc("computeA");
        this->prepareNextLevelMatrix(A, Ac);
        A.setView(OWNED);
        Ac.setView(OWNED);
    }

    this->m_next_level_size = this->m_num_all_aggregates * Ac.get_block_dimy();

    if (this->m_print_aggregation_info)
    {
        MatrixAnalysis<TConfig> ana(&Ac);
        ana.aggregatesQuality2(this->m_aggregates, this->m_num_aggregates, A);
    }
}


// -------------------------------------------------------------
// Explicit instantiations
// -------------------------------------------------------------

#define AMGX_CASE_LINE(CASE) template class Aggregation_AMG_Level<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE
}

}
