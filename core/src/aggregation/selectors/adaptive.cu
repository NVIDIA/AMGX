#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <aggregation/selectors/adaptive.h>
#include <cutil.h>
#include <util.h>
#include <types.h>
#include <basic_types.h>
#include <matrix_analysis.h>
#include <transpose.h>

#include <async_event.h>

#include <thrust/count.h> //count
#include <thrust/sort.h> //sort
#include <thrust/binary_search.h> //lower_bound
#include <thrust/unique.h> //unique
#include <thrust/host_vector.h>
#include <cusp/detail/format_utils.h> //offsets_to_indices
#include <determinism_checker.h>
#include <hiprand.h>

namespace amgx
{
namespace aggregation
{
namespace adaptive
{

template <typename IndexType, typename ValueType>
__global__
void checkDiagonalDominance( const IndexType *row_offsets, const IndexType *col_indices, const ValueType *values, const IndexType *diag, IndexType *aggregates, int num_owned, int bsize, double diag_dom )
{
    //this time tid = i
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int bsize_sq = bsize * bsize;

    while ( i < num_owned )
    {
        bool dd = false;

        for ( int row = 0; row < bsize; row++ )
        {
            ValueType sum = 0.0;
            int iimax = row_offsets[i + 1];

            for ( int ii = row_offsets[i]; ii < iimax; ii++ )
            {
                //dont add the diagonal
                if ( col_indices[ii] == i )
                {
                    for ( int col = 0; col < bsize; col++ )
                    {
                        sum += fabs( values[ii * bsize_sq + bsize * row + col] );

                        if ( col == row )
                        {
                            continue;
                        }
                    }
                }
                else
                {
                    for ( int col = 0; col < bsize; col++ )
                    {
                        sum += fabs( values[ii * bsize_sq + bsize * row + col] );
                    }
                }
            }

            //check for DD condition and sort out
            dd = sum <= diag_dom * fabs( values[diag[i] * bsize_sq + row * bsize + row] );

            if ( !dd )
            {
                break;
            }
        }

        if ( dd )
        {
            aggregates[i] = num_owned;
        }

        i += gridDim.x * blockDim.x;
    }
}


template <typename IndexType, typename ValueType>
__global__
void assignNodeToBin( const ValueType *x, IndexType *aggregates, ValueType min, ValueType max, IndexType numRows, IndexType numAggregates )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    double range = (max - min) / numAggregates;

    while ( tid < numRows )
    {
        aggregates[tid] = floor( (x[tid] - min) / range );
        tid += gridDim.x * blockDim.x;
    }
}


template <typename ValueType, typename IndexType>
__global__
void rescaleVector( ValueType *x, IndexType numRows )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < numRows )
    {
        x[tid] = 2 * x[tid] - 1;
        tid += gridDim.x * blockDim.x;
    }
}

// -----------------
//    Methods
// ----------------

// Constructor
template<class T_Config>
AdaptiveSelectorBase<T_Config>::AdaptiveSelectorBase(AMG_Config &cfg, const std::string &cfg_scope)
{
    smoother = SolverFactory<T_Config>::allocate( cfg, cfg_scope, "smoother" );
}
/*
template <typename ValueType>
void curandGenerateUniformWrapper( hiprandGenerator_t generator, ValueType *outputPtr, size_t num );
template<> void curandGenerateUniformWrapper<float>( hiprandGenerator_t generator, float *outputPtr, size_t num )
{
    hiprandGenerateUniform( generator, outputPtr, num);
}
template<> void curandGenerateUniformWrapper<double>( hiprandGenerator_t generator, double *outputPtr, size_t num )
{
    hiprandGenerateUniformDouble( generator, outputPtr, num);
}
*/

template<class T_Config>
void AdaptiveSelectorBase<T_Config>::setAggregates(Matrix<T_Config> &A,
        IVector &aggregates, IVector &aggregates_global, int &numAggregates)
{
    FatalError("Adaptive selector is still in development", AMGX_ERR_NOT_IMPLEMENTED);
    /*
        if (A.get_block_dimx() == A.get_block_dimy())
        {
            IndexType numRows = A.get_num_rows();


            // This is how adaptive aggregation works:
            // 1. init random x vector (random values would preferably be between -1 and 1) and 0 rhs
            // 2. apply smoother a certain number of steps
            // 3. solve "binning problem":
            //      1. find min and max of x
            //      2. create N*coarseningrate bins with linear range
            //      3. assign nodes to bin, this is the final aggregates vector

            // allocate
            VVector& x = *Allocator<VVector>::allocate( numRows );
            VVector& rhs = *Allocator<VVector>::allocate( numRows );

            // initialize rhs
            thrust::fill( rhs.begin(), rhs.end(), 0.0 );

            //compute random numbers
            hiprandGenerator_t generator;
            hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
            curandGenerateUniformWrapper(generator, x.raw(), numRows);

            hipStream_t str = 0;
            const int threads_per_block = 256;
            const int num_blocks = min( AMGX_GRID_MAX_SIZE, (numRows-1)/threads_per_block + 1 );
            rescaleVector<<<num_blocks, threads_per_block, 0, str>>>( x.raw(), numRows );


            // smooth
            smoother->setup( A, false );
            smoother->set_max_iters( 15 ); //TODO: make this a parameter
            smoother->solve( x, rhs, false );

            //free rhs
            Allocator<VVector>::free( &rhs, numRows );

            //find min, max
            ValueType min = thrust::reduce( x.begin(), x.end(), 0.0, thrust::minimum<ValueType>() );
            ValueType max = thrust::reduce( x.begin(), x.end(), 0.0, thrust::maximum<ValueType>() );

            //bin
            aggregates.resize( numRows );
            numAggregates = numRows / 4;
            assignNodeToBin<<<num_blocks, threads_per_block, 0, str>>>( x.raw(), aggregates.raw(), min, max, numRows, numAggregates );

            //sync
            hipStreamSynchronize( str );
            cudaCheckError();

            //free
            Allocator<VVector>::free( &x, numRows );

            //fill "gaps" of empty bins
            this->renumberAndCountAggregates(aggregates, aggregates_global, numRows, numAggregates);
        }
        else
            FatalError("Unsupported block size for Adaptive Aggregation", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
        */
}

// -------------------------
//    Explict instantiations
// -------------------------
#define AMGX_CASE_LINE(CASE) template class AdaptiveSelectorBase<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE
#define AMGX_CASE_LINE(CASE) template class AdaptiveSelector<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

}
}
}
