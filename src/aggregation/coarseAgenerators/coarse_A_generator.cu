#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <types.h>
#include <aggregation/coarseAgenerators/coarse_A_generator.h>
#include <thrust/remove.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/count.h>

#include <thrust/extrema.h>

#include <assert.h>
#include <matrix.h>

namespace amgx
{

namespace aggregation
{

// ---------------------------------------------------------------------
// Method to print the distribution of number of nonzeros in matrix Ac
// ---------------------------------------------------------------------
template <class T_Config>
void CoarseAGenerator<T_Config>::printNonzeroStats(const typename Matrix<T_Config>::IVector &Ac_row_offsets, const int num_aggregates)
{
    // Printing the number of nonzeros per row
    Vector<typename TConfig::template setVecPrec<AMGX_vecBool>::Type> temporary(num_aggregates, 0);
    int max_nonzero = *thrust::max_element(Ac_row_offsets.begin(), Ac_row_offsets.end()) + 1;
    amgx_printf("\nnew level, max number of nonzeros per row = %d\n", max_nonzero);
    double *breakdown = new double[max_nonzero];

    for (int i = 0; i < max_nonzero; i++)
    {
        thrust::transform(Ac_row_offsets.begin(), Ac_row_offsets.end(), thrust::make_constant_iterator(i + 1), temporary.begin(), thrust::less<int>());
        breakdown[i] = 1.0 * (thrust::count(temporary.begin(), temporary.end(), true)) / num_aggregates;
        amgx_printf("Percentage of rows with less than %d nonzeros is %d\n", (i + 1), breakdown[i]);
    }

    delete[] breakdown;
}

template<class T_Config>
std::map<std::string, CoarseAGeneratorFactory<T_Config>*> &
CoarseAGeneratorFactory<T_Config>::getFactories( )
{
    static std::map<std::string, CoarseAGeneratorFactory<T_Config>*> s_factories;
    return s_factories;
}

template<class T_Config>
void CoarseAGeneratorFactory<T_Config>::registerFactory(std::string name, CoarseAGeneratorFactory<T_Config> *f)
{
    std::map<std::string, CoarseAGeneratorFactory<T_Config>*> &factories = getFactories( );
    typename std::map<std::string, CoarseAGeneratorFactory<T_Config> *>::const_iterator it = factories.find(name);

    if (it != factories.end())
    {
        std::string error = "CoarseAGeneratorFactory '" + name + "' has already been registered\n";
        FatalError(error.c_str(), AMGX_ERR_CORE);
    }

    factories[name] = f;
}

template<class T_Config>
void CoarseAGeneratorFactory<T_Config>::unregisterFactory(std::string name)
{
    std::map<std::string, CoarseAGeneratorFactory<T_Config>*> &factories = getFactories( );
    typename std::map<std::string, CoarseAGeneratorFactory<T_Config> *>::iterator it = factories.find(name);

    if (it == factories.end())
    {
        std::string error = "CoarseAGeneratorFactory '" + name + "' has not been registered\n";
        FatalError(error.c_str(), AMGX_ERR_CORE);
    }

    CoarseAGeneratorFactory<T_Config> *factory = it->second;
    assert( factory != NULL );
    delete factory;
    factories.erase(it);
}

template<class T_Config>
void CoarseAGeneratorFactory<T_Config>::unregisterFactories( )
{
    std::map<std::string, CoarseAGeneratorFactory<T_Config>*> &factories = getFactories( );
    typename std::map<std::string, CoarseAGeneratorFactory<T_Config> *>::iterator it = factories.begin( );

    for ( ; it != factories.end( ) ; )
    {
        CoarseAGeneratorFactory<T_Config> *factory = it->second;
        assert( factory != NULL );
        it++;
        delete factory;
    }

    factories.clear( );
}

template<class T_Config>
CoarseAGenerator<T_Config> *CoarseAGeneratorFactory<T_Config>::allocate(AMG_Config &cfg, const std::string &cfg_scope)
{
    std::map<std::string, CoarseAGeneratorFactory<T_Config>*> &factories = getFactories( );
    int agg_lvl_change = cfg.AMG_Config::getParameter<int>("fine_levels", cfg_scope);
    std::string generator;
    generator = cfg.getParameter<std::string>("coarseAgenerator", cfg_scope);
    typename std::map<std::string, CoarseAGeneratorFactory<T_Config> *>::const_iterator it = factories.find(generator);

    if (it == factories.end())
    {
        std::string error = "CoarseAGeneratorFactory '" + generator + "' has not been registered\n";
        FatalError(error.c_str(), AMGX_ERR_CORE);
    }

    return it->second->create(cfg, cfg_scope);
};

// ---------------------------------
// Explict instantiations
// ---------------------------------
#define AMGX_CASE_LINE(CASE) template class CoarseAGenerator<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

#define AMGX_CASE_LINE(CASE) template class CoarseAGeneratorFactory<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

}
}
