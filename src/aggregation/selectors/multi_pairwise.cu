#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <aggregation/selectors/multi_pairwise.h>
#include <cutil.h>
#include <util.h>
#include <types.h>
#include <basic_types.h>
#include <texture.h>
#include <matrix_analysis.h>
#include <transpose.h>

#include <async_event.h>

#include <thrust/count.h> //count
#include <thrust/sort.h> //sort
#include <thrust/binary_search.h> //lower_bound
#include <thrust/unique.h> //unique
#include <thrust/host_vector.h>
#include <cusp/detail/format_utils.h> //offsets_to_indices
#include <determinism_checker.h>
#include <solvers/solver.h>

#include <aggregation/coarseAgenerators/thrust_coarse_A_generator.h>
#include <aggregation/coarseAgenerators/low_deg_coarse_A_generator.h>

#include <omp.h>

#define EXPERIMENTAL_ITERATIVE_MATCHING

namespace amgx
{
namespace aggregation
{
namespace multi_pairwise
{

// include common routines for all selectors
#include <aggregation/selectors/common_selector.h>

// ------------------------
//    Kernels
// ------------------------

__device__
float random_weight2(int i, int j)
{
#define RAND_MULTIPLIER                 1145637293
    unsigned long i_min = (min(i, j) * RAND_MULTIPLIER);
    unsigned long i_max = (max(i, j) * RAND_MULTIPLIER);
    return ((float)i_min / i_max);
}

__device__
unsigned long random_weight3(int i, int j)
{
    unsigned long a;
    a = (i + j) ^ 8;
    a = (a + 0x7ed55d16) + (a << 12);
    a = (a ^ 0xc761c23c) + (a >> 19);
    a = (a + 0x165667b1) + (a << 5);
    a = (a ^ 0xd3a2646c) + (a << 9);
    a = (a + 0xfd7046c5) + (a << 3);
    a = (a ^ 0xb55a4f09) + (a >> 16);
    return a;
}


// findStrongestNeighbour kernel for block_dia_csr_matrix format
// Reads the weight from edge_weights array
template <typename IndexType, typename MatrixValueType>
__global__
void findStrongestNeighbourBlockDiaCsr_V2(const IndexType *row_offsets, const IndexType *column_indices,
        MatrixValueType *edge_weights, IndexType num_block_rows, IndexType *aggregates,
        IndexType *strongest_neighbour_1phase, IndexType *strongest_neighbour,
        const size_t bsize, int phase, int merge_singletons)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    MatrixValueType weight;
    int jcol;

    while (tid < num_block_rows)
    {
        int strongest_unaggregated = -1;
        int strongest_aggregated = -1;
        MatrixValueType     max_weight_unaggregated = 0.;
        MatrixValueType     max_weight_aggregated = 0.;

        if (aggregates[tid] == -1) // Unaggregated row
        {
            for (int j = row_offsets[tid]; j < row_offsets[tid + 1]; j++)
            {
                jcol = column_indices[j];

                if (phase == 1) { weight = edge_weights[j]; }
                else { weight = random_weight2(tid, jcol); }

                if (tid == jcol || jcol >= num_block_rows) { continue; }    // skip diagonal and halo

                if (phase == 2 && strongest_neighbour_1phase[jcol] != tid) { continue; } // if 2nd phase only accept those who gave a hand on the 1st phase

                // Identify strongest aggregated and unaggregated neighbours (method by multi_pairwise)
                if (aggregates[jcol] == -1 && weight > 0.0 && (weight > max_weight_unaggregated || (weight == max_weight_unaggregated && random_weight3(tid, jcol) > random_weight3(tid, strongest_unaggregated)))) // unaggregated
                {
                    max_weight_unaggregated = weight;
                    strongest_unaggregated = jcol;
                }
                else if (aggregates[jcol] != -1 && weight > 0.0 && (weight > max_weight_aggregated || (weight == max_weight_aggregated && random_weight3(tid, jcol) > random_weight3(tid, strongest_aggregated)))) // aggregated
                {
                    max_weight_aggregated = weight;
                    strongest_aggregated = jcol;
                }
            }

            if (strongest_unaggregated == -1 && strongest_aggregated != -1) // All neighbours are aggregated
            {
                if ( merge_singletons == 1 )
                    // Put in same aggregate as strongest neighbour
                {
                    aggregates[tid] = aggregates[strongest_aggregated];
                }
                else
                    // create singleton
                {
                    aggregates[tid] = tid;
                }
            }
            else if (strongest_unaggregated != -1)
            {
                if (phase == 2)
                {
                    MatrixValueType rand_w1 = random_weight2(tid, strongest_neighbour_1phase[tid]);
                    strongest_neighbour[tid] = max_weight_unaggregated > rand_w1 ? strongest_unaggregated : strongest_neighbour_1phase[tid];
                }
                else { strongest_neighbour_1phase[tid] = strongest_unaggregated; }
            }
            else
            {
                if (phase == 2) { strongest_neighbour[tid] = strongest_neighbour_1phase[tid]; }
                else { strongest_neighbour_1phase[tid] = tid; }
            }
        }

        tid += gridDim.x * blockDim.x;
    }
}


template <typename IndexType>
__device__
bool atomicJoin( IndexType node, IndexType aggregate, IndexType *aggregates, int *sizes, int allowed )
{
    int mySize = sizes[node];
    int theirSize = sizes[aggregate];
    int theirSizeOld = theirSize;

    do
    {
        int newSize = mySize + theirSize;

        if ( newSize > allowed )
        {
            return false;
        }

        theirSizeOld = theirSize;
        theirSize = atomicCAS( &sizes[aggregate], theirSize, newSize );
    }
    while ( theirSize != theirSizeOld );

    aggregates[node] = aggregate;
    return true;
}

template <typename IndexType, typename MatrixValueType, bool use_degree>
__global__
void findStrongestNeighbourBlockDiaCsr_V3(const IndexType *row_offsets,
        const IndexType *column_indices,
        MatrixValueType *edge_weights,
        IndexType num_block_rows,
        IndexType *aggregates,
        IndexType *strongest_neighbour,
        int *sizes,
        int *degree,
        const size_t bsize,
        int max_aggregate_size,
        int merge_singletons)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    MatrixValueType weight;
    int jcol;

    while (tid < num_block_rows)
    {
        int strongest_unaggregated = -1;
        int strongest_aggregated = -1;
        int lowest_degree;

        if ( use_degree )
        {
            lowest_degree = degree[tid];    //only interested in finding lower degree than self
        }
        else
        {
            lowest_degree = 0;    //if we decide to not use degree than just propose to the strongest edge
        }

        int lowest_degree_neighbor = tid;
        MatrixValueType     lowest_degree_weight = 1e100; //high start value, so that same degree neighbor won't become lowest degree neighbor
        MatrixValueType     max_weight_unaggregated = 0.;
        MatrixValueType     max_weight_aggregated = 0.;
        int mySize;

        if ( merge_singletons == 2 )
        {
            mySize = sizes[tid];
        }
        else
        {
            mySize = 0;
        }

        if ( merge_singletons != 2 )
        {
            max_aggregate_size = 100000;
        }

        //this aggregate is already full
        if (mySize == max_aggregate_size)
        {
            aggregates[tid] = tid;
        }

        if (aggregates[tid] == -1) // Unaggregated row
        {
            for (int j = row_offsets[tid]; j < row_offsets[tid + 1]; j++)
            {
                jcol = column_indices[j];

                if (tid == jcol || jcol >= num_block_rows) { continue; }    // skip diagonal and halo

                weight = edge_weights[j];

                if (weight <= 0.0) { continue; }

                if ( aggregates[jcol] != -1 ) //aggregated neighbor
                {
                    int theirSize;

                    if ( merge_singletons == 2 )
                    {
                        theirSize = aggregates[sizes[jcol]];
                    }
                    else
                    {
                        theirSize = 0;
                    }

                    //if all neighbors are aggregated, find the strongest edge to neighbor aggregate that is not full yet
                    if (mySize + theirSize <= max_aggregate_size &&
                            (weight > max_weight_aggregated)) // aggregated
                    {
                        max_weight_aggregated = weight;
                        strongest_aggregated = jcol;
                    }
                }
                else //unaggregated neighbor
                {
                    if ( use_degree && merge_singletons == 2 )
                    {
                        int theirSize = sizes[jcol];
                        //get lowest degree neighbor or find out that there is no lower degree neighbor
                        int current_degree = degree[jcol];

                        if ( mySize + theirSize <= max_aggregate_size && (current_degree < lowest_degree || current_degree == lowest_degree && weight > lowest_degree_weight) )
                        {
                            lowest_degree = current_degree;
                            lowest_degree_weight = weight;
                            lowest_degree_neighbor = jcol;
                        }

                        //get highest weight neighbor
                        if ( mySize + theirSize <= max_aggregate_size && (weight > max_weight_unaggregated) )
                        {
                            strongest_unaggregated = jcol;
                            max_weight_unaggregated = weight;
                        }
                    }

                    if ( use_degree && merge_singletons != 2 ) //same as above but ignore sizes
                    {
                        //get lowest degree neighbor or find out that there is no lower degree neighbor
                        int current_degree = degree[jcol];

                        if ( current_degree < lowest_degree || current_degree == lowest_degree && weight > lowest_degree_weight)
                        {
                            lowest_degree = current_degree;
                            lowest_degree_weight = weight;
                            lowest_degree_neighbor = jcol;
                        }

                        //get highest weight neighbor
                        if (weight > max_weight_unaggregated)
                        {
                            strongest_unaggregated = jcol;
                            max_weight_unaggregated = weight;
                        }
                    }

                    if ( !use_degree && merge_singletons == 2 )
                    {
                        //get highest weight neighbor only but pay attention to the aggregate sizes
                        int theirSize = sizes[jcol];                        //get highest weight neighbor

                        if ( mySize + theirSize <= max_aggregate_size && (weight > max_weight_unaggregated) )
                        {
                            strongest_unaggregated = jcol;
                            max_weight_unaggregated = weight;
                        }
                    }

                    if ( !use_degree && merge_singletons != 2 )
                    {
                        //just highest weight
                        if (weight > max_weight_unaggregated)
                        {
                            strongest_unaggregated = jcol;
                            max_weight_unaggregated = weight;
                        }
                    }
                }
            }

            //prefer lowest degree neighbor
            if ( lowest_degree_neighbor != tid )
            {
                strongest_unaggregated = lowest_degree_neighbor;
            }

            if (strongest_unaggregated != -1) //Unaggregated neighbor exists
            {
                strongest_neighbour[tid] = strongest_unaggregated;    //assign strongest aggregated
            }

            if (strongest_unaggregated == -1 && strongest_aggregated != -1) // All neighbours are aggregated but small enough aggregated neighbors exist
            {
                if ( merge_singletons == 0 )
                {
                    aggregates[tid] = tid;
                }

                if ( merge_singletons == 1 )
                {
                    aggregates[tid] = aggregates[strongest_aggregated];
                }

                if ( merge_singletons == 2)
                {
                    atomicJoin( tid, aggregates[strongest_aggregated], aggregates, sizes, max_aggregate_size );    //try to join, can fail. maybe it works next round.
                }
            }

            if (strongest_unaggregated == -1 && strongest_aggregated == -1) //no feasable neighbor at all, become singleton
            {
                strongest_neighbour[tid] = tid;    //become singleton
            }
        }

        tid += gridDim.x * blockDim.x;
    }
}

template <typename IndexType, typename ValueType>
__global__
void computeDegree( const IndexType *ia, const IndexType *ja, const ValueType *weights, IndexType *aggregates, IndexType *sizes, IndexType *degree, IndexType numRows, IndexType max_aggregate_size)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    while ( i < numRows )
    {
        int myDegree = 0;
        IndexType ia_ip1 = ia[i + 1];

        for ( IndexType ii = ia[i]; ii < ia_ip1; ii++ )
        {
            IndexType j = ja[ii];

            if ( j == i )
            {
                continue;
            }

            int mySize, theirSize;

            if ( sizes != NULL )
            {
                mySize = sizes[i];
                theirSize = sizes[j];
            }
            else
            {
                mySize = theirSize = 0;
            }

            if ( weights[ii] > 0.0 && aggregates[j] == -1 && mySize + theirSize <= max_aggregate_size )
            {
                myDegree++;
            }
        }

        degree[i] = myDegree;
        i += gridDim.x * blockDim.x;
    }
}


template <typename IndexType, typename ValueType>
__global__
void mergeSingletonsSmart(const IndexType *ia, const IndexType *ja, const ValueType *weights, IndexType *aggregates, IndexType *sizes, IndexType numRows, int max_aggregate_size)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < numRows )
    {
        //unaggregated nodes try to join or create their own aggregate
        if ( aggregates[tid] == -1 )
        {
            bool joined = false;

            while ( !joined )
            {
                int neighbor_aggregate = -1;
                ValueType max_weight = 0.0;
                IndexType mySize = sizes[tid];

                for (IndexType ii = ia[tid]; ii < ia[tid + 1]; ii++)
                {
                    IndexType j = ja[ii];

                    if (j == tid || j >= numRows) { continue; }

                    if ( aggregates[j] != -1 && sizes[aggregates[j]] + mySize <= max_aggregate_size && weights[ii] > max_weight )
                    {
                        neighbor_aggregate = aggregates[j];
                        max_weight = weights[ii];
                    }
                }

                //no possible neighbor found
                if ( neighbor_aggregate == -1 )
                {
                    //create own aggregate
                    aggregates[tid] = tid;
                    joined = true;
                }
                else
                {
                    //try to join
                    joined = atomicJoin( tid, neighbor_aggregate, aggregates, sizes, max_aggregate_size );
                }
            }
        }

        tid += gridDim.x * blockDim.x;
    }
}


template <typename IndexType>
__global__
void updateAggregateSizes( IndexType *sizesSource, IndexType *sizes, IndexType *aggregates, IndexType numRows )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < numRows )
    {
        IndexType agg = aggregates[tid];
        IndexType aggregateSize = sizes[agg];
        IndexType mySize = sizesSource[tid];

        while ( mySize > aggregateSize )
        {
            aggregateSize = atomicCAS( &sizes[agg], aggregateSize, mySize );
        }

        tid += gridDim.x * blockDim.x;
    }
}


// Kernel that checks if perfect matchs exist
template <typename IndexType>
__global__
void matchEdges(const IndexType num_rows, IndexType *aggregates, int *strongest_neighbour, IndexType *sizes)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int potential_match, potential_match_neighbour;

    while (tid < num_rows)
    {
        if (aggregates[tid] == -1) // Unaggregated row
        {
            potential_match = strongest_neighbour[tid];
            potential_match_neighbour = strongest_neighbour[potential_match];

            if ( potential_match == tid )
            {
                aggregates[tid] = tid;
            }
            else if (potential_match != -1 && potential_match_neighbour == tid && tid < potential_match) // we have a match
            {
                aggregates[tid] = tid;
                aggregates[potential_match] = tid;

                if ( sizes != NULL)
                {
                    sizes[tid] += sizes[potential_match];
                }
            }
        }

        tid += gridDim.x * blockDim.x;
    }
}

template <typename IndexType, int block_size>
__global__
void countAggregates(const IndexType num_rows, IndexType *aggregates, int *num_unaggregated)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int c = 0;
    int i = tid;

    while ( i < num_rows )
    {
        c += ( aggregates[i] == -1 );
        i += gridDim.x * blockDim.x;
    }

    __shared__ volatile int smem[block_size];
    smem[threadIdx.x] = c;
    __syncthreads();

    for ( int off = blockDim.x / 2; off >= 32; off = off / 2 )
    {
        if ( threadIdx.x < off )
        {
            smem[threadIdx.x] += smem[threadIdx.x + off];
        }

        __syncthreads();
    }

    // warp reduce
    if ( threadIdx.x < 32 )
    {
        smem[threadIdx.x] += smem[threadIdx.x + 16];
        smem[threadIdx.x] += smem[threadIdx.x + 8];
        smem[threadIdx.x] += smem[threadIdx.x + 4];
        smem[threadIdx.x] += smem[threadIdx.x + 2];
        smem[threadIdx.x] += smem[threadIdx.x + 1];
    }

    if ( threadIdx.x == 0 )
    {
        atomicAdd(num_unaggregated, smem[0]);
    }
}

template <typename IndexType>
__global__
void joinExistingAggregates(IndexType num_rows, IndexType *aggregates, IndexType *aggregates_candidate)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while (tid < num_rows)
    {
        if (aggregates[tid] == -1 && aggregates_candidate[tid] != -1) // Unaggregated row
        {
            aggregates[tid] = aggregates_candidate[tid];
        }

        tid += gridDim.x * blockDim.x;
    }
}

// Kernel that merges unaggregated vertices its strongest aggregated neighbour
// Weights are read from edge_weights array
// For block_dia_csr_matrix_format
template <typename IndexType, typename MatrixValueType>
__global__
void mergeWithExistingAggregatesBlockDiaCsr_V2(const IndexType *row_offsets, const IndexType *column_indices, const MatrixValueType *edge_weights,
        const int num_block_rows, IndexType *aggregates, int bsize, const int deterministic, IndexType *aggregates_candidate)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int jcol;
    MatrixValueType weight;

    while (tid < num_block_rows)
    {
        MatrixValueType max_weight_aggregated = 0.;
        int strongest_aggregated = -1;

        if (aggregates[tid] == -1) // Unaggregated row
        {
            for (int j = row_offsets[tid]; j < row_offsets[tid + 1]; j++)
            {
                // Compute edge weight
                weight = edge_weights[j];
                jcol = column_indices[j];

                if (jcol == tid || jcol >= num_block_rows) { continue; }    // skip diagonal

                if ( aggregates[jcol] == num_block_rows ) { continue; } // skip dd rows

                // Identify strongest aggregated neighbour
                if (aggregates[jcol] != -1 && weight > 0 && (weight > max_weight_aggregated || (weight == max_weight_aggregated && random_weight3( tid, jcol ) > random_weight3( tid, strongest_aggregated )))) //
                {
                    max_weight_aggregated = weight;
                    strongest_aggregated = jcol;
                }
            }

            if (strongest_aggregated != -1) // Found a neighbour to aggregate to
            {
                if (deterministic == 1)
                {
                    aggregates_candidate[tid] = aggregates[strongest_aggregated];
                }
                else
                {
                    // Put in same aggregate as strongest neighbour
                    aggregates[tid] = aggregates[strongest_aggregated];
                }
            }
            else // All neighbours are unaggregated, leave alone
            {
                if (deterministic == 1)
                {
                    aggregates_candidate[tid] = tid;
                }
                else
                {
                    aggregates[tid] = tid;
                }
            }
        }

        tid += gridDim.x * blockDim.x;
    }
}

// Kernel to extract diagonal for csr_matrix format
template <typename IndexType, typename ValueType>
__global__
void getDiagonalKernel(const IndexType *offsets, const IndexType *column_indices,
                       const ValueType *values, const IndexType numRows, ValueType *diagonal)
{
    int tIdx = threadIdx.x + blockDim.x * blockIdx.x;

    while (tIdx < numRows)
    {
        const int offset = offsets[tIdx];
        const int numj = offsets[tIdx + 1] - offset;

        for (int j = offset; j < offset + numj; j++)
        {
            int jcol = column_indices[j];

            if (tIdx == jcol)
            {
                diagonal[tIdx] = values[j];
            }
        }

        tIdx += gridDim.x * blockDim.x;
    }
}

// Kernel to extract diagonal for csr_matrix format
template <typename IndexType, typename ValueType>
__global__
void getDiagonalKernelNoDiaProp(const IndexType *dia_idx, const ValueType *values, const IndexType numRows, ValueType *diagonal)
{
    int tIdx = threadIdx.x + blockDim.x * blockIdx.x;

    while (tIdx < numRows)
    {
        diagonal[tIdx] = values[dia_idx[tIdx]];
        tIdx += gridDim.x * blockDim.x;
    }
}

// filter edge weights like this:
// set w_ij = 0 iff
// w_ij < alpha * sqrt( max_k{w_ik} * max_l{w_jl} )
// alpha is some constant, 0.25 or 0.5 should work fine
template<typename IndexType, typename ValueType>
__global__
void filterWeights( const IndexType *row_offsets, const IndexType *row_indices, const IndexType *col_indices, const IndexType *diag, const ValueType *old_weights, ValueType *new_weights, IndexType num_nonzero_blocks, IndexType num_owned, ValueType alpha )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int i, j, kmin, kmax;
    ValueType max_ik, max_jl;

    while ( tid < num_nonzero_blocks )
    {
        i = row_indices[tid];
        j = col_indices[tid];

        if ( i != j && j < num_owned )
        {
            //find max_k{w_ik}
            kmin = row_offsets[i];
            kmax = row_offsets[i + 1];
            max_ik = 0.0;

            for (int k = kmin; k < kmax; k++)
            {
                if ( col_indices[k] != i && old_weights[k] > max_ik )
                {
                    max_ik = old_weights[k];
                }
            }

            //find max_l{w_jl}
            kmin = row_offsets[j];
            kmax = row_offsets[j + 1];
            max_jl = 0.0;

            for (int l = kmin; l < kmax; l++)
            {
                if ( col_indices[l] != j && old_weights[l] > max_jl )
                {
                    max_jl = old_weights[l];
                }
            }

            //test squared inequality
            if ( old_weights[tid] * old_weights[tid] < alpha * alpha * max_ik * max_jl )
            {
                new_weights[tid] = 0.0;
            }
            else //rescale to relative importance. this should also increase the chance of a handshake
            {
                new_weights[tid] = old_weights[tid];
            }

            //              new_weights[tid] = old_weights[tid] / sqrt(max_ik*max_jl);
        }

        tid += gridDim.x * blockDim.x;
    }
}


template<typename IndexType, typename ValueType>
__global__
void gatherValuesInterleaved( const ValueType *inValues, ValueType *outValues, IndexType nnz, int sq_blocksize, int index_offset )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < nnz )
    {
        //at least the write is coalesced
        outValues[tid] = inValues[tid * sq_blocksize + index_offset];
        tid += gridDim.x * blockDim.x;
    }
}


template<typename IndexType, typename ValueTypeV, typename ValueTypeM>
__global__
void addToWeights( ValueTypeM *edge_weights, const ValueTypeV *x, const IndexType *row_indices, IndexType *col_indices, IndexType nnz, double scale )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < nnz )
    {
        int i = row_indices[tid];
        int j = col_indices[tid];
        edge_weights[tid] -= static_cast<ValueTypeM>( scale * fabs( x[i] - x[j] ) );
        tid += gridDim.x * blockDim.x;
    }
}

template <typename ValueType, typename IndexType>
__global__
void rescaleVector( ValueType *x, IndexType numRows )
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < numRows )
    {
        x[tid] = 2 * x[tid] - 1;
        tid += gridDim.x * blockDim.x;
    }
}

// -----------------
//    Methods
// ----------------

// Constructor
template<class T_Config>
MultiPairwiseSelectorBase<T_Config>::MultiPairwiseSelectorBase(AMG_Config &cfg, const std::string &cfg_scope)
{
    deterministic = cfg.AMG_Config::getParameter<IndexType>("determinism_flag", "default");
    max_iterations = cfg.AMG_Config::getParameter<IndexType>("max_matching_iterations", cfg_scope);
    numUnassigned_tol = cfg.AMG_Config::getParameter<double>("max_unassigned_percentage", cfg_scope);
    two_phase = cfg.AMG_Config::getParameter<int>("handshaking_phases", cfg_scope) == 2;
    m_aggregation_edge_weight_component = cfg.AMG_Config::getParameter<int>("aggregation_edge_weight_component", cfg_scope);
    aggregation_passes = cfg.AMG_Config::getParameter<int>("aggregation_passes", cfg_scope); //default to size 8 aggregates. maybe its more convenient to have that as a config parameter
    filter_weights = cfg.AMG_Config::getParameter<int>("filter_weights", cfg_scope); //by default: no filtering
    filter_weights_alpha = cfg.AMG_Config::getParameter<double>( "filter_weights_alpha", cfg_scope ); //default to 0.25
    full_ghost_level = cfg.AMG_Config::getParameter<int>( "full_ghost_level", cfg_scope ); //defaults to 0
    notay_weights = cfg.AMG_Config::getParameter<int>( "notay_weights", cfg_scope ); //defaults to 0
    ghost_offdiag_limit = cfg.AMG_Config::getParameter<int>( "ghost_offdiag_limit", cfg_scope ); //defaults to 0
    merge_singletons = cfg.AMG_Config::getParameter<int>( "merge_singletons", cfg_scope ); //defaults to 1
    weight_formula = cfg.AMG_Config::getParameter<int>( "weight_formula", cfg_scope ); //wheight formula defaults to 0
    serial_matching = cfg.AMG_Config::getParameter<int>( "serial_matching", cfg_scope ) != 0; //will use a serial matching algorithm instead of handshake
    modified_handshake = cfg.AMG_Config::getParameter<int>("modified_handshake", cfg_scope ) == 1;
    //passes = 1 -> max = 3
    //passes = 2 -> max = 5
    //passes = 3 -> max = 10
    //passes = 4 -> max = 18
    max_aggregate_size = 2;

    for (int i = 1; i < aggregation_passes; i ++)
    {
        max_aggregate_size *= 2;
    }

    max_aggregate_size += aggregation_passes - (aggregation_passes / 2);
    mCfg = cfg;
    mCfg_scope = cfg_scope;
}

// setAggregates for block_dia_csr_matrix_h format
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MultiPairwiseSelector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::setAggregates_common_sqblocks(Matrix_h &A,
        typename Matrix_h::IVector &aggregates, typename Matrix_h::IVector &aggregates_global, int &num_aggregates, MVector &edge_weights, IVector &sizes)
{
    FatalError("MultiPairwise selector: setAggregates not implemented on CPU, exiting", AMGX_ERR_NOT_SUPPORTED_TARGET);
}

// device specialization

//edge_weights is an in/out parameter:
//if its size is zero, the edge_weights will be computed from A and stored into edge_weights
//else the edge_weights will not be computed and assumed to be valid for the given A. the value array of A is not used in this case
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MultiPairwiseSelector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::setAggregates_common_sqblocks(Matrix_d &A,
        typename Matrix_d::IVector &aggregates,
        typename Matrix_d::IVector &aggregates_global,
        int &num_aggregates,
        MVector &edge_weights,
        IVector &sizes)
{
    IndexType num_block_rows = (int) A.get_num_rows();
    IndexType num_nonzero_blocks = (int) A.get_num_nz();
    // both ways are supported
    IndexType total_nz = (A.is_matrix_singleGPU()) ? num_nonzero_blocks : A.manager->num_nz_all();
    typename Matrix_d::IVector &row_indices = A.row_indices;
    row_indices.resize( total_nz);
    cusp::detail::offsets_to_indices(A.row_offsets, row_indices);
    IndexType total_rows = (A.is_matrix_singleGPU()) ? A.get_num_rows() : A.manager->num_rows_all();
    aggregates.resize(total_rows);
    thrust::fill(aggregates.begin(), aggregates.end(), -1);
    cudaCheckError();

    if ( this->merge_singletons == 2 && sizes.size() == 0 )
    {
        sizes.resize( total_rows, 1 );    //init with all ones
    }

    const IndexType *A_row_offsets_ptr = A.row_offsets.raw();
    const IndexType *A_row_indices_ptr = row_indices.raw();
    const IndexType *A_column_indices_ptr = A.col_indices.raw();
    const IndexType *A_dia_idx_ptr = A.diag.raw();
    const ValueType *A_nonzero_values_ptr = A.values.raw();
    typename Matrix_d::IVector strongest_neighbour(num_block_rows, -1);
    typename Matrix_d::IVector strongest_neighbour_1phase(num_block_rows, -1);
    Vector<TemplateConfig<AMGX_device, AMGX_vecUInt, t_matPrec, t_indPrec> > aggregated(num_block_rows, 0);
    IndexType *strongest_neighbour_ptr = strongest_neighbour.raw();
    IndexType *strongest_neighbour_1phase_ptr = strongest_neighbour_1phase.raw();
    IndexType *aggregates_ptr = aggregates.raw();
    const int threads_per_block = 256;
    const int num_blocks = min( AMGX_GRID_MAX_SIZE, (num_block_rows - 1) / threads_per_block + 1 );
    int numUnassigned = num_block_rows;
    int numUnassigned_previous = numUnassigned;
    bool computeWeights = ( edge_weights.size() == 0 );

    if (computeWeights)
    {
        if ( A.hasProps( DIAG ) )
        {
            edge_weights.resize( num_nonzero_blocks + num_block_rows, 0.0 );
        }
        else
        {
            edge_weights.resize( num_nonzero_blocks + 1, -1 );    //+1 is important to some algorithms
        }
    }

    ValueType *edge_weights_ptr = edge_weights.raw();
    ValueType *rand_edge_weights_ptr = NULL;
    hipStream_t str = thrust::global_thread_handle::get_stream();

    // Compute the edge weights
    if ( computeWeights )
    {
        const int num_blocks_V2 = min( AMGX_GRID_MAX_SIZE, (num_nonzero_blocks - 1) / threads_per_block + 1);
        //compute with std formula
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(computeEdgeWeightsBlockDiaCsr_V2<IndexType), ValueType, ValueType>, hipFuncCachePreferL1);
        computeEdgeWeightsBlockDiaCsr_V2 <<< num_blocks_V2, threads_per_block, 0, str>>>(A_row_offsets_ptr,
                A_row_indices_ptr,
                A_column_indices_ptr,
                A_dia_idx_ptr,
                A_nonzero_values_ptr,
                num_nonzero_blocks,
                edge_weights_ptr,
                rand_edge_weights_ptr,
                num_block_rows,
                A.get_block_dimy(),
                this->m_aggregation_edge_weight_component,
                this->weight_formula);
        cudaCheckError();
    }

    //filter weights if desired
    if ( this->filter_weights == 1 )
    {
        MVector tmp( edge_weights.size() );
        const int num_blocks_filter = min( AMGX_GRID_MAX_SIZE, (num_nonzero_blocks - 1) / threads_per_block + 1);
        hipStreamSynchronize(str);
        cudaCheckError();
        filterWeights <<< num_blocks_filter, threads_per_block, 0, str>>>( A_row_offsets_ptr,
                A_row_indices_ptr,
                A_column_indices_ptr,
                A_dia_idx_ptr,
                edge_weights_ptr,
                tmp.raw(),
                num_nonzero_blocks,
                num_block_rows,
                this->filter_weights_alpha);
        hipStreamSynchronize(str);
        cudaCheckError();
        tmp.swap( edge_weights );
        edge_weights_ptr = edge_weights.raw();
    }

// compute matching
    if ( !this->serial_matching )
    {
        IVector degree;

        if ( this->modified_handshake )
        {
            degree.resize( num_block_rows );
        }

#ifdef EXPERIMENTAL_ITERATIVE_MATCHING
        // TODO: allocate host pinned memory
        AsyncEvent *throttle_event = new AsyncEvent;
        throttle_event->create();
        typename Matrix_h::IVector h_unagg_vec(1);
        typename Matrix_d::IVector d_unagg_vec(1);
        int *unaggregated = h_unagg_vec.raw();
        int *d_unaggregated = d_unagg_vec.raw();
#endif
        int icount, s = 1;
        {
            icount = 0;
            ValueType *weights_ptr = edge_weights_ptr;

            do
            {
                if ( !this->two_phase )
                {
                    if ( this->modified_handshake )
                        computeDegree <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr,
                                A_column_indices_ptr,
                                weights_ptr,
                                aggregates_ptr,
                                sizes.raw(),
                                degree.raw(),
                                num_block_rows,
                                this->max_aggregate_size );

                    // 1-phase handshaking
                    if ( this->modified_handshake )
                        findStrongestNeighbourBlockDiaCsr_V3<IndexType, ValueType, true>
                        <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr,
                                A_column_indices_ptr,
                                weights_ptr,
                                num_block_rows,
                                aggregates_ptr,
                                strongest_neighbour_ptr,
                                sizes.raw(),
                                degree.raw(),
                                A.get_block_dimy(),
                                this->max_aggregate_size,
                                this->merge_singletons);
                    else
                        findStrongestNeighbourBlockDiaCsr_V3<IndexType, ValueType, false>
                        <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr,
                                A_column_indices_ptr,
                                weights_ptr,
                                num_block_rows,
                                aggregates_ptr,
                                strongest_neighbour_ptr,
                                sizes.raw(),
                                degree.raw(),
                                A.get_block_dimy(),
                                this->max_aggregate_size,
                                this->merge_singletons);

                    cudaCheckError();
                }
                else
                {
                    // 2-phase handshaking
                    findStrongestNeighbourBlockDiaCsr_V2 <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr, A_column_indices_ptr, weights_ptr, num_block_rows, aggregates_ptr, strongest_neighbour_1phase_ptr, strongest_neighbour_ptr, A.get_block_dimy(), 1, this->merge_singletons);
                    cudaCheckError();
                    // 2nd phase: for each block_row, find the strongest neighbour among those who gave hand on 1st phase
                    findStrongestNeighbourBlockDiaCsr_V2 <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr, A_column_indices_ptr, weights_ptr, num_block_rows, aggregates_ptr, strongest_neighbour_1phase_ptr, strongest_neighbour_ptr, A.get_block_dimy(), 2, this->merge_singletons);
                    cudaCheckError();
                }

                // Look for perfect matches. Also, for nodes without unaggregated neighbours, merge with aggregate containing strongest neighbour
                if ( this->merge_singletons == 2 )
                {
                    matchEdges <<< num_blocks, threads_per_block, 0, str>>>(num_block_rows, aggregates_ptr, strongest_neighbour_ptr, sizes.raw());
                }
                else
                {
                    matchEdges <<< num_blocks, threads_per_block, 0, str>>>(num_block_rows, aggregates_ptr, strongest_neighbour_ptr, (int *)NULL);
                }

                cudaCheckError();
#ifdef EXPERIMENTAL_ITERATIVE_MATCHING
                s = (icount & 1);

                if ( s == 0 )
                {
                    // count unaggregated vertices
                    hipMemsetAsync(d_unaggregated, 0, sizeof(int), str);
                    countAggregates<IndexType, threads_per_block> <<< num_blocks, threads_per_block, 0, str>>>(num_block_rows, aggregates_ptr, d_unaggregated);
                    cudaCheckError();
                    hipMemcpyAsync(unaggregated, d_unaggregated, sizeof(int), hipMemcpyDeviceToHost, str);
                    throttle_event->record(str);
                }
                else
                {
                    throttle_event->sync();
                    numUnassigned_previous = numUnassigned;
                    numUnassigned = *unaggregated;
                }

#else
                hipStreamSynchronize(str);
                numUnassigned_previous = numUnassigned;
                numUnassigned = (int)thrust::count(aggregates.begin(), aggregates.begin() + num_block_rows, -1);
                cudaCheckError();
#endif
                icount++;
            }
            while ( (s == 0) || !(numUnassigned == 0 || icount > this->max_iterations || 1.0 * numUnassigned / num_block_rows < this->numUnassigned_tol || numUnassigned == numUnassigned_previous));

        }
        //    printf("%i,\n", icount);
#ifdef EXPERIMENTAL_ITERATIVE_MATCHING
        delete throttle_event;
#endif
    }
    else
    {
        computeMatchingSerialGreedy( A, aggregates, num_aggregates, edge_weights );
    }

    if ( this->merge_singletons == 1 )
    {
        // Merge remaining vertices with current aggregates
        if (this->deterministic != 1)
        {
            while (numUnassigned != 0)
            {
                mergeWithExistingAggregatesBlockDiaCsr_V2 <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr, A_column_indices_ptr, edge_weights_ptr, num_block_rows, aggregates_ptr, A.get_block_dimy(), this->deterministic, (IndexType *) NULL);
                cudaCheckError();
                numUnassigned = (int)thrust::count(aggregates.begin(), aggregates.begin() + num_block_rows, -1);
                cudaCheckError();
            }
        }
        else
        {
            typename Matrix_d::IVector aggregates_candidate(num_block_rows, -1);

            while (numUnassigned != 0)
            {
                mergeWithExistingAggregatesBlockDiaCsr_V2 <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr, A_column_indices_ptr, edge_weights_ptr, num_block_rows, aggregates_ptr, A.get_block_dimy(), this->deterministic, aggregates_candidate.raw());
                cudaCheckError();
                joinExistingAggregates <<< num_blocks, threads_per_block, 0, str>>>(num_block_rows, aggregates_ptr, aggregates_candidate.raw());
                cudaCheckError();
                numUnassigned = (int)thrust::count(aggregates.begin(), aggregates.begin() + num_block_rows, -1);
                cudaCheckError();
            }

            aggregates_candidate.resize(0);
        }
    }
    else if (this->merge_singletons == 0 )
    {
        //make singletons
        aggregateSingletons <<< num_blocks, threads_per_block, 0, str>>>( aggregates_ptr, num_block_rows );
        cudaCheckError();
    }
    else if ( this->merge_singletons == 2 )
    {
        //merges all remaining singletons into adequate neighbors if possible
        mergeSingletonsSmart <<< num_blocks, threads_per_block, 0, str>>>(A_row_offsets_ptr,
                A_column_indices_ptr,
                edge_weights_ptr,
                aggregates_ptr,
                sizes.raw(),
                num_block_rows,
                this->max_aggregate_size);
        cudaCheckError();
    }

    //This will assign num_aggregates to the pseudo aggregate without counting it. Perfect!
    this->renumberAndCountAggregates(aggregates, aggregates_global, num_block_rows, num_aggregates);

    if ( this->merge_singletons == 2 )
    {
        //udpate the sizes vector, so it matches the renumbered aggregates size
        IVector sizesSource;
        sizesSource.swap( sizes );
        sizes.resize( num_aggregates, 1 );
        updateAggregateSizes <<< num_blocks, threads_per_block, 0, str>>>( sizesSource.raw(), sizes.raw(), aggregates_ptr, num_block_rows );
        cudaCheckError();
    }

}


//instead of a handshake, we use a serial greedy algorithm to compute a better matching
//the algorithm:
// 1. compute degree of every node and sort nodes by degree into double linked list
// 2. while non-isolated nodes left:
//        take node with minimum degree > 0
//        find strongest edge to unaggregated node and assign to new aggregate
//        remove both nodes from linked list
//        decrease degree of each neighbor by one for each of the two nodes
//        update list
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MultiPairwiseSelector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::computeMatchingSerialGreedy( const Matrix_d &A, IVector &aggregates, int &numAggregates, MVector &edge_weights)
{
    IndexType numRows = A.row_offsets.size() - 1;
    IndexType nnz = A.col_indices.size();
    //allocate memory on host
    IndexType *ia = new IndexType[numRows + 1];
    IndexType *ja = new IndexType[nnz];
    ValueType *w = new ValueType[nnz];
    IndexType *agg = new IndexType[numRows];
    IndexType *deg = new IndexType[numRows];
    //copy
    hipMemcpy( ia, A.row_offsets.raw(), sizeof(IndexType) * (numRows + 1), hipMemcpyDeviceToHost );
    hipMemcpy( ja, A.col_indices.raw(), sizeof(IndexType)*nnz, hipMemcpyDeviceToHost );
    hipMemcpy( w, edge_weights.raw(), sizeof(ValueType)*nnz, hipMemcpyDeviceToHost );
    //init agg and compute the degree of each aggregate
    int max_degree = 0;

    for (IndexType i = 0; i < numRows; i++)
    {
        agg[i] = -1;
        int degree = 0;

        for (IndexType ii = ia[i]; ii < ia[i + 1]; ii++) //only care for positive weights
        {
            if ( ja[ii] != i && w[ii] > 0.0 )
            {
                degree++;
            }
        }

        if ( degree > max_degree )
        {
            max_degree = degree;
        }

        deg[i] = degree;
    }

    if ( max_degree >= numRows )
    {
        FatalError( "max degree is greater than numRows.", AMGX_ERR_UNKNOWN );
    }

    //init double linked list
    IndexType *fwd = new IndexType[numRows + max_degree + 1];
    IndexType *bwd = new IndexType[numRows + max_degree + 1];

    for (IndexType i = 0; i < numRows + max_degree + 1; i++)
    {
        fwd[i] = i;
        bwd[i] = i;
    }

    IndexType nodesLeft = numRows;
    numAggregates = 0;

    //insert nodes into list
    for (IndexType i = numRows - 1; i >= 0; i--) //inserting in backward order the nodes will be sorted by index in case of same degree
    {
        //insert forward following root
        fwd[i] = fwd[numRows + deg[i]];
        fwd[numRows + deg[i]] = i;
        //insert backward
        bwd[i] = numRows + deg[i];
        bwd[fwd[i]] = i;

        //isolated nodes cannot be aggregated
        if ( deg[i] == 0 )
        {
            nodesLeft--;
        }
    }

    while ( nodesLeft > 0 )
    {
        IndexType node = numRows;
        int degree;

        for (degree = 1; degree <= max_degree; degree++)
        {
            //list not empty -> select node
            if ( fwd[numRows + degree] < numRows ) //selecting the first node will select the most recently inserted one or the one with lowest index. both is preferable
            {
                node = fwd[numRows + degree];
            }

            if ( node < numRows )
            {
                break;
            }
        }

        //no node with degree > 1 found even though nodesLeft > 0
        if ( node == numRows )
        {
            FatalError("nodeLeft counting or list invalid", AMGX_ERR_UNKNOWN );
        }

        if ( agg[node] != -1 )
        {
            FatalError("node is already aggregated", AMGX_ERR_UNKNOWN );
        }

        //find strongest edge
        ValueType max_weight = 0.0;
        IndexType max_node = numRows; //use this as gatekeeper, so if weight == 0 the node index will not be greater than this

        for (IndexType ii = ia[node]; ii < ia[node + 1]; ii++)
        {
            IndexType j = ja[ii];

            if ( agg[j] != -1 || j == node)
            {
                continue;
            }

            if ( w[ii] > 0.0 )
            {
                degree--;
            }

            //deterministic, doesn't selects 0 weight.
            if ( w[ii] > max_weight || (w[ii] == max_weight && j > max_node) ) //always taking the edge pointing to the max node can give good alignment if numbering is structured
            {
                max_node = j;
                max_weight = w[ii];
            }
        } //Note that there has to be at least one neighbor node because degree of node is at least 1.

        if ( max_node == numRows )
        {
            FatalError( "node  has no neighbor although degree of node is at least 1", AMGX_ERR_UNKNOWN );
        }

        if ( degree != 0 )
        {
            FatalError( "node degree corrupted", AMGX_ERR_UNKNOWN );
        }

        //aggregate
        agg[node] = node;
        agg[max_node] = node;
        numAggregates++;
        nodesLeft -= 2;
        //remove from list
        fwd[bwd[node]] = fwd[node];
        bwd[fwd[node]] = bwd[node];
        fwd[bwd[max_node]] = fwd[max_node];
        bwd[fwd[max_node]] = bwd[max_node];

        //update neighbors and list
        //max_node first
        for (IndexType ii = ia[max_node]; ii < ia[max_node + 1]; ii++)
        {
            IndexType j = ja[ii];

            if ( agg[j] != -1  || w[ii] <= 0.0)
            {
                continue;
            }

            //remove j from list
            fwd[bwd[j]] = fwd[j];
            bwd[fwd[j]] = bwd[j];
            //update degree of j
            deg[j]--;
            //add j back to start of the list
            fwd[j] = fwd[numRows + deg[j]];
            bwd[j] = numRows + deg[j];
            bwd[fwd[j]] = j;
            fwd[bwd[j]] = j;

            if (deg[j] == 0)
            {
                nodesLeft--;
            }
        }

        //node second, this will prefer nodes neighbors over max_nodes neighbors when choosing the next node
        for (IndexType ii = ia[node]; ii < ia[node + 1]; ii++)
        {
            IndexType j = ja[ii];

            if ( agg[j] != -1  || w[ii] <= 0.0)
            {
                continue;
            }

            //remove j from list
            fwd[bwd[j]] = fwd[j];
            bwd[fwd[j]] = bwd[j];
            //update degree of j
            deg[j]--;
            //add j back to start of the list
            fwd[j] = fwd[numRows + deg[j]];
            bwd[j] = numRows + deg[j];
            bwd[fwd[j]] = j;
            fwd[bwd[j]] = j;

            if (deg[j] == 0)
            {
                nodesLeft--;
            }
        }
    }

    //copy result back to device
    hipMemcpy( aggregates.raw(), agg, sizeof(IndexType)*numRows, hipMemcpyHostToDevice );

    //assert matching
    for (IndexType node = 0; node < numRows; node++)
    {
        if ( agg[node] == -1 )
        {
            continue;
        }

        for ( IndexType partner = 0; partner < numRows; partner++)
        {
            if ( agg[partner] == agg[node] )
            {
                if ( partner == node )
                {
                    continue;
                }

                bool neighbor = false;

                for (IndexType ii = ia[node]; ii < ia[node + 1]; ii++)
                    if ( ja[ii] == partner )
                    {
                        neighbor = true;
                        break;
                    }

                if ( !neighbor )
                {
                    for (IndexType ii = ia[partner]; ii < ia[partner + 1]; ii++)
                        if ( ja[ii] == node )
                        {
                            neighbor = true;
                            break;
                        }
                }

                if ( !neighbor )
                {
                    FatalError("Internal error in aggregation selector", AMGX_ERR_INTERNAL);
                }
            }
        }
    }

    //you shall not leak memory
    delete[] ia;
    delete[] ja;
    delete[] w;
    delete[] agg;
    delete[] deg;
    delete[] fwd;
    delete[] bwd;
}




//this kernel merges aggregate2 into aggregate1
template<typename IndexType>
__global__
void mergeAggregates(IndexType *aggregate1, const IndexType *aggregate2, IndexType sizeAggregate1, IndexType sizeAggregate2, IndexType sizeAggregate3)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    while ( tid < sizeAggregate1 )
    {
        if ( aggregate1[tid] == sizeAggregate2 )
        {
            aggregate1[tid] = sizeAggregate3;
        }
        else
        {
            aggregate1[tid] = aggregate2[aggregate1[tid]];
        }

        tid += gridDim.x * blockDim.x;
    }
}



template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MultiPairwiseSelector<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::computeIncompleteGalerkin( const Matrix_h &A,
        Matrix_h &Ac,
        const typename Matrix_h::IVector &aggregates,
        const typename Matrix_h::IVector &R_row_offsets,
        const typename Matrix_h::IVector &R_column_indices,
        const int num_aggregates )
{
    FatalError("computeIncomlpetegalerkin is not supported on host. Run with ghost_offdiag_limit=0 instead.", AMGX_ERR_NOT_SUPPORTED_TARGET);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MultiPairwiseSelector<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::computeIncompleteGalerkin( const Matrix_d &A,
        Matrix_d &Ac,
        const typename Matrix_d::IVector &aggregates,
        const typename Matrix_d::IVector &R_row_offsets,
        const typename Matrix_d::IVector &R_column_indices,
        const int num_aggregates )
{
    FatalError("computeIncomlpetegalerkin is not implemented yet. run with ghost_offdiag_limit=0 instead.", AMGX_ERR_NOT_IMPLEMENTED);
}

template<class TConfig>
void MultiPairwiseSelectorBase<TConfig>::assertRestriction( const IVector &R_row_offsets, const IVector &R_col_indices, const IVector &aggregates )
{
    int *r_ia = new int[R_row_offsets.size()];
    int *r_ja = new int[R_col_indices.size()];
    int *agg = new int[aggregates.size()];
    int *used_col = new int[aggregates.size()];

    for ( int i = 0; i < aggregates.size(); i++ )
    {
        used_col[i] = 0;
    }

    hipMemcpy( r_ia, R_row_offsets.raw(), sizeof(int)*R_row_offsets.size(), hipMemcpyDeviceToHost );
    hipMemcpy( r_ja, R_col_indices.raw(), sizeof(int)*R_col_indices.size(), hipMemcpyDeviceToHost );
    hipMemcpy( agg, aggregates.raw(), sizeof(int)*aggregates.size(), hipMemcpyDeviceToHost );

    for ( int i = 0; i < R_row_offsets.size() - 1; i++ )
    {
        for ( int ii = r_ia[i]; ii < r_ia[i + 1]; ii++ )
        {
            int j = r_ja[ii];
            used_col[j]++;

            if ( used_col[j] > 1 )
            {
                std::cout << "column " << j << " is present at least " << used_col[j] << " times" << std::endl;
            }

            if ( j < 0 || j >= aggregates.size() )
            {
                std::cout << "Error: j out of bounds, j = " << j << " and numRows = " << aggregates.size() << std::endl;
            }
            else if  ( agg[j] != i )
            {
                std::cout << "Error: agg[" << j << "] = " << agg[j] << " != " << i << std::endl;
            }
        }
    }

    std::cout << "assert restriction done" << std::endl;
}

template<class T_Config>
void MultiPairwiseSelectorBase<T_Config>::setAggregates(Matrix<T_Config> &A,
        IVector &aggregates, IVector &aggregates_global, int &num_aggregates)
{
    if (A.get_block_dimx() == A.get_block_dimy())
    {
        //ghost level matrix. this is a probably a weight matrix
        Matrix<TConfig> ghostA;
        ghostA.values.resize(0);
        //prolongation and restriction operator. this is only needed in when LowDegCoarseAGenerator is used
        IVector R_row_offsets;
        IVector R_col_indices;
        //holds the size of each aggregate
        IVector sizes;
        sizes.resize(0);
        //aggregates for ghost level
        IVector aggregates_current;
        IVector aggregates_global_current;
        bool aggregates_initialized = true;

        if (aggregates.size() == 0)
        {
            aggregates_initialized = false;

            if (!A.is_matrix_singleGPU())
            {
                aggregates.resize(A.manager->halo_offset(A.manager->num_neighbors()));
            }
            else
            {
                aggregates.resize(A.get_num_rows());
            }
        }

        //for mergeAggregates kernel
        const int threads_per_block = 256;
        const int num_blocks = min( AMGX_GRID_MAX_SIZE, (A.get_num_rows() - 1) / threads_per_block + 1 );
        hipStream_t stream = thrust::global_thread_handle::get_stream();
        //initialize and prepare weight matrix
        Matrix<TConfig> w;
        w.set_initialized(0);
        w.addProps(CSR);
        w.delProps(COO);
        w.setColsReorderedByColor(false);
        w.resize( 0, 0, 0, 1, 1, true ); //empty scalar 0x0 matrix
        w.values.resize(0); //matrix resize sets the values array to nnz+1 for no apparent reason
        IndexType targetSize = 1;
        //initialize coarse A generator
        CoarseAGenerator<TConfig> *cag;
        const bool use_restriction = true;
        const bool shrink_ghost_level = false;
        cag = new LowDegCoarseAGenerator<TConfig>(mCfg, mCfg_scope);
        // This will make coarseAGenerator to allocate more memory inside of  galerkin
        ghostA.manager = new DistributedManager<TConfig>();
        w.manager = new DistributedManager<TConfig>();
        Matrix<TConfig> *curA = &A;

        //foreach pass do:
        // 1. build aggregates and weights
        // 2. create weight matrix (in full_ghost_level mode this is the input matrix or the last ghostlevel matrix)
        // 3. if in full ghost level mode, build R
        // 4. compute next level
        for (int current_pass = 1; true; current_pass++)
        {
            const IndexType numRows = curA->get_num_rows();
            const IndexType nnz = curA->get_num_nz();
            targetSize *= 2;

            if ( full_ghost_level )
            {
                w.values.resize(0);    //compute weights from curA
            }
            else
            {
                w.values.swap( ghostA.values );    //use the weights computed with the galerkin operator (this will do nothing in the first pass, both values have size 0)
            }

            // create aggregates from correct input matrix
            setAggregates_common_sqblocks( *curA, aggregates_current, aggregates_global_current, num_aggregates, w.values, sizes );


            if ( current_pass > 1 )
            {
                //merge original aggregate with the newly created ones
                mergeAggregates <<< num_blocks, threads_per_block, 0, stream >>>( aggregates.raw(), aggregates_current.raw(), A.get_num_rows(), numRows, num_aggregates );
                cudaCheckError();
                //mergeAggregates<<< num_blocks, threads_per_block, 0, stream >>>( aggregates_global.raw(), aggregates_global_current.raw(), A.get_num_rows() );
                //cudaCheckError();
            }

            //try to free memory
            if ( full_ghost_level )
            {
                //then we don't need to save the weights, only for original level to do post processing
                w.values.resize(0);
            }
            else
            {
                //save edge weights for original level later
                //in that case we can throw away the values of ghostA as we will use the values to compute the next ghost level
                ghostA.values.resize(0);
            }

            // this is the break condition for the loop
            if ( current_pass >= aggregation_passes || num_aggregates <= 1 || num_aggregates == numRows)
            {
                //this means, aggregates has not been initialized yet
                if ( !aggregates_initialized )
                {
                    aggregates.swap( aggregates_current );
                }

                if ( !aggregates_initialized )
                {
                    aggregates_global.swap( aggregates_global_current );
                }

                hipStreamSynchronize( stream );
                cudaCheckError();

                break;
            }

            //prepare A to be corrupted
            curA->set_initialized(0);
            //swap in ia, ja from curA
            w.row_offsets.swap( curA->row_offsets );
            w.col_indices.swap( curA->col_indices );

            if ( full_ghost_level )
            {
                if ( shrink_ghost_level && curA->get_block_dimx() > 1)
                {
                    //set w to correct size
                    w.values.resize( nnz );
                    //define grid and offsets
                    const int num_blocks_inter = min( (int)AMGX_GRID_MAX_SIZE, (int)(nnz - 1) / threads_per_block + 1 );
                    const int sq_blocksize = A.get_block_dimx() * A.get_block_dimy();
                    const int index_offset = A.get_block_dimy() * m_aggregation_edge_weight_component + m_aggregation_edge_weight_component;
                    //do the interleaved copy
                    gatherValuesInterleaved <<< num_blocks_inter, threads_per_block, 0, stream>>>( A.values.raw(), w.values.raw(), nnz, sq_blocksize, index_offset );
                    hipStreamSynchronize( stream );
                    cudaCheckError();
                }
                else
                {
                    w.values.swap( curA->values );
                }
            }

            w.diag.swap( curA->diag );

            //resize to inform the matrix of its new size
            if ( full_ghost_level && !shrink_ghost_level )
            {
                w.set_block_dimx( A.get_block_dimx() );
                w.set_block_dimy( A.get_block_dimy() );
            }
            else
            {
                w.set_block_dimx( 1 );
                w.set_block_dimy( 1 );
            }

            w.set_num_rows( numRows );
            w.set_num_cols( numRows );
            w.set_num_nz( nnz );
            w.set_allow_recompute_diag( false );

            if ( curA->hasProps( DIAG ) )
            {
                w.addProps( DIAG );
            }

            //ready to use
            w.set_initialized(1);

            //compute restriction operator
            if ( use_restriction )
            {
                IVector R_row_indices(aggregates_current);
                R_row_offsets.resize(num_aggregates + 2);
                R_col_indices.resize(numRows);
                thrust::sequence(R_col_indices.begin(), R_col_indices.end());
                cudaCheckError();
                thrust::sort_by_key(R_row_indices.begin(), R_row_indices.end(), R_col_indices.begin());
                cudaCheckError();
                cusp::detail::indices_to_offsets(R_row_indices, R_row_offsets);
                cudaCheckError();
                //delete last row, which holds the pseudo aggregate
                R_row_offsets.resize( num_aggregates + 1);
                R_col_indices.resize( R_row_offsets[num_aggregates] );
            }

            // 3. compute galerkin ghost level
            if ( ghost_offdiag_limit == 0 )
            {
                //compute full galerkin
                cag->computeAOperator(w,
                                      ghostA,
                                      aggregates_current,
                                      R_row_offsets,
                                      R_col_indices,
                                      num_aggregates);
            }
            else
            {
                //compute incomplete galerkin
                computeIncompleteGalerkin(w,
                                          ghostA,
                                          aggregates_current,
                                          R_row_offsets,
                                          R_col_indices,
                                          num_aggregates);
            }

            //from now on w will be destroyed again.
            w.set_initialized(0);

            //repair the original A matrix. its ia and ja are in w
            if ( current_pass == 1 )
            {
                //swap back
                w.row_offsets.swap( A.row_offsets );
                w.col_indices.swap( A.col_indices );

                //only in that case we have swapped the values
                if ( full_ghost_level && !shrink_ghost_level )
                {
                    w.values.swap( A.values );
                }

                //save the edge weights of the original level
                A.diag.swap( w.diag );
                A.set_initialized(1); //A is repaired now
                //save the first aggregates into the original aggregate vector so we can merge them later
                aggregates.swap( aggregates_current );
                aggregates_global.swap( aggregates_global_current );
                aggregates_initialized = true;
                curA = &ghostA;
            }
        }

        delete cag;
    }
    else
    {
        FatalError("Unsupported block size for MultiPairwise", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
    }
}

// -------------------------
//    Explict instantiations
// -------------------------
#define AMGX_CASE_LINE(CASE) template class MultiPairwiseSelectorBase<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE
#define AMGX_CASE_LINE(CASE) template class MultiPairwiseSelector<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

}
}
}
