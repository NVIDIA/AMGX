/* Copyright (c) 2013-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <amgx_cublas.h>
#ifdef AMGX_USE_LAPACK
#include "mkl.h"
#endif
namespace amgx
{

hipblasHandle_t Cublas::m_handle = 0;

namespace
{
// real valued calls
hipblasStatus_t cublas_axpy(hipblasHandle_t handle, int n,
                           const float *alpha,
                           const float *x, int incx,
                           float *y, int incy)
{
    return hipblasSaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublas_axpy(hipblasHandle_t handle, int n,
                           const double *alpha,
                           const double *x, int incx,
                           double *y, int incy)
{
    return hipblasDaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublas_copy(hipblasHandle_t handle, int n,
                           const float *x, int incx,
                           float *y, int incy)
{
    return hipblasScopy(handle, n, x, incx, y, incy);
}

hipblasStatus_t cublas_copy(hipblasHandle_t handle, int n,
                           const double *x, int incx,
                           double *y, int incy)
{
    return hipblasDcopy(handle, n, x, incx, y, incy);
}

hipblasStatus_t cublas_dot(hipblasHandle_t handle, int n,
                          const float *x, int incx, const float *y, int incy,
                          float *result)
{
    return hipblasSdot(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublas_dot(hipblasHandle_t handle, int n,
                          const double *x, int incx, const double *y, int incy,
                          double *result)
{
    return hipblasDdot(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublas_dotc(hipblasHandle_t handle, int n,
                           const float *x, int incx, const float *y, int incy,
                           float *result)
{
    return hipblasSdot(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublas_dotc(hipblasHandle_t handle, int n,
                           const double *x, int incx, const double *y, int incy,
                           double *result)
{
    return hipblasDdot(handle, n, x, incx, y, incy, result);
}


hipblasStatus_t cublas_trsv_v2(hipblasHandle_t handle,
                              hipblasFillMode_t uplo,
                              hipblasOperation_t trans,
                              hipblasDiagType_t diag,
                              int n,
                              const float *A,
                              int lda,
                              float *x,
                              int incx)
{
    return hipblasStrsv (handle, uplo, trans, diag, n, A, lda, x, incx);
}
hipblasStatus_t cublas_trsv_v2(hipblasHandle_t handle,
                              hipblasFillMode_t uplo,
                              hipblasOperation_t trans,
                              hipblasDiagType_t diag,
                              int n,
                              const double *A,
                              int lda,
                              double *x,
                              int incx)
{
    return hipblasDtrsv (handle, uplo, trans, diag, n, A, lda, x, incx);
}

hipblasStatus_t cublas_gemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const float           *alpha,
                           const float           *A, int lda,
                           const float           *B, int ldb,
                           const float           *beta,
                           float           *C, int ldc)
{
    return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

hipblasStatus_t cublas_gemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const double          *alpha,
                           const double          *A, int lda,
                           const double          *B, int ldb,
                           const double          *beta,
                           double          *C, int ldc)
{
    return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

hipblasStatus_t cublas_gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
                           const float *alpha, const float *A, int lda,
                           const float *x, int incx,
                           const float *beta, float *y, int incy)
{
    return hipblasSgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

hipblasStatus_t cublas_gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
                           const double *alpha, const double *A, int lda,
                           const double *x, int incx,
                           const double *beta, double *y, int incy)
{
    return hipblasDgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

hipblasStatus_t cublas_ger(hipblasHandle_t handle, int m, int n,
                          const float *alpha,
                          const float *x, int incx,
                          const float *y, int incy,
                          float *A, int lda)
{
    return hipblasSger(handle, m, n, alpha, x, incx, y, incy, A, lda);
}

hipblasStatus_t cublas_ger(hipblasHandle_t handle, int m, int n,
                          const double *alpha,
                          const double *x, int incx,
                          const double *y, int incy,
                          double *A, int lda)
{
    return hipblasDger(handle, m, n, alpha, x, incx, y, incy, A, lda);
}
hipblasStatus_t cublas_gerc(hipblasHandle_t handle, int m, int n,
                           const float *alpha,
                           const float *x, int incx,
                           const float *y, int incy,
                           float *A, int lda)
{
    return hipblasSger(handle, m, n, alpha, x, incx, y, incy, A, lda);
}

hipblasStatus_t cublas_gerc(hipblasHandle_t handle, int m, int n,
                           const double *alpha,
                           const double *x, int incx,
                           const double *y, int incy,
                           double *A, int lda)
{
    return hipblasDger(handle, m, n, alpha, x, incx, y, incy, A, lda);
}

hipblasStatus_t cublas_nrm2(hipblasHandle_t handle, int n,
                           const float *x, int incx, float *result)
{
    return hipblasSnrm2(handle, n, x, incx, result);
}

hipblasStatus_t cublas_nrm2(hipblasHandle_t handle, int n,
                           const double *x, int incx, double *result)
{
    return hipblasDnrm2(handle, n, x, incx, result);
}

hipblasStatus_t cublas_scal(hipblasHandle_t handle, int n,
                           const float *alpha,
                           float *x, int incx)
{
    return hipblasSscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublas_scal(hipblasHandle_t handle, int n,
                           const double *alpha,
                           double *x, int incx)
{
    return hipblasDscal(handle, n, alpha, x, incx);
}


// complex valued calls
hipblasStatus_t cublas_axpy(hipblasHandle_t handle, int n,
                           const hipComplex *alpha,
                           const hipComplex *x, int incx,
                           hipComplex *y, int incy)
{
    return hipblasCaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublas_axpy(hipblasHandle_t handle, int n,
                           const hipDoubleComplex *alpha,
                           const hipDoubleComplex *x, int incx,
                           hipDoubleComplex *y, int incy)
{
    return hipblasZaxpy(handle, n, alpha, x, incx, y, incy);
}

hipblasStatus_t cublas_copy(hipblasHandle_t handle, int n,
                           const hipComplex *x, int incx,
                           hipComplex *y, int incy)
{
    return hipblasCcopy(handle, n, x, incx, y, incy);
}

hipblasStatus_t cublas_copy(hipblasHandle_t handle, int n,
                           const hipDoubleComplex *x, int incx,
                           hipDoubleComplex *y, int incy)
{
    return hipblasZcopy(handle, n, x, incx, y, incy);
}

hipblasStatus_t cublas_dot(hipblasHandle_t handle, int n,
                          const hipComplex *x, int incx, const hipComplex *y, int incy,
                          hipComplex *result)
{
    return hipblasCdotu(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublas_dot(hipblasHandle_t handle, int n,
                          const hipDoubleComplex *x, int incx, const hipDoubleComplex *y, int incy,
                          hipDoubleComplex *result)
{
    return hipblasZdotu(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublas_dotc(hipblasHandle_t handle, int n,
                           const hipComplex *x, int incx, const hipComplex *y, int incy,
                           hipComplex *result)
{
    return hipblasCdotc(handle, n, x, incx, y, incy, result);
}

hipblasStatus_t cublas_dotc(hipblasHandle_t handle, int n,
                           const hipDoubleComplex *x, int incx, const hipDoubleComplex *y, int incy,
                           hipDoubleComplex *result)
{
    return hipblasZdotc(handle, n, x, incx, y, incy, result);
}


hipblasStatus_t cublas_trsv_v2(hipblasHandle_t handle,
                              hipblasFillMode_t uplo,
                              hipblasOperation_t trans,
                              hipblasDiagType_t diag,
                              int n,
                              const hipComplex *A,
                              int lda,
                              hipComplex *x,
                              int incx)
{
    return hipblasCtrsv (handle, uplo, trans, diag, n, A, lda, x, incx);
}
hipblasStatus_t cublas_trsv_v2(hipblasHandle_t handle,
                              hipblasFillMode_t uplo,
                              hipblasOperation_t trans,
                              hipblasDiagType_t diag,
                              int n,
                              const hipDoubleComplex *A,
                              int lda,
                              hipDoubleComplex *x,
                              int incx)
{
    return hipblasZtrsv (handle, uplo, trans, diag, n, A, lda, x, incx);
}

hipblasStatus_t cublas_gemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const hipComplex           *alpha,
                           const hipComplex           *A, int lda,
                           const hipComplex           *B, int ldb,
                           const hipComplex           *beta,
                           hipComplex           *C, int ldc)
{
    return hipblasCgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

hipblasStatus_t cublas_gemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const hipDoubleComplex          *alpha,
                           const hipDoubleComplex          *A, int lda,
                           const hipDoubleComplex          *B, int ldb,
                           const hipDoubleComplex          *beta,
                           hipDoubleComplex          *C, int ldc)
{
    return hipblasZgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

hipblasStatus_t cublas_gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
                           const hipComplex *alpha, const hipComplex *A, int lda,
                           const hipComplex *x, int incx,
                           const hipComplex *beta, hipComplex *y, int incy)
{
    return hipblasCgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

hipblasStatus_t cublas_gemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
                           const hipDoubleComplex *alpha, const hipDoubleComplex *A, int lda,
                           const hipDoubleComplex *x, int incx,
                           const hipDoubleComplex *beta, hipDoubleComplex *y, int incy)
{
    return hipblasZgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

hipblasStatus_t cublas_ger(hipblasHandle_t handle, int m, int n,
                          const hipComplex *alpha,
                          const hipComplex *x, int incx,
                          const hipComplex *y, int incy,
                          hipComplex *A, int lda)
{
    return hipblasCgeru(handle, m, n, alpha, x, incx, y, incy, A, lda);
}

hipblasStatus_t cublas_ger(hipblasHandle_t handle, int m, int n,
                          const hipDoubleComplex *alpha,
                          const hipDoubleComplex *x, int incx,
                          const hipDoubleComplex *y, int incy,
                          hipDoubleComplex *A, int lda)
{
    return hipblasZgeru(handle, m, n, alpha, x, incx, y, incy, A, lda);
}
hipblasStatus_t cublas_gerc(hipblasHandle_t handle, int m, int n,
                           const hipComplex *alpha,
                           const hipComplex *x, int incx,
                           const hipComplex *y, int incy,
                           hipComplex *A, int lda)
{
    return hipblasCgerc(handle, m, n, alpha, x, incx, y, incy, A, lda);
}

hipblasStatus_t cublas_gerc(hipblasHandle_t handle, int m, int n,
                           const hipDoubleComplex *alpha,
                           const hipDoubleComplex *x, int incx,
                           const hipDoubleComplex *y, int incy,
                           hipDoubleComplex *A, int lda)
{
    return hipblasZgerc(handle, m, n, alpha, x, incx, y, incy, A, lda);
}

hipblasStatus_t cublas_nrm2(hipblasHandle_t handle, int n,
                           const hipComplex *x, int incx, float *result)
{
    return hipblasScnrm2(handle, n, x, incx, result);
}

hipblasStatus_t cublas_nrm2(hipblasHandle_t handle, int n,
                           const hipDoubleComplex *x, int incx, double *result)
{
    return hipblasDznrm2(handle, n, x, incx, result);
}

hipblasStatus_t cublas_scal(hipblasHandle_t handle, int n,
                           const hipComplex *alpha,
                           hipComplex *x, int incx)
{
    return hipblasCscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublas_scal(hipblasHandle_t handle, int n,
                           const hipDoubleComplex *alpha,
                           hipDoubleComplex *x, int incx)
{
    return hipblasZscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublas_scal(hipblasHandle_t handle, int n,
                           const float *alpha,
                           hipComplex *x, int incx)
{
    return hipblasCsscal(handle, n, alpha, x, incx);
}

hipblasStatus_t cublas_scal(hipblasHandle_t handle, int n,
                           const double *alpha,
                           hipDoubleComplex *x, int incx)
{
    return hipblasZdscal(handle, n, alpha, x, incx);
}

} // anonymous namespace.

void Cublas::set_pointer_mode_device()
{
    hipblasHandle_t handle = Cublas::get_handle();
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
}

void Cublas::set_pointer_mode_host()
{
    hipblasHandle_t handle = Cublas::get_handle();
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
}

template <class TConfig>
void Cublas::gemm(typename TConfig::VecPrec alpha,
                  const Vector<TConfig> &A, const Vector<TConfig> &B,
                  typename TConfig::VecPrec beta, Vector<TConfig> &C,
                  bool A_transposed, bool B_transposed)
{
    hipblasOperation_t trans_A = A_transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t trans_B = B_transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    int m = A_transposed ? A.get_num_cols() : A.get_num_rows();
    int n = B_transposed ? B.get_num_rows() : B.get_num_cols();
    int k = A_transposed ? A.get_num_rows() : A.get_num_cols();
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_gemm(handle, trans_A, trans_B,
                                 m, n, k,
                                 &alpha, A.raw(), A.get_lda(),
                                 B.raw(), B.get_lda(),
                                 &beta, C.raw(), C.get_lda()));
    C.dirtybit = 1;
}

template <typename T>
void Cublas::axpy(int n, T alpha,
                  const T *x, int incx,
                  T *y, int incy)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_axpy(handle, n, &alpha, x, incx, y, incy));
}

template <typename T>
void Cublas::copy(int n, const T *x, int incx,
                  T *y, int incy)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_copy(handle, n, x, incx, y, incy));
}

template <typename T>
void Cublas::dot(int n, const T *x, int incx,
                 const T *y, int incy,
                 T *result)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_dot(handle, n, x, incx, y, incy, result));
}

template <typename T>
void Cublas::dotc(int n, const T *x, int incx,
                  const T *y, int incy,
                  T *result)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_dotc(handle, n, x, incx, y, incy, result));
}

template <typename T, typename V>
V Cublas::nrm2(int n, const T *x, int incx)
{
    hipblasHandle_t handle = Cublas::get_handle();
    V result;
    Cublas::nrm2(n, x, incx, &result);
    return result;
}

template <typename T, typename V>
void Cublas::nrm2(int n, const T *x, int incx, V *result)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_nrm2(handle, n, x, incx, result));
}

template <typename T, typename V>
void Cublas::scal(int n, T alpha, V *x, int incx)
{
    Cublas::scal(n, &alpha, x, incx);
}

template <typename T, typename V>
void Cublas::scal(int n, T *alpha, V *x, int incx)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_scal(handle, n, alpha, x, incx));
}

template <typename T>
void Cublas::gemv(bool transposed, int m, int n,
                  const T *alpha, const T *A, int lda,
                  const T *x, int incx,
                  const T *beta, T *y, int incy)
{
    hipblasHandle_t handle = Cublas::get_handle();
    hipblasOperation_t trans = transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    cublasCheckError(cublas_gemv(handle, trans, m, n, alpha, A, lda,
                                 x, incx, beta, y, incy));
}

template <typename T>
void Cublas::gemv_ext(bool transposed, const int m, const int n,
                      const T *alpha, const T *A, const int lda,
                      const T *x, const int incx,
                      const T *beta, T *y, const int incy, const int offsetx, const int offsety, const int offseta)
{
    hipblasHandle_t handle = Cublas::get_handle();
    hipblasOperation_t trans = transposed ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    cublasCheckError(cublas_gemv(handle, trans, m, n, alpha, A + offseta, lda,
                                 x + offsetx, incx, beta, y + offsety, incy));
}

template <typename T>
void Cublas::trsv_v2( hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n,
                      const T *A, int lda, T *x, int incx, int offseta)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError( cublas_trsv_v2(handle, uplo, trans, diag, n, A + offseta, lda, x, incx));
}


template <typename T>
void Cublas::ger(int m, int n, const T *alpha,
                 const T *x, int incx,
                 const T *y, int incy,
                 T *A, int lda)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_ger(handle, m, n, alpha, x, incx, y, incy, A, lda));
}

template <typename T>
void Cublas::gerc(int m, int n, const T *alpha,
                  const T *x, int incx,
                  const T *y, int incy,
                  T *A, int lda)
{
    hipblasHandle_t handle = Cublas::get_handle();
    cublasCheckError(cublas_gerc(handle, m, n, alpha, x, incx, y, incy, A, lda));
}

#define AMGX_CASE_LINE(CASE) \
    template void Cublas::gemm(typename TemplateMode<CASE>::Type::VecPrec, const Vector<TemplateMode<CASE>::Type>&, const Vector<TemplateMode<CASE>::Type>&, typename TemplateMode<CASE>::Type::VecPrec, Vector<TemplateMode<CASE>::Type>&, bool, bool);
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

// real valued instantiaions
template void Cublas::axpy(int n, float alpha,
                           const float *x, int incx,
                           float *y, int incy);
template void Cublas::axpy(int n, double alpha,
                           const double *x, int incx,
                           double *y, int incy);

template void Cublas::copy(int n, const float *x, int incx, float *y, int incy);
template void Cublas::copy(int n, const double *x, int incx, double *y, int incy);

template void Cublas::dot(int n, const float *x, int incx,
                          const float *y, int incy,
                          float *result);
template void Cublas::dot(int n, const double *x, int incx,
                          const double *y, int incy,
                          double *result);
template void Cublas::dotc(int n, const float *x, int incx,
                           const float *y, int incy,
                           float *result);
template void Cublas::dotc(int n, const double *x, int incx,
                           const double *y, int incy,
                           double *result);

template void Cublas::gemv(bool transposed, int m, int n,
                           const float *alpha, const float *A, int lda,
                           const float *x, int incx,
                           const float *beta, float *y, int incy);
template void Cublas::gemv(bool transposed, int m, int n,
                           const double *alpha, const double *A, int lda,
                           const double *x, int incx,
                           const double *beta, double *y, int incy);

template void Cublas::ger(int m, int n, const float *alpha,
                          const float *x, int incx,
                          const float *y, int incy,
                          float *A, int lda);
template void Cublas::ger(int m, int n, const double *alpha,
                          const double *x, int incx,
                          const double *y, int incy,
                          double *A, int lda);
template void Cublas::gerc(int m, int n, const float *alpha,
                           const float *x, int incx,
                           const float *y, int incy,
                           float *A, int lda);
template void Cublas::gerc(int m, int n, const double *alpha,
                           const double *x, int incx,
                           const double *y, int incy,
                           double *A, int lda);


template void Cublas::gemv_ext(bool transposed, const int m, const int n,
                               const float *alpha, const float *A, const int lda,
                               const float *x, const int incx,
                               const float *beta, float *y, const int incy, const int offsetx, const int offsety, const int offseta);
template void Cublas::gemv_ext(bool transposed, const int m, const int n,
                               const double *alpha, const double *A, const int lda,
                               const double *x, const int incx,
                               const double *beta, double *y, const int incy, const int offsetx, const int offsety, const int offseta);


template void Cublas::trsv_v2( hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n,
                               const float *A, int lda, float *x, int incx, int offseta);
template void Cublas::trsv_v2( hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n,
                               const double *A, int lda, double *x, int incx, int offseta);

template double Cublas::nrm2(int n, const double *x, int incx);
template float Cublas::nrm2(int n, const float *x, int incx);

template void Cublas::scal(int n, float alpha, float *x, int incx);
template void Cublas::scal(int n, double alpha, double *x, int incx);

// complex valued instantiaions
template void Cublas::axpy(int n, hipComplex alpha,
                           const hipComplex *x, int incx,
                           hipComplex *y, int incy);
template void Cublas::axpy(int n, hipDoubleComplex alpha,
                           const hipDoubleComplex *x, int incx,
                           hipDoubleComplex *y, int incy);

template void Cublas::copy(int n, const hipComplex *x, int incx, hipComplex *y, int incy);
template void Cublas::copy(int n, const hipDoubleComplex *x, int incx, hipDoubleComplex *y, int incy);

template void Cublas::dot(int n, const hipComplex *x, int incx,
                          const hipComplex *y, int incy,
                          hipComplex *result);
template void Cublas::dot(int n, const hipDoubleComplex *x, int incx,
                          const hipDoubleComplex *y, int incy,
                          hipDoubleComplex *result);
template void Cublas::dotc(int n, const hipComplex *x, int incx,
                           const hipComplex *y, int incy,
                           hipComplex *result);
template void Cublas::dotc(int n, const hipDoubleComplex *x, int incx,
                           const hipDoubleComplex *y, int incy,
                           hipDoubleComplex *result);

template void Cublas::gemv(bool transposed, int m, int n,
                           const hipComplex *alpha, const hipComplex *A, int lda,
                           const hipComplex *x, int incx,
                           const hipComplex *beta, hipComplex *y, int incy);
template void Cublas::gemv(bool transposed, int m, int n,
                           const hipDoubleComplex *alpha, const hipDoubleComplex *A, int lda,
                           const hipDoubleComplex *x, int incx,
                           const hipDoubleComplex *beta, hipDoubleComplex *y, int incy);

template void Cublas::ger(int m, int n, const hipComplex *alpha,
                          const hipComplex *x, int incx,
                          const hipComplex *y, int incy,
                          hipComplex *A, int lda);
template void Cublas::ger(int m, int n, const hipDoubleComplex *alpha,
                          const hipDoubleComplex *x, int incx,
                          const hipDoubleComplex *y, int incy,
                          hipDoubleComplex *A, int lda);
template void Cublas::gerc(int m, int n, const hipComplex *alpha,
                           const hipComplex *x, int incx,
                           const hipComplex *y, int incy,
                           hipComplex *A, int lda);
template void Cublas::gerc(int m, int n, const hipDoubleComplex *alpha,
                           const hipDoubleComplex *x, int incx,
                           const hipDoubleComplex *y, int incy,
                           hipDoubleComplex *A, int lda);


template void Cublas::gemv_ext(bool transposed, const int m, const int n,
                               const hipComplex *alpha, const hipComplex *A, const int lda,
                               const hipComplex *x, const int incx,
                               const hipComplex *beta, hipComplex *y, const int incy, const int offsetx, const int offsety, const int offseta);
template void Cublas::gemv_ext(bool transposed, const int m, const int n,
                               const hipDoubleComplex *alpha, const hipDoubleComplex *A, const int lda,
                               const hipDoubleComplex *x, const int incx,
                               const hipDoubleComplex *beta, hipDoubleComplex *y, const int incy, const int offsetx, const int offsety, const int offseta);


template void Cublas::trsv_v2( hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n,
                               const hipComplex *A, int lda, hipComplex *x, int incx, int offseta);
template void Cublas::trsv_v2( hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n,
                               const hipDoubleComplex *A, int lda, hipDoubleComplex *x, int incx, int offseta);

template double Cublas::nrm2(int n, const hipDoubleComplex *x, int incx);
template float Cublas::nrm2(int n, const hipComplex *x, int incx);

template void Cublas::scal(int n, hipComplex alpha, hipComplex *x, int incx);
template void Cublas::scal(int n, hipDoubleComplex alpha, hipDoubleComplex *x, int incx);
template void Cublas::scal(int n, float alpha, hipComplex *x, int incx);
template void Cublas::scal(int n, double alpha, hipDoubleComplex *x, int incx);

} // namespace amgx

