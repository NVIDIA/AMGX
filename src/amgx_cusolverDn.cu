// SPDX-FileCopyrightText: 2011 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include <amgx_cusolverDn.h>

namespace amgx
{

//
// LU factorization
//
hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        float *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnSgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        double *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnDgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        hipComplex *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnCgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        hipDoubleComplex *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnZgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   float *A,
                                   int lda,
                                   float *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnSgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   double *A,
                                   int lda,
                                   double *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnDgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   hipComplex *A,
                                   int lda,
                                   hipComplex *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnCgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   hipDoubleComplex *A,
                                   int lda,
                                   hipDoubleComplex *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnZgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

//
// solve
//
hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const float *A,
                                  int lda,
                                  const int *devIpiv,
                                  float *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnSgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const double *A,
                                  int lda,
                                  const int *devIpiv,
                                  double *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnDgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const hipComplex *A,
                                  int lda,
                                  const int *devIpiv,
                                  hipComplex *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnCgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const hipDoubleComplex *A,
                                  int lda,
                                  const int *devIpiv,
                                  hipDoubleComplex *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnZgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

} // namespace amgx