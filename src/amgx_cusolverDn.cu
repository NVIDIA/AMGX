/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <amgx_cusolverDn.h>

namespace amgx
{

//
// LU factorization
//
hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        float *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnSgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        double *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnDgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        hipComplex *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnCgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf_bufferSize (hipsolverHandle_t handle,
        int m,
        int n,
        hipDoubleComplex *A,
        int lda,
        int *Lwork )
{
    return hipsolverDnZgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   float *A,
                                   int lda,
                                   float *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnSgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   double *A,
                                   int lda,
                                   double *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnDgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   hipComplex *A,
                                   int lda,
                                   hipComplex *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnCgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

hipsolverStatus_t hipsolverDnXgetrf (hipsolverHandle_t handle,
                                   int m,
                                   int n,
                                   hipDoubleComplex *A,
                                   int lda,
                                   hipDoubleComplex *wspace,
                                   int *devIpiv,
                                   int *info)
{
    return hipsolverDnZgetrf(handle, m, n, A, lda, wspace, devIpiv, info);
}

//
// solve
//
hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const float *A,
                                  int lda,
                                  const int *devIpiv,
                                  float *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnSgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const double *A,
                                  int lda,
                                  const int *devIpiv,
                                  double *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnDgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const hipComplex *A,
                                  int lda,
                                  const int *devIpiv,
                                  hipComplex *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnCgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

hipsolverStatus_t hipsolverDnXgetrs(hipsolverHandle_t handle,
                                  hipblasOperation_t trans,
                                  int n,
                                  int nrhs,
                                  const hipDoubleComplex *A,
                                  int lda,
                                  const int *devIpiv,
                                  hipDoubleComplex *B,
                                  int ldb,
                                  int *devInfo )
{
    return hipsolverDnZgetrs(handle, trans, n, 1, A, lda, devIpiv, B, ldb, devInfo );
}

} // namespace amgx