// SPDX-FileCopyrightText: 2011 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include <amgx_types/util.h>

std::ostream &operator<<(std::ostream &os, const hipComplex &x)
{
    os << amgx::types::get_re(x) << " " << amgx::types::get_im(x);
    return os;
}

std::ostream &operator<<(std::ostream &os, const hipDoubleComplex &x)
{
    os << amgx::types::get_re(x) << " " << amgx::types::get_im(x);
    return os;
}