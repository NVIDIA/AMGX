#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <fstream>
#include <cutil.h>
#include <types.h>
#include <classical/strength/ahat.h>
#include <classical/strength/all.h>
#include <thrust/detail/integer_traits.h>
#include <float.h>
#include <specific_spmv.h>
#include <sm_utils.inl>

#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>
#include <thrust/logical.h>

/*
* Note:
* This implementation assumes that off-diag entries all have the opposite sign
* comparing to the diag entry. This is true for most practical cases.
* It would even work if the offending off-diag entries are just a few.
* But if there are many off-diag entries violate this assumption,
* the interpolation based on this strength would be inaccurate.
* This is explained in "Intro to Algebraic multigrid" by K. Stuben.
*/


namespace amgx
{

template< typename T_Config >
Strength_BaseBase<T_Config>::Strength_BaseBase(AMG_Config &cfg,
        const std::string &cfg_scope)
{
    alpha = cfg.AMG_Config::getParameter<double>("strength_threshold", cfg_scope);
}

/*************************************************************************
* "random" hash function for both device and host
************************************************************************/
__host__ __device__ __forceinline__
static float ourHash(const int i)
{
    unsigned int a = i;
    a = (a + 0x7ed55d16) + (a << 12);
    a = (a ^ 0xc761c23c) + (a >> 19);
    a = (a + 0x165667b1) + (a << 5);
    a = (a ^ 0xd3a2646c) + (a << 9);
    a = (a + 0xfd7046c5) + (a << 3);
    a = (a ^ 0xb55a4f09) + (a >> 16);
    return (a ^ 0x4a51e590) / (float)UINT_MAX;
}

/*************************************************************************
* Computes the strength matrix and the connection weights
* Described in \S 4.1 of:
* "Reducing complexity in parallel algebraic multigrid preconditioners"
*
************************************************************************/
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec,
          AMGX_IndPrecision t_indPrec>
void
Strength_Base<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::
computeStrongConnectionsAndWeights_1x1(Matrix_h &A,
                                       BVector &s_con,
                                       FVector &weights,
                                       const double max_row_sum)
{
    bool compute_row_sum = (max_row_sum < 1.0);
    VVector sums_ptr;

    // get the (normalised) row sums
    if (compute_row_sum)
    {
        sums_ptr.resize(A.get_num_rows());
        weightedRowSum(A, sums_ptr);
        cudaCheckError();
    }

    // get min/max off-diag, depending on sign of diagonal
    for (int row = 0; row < A.get_num_rows(); row++)
    {
        ValueType diag(0), minVal(0), maxVal(0);
        ValueType row_sum = compute_row_sum ? sums_ptr[row] : -1;
        int rowEnd = A.row_offsets[row + 1];

        for (int j = A.row_offsets[row]; j < rowEnd; j++)
        {
            int col = A.col_indices[j];
            ValueType val = A.values[j];

            if (col == row)
            {
                diag = val;
            }
            else
            {
                minVal = min(minVal, val);
                maxVal = max(maxVal, val);
            }
        }

        //set the threshold for being strongly connected
        ValueType threshold = (diag < 0) ? maxVal : minVal;
        threshold *= this->alpha;

        //initialize the weight to a small random number
        if (A.is_matrix_singleGPU())
        {
            weights[row] += ourHash(row);
        }
        else
        {
            weights[row] += ourHash((int)A.manager->base_index() + row);
        }

        // sum the column of S - # of points that strongly connect to me
        for (int j = A.row_offsets[row]; j < rowEnd; j++)
        {
            bool is_strongly_connected = false;

            if (compute_row_sum && row_sum > max_row_sum)
            {
                is_strongly_connected = false;
            }
            else
                is_strongly_connected =
                    this->strongly_connected(A.values[j], threshold, diag);

            int col = A.col_indices[j];
            s_con[j] = (col != row) && is_strongly_connected;
            weights[A.col_indices[j]] += s_con[j] ? 1. : 0.;
        }
    }
}


template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec,
          AMGX_IndPrecision t_indPrec>
void
Strength_Base<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::
computeWeights_1x1(Matrix_h &S,
                   FVector &weights)
{
    for (int row = 0; row < S.get_num_rows(); row++)
    {
        //initialize the weight to a small random number
        if (S.is_matrix_singleGPU())
        {
            weights[row] += ourHash(row);
        }
        else
        {
            weights[row] += ourHash(S.manager->base_index() + row);
        }

        int rowEnd = S.row_offsets[row + 1];

        for (int j = S.row_offsets[row]; j < rowEnd; j++)
        {
            int col = S.col_indices[j];

            if (col != row) { weights[col] += 1.; }
        }
    }
}


template <typename ValueType>
__device__
bool stronglyConnectedAHat(ValueType value, ValueType threshold, ValueType diag)
{
    return (diag < ValueType(0)) ? value > threshold : value < threshold;
}

/*************************************************************************
* Computes the strength matrix and the connection weights (device)
************************************************************************/
template< typename IndexType, typename ValueType, int kCtaSize, bool singleGPU >
__global__
void computeStrongConnectionsAndWeightsKernel( const IndexType *A_rows,
        const IndexType *A_cols,
        const ValueType *A_vals,
        int A_num_rows,
        bool *s_con,
        float *weights,
        ValueType alpha,
        ValueType *row_sum,
        const double max_row_sum,
        int64_t base_index)
{
    // One warp works on each row and hence one iteration handles
    // num_warps*numBlock rows. This means atomicAdd() is inevitable.
    const int num_warps = kCtaSize / 32;
    const int num_rows_per_iter = num_warps * gridDim.x;
    __shared__ volatile ValueType smem[kCtaSize];
    __shared__ volatile ValueType s_diag[num_warps];
    __shared__ volatile ValueType s_threshold[num_warps];
    const int warpId = threadIdx.x / 32;
    const int laneId = threadIdx.x % 32;

    for ( int aRowId = blockIdx.x * num_warps + warpId ; aRowId < A_num_rows ;
            aRowId += num_rows_per_iter )
    {
        ValueType minVal(0), maxVal(0);

        if ( laneId == 0 ) // Reset the diagonal
        {
            s_diag[warpId] = ValueType(0);
        }

        utils::syncwarp();

        // Row sum
        ValueType rowSum = -1.0;

        if (max_row_sum < 1.0) { rowSum = row_sum[aRowId]; }

        // get diagonal, min/max off-diagonals
        const int aRowBegin = A_rows[aRowId  ];
        const int aRowEnd   = A_rows[aRowId + 1];

        for ( IndexType aRowIt = aRowBegin + laneId ; utils::any( aRowIt < aRowEnd ) ;
                aRowIt += 32 )
        {
            IndexType aColId = aRowIt < aRowEnd ? A_cols[aRowIt] : -1;
            ValueType aValue = aRowIt < aRowEnd ? A_vals[aRowIt] : ValueType(0);

            if ( aColId == aRowId ) // only one thread evaluates to true.
            {
                s_diag[warpId] = aValue;
            }

            bool is_off_diagonal = aRowIt < aRowEnd && aColId != aRowId;

            if ( is_off_diagonal )
            {
                minVal = min( minVal, aValue );
                maxVal = max( maxVal, aValue );
            }
        }

        // init weights[] with a random number
        if ( laneId == 0 )
        {
            if ( singleGPU )
            {
                atomicAdd( &weights[aRowId], ourHash(aRowId) );
            }
            else
            {
                atomicAdd( &weights[aRowId], ourHash( (int) base_index + aRowId) );
            }
        }

        utils::syncwarp();

        // Big assumption: diag and off-diag always have the opposite sign.
        // If diag entry is negative, then all off-diag entries must be positive.
        // This means max off-diag is to be used to compute the threshold.
        // If diag entry is positve, the min off-diag is used instead.
        if ( s_diag[warpId] < ValueType(0) )
        {
            smem[threadIdx.x] = maxVal;

            utils::syncwarp();

#pragma unroll
            for ( int offset = 16 ; offset > 0 ; offset /= 2 )
            {
                if ( laneId < offset )
                {
                    smem[threadIdx.x] = maxVal = max( maxVal, smem[threadIdx.x + offset] );
                }
                utils::syncwarp();
            }
        }
        else
        {
            smem[threadIdx.x] = minVal;

            utils::syncwarp();

#pragma unroll
            for ( int offset = 16 ; offset > 0 ; offset /= 2 )
            {
                if ( laneId < offset )
                {
                    smem[threadIdx.x] = minVal = min( minVal, smem[threadIdx.x + offset] );
                }
                utils::syncwarp();
            }
        }

        if ( laneId == 0 )
        {
            // If laneId=0, then maxVal or minVal is in smem[threadIdx.x].
            s_threshold[warpId] = smem[threadIdx.x] * alpha;
        }

        utils::syncwarp();

        // sum of the column of S
        for ( IndexType aRowIt = aRowBegin + laneId ; utils::any( aRowIt < aRowEnd ) ;
                aRowIt += 32 )
        {
            IndexType aColId = aRowIt < aRowEnd ? A_cols[aRowIt] : -1;
            ValueType aValue = aRowIt < aRowEnd ? A_vals[aRowIt] : ValueType(0);
            bool is_strongly_connected = false;

            if (max_row_sum < 1.0 && rowSum > max_row_sum)
            {
                is_strongly_connected = false;
            }
            else
            {
                bool is_off_diagonal = aRowIt < aRowEnd && aColId != aRowId;
                is_strongly_connected = is_off_diagonal &&
                                        stronglyConnectedAHat( aValue, s_threshold[warpId], s_diag[warpId] );
            }

            if ( is_strongly_connected && aRowIt < aRowEnd && aColId < A_num_rows)
            {
                atomicAdd( &weights[aColId], 1.0f );
            }

            if ( aRowIt < aRowEnd )
            {
                s_con[aRowIt] = is_strongly_connected;
            }
        }
    }
}


template< typename IndexType, typename ValueType, int kCtaSize, bool singleGPU >
__global__
void computeWeightsKernel( const IndexType *A_rows,
                           const IndexType *A_cols,
                           int A_num_rows,
                           float *weights,
                           int64_t base_index)
{
    const int num_warps = kCtaSize / 32;
    const int num_rows_per_iter = num_warps * gridDim.x;
    const int warpId = threadIdx.x / 32;
    const int laneId = threadIdx.x % 32;

    for ( int aRowId = blockIdx.x * num_warps + warpId ; aRowId < A_num_rows ;
            aRowId += num_rows_per_iter )
    {
        if ( laneId == 0 )
        {
            if ( singleGPU )
            {
                atomicAdd( &weights[aRowId], ourHash(aRowId) );
            }
            else
            {
                atomicAdd( &weights[aRowId], ourHash( (int) base_index + aRowId) );
            }
        }

        const int aRowBegin = A_rows[aRowId  ];
        const int aRowEnd   = A_rows[aRowId + 1];

        for ( IndexType aRowIt = aRowBegin + laneId ; utils::any( aRowIt < aRowEnd ) ;
                aRowIt += 32 )
        {
            IndexType aColId = aRowIt < aRowEnd ? A_cols[aRowIt] : -1;
            bool is_off_diagonal = aRowIt < aRowEnd && aColId != aRowId;

            if (is_off_diagonal)
            {
                atomicAdd( &weights[aColId], 1.0f );
            }
        }
    }
}


/*************************************************************************
* Computes the strength matrix and the connection weights (device)
************************************************************************/
template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec,
          AMGX_IndPrecision t_indPrec>
void Strength_Base<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::
computeStrongConnectionsAndWeights_1x1(Matrix_d &A,
                                       BVector &s_con,
                                       FVector &weights,
                                       const double max_row_sum)
{
    typedef TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> TConfig_d;
    typedef typename Matrix_d::index_type IndexType;
    typedef typename Matrix_d::value_type ValueType;
    // get the raw pointers for everything I need
    const IndexType *offsets_ptr = A.row_offsets.raw();
    const IndexType *column_indices_ptr = A.col_indices.raw();
    const ValueType *values_ptr = A.values.raw();
    bool *s_con_ptr = s_con.raw();
    float *weights_ptr = weights.raw();
    bool compute_row_sum = (max_row_sum < 1.0);

    if (A.get_num_rows() == 0) { compute_row_sum = false; }

    VVector sums_ptr;

    if (compute_row_sum)
    {
        sums_ptr.resize(A.get_num_rows());
        weightedRowSum(A, sums_ptr);
        cudaCheckError();
    }

    // choose a blocksize. Use 1 warp per row
    const int blockSize = 256;
    const int numWarps  = blockSize / 32;
    const int numBlocks = min( 4096, (int) (A.get_num_rows() + numWarps - 1) / numWarps );

    if (A.get_num_rows() > 0)
    {
        if (A.is_matrix_singleGPU())
            computeStrongConnectionsAndWeightsKernel<IndexType, ValueType, blockSize, true>
            <<< numBlocks, blockSize>>>(
                A.row_offsets.raw(),
                A.col_indices.raw(),
                A.values.raw(),
                A.get_num_rows(),
                s_con.raw(),
                weights.raw(),
                this->alpha,
                compute_row_sum ? sums_ptr.raw() : NULL,
                max_row_sum,
                0);
        else
            computeStrongConnectionsAndWeightsKernel<IndexType, ValueType, blockSize, false>
            <<< numBlocks, blockSize>>>(
                A.row_offsets.raw(),
                A.col_indices.raw(),
                A.values.raw(),
                A.get_num_rows(),
                s_con.raw(),
                weights.raw(),
                this->alpha,
                compute_row_sum ? sums_ptr.raw() : NULL,
                max_row_sum,
                A.manager->base_index());
    }

    if (!A.is_matrix_singleGPU() && A.currentView() == OWNED)
    {
        // Need to add neighbors contribution to my weights
        weights.dirtybit = 1;
        A.manager->add_from_halo(weights, weights.tag);
    }

    cudaCheckError();
}


template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec,
          AMGX_IndPrecision t_indPrec>
void Strength_Base<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::
computeWeights_1x1(Matrix_d &S,
                   FVector &weights)
{
    typedef TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> TConfig_d;
    typedef typename Matrix_d::index_type IndexType;
    typedef typename Matrix_d::value_type ValueType;
    // get the raw pointers for everything I need
    const IndexType *offsets_ptr = S.row_offsets.raw();
    const IndexType *column_indices_ptr = S.col_indices.raw();
    float *weights_ptr = weights.raw();
    // choose a blocksize. Use 1 thread per row
    const int blockSize = 256;
    const int numWarps  = blockSize / 32;
    const int numBlocks = min( 4096, (int) (S.get_num_rows() + numWarps - 1) / numWarps );
    hipDeviceSynchronize();
    cudaCheckError();

    // call the CUDA kernel
    if (S.is_matrix_singleGPU())
        computeWeightsKernel<IndexType, ValueType, blockSize, true>
        <<< numBlocks, blockSize>>>(
            S.row_offsets.raw(),
            S.col_indices.raw(),
            S.get_num_rows(),
            weights.raw(),
            0);
    else
        computeWeightsKernel<IndexType, ValueType, blockSize, false>
        <<< numBlocks, blockSize>>>(
            S.row_offsets.raw(),
            S.col_indices.raw(),
            S.get_num_rows(),
            weights.raw(),
            S.manager->base_index());

    cudaCheckError();

    if (!S.is_matrix_singleGPU() && S.currentView() == OWNED)
    {
        // Need to add neighbors contribution to my weights
        weights.dirtybit = 1;
        S.manager->add_from_halo(weights, weights.tag);
    }

    cudaCheckError();
}

template<class T_Config>
void Strength_BaseBase<T_Config>::
computeStrongConnectionsAndWeights(Matrix<T_Config> &A,
                                   BVector &s_con,
                                   FVector &weights,
                                   const double max_row_sum
                                  )
{
    if (A.get_block_size() == 1)
    {
        computeStrongConnectionsAndWeights_1x1(A, s_con, weights, max_row_sum);
    }
    else
        FatalError("Unsupported block size for strong connections",
                   AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
}

template<class T_Config>
void Strength_BaseBase<T_Config>::computeWeights(Matrix<T_Config> &S,
        FVector &weights
                                                )
{
    if (S.get_block_size() == 1)
    {
        computeWeights_1x1(S, weights);
    }
    else
        FatalError("Unsupported block size for strong connections",
                   AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
}


/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class Strength_Base<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

#define AMGX_CASE_LINE(CASE) template class Strength_BaseBase<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

} // namespace amgx

