/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "amg_config.h"
#include "convergence/absolute.h"

namespace amgx
{

template<class TConfig>
AbsoluteConvergence<TConfig>::AbsoluteConvergence(AMG_Config &cfg, const std::string &cfg_scope) : Convergence<TConfig>(cfg, cfg_scope)
{
}

template<class TConfig>
void AbsoluteConvergence<TConfig>::convergence_init()
{
    this->m_tolerance = this->m_cfg->AMG_Config::getParameter<double>("tolerance", this->m_cfg_scope);
}


template<class TConfig>
bool AbsoluteConvergence<TConfig>::convergence_update_and_check(const PODVec_h &nrm, const PODVec_h &nrm_ini)
{
    //printf("Check tolerance: %f\n", this->m_tolerance);
    bool res_converged = true;
    bool res_converged_rel = true;

    for (int i = 0; i < nrm.size(); i++)
    {
        bool conv = nrm[i] < this->m_tolerance;
        res_converged = res_converged && conv;
        bool conv_rel = nrm[i] < Epsilon_conv<ValueTypeB>::value() * nrm_ini[i];
        res_converged_rel = res_converged_rel && conv_rel;
    }

    if (res_converged_rel)
    {
        std::stringstream ss;
        ss << "Relative residual has reached machine precision" << std::endl;
        amgx_output(ss.str().c_str(), static_cast<int>(ss.str().length()));
        return true;
    }

    return res_converged;
}

/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class AbsoluteConvergence<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

} // end namespace

