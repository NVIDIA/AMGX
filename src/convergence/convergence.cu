/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <convergence/convergence.h>


namespace amgx
{

// Constructor
template<class TConfig>
Convergence<TConfig>::Convergence(AMG_Config &cfg, const std::string &cfg_scope) : m_convergence_name("ConvergenceNameNotSet"), m_cfg(&cfg), m_cfg_scope(cfg_scope)
{
    setTolerance(cfg.getParameter<double>("tolerance", cfg_scope));
}

template<class TConfig>
void Convergence<TConfig>::convergence_init()
{
}

template<class TConfig>
bool Convergence<TConfig>::convergence_update_and_check(const PODVec_h &nrm, const PODVec_h &nrm_ini)
{
    FatalError("Convergence::converge_and_update_check not implemented for this type", AMGX_ERR_NOT_IMPLEMENTED);
}



template<class TConfig>
void ConvergenceFactory<TConfig>::registerFactory(std::string name, ConvergenceFactory<TConfig> *f)
{
    std::map<std::string, ConvergenceFactory<TConfig>*> &factories = getFactories( );
    typename std::map<std::string, ConvergenceFactory<TConfig> *>::const_iterator it = factories.find(name);

    if (it != factories.end())
    {
        std::string error = "ConvergenceFactory '" + name + "' has already been registered\n";
        FatalError(error.c_str(), AMGX_ERR_CORE);
    }

    factories[name] = f;
}

template<class TConfig>
void ConvergenceFactory<TConfig>::unregisterFactory(std::string name)
{
    std::map<std::string, ConvergenceFactory<TConfig>*> &factories = getFactories( );
    typename std::map<std::string, ConvergenceFactory<TConfig> *>::iterator it = factories.find(name);

    if (it == factories.end())
    {
        std::string error = "ConvergenceFactory '" + name + "' has not been registered\n";
        FatalError(error.c_str(), AMGX_ERR_CORE);
    }

    ConvergenceFactory<TConfig> *factory = it->second;
    assert( factory != NULL );
    delete factory;
    factories.erase(it);
}

template<class TConfig>
void ConvergenceFactory<TConfig>::unregisterFactories( )
{
    std::map<std::string, ConvergenceFactory<TConfig>*> &factories = getFactories( );
    typename std::map<std::string, ConvergenceFactory<TConfig> *>::iterator it = factories.begin( );

    for ( ; it != factories.end( ) ; )
    {
        ConvergenceFactory<TConfig> *factory = it->second;
        assert( factory != NULL );
        it++;
        delete factory;
    }

    factories.clear( );
}

template<class TConfig>
Convergence<TConfig> *ConvergenceFactory<TConfig>::allocate(AMG_Config &cfg, const std::string &current_scope)
{
    std::map<std::string, ConvergenceFactory<TConfig>*> &factories = getFactories( );
    std::string conv = cfg.getParameter<std::string>("convergence", current_scope);
    typename std::map<std::string, ConvergenceFactory<TConfig> *>::const_iterator it = factories.find(conv);

    if (it == factories.end())
    {
        std::string error = "ConvergenceFactory '" + conv + "' has not been registered\n";
        FatalError(error.c_str(), AMGX_ERR_CORE);
    }

    Convergence<TConfig> *convergence = it->second->create(cfg, current_scope);
    convergence->setName(conv);
    return convergence;
};

template<class TConfig>
std::map<std::string, ConvergenceFactory<TConfig>*> &
ConvergenceFactory<TConfig>::getFactories( )
{
    static std::map<std::string, ConvergenceFactory<TConfig>*> s_factories;
    return s_factories;
}

/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class ConvergenceFactory<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

#define AMGX_CASE_LINE(CASE) template class Convergence<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

} // end namespace

