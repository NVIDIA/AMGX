#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <algorithm> // std::max
#include "amg_config.h"
#include "convergence/relative_ini.h"

namespace amgx
{
template<typename TConfig>
RelativeIniConvergence<TConfig>::RelativeIniConvergence(AMG_Config &cfg, const std::string &cfg_scope) : Convergence<TConfig>(cfg, cfg_scope)
{
}

template<class TConfig>
void RelativeIniConvergence<TConfig>::convergence_init()
{
    this->setTolerance(this->m_cfg->template getParameter<double>("tolerance", this->m_cfg_scope));
}

template<class TConfig>
bool RelativeIniConvergence<TConfig>::convergence_update_and_check(const PODVec_h &nrm, const PODVec_h &nrm_ini)
{
    bool res_converged = true;
    bool res_converged_abs = true;

    for (int i = 0; i < nrm.size(); i++)
    {
        bool conv = (nrm[i] / nrm_ini[i] <= this->m_tolerance);
        res_converged = res_converged && conv ;
        bool conv_abs = (nrm[i] <= std::max(nrm_ini[i] * Epsilon_conv<ValueTypeB>::value(), (PODValueTypeB)(1e-20)));
        res_converged_abs = res_converged_abs && conv_abs ;
    }

    if (res_converged_abs)
    {
        return true;
    }

    return res_converged;
}


/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class RelativeIniConvergence<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

} // end namespace

