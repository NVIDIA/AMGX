#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2013 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include <cassert>
#include <iostream>
#include <thrust/scan.h>
#include <util.h>
#include <csr_multiply.h>
#include <csr_multiply_detail.h>
#include <device_properties.h>
#include <thrust_wrapper.h>
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace csr_multiply_detail
{

#include <amgx_types/util.h>

#include <sm_utils.inl>
#include <hash_containers_detail.inl> // Included inside the namespace to solve name colisions.

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__
void flag_halo_rows(int *row_ids, int size, int *flagArray, int neighbor, int global_id)
{
    for (int tidx = threadIdx.x + blockIdx.x * blockDim.x; tidx < size; tidx += blockDim.x * gridDim.x)
    {
        int row_id = row_ids[tidx];
        flagArray[row_id] = tidx;
    }
}

__device__ __forceinline__ int get_work( int *queue, int warp_id )
{
    int offset = -1;

    if ( utils::lane_id() == 0 )
    {
        offset = atomicAdd( queue, 1 );
    }

    return utils::shfl( offset, 0 );
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template< int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE, bool COUNT_ONLY >
__global__ __launch_bounds__( CTA_SIZE )
void
count_non_zeroes_kernel( const int A_num_rows,
                         const int *A_rows,
                         const int *A_cols,
                         const int *B_rows,
                         const int *B_cols,
                         int *C_rows,
                         int *C_cols,
                         int *Aq1,
                         int *Bq1,
                         int *Aq2,
                         int *Bq2,
                         const int gmem_size,
                         int *g_keys,
                         int *wk_work_queue,
                         int *wk_status )
{
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // The hash keys stored in shared memory.
    __shared__ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id();
    const int lane_id = utils::lane_id();
    // First threads load the row IDs of A needed by the CTA...
    int a_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_set<int, SMEM_SIZE, 4, WARP_SIZE> set( &s_keys[warp_id * SMEM_SIZE], &g_keys[a_row_id * gmem_size], gmem_size );

    // Loop over rows of A.
    for ( ; a_row_id < A_num_rows ; a_row_id = get_work( wk_work_queue, warp_id ) )
    {
        int c_row_id = a_row_id;

        if (Aq1 != NULL)
        {
            a_row_id = Aq1[a_row_id];
        }

        // Make sure we have to proceed.
        if ( COUNT_ONLY )
        {
            volatile int *status = reinterpret_cast<volatile int *>( wk_status );

            if ( set.has_failed() || *status != 0 )
            {
                return;
            }
        }

        // Clear the set.
        set.clear();
        // Load the range of the row.
        int a_col_tmp = -1;

        if ( lane_id < 2 )
        {
            a_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[a_row_id + lane_id] );
        }

        int a_col_it  = utils::shfl( a_col_tmp, 0 );
        int a_col_end = utils::shfl( a_col_tmp, 1 );

        // Iterate over the columns of A.
        for ( a_col_it += lane_id ; utils::any(a_col_it < a_col_end) ; a_col_it += WARP_SIZE )
        {
            // Is it an active thread.
            const bool is_active = a_col_it < a_col_end;
            // Columns of A maps to rows of B. Each thread of the warp loads its A-col/B-row ID.
            int b_row_id = -1;

            if ( is_active )
            {
                b_row_id = utils::Ld<utils::LD_NC>::load( &A_cols[a_col_it] );

                //b_row_id is actually column of A
                if (Aq2 != NULL)
                {
                    b_row_id = Aq2[b_row_id];
	    		
                }

                if (Bq1 != NULL)
                {
                    b_row_id = Bq1[b_row_id];
                }
            }

            // The number of valid rows.
            const int num_rows = __popc( utils::ballot(is_active) );

            // Uniform loop: threads collaborate to load other elements.
            for ( int k = 0 ; k < num_rows ; ++k )
            {
                // Threads in the warp proceeds columns of B in the range [bColIt, bColEnd).
                const int uniform_b_row_id = utils::shfl( b_row_id, k );
                // Load the range of the row of B.
                int b_col_tmp = -1;

                if ( lane_id < 2 )
                {
                    b_col_tmp = utils::Ld<utils::LD_NC>::load( &B_rows[uniform_b_row_id + lane_id] );
                }

                int b_col_it  = utils::shfl( b_col_tmp, 0 );
                int b_col_end = utils::shfl( b_col_tmp, 1 );

                // Iterate over the range of columns of B.
                for ( b_col_it += lane_id ; utils::any(b_col_it < b_col_end) ; b_col_it += WARP_SIZE )
                {
                    int b_col_id = -1;

                    if ( b_col_it < b_col_end )
                    {
                        b_col_id = utils::Ld<utils::LD_NC>::load( &B_cols[b_col_it] );

                        // b_col_id is actually column of B
                        if (Bq2 != NULL)
                        {
                            b_col_id = Bq2[b_col_id];
                        }
                    }

                    set.insert( b_col_id, COUNT_ONLY ? wk_status : NULL );
                }
            }
        }

        // Store the results.
        if ( COUNT_ONLY )
        {
            int count = set.compute_size();

            if ( lane_id == 0 )
            {
                C_rows[c_row_id] = count;
            }
        }
        else
        {
            int c_col_tmp = -1;

            if ( lane_id < 2 )
            {
                c_col_tmp = utils::Ld<utils::LD_NC>::load( &C_rows[c_row_id + lane_id] );
            }

            int c_col_it  = utils::shfl( c_col_tmp, 0 );
            int c_col_end = utils::shfl( c_col_tmp, 1 );
            // Store the results.
            int count = c_col_end - c_col_it;

            if ( count == 0 )
            {
                continue;
            }

            set.store( count, &C_cols[c_col_it] );
        }
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template< int NUM_THREADS_PER_ROW, int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE, bool COUNT_ONLY >
__global__ __launch_bounds__( CTA_SIZE )
void
count_non_zeroes_kernel( const int A_num_rows,
                         const int *__restrict A_rows,
                         const int *__restrict A_cols,
                         const int *__restrict B_rows,
                         const int *__restrict B_cols,
                         int *__restrict C_rows,
                         int *__restrict C_cols,
                         int *Aq1,
                         int *Bq1,
                         int *Aq2,
                         int *Bq2,
                         const int gmem_size,
                         int *g_keys,
                         int *wk_work_queue,
                         int *wk_status )
{
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    const int NUM_LOADED_ROWS = WARP_SIZE / NUM_THREADS_PER_ROW;
    // The hash keys stored in shared memory.
    __shared__ /*volatile*/ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id( );
    const int lane_id = utils::lane_id( );
    // Constants.
    const int lane_id_div_num_threads = lane_id / NUM_THREADS_PER_ROW;
    const int lane_id_mod_num_threads = lane_id % NUM_THREADS_PER_ROW;
    // First threads load the row IDs of A needed by the CTA...
    int a_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_set<int, SMEM_SIZE, 4, WARP_SIZE> set( &s_keys[warp_id * SMEM_SIZE], &g_keys[a_row_id * gmem_size], gmem_size );

    for ( ; a_row_id < A_num_rows ; a_row_id = get_work( wk_work_queue, warp_id ) )
    {
        int c_row_id = a_row_id;

        if (Aq1 != NULL)
        {
            a_row_id = Aq1[a_row_id];
        }

        // Make sure we have to proceed.
        if ( COUNT_ONLY )
        {
            volatile int *status = reinterpret_cast<volatile int *>( wk_status );

            if ( set.has_failed() || *status != 0 )
            {
                return;
            }
        }

        // Clear the set.
        set.clear();
        // Load the range of the row.
        int a_col_tmp = -1;

        if ( lane_id < 2 )
        {
            a_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[a_row_id + lane_id] );
        }

        int a_col_it  = utils::shfl( a_col_tmp, 0 );
        int a_col_end = utils::shfl( a_col_tmp, 1 );

        // Iterate over the columns of A.
        for ( a_col_it += lane_id ; utils::any(a_col_it < a_col_end) ; a_col_it += WARP_SIZE )
        {
            // Is it an active thread.
            const bool is_active = a_col_it < a_col_end;
            // Columns of A maps to rows of B. Each thread of the warp loads its A-col/B-row ID.
            int b_row_id = -1;

            if ( is_active )
            {
                b_row_id = utils::Ld<utils::LD_NC>::load( &A_cols[a_col_it] );

                //b_row_id is actually column of A
                if (Aq2 != NULL)
                {
                    b_row_id = Aq2[b_row_id];
                }

                if (Bq1 != NULL)
                {
                    b_row_id = Bq1[b_row_id];
                }
            }

            const int num_rows = __popc( utils::ballot(is_active) );

            // Uniform loop: threads collaborate to load other elements.
            for ( int k = 0 ; k < num_rows ; k += NUM_LOADED_ROWS )
            {
                int local_k = k + lane_id_div_num_threads;
                // Is it an active thread.
                bool is_active_k = local_k < num_rows;
                // Threads in the warp proceeds columns of B in the range [bColIt, bColEnd).
                const int uniform_b_row_id = utils::shfl( b_row_id, local_k );
                // Load the range of the row of B.
                int b_col_tmp = -1;

                if ( is_active_k && lane_id_mod_num_threads < 2 )
                {
                    b_col_tmp = utils::Ld<utils::LD_NC>::load( &B_rows[uniform_b_row_id + lane_id_mod_num_threads] );
                }

                int b_col_it  = utils::shfl( b_col_tmp, lane_id_div_num_threads * NUM_THREADS_PER_ROW + 0 );
                int b_col_end = utils::shfl( b_col_tmp, lane_id_div_num_threads * NUM_THREADS_PER_ROW + 1 );

                // Iterate over the range of columns of B.
                for ( b_col_it += lane_id_mod_num_threads ; utils::any(b_col_it < b_col_end) ; b_col_it += NUM_THREADS_PER_ROW )
                {
                    int b_col_id = -1;

                    if ( b_col_it < b_col_end )
                    {
                        b_col_id = utils::Ld<utils::LD_NC>::load( &B_cols[b_col_it] );

                        // b_col_id is actually column of B
                        if (Bq2 != NULL)
                        {
                            b_col_id = Bq2[b_col_id];
                        }
                    }

                    set.insert( b_col_id, COUNT_ONLY ? wk_status : NULL );
                }
            }
        }

        // Store the results.
        if ( COUNT_ONLY )
        {
            int count = set.compute_size();

            if ( lane_id == 0 )
            {
                C_rows[c_row_id] = count;
            }
        }
        else
        {
            int c_col_tmp = -1;

            if ( lane_id < 2 )
            {
                c_col_tmp = utils::Ld<utils::LD_NC>::load( &C_rows[c_row_id + lane_id] );
            }

            int c_col_it  = utils::shfl( c_col_tmp, 0 );
            int c_col_end = utils::shfl( c_col_tmp, 1 );
            // Store the results.
            int count = c_col_end - c_col_it;

            if ( count == 0 )
            {
                continue;
            }

            set.store( count, &C_cols[c_col_it] );
        }
    }
}

template <int SMEM_SIZE, int NUM_HASH_FCTS, int WARP_SIZE, bool COUNT_ONLY >
__device__ __forceinline__ void sparse_add_process_row(int row_id, const int *__restrict__ row_offsets, const int *__restrict__ col_indices, int lane_id, Hash_set<int, SMEM_SIZE, NUM_HASH_FCTS, WARP_SIZE> &set, int *wk_status, int global_id, bool print_flag)
{
    // Load the range of the row of RAP_int
    int col_tmp = -1;

    if ( lane_id < 2 )
    {
        col_tmp = utils::Ld<utils::LD_NC>::load( &row_offsets[row_id + lane_id] );
    }

    int col_it  = utils::shfl( col_tmp, 0 );
    int col_end = utils::shfl( col_tmp, 1 );

    // Iterate over the columns of RAP_int
    for ( col_it += lane_id ; utils::any(col_it < col_end) ; col_it += WARP_SIZE )
    {
        int col_id = -1;

        if ( col_it < col_end )
        {
            col_id = utils::Ld<utils::LD_NC>::load( &col_indices[col_it] );
        }

        set.insert( col_id, COUNT_ONLY ? wk_status : NULL );
    }
}


template <typename Value_type, int SMEM_SIZE, int NUM_HASH_FCTS, int WARP_SIZE>
__device__ __forceinline__ void sparse_add_process_row_values(int row_id, const int *__restrict__ row_offsets, const int *__restrict__ col_indices, const Value_type *__restrict vals, int lane_id, Hash_map<int, Value_type, SMEM_SIZE, NUM_HASH_FCTS, WARP_SIZE> &map, int *wk_status)
{
    // Load the range of the row.
    int col_tmp = -1;

    if ( lane_id < 2 )
    {
        col_tmp = utils::Ld<utils::LD_NC>::load( &row_offsets[row_id + lane_id] );
    }

    int col_it  = utils::shfl( col_tmp, 0 );
    int col_end = utils::shfl( col_tmp, 1 );

    // Iterate over the columns of A.
    for ( col_it += lane_id ; utils::any(col_it < col_end) ; col_it += WARP_SIZE )
    {
        const bool is_active = col_it < col_end;
        // Columns of A maps to rows of B. Each thread of the warp loads its A-col/B-row ID.
        int col_id(-1);
        Value_type value = amgx::types::util<Value_type>::get_zero();

        if ( is_active )
        {
            col_id = utils::Ld<utils::LD_NC>::load( &col_indices[col_it] );
            value  = utils::Ld<utils::LD_NC>::load( &vals[col_it] );
        }

        map.insert( col_id, value, wk_status );
    }
}




template<int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE, bool COUNT_ONLY >
__global__ __launch_bounds__( CTA_SIZE )
void
count_non_zeroes_RAP_ext_kernel( const int RAP_int_num_rows,
                                 const int *__restrict RAP_int_rows,
                                 const int *__restrict RAP_int_cols,
                                 int **RAP_ext_row_ptrs,
                                 int **RAP_ext_col_ptrs,
                                 int *__restrict RAP_rows,
                                 int *__restrict RAP_cols,
                                 int **flagArray_ptrs,
                                 const int gmem_size,
                                 int *g_keys,
                                 int *wk_work_queue,
                                 int *wk_status,
                                 int num_neighbors,
                                 int global_id )
{
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // The hash keys stored in shared memory.
    __shared__ /*volatile*/ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id( );
    const int lane_id = utils::lane_id( );
    // First threads load the row IDs of A needed by the CTA...
    int rap_int_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_set<int, SMEM_SIZE, 4, WARP_SIZE> set( &s_keys[warp_id * SMEM_SIZE], &g_keys[rap_int_row_id * gmem_size], gmem_size );

    // Loop over rows of RAP_ext
    for ( ; rap_int_row_id < RAP_int_num_rows; rap_int_row_id = get_work( wk_work_queue, warp_id ) )
    {
        // Make sure we have to proceed.
        if ( COUNT_ONLY )
        {
            volatile int *status = reinterpret_cast<volatile int *>( wk_status );

            if ( set.has_failed() || *status != 0 )
            {
                return;
            }
        }

        // Clear the set.
        set.clear();
        // ---------------------------------
        // First process RAP_int
        // ---------------------------------
        bool print_flag = false;

        if (rap_int_row_id == 3 && global_id == 0)
        {
            print_flag = true;
        }

        sparse_add_process_row<SMEM_SIZE, 4, WARP_SIZE, COUNT_ONLY>(rap_int_row_id, RAP_int_rows, RAP_int_cols, lane_id, set, wk_status, global_id, print_flag);

        // ---------------------------------
        // Then process RAP_ext
        // ---------------------------------

        for (int i = 0; i < num_neighbors; i++)
        {
            int flag = flagArray_ptrs[i][rap_int_row_id];

            if (flag != -1)
            {
                int *RAP_ext_rows = RAP_ext_row_ptrs[i];
                int *RAP_ext_cols = RAP_ext_col_ptrs[i];
                int pos_in_row_ext = flag;
                sparse_add_process_row<SMEM_SIZE, 4, WARP_SIZE, COUNT_ONLY>(pos_in_row_ext, RAP_ext_rows, RAP_ext_cols, lane_id, set, wk_status, global_id, print_flag);
            }
        }

        // Store the results.
        if ( COUNT_ONLY )
        {
            int count = set.compute_size();

            if ( lane_id == 0 )
            {
                RAP_rows[rap_int_row_id] = count;
            }
        }
        else
        {
            int rap_col_tmp = -1;

            if ( lane_id < 2 )
            {
                rap_col_tmp = utils::Ld<utils::LD_NC>::load( &RAP_rows[rap_int_row_id + lane_id] );
            }

            int rap_col_it  = utils::shfl( rap_col_tmp, 0 );
            int rap_col_end = utils::shfl( rap_col_tmp, 1 );
            // Store the results.
            int count = rap_col_end - rap_col_it;

            if ( count == 0 )
            {
                continue;
            }

            set.store( count, &RAP_cols[rap_col_it] );
        }
    }
}




///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

struct Without_external_diag
{
    static __device__ __forceinline__ bool is_active  ( int a_col_it, int a_col_end ) { return a_col_it < a_col_end; }
    static __device__ __forceinline__ bool is_boundary( int a_col_it, int a_col_end ) { return false; }
};

// ====================================================================================================================

struct With_external_diag
{
    static __device__ __forceinline__ bool is_active  ( int a_col_it, int a_col_end ) { return a_col_it <= a_col_end; }
    static __device__ __forceinline__ bool is_boundary( int a_col_it, int a_col_end ) { return a_col_it == a_col_end; }
};

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template< int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE, bool COUNT_ONLY, typename Diag_traits >
__global__ __launch_bounds__( CTA_SIZE )
void
count_non_zeroes_ilu1_kernel( const int A_num_rows,
                              const int *__restrict A_rows,
                              const int *__restrict A_cols,
                              const int *__restrict A_coloring,
                              int *__restrict C_rows,
                              int *__restrict C_cols,
                              const int gmem_size,
                              int *g_keys,
                              int *wk_work_queue,
                              int *wk_status )
{
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Tables to broadcast values.
    __shared__ volatile int s_b_rows[CTA_SIZE], s_b_colors[CTA_SIZE];
    // The hash keys stored in shared memory.
    __shared__ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id( );
    const int lane_id = utils::lane_id( );
    // First threads load the row IDs of A needed by the CTA...
    int a_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_set<int, SMEM_SIZE, 4, WARP_SIZE> set( &s_keys[warp_id * SMEM_SIZE], &g_keys[a_row_id * gmem_size], gmem_size );

    // Loop over rows of A.
    for ( ; a_row_id < A_num_rows ; a_row_id = get_work( wk_work_queue, warp_id ) )
    {
        // Make sure we have to proceed.
        if ( COUNT_ONLY )
        {
            volatile int *status = reinterpret_cast<volatile int *>( wk_status );

            if ( set.has_failed() || *status != 0 )
            {
                return;
            }
        }

        // Clear the set.
        set.clear();
        // The color of the row.
        int a_row_color = A_coloring[a_row_id];
        // Load the range of the row.
        int a_col_tmp = -1;

        if ( lane_id < 2 )
        {
            a_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[a_row_id + lane_id] );
        }

        int a_col_it  = utils::shfl( a_col_tmp, 0 );
        int a_col_end = utils::shfl( a_col_tmp, 1 );

        // Iterate over the columns of A.
        for ( a_col_it += lane_id ; utils::any(Diag_traits::is_active(a_col_it, a_col_end)) ; a_col_it += WARP_SIZE )
        {
            // Is it an active thread.
            const bool is_active = a_col_it < a_col_end;
            // Columns of A maps to rows of B. Each thread of the warp loads its A-col/B-row ID.
            int b_row_id = -1;

            if ( is_active )
            {
                b_row_id = utils::Ld<utils::LD_NC>::load( &A_cols[a_col_it] );
            }

            if ( Diag_traits::is_boundary(a_col_it, a_col_end) )
            {
                b_row_id = a_row_id;
            }

            // Push the columns in the set.
            set.insert( b_row_id, COUNT_ONLY ? wk_status : NULL );

            // Skip computation if the color of the row is 0.
            if ( a_row_color != 0 )
            {
                // Gather the colors of the columns.
                int b_row_color = -1;

                if ( is_active )
                {
                    b_row_color = A_coloring[b_row_id];
                }

                // The number of valid rows.
                int pred = is_active && b_row_color < a_row_color;
                int vote = utils::ballot( pred );
                int dest = __popc( vote & utils::lane_mask_lt() );

                if ( pred )
                {
                    s_b_rows  [warp_id * WARP_SIZE + dest] = b_row_id;
                    s_b_colors[warp_id * WARP_SIZE + dest] = b_row_color; // TODO: store an int2 rather than 2 ints of SM35 with 64bit banks.
                }

                const int num_rows = __popc( vote );

                // Uniform loop: threads collaborate to load other elements.
                for ( int k = 0 ; k < num_rows ; ++k )
                {
                    // Threads in the warp proceeds columns of B in the range [bColIt, bColEnd).
                    const int uniform_b_row_id = s_b_rows  [warp_id * WARP_SIZE + k];
                    const int uniform_b_color  = s_b_colors[warp_id * WARP_SIZE + k];
                    // Load the range of the row of B.
                    int b_col_tmp = -1;

                    if ( lane_id < 2 )
                    {
                        b_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[uniform_b_row_id + lane_id] );
                    }

                    int b_col_it  = utils::shfl( b_col_tmp, 0 );
                    int b_col_end = utils::shfl( b_col_tmp, 1 );

                    // Iterate over the range of columns of B.
                    for ( b_col_it += lane_id ; utils::any(b_col_it < b_col_end) ; b_col_it += WARP_SIZE )
                    {
                        int b_col_id = -1, b_col_color = -1;

                        if ( b_col_it < b_col_end )
                        {
                            b_col_id    = utils::Ld<utils::LD_NC>::load( &A_cols[b_col_it] );
                            b_col_color = utils::Ld<utils::LD_NC>::load( &A_coloring[b_col_id] );
                        }

                        int item = -1;

                        if ( b_col_color >= uniform_b_color && b_col_color != a_row_color )
                        {
                            item = b_col_id;
                        }

                        set.insert( item, COUNT_ONLY ? wk_status : NULL );
                    }
                }
            }
        }

        // Store the results.
        if ( COUNT_ONLY )
        {
            int count = set.compute_size();

            if ( lane_id == 0 )
            {
                C_rows[a_row_id] = count;
            }
        }
        else
        {
            int c_col_tmp = -1;

            if ( lane_id < 2 )
            {
                c_col_tmp = utils::Ld<utils::LD_NC>::load( &C_rows[a_row_id + lane_id] );
            }

            int c_col_it  = utils::shfl( c_col_tmp, 0 );
            int c_col_end = utils::shfl( c_col_tmp, 1 );
            // Store the results.
            int count = c_col_end - c_col_it;

            if ( count == 0 )
            {
                continue;
            }

            set.store( count, &C_cols[c_col_it] );
        }
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template< int NUM_THREADS_PER_ROW, int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE, bool COUNT_ONLY, typename Diag_traits >
__global__ __launch_bounds__( CTA_SIZE )
void
count_non_zeroes_ilu1_kernel( const int A_num_rows,
                              const int *__restrict A_rows,
                              const int *__restrict A_cols,
                              const int *__restrict A_coloring,
                              int *__restrict C_rows,
                              int *__restrict C_cols,
                              const int gmem_size,
                              int *g_keys,
                              int *wk_work_queue,
                              int *wk_status )
{
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    const int NUM_LOADED_ROWS = WARP_SIZE / NUM_THREADS_PER_ROW;
    // Tables to broadcast values.
    __shared__ volatile int s_b_rows[CTA_SIZE], s_b_colors[CTA_SIZE];
    // The hash keys stored in shared memory.
    __shared__ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id( );
    const int lane_id = utils::lane_id( );
    // Constants.
    const int lane_id_div_num_threads = lane_id / NUM_THREADS_PER_ROW;
    const int lane_id_mod_num_threads = lane_id % NUM_THREADS_PER_ROW;
    // First threads load the row IDs of A needed by the CTA...
    int a_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_set<int, SMEM_SIZE, 4, WARP_SIZE> set( &s_keys[warp_id * SMEM_SIZE], &g_keys[a_row_id * gmem_size], gmem_size );

    // Loop over rows of A.
    for ( ; a_row_id < A_num_rows ; a_row_id = get_work( wk_work_queue, warp_id ) )
    {
        // Make sure we have to proceed.
        if ( COUNT_ONLY )
        {
            volatile int *status = reinterpret_cast<volatile int *>( wk_status );

            if ( set.has_failed() || *status != 0 )
            {
                return;
            }
        }

        // Clear the set.
        set.clear();
        // The color of the row.
        int a_row_color = A_coloring[a_row_id];
        // Load the range of the row.
        int a_col_tmp = -1;

        if ( lane_id < 2 )
        {
            a_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[a_row_id + lane_id] );
        }

        int a_col_it  = utils::shfl( a_col_tmp, 0 );
        int a_col_end = utils::shfl( a_col_tmp, 1 );

        // Iterate over the columns of A.
        for ( a_col_it += lane_id ; utils::any(Diag_traits::is_active(a_col_it, a_col_end)) ; a_col_it += WARP_SIZE )
        {
            // Is it an active thread.
            const bool is_active = a_col_it < a_col_end;
            // Columns of A maps to rows of B. Each thread of the warp loads its A-col/B-row ID.
            int b_row_id = -1;

            if ( is_active )
            {
                b_row_id = utils::Ld<utils::LD_NC>::load( &A_cols[a_col_it] );
            }

            if ( Diag_traits::is_boundary(a_col_it, a_col_end) )
            {
                b_row_id = a_row_id;
            }

            // Push the columns in the set.
            set.insert( b_row_id, COUNT_ONLY ? wk_status : NULL );

            // Skip computation if the color of the row is 0.
            if ( a_row_color != 0 )
            {
                // Gather the colors of the columns.
                int b_row_color = -1;

                if ( is_active )
                {
                    b_row_color = A_coloring[b_row_id];
                }

                // The number of valid rows.
                int pred = is_active && b_row_color < a_row_color;
                int vote = utils::ballot( pred );
                int dest = __popc( vote & utils::lane_mask_lt() );

                if ( pred )
                {
                    s_b_rows  [warp_id * WARP_SIZE + dest] = b_row_id;
                    s_b_colors[warp_id * WARP_SIZE + dest] = b_row_color;
                }

                const int num_rows = __popc( vote );

                // Uniform loop: threads collaborate to load other elements.
                for ( int k = 0 ; k < num_rows ; k += NUM_LOADED_ROWS )
                {
                    int local_k = k + lane_id_div_num_threads;
                    // Is it an active thread.
                    bool is_active_k = local_k < num_rows;
                    // Threads in the warp proceeds columns of B in the range [bColIt, bColEnd).
                    int uniform_b_row_id = -1, uniform_b_color = -1;

                    if ( is_active_k )
                    {
                        uniform_b_row_id = s_b_rows  [warp_id * WARP_SIZE + local_k];
                        uniform_b_color  = s_b_colors[warp_id * WARP_SIZE + local_k];
                    }

                    // Load the range of the row of B.
                    int b_col_tmp = -1;

                    if ( is_active_k && lane_id_mod_num_threads < 2 )
                    {
                        b_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[uniform_b_row_id + lane_id_mod_num_threads] );
                    }

                    int b_col_it  = utils::shfl( b_col_tmp, lane_id_div_num_threads * NUM_THREADS_PER_ROW + 0 );
                    int b_col_end = utils::shfl( b_col_tmp, lane_id_div_num_threads * NUM_THREADS_PER_ROW + 1 );

                    // Iterate over the range of columns of B.
                    for ( b_col_it += lane_id_mod_num_threads ; utils::any(b_col_it < b_col_end) ; b_col_it += NUM_THREADS_PER_ROW )
                    {
                        int b_col_id = -1, b_col_color = -1;

                        if ( b_col_it < b_col_end )
                        {
                            b_col_id    = utils::Ld<utils::LD_NC>::load( &A_cols[b_col_it] );
                            b_col_color = utils::Ld<utils::LD_NC>::load( &A_coloring[b_col_id] );
                        }

                        int item = -1;

                        if ( b_col_color >= uniform_b_color && b_col_color != a_row_color )
                        {
                            item = b_col_id;
                        }

                        set.insert( item, COUNT_ONLY ? wk_status : NULL );
                    }
                }
            }
        }

        // Store the results.
        if ( COUNT_ONLY )
        {
            int count = set.compute_size();

            if ( lane_id == 0 )
            {
                C_rows[a_row_id] = count;
            }
        }
        else
        {
            int c_col_tmp = -1;

            if ( lane_id < 2 )
            {
                c_col_tmp = utils::Ld<utils::LD_NC>::load( &C_rows[a_row_id + lane_id] );
            }

            int c_col_it  = utils::shfl( c_col_tmp, 0 );
            int c_col_end = utils::shfl( c_col_tmp, 1 );
            // Store the results.
            int count = c_col_end - c_col_it;

            if ( count == 0 )
            {
                continue;
            }

            set.store( count, &C_cols[c_col_it] );
        }
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template< typename Value_type, int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE >
__global__ __launch_bounds__( CTA_SIZE, 6 )
void
compute_values_kernel( const int A_num_rows,
                       const int *__restrict A_rows,
                       const int *__restrict A_cols,
                       const Value_type *__restrict A_vals,
                       const int *__restrict B_rows,
                       const int *__restrict B_cols,
                       const Value_type *__restrict B_vals,
                       const int *__restrict C_rows,
                       int *__restrict C_cols,
                       Value_type *__restrict C_vals,
                       int *Aq1,
                       int *Bq1,
                       int *Aq2,
                       int *Bq2,
                       const int gmem_size,
                       int *g_keys,
                       Value_type *g_vals,
                       int *wk_work_queue,
                       int *wk_status )
{
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // The hash keys stored in shared memory.
    __shared__ /*volatile*/ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The hash values stored in shared memory.
    __shared__ Value_type s_vals[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id();
    const int lane_id = utils::lane_id();
    // First threads load the row IDs of A needed by the CTA...
    int a_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_map<int, Value_type, SMEM_SIZE, 4, WARP_SIZE> map(&s_keys[warp_id * SMEM_SIZE],
            &g_keys[a_row_id * gmem_size],
            &s_vals[warp_id * SMEM_SIZE],
            &g_vals[a_row_id * gmem_size],
            gmem_size );

    // Loop over rows of A.
    for ( ; a_row_id < A_num_rows ; a_row_id = get_work( wk_work_queue, warp_id ) )
    {
        int c_row_id = a_row_id;

        if (Aq1 != NULL)
        {
            a_row_id = Aq1[a_row_id];
        }

        // Clear the map.
        map.clear();
        // Load the range of the row.
        int a_col_tmp = -1;

        if ( lane_id < 2 )
        {
            a_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[a_row_id + lane_id] );
        }

        int a_col_it  = utils::shfl( a_col_tmp, 0 );
        int a_col_end = utils::shfl( a_col_tmp, 1 );

        // Iterate over the columns of A.
        for ( a_col_it += lane_id ; utils::any(a_col_it < a_col_end) ; a_col_it += WARP_SIZE )
        {
            // Is it an active thread.
            const bool is_active = a_col_it < a_col_end;
            // Columns of A maps to rows of B. Each thread of the warp loads its A-col/B-row ID.
            int b_row_id = -1;
            Value_type a_value = amgx::types::util<Value_type>::get_zero();

            if ( is_active )
            {
                b_row_id = utils::Ld<utils::LD_NC>::load( &A_cols[a_col_it] );
                a_value  = utils::Ld<utils::LD_NC>::load( &A_vals[a_col_it] );

                //b_row_id is actually column of A
                if (Aq2 != NULL)
                {
                    b_row_id = Aq2[b_row_id];
                }

                if (Bq1 != NULL)
                {
                    b_row_id = Bq1[b_row_id];
                }
            }

            const int num_rows = __popc( utils::ballot(is_active) );

            // Uniform loop: threads collaborate to load other elements.
            for ( int k = 0 ; k < num_rows ; ++k )
            {
                // Threads in the warp proceeds columns of B in the range [bColIt, bColEnd).
                const int uniform_b_row_id = utils::shfl( b_row_id, k );
                // The value of A.
                const Value_type uniform_a_value = utils::shfl( a_value, k );
                // Load the range of the row of B.
                int b_col_tmp = -1;

                if ( lane_id < 2 )
                {
                    b_col_tmp = utils::Ld<utils::LD_NC>::load( &B_rows[uniform_b_row_id + lane_id] );
                }

                int b_col_it  = utils::shfl( b_col_tmp, 0 );
                int b_col_end = utils::shfl( b_col_tmp, 1 );

                // Iterate over the range of columns of B.
                for ( b_col_it += lane_id ; utils::any(b_col_it < b_col_end) ; b_col_it += WARP_SIZE )
                {
                    int b_col_id = -1;
                    Value_type b_value = amgx::types::util<Value_type>::get_zero();

                    if ( b_col_it < b_col_end )
                    {
                        b_col_id = utils::Ld<utils::LD_NC>::load( &B_cols[b_col_it] );
                        b_value  = utils::Ld<utils::LD_NC>::load( &B_vals[b_col_it] );

                        if (Bq2 != NULL)
                        {
                            b_col_id = Bq2[b_col_id];
                        }
                    }

                    map.insert( b_col_id, uniform_a_value * b_value, wk_status );
                }
            }
        }

        // Store the results.
        int c_col_tmp = -1;

        if ( lane_id < 2 )
        {
            c_col_tmp = utils::Ld<utils::LD_NC>::load( &C_rows[c_row_id + lane_id] );
        }

        int c_col_it  = utils::shfl( c_col_tmp, 0 );
        int c_col_end = utils::shfl( c_col_tmp, 1 );
        // Store the results.
        int count = c_col_end - c_col_it;

        if ( count == 0 )
        {
            continue;
        }

        map.store( count, &C_cols[c_col_it], &C_vals[c_col_it] );
    }

}

template <int GROUP_SIZE, int CTA_SIZE, int HASH_SIZE>
__global__ 
void count_non_zeroes_kernel_opt( const int A_num_rows,
                       const int *__restrict A_rows,
                       const int *__restrict A_cols,
                       const int *__restrict B_rows,
                       const int *__restrict B_cols,
                       int *__restrict C_row_counts)
{
    // Defines for hash container
    constexpr int SLOT_VACANT = -1;

    // Group indices
    constexpr int ngroups = CTA_SIZE / GROUP_SIZE;
    const int group_id = threadIdx.x / GROUP_SIZE;
    const int lane_id = threadIdx.x % GROUP_SIZE;

    // One row of A per group
    const int a_row_id = blockIdx.x * ngroups + group_id;

    // Block-level hash container storage
    __shared__ int key_s[ngroups*HASH_SIZE];
    __shared__ int counts[ngroups];

    // Initialise the keys and values.
#pragma unroll
    for(int i = threadIdx.x; i < ngroups*HASH_SIZE; i += CTA_SIZE)
    {
        key_s[i] = SLOT_VACANT; // Inserted keys will be in range [0,N]
    }

    for(int i = 0; i < ngroups; ++i)
    {
        counts[i] = 0;
    }

    //__syncwarp();
    __syncthreads();

    int* key_group_s = &key_s[group_id*HASH_SIZE];

    if(a_row_id < A_num_rows)
    {
        // Distributed columns of row of A over threads in group.
        for (int a_col_it = A_rows[a_row_id] + lane_id; a_col_it < A_rows[a_row_id + 1]; a_col_it += GROUP_SIZE)
        {
            int a_col_id = A_cols[a_col_it];

            // Outer product of element of A and row of B.
            for (int b_col_it = B_rows[a_col_id]; b_col_it < B_rows[a_col_id + 1] ; ++b_col_it)
            {
                int b_col_id = B_cols[b_col_it];

                int hash = b_col_id % HASH_SIZE;

                // By construction this algorithm should guarantee 
                // all keys can be inserted
                while(true)
                {
                    // If the slot is vacant, then attempt acquire
                    int key = key_group_s[hash];

                    // Check if the key was already b_col_id, or was already set 
                    // Insert the product
                    if(key == b_col_id)
                    {
                        break;
                    }

                    if(key == SLOT_VACANT)
                    {
                        int new_key = atomicCAS(&key_group_s[hash], SLOT_VACANT, b_col_id);

                        if(new_key == SLOT_VACANT || new_key == b_col_id)
                        {
                            break;
                        }
                    }

                    // We did not secure a slot, so linear probe to next slot
                    hash = (hash + 1) % HASH_SIZE;
                }
            }
        }
    }

    //__syncwarp();
    __syncthreads();

    // Store the results.
    if(a_row_id < A_num_rows)
    {
#pragma unroll
        for(int i = lane_id; i < HASH_SIZE; i += GROUP_SIZE)
        {
            if(key_group_s[i] != SLOT_VACANT)
            {
                atomicAdd(&counts[group_id], 1);
            }
        }
    }

    __syncthreads();

    if(a_row_id < A_num_rows && lane_id == 0)
    {
        C_row_counts[a_row_id] = counts[group_id];
    }
}

template <int GROUP_SIZE, int CTA_SIZE, int HASH_SIZE, class ValueType>
__global__ 
void compute_values_kernel_opt( const int A_num_rows,
                       const int *__restrict A_rows,
                       const int *__restrict A_cols,
                       const ValueType *__restrict A_vals,
                       const int *__restrict B_rows,
                       const int *__restrict B_cols,
                       const ValueType *__restrict B_vals,
                       const int *__restrict C_rows,
                       int *__restrict C_cols,
                       ValueType *__restrict C_vals)
{
    // Defines for hash container
    constexpr int SLOT_VACANT = -1;

    // Group indices
    constexpr int ngroups = CTA_SIZE / GROUP_SIZE;
    const int group_id = threadIdx.x / GROUP_SIZE;
    const int lane_id = threadIdx.x % GROUP_SIZE;

    // One row of A per group
    const int a_row_id = blockIdx.x * ngroups + group_id;

    // Dynamic sized shared memory
    extern __shared__ int s[];

    // Block-level hash container storage
    int* key_s = s;
    ValueType* data_s = (ValueType*)&key_s[ngroups*HASH_SIZE];
    int* col_ind_s = (int*)&data_s[ngroups*HASH_SIZE];

    // Group-level hash containers
    int* key_group_s = &key_s[group_id*HASH_SIZE];
    ValueType* data_group_s = &data_s[group_id*HASH_SIZE];

    // Initialise the keys and values.
#pragma unroll
    for(int i = threadIdx.x; i < ngroups*HASH_SIZE; i += CTA_SIZE)
    {
        key_s[i] = SLOT_VACANT; // Inserted keys will be in range [0,N]
        data_s[i] = amgx::types::util<ValueType>::get_zero(); // We will sum into values
    }

    if(lane_id == 0)
    {
        col_ind_s[group_id] = 0;
    }

    //__syncwarp();
    __syncthreads();

    if(a_row_id < A_num_rows)
    {
        // Distributed columns of row of A over threads in group.
        for (int a_col_it = A_rows[a_row_id] + lane_id; a_col_it < A_rows[a_row_id + 1]; a_col_it += GROUP_SIZE)
        {
            int a_col_id = A_cols[a_col_it];

            // Outer product of element of A and row of B.
            for (int b_col_it = B_rows[a_col_id]; b_col_it < B_rows[a_col_id + 1] ; ++b_col_it)
            {
                ValueType val = A_vals[a_col_it]*B_vals[b_col_it];

                // Short circuit if zero
                if(amgx::types::util<ValueType>::is_zero(val)) continue;

                int b_col_id = B_cols[b_col_it];

                // XXX UPDATE HASHING APPROACH
                int hash = b_col_id % HASH_SIZE;

                // By construction this algorithm should guarantee 
                // all keys can be inserted
                while(true)
                {
                    // If the slot is vacant, then attempt acquire
                    int key = key_group_s[hash];
                    if(key == SLOT_VACANT)
                    {
                        int new_key = atomicCAS(&key_group_s[hash], SLOT_VACANT, b_col_id);
                        if(new_key == SLOT_VACANT || new_key == b_col_id)
                        {
                            key = b_col_id;
                        }
                    }

                    // Check if the key was already b_col_id, or was already set 
                    // Insert the product
                    if(key == b_col_id)
                    {
                        utils::atomic_add(&data_group_s[hash], val);
                        break;
                    }

                    // We did not secure a slot, so linear probe to next slot
                    hash = (hash + 1) % HASH_SIZE;
                }
            }
        }
    }

    //__syncwarp();
    __syncthreads();

    // Store the results.
    int c_row_id  = a_row_id;
    int c_col_it  = C_rows[c_row_id];

    if(a_row_id < A_num_rows)
    {
#pragma unroll
        for(int i = lane_id; i < HASH_SIZE; i += GROUP_SIZE)
        {
            if(key_group_s[i] != SLOT_VACANT)
            {
                // Short circuit if zero
                if(amgx::types::util<ValueType>::is_zero(data_group_s[i])) continue;

                int ind = atomicAdd(&col_ind_s[group_id], 1);

                C_cols[c_col_it + ind] = key_group_s[i];
                C_vals[c_col_it + ind] = data_group_s[i];
            }
        }
    }
}

template< int NUM_THREADS_PER_ROW, typename Value_type, int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE >
__global__ __launch_bounds__( CTA_SIZE, 6 )
void
compute_values_kernel( const int A_num_rows,
                       const int *__restrict A_rows,
                       const int *__restrict A_cols,
                       const Value_type *__restrict A_vals,
                       const int *__restrict B_rows,
                       const int *__restrict B_cols,
                       const Value_type *__restrict B_vals,
                       const int *__restrict C_rows,
                       int *__restrict C_cols,
                       Value_type *__restrict C_vals,
                       int *Aq1,
                       int *Bq1,
                       int *Aq2,
                       int *Bq2,
                       const int gmem_size,
                       int *g_keys,
                       Value_type *g_vals,
                       int *wk_work_queue,
                       int *wk_status )
{

    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    const int NUM_LOADED_ROWS = WARP_SIZE / NUM_THREADS_PER_ROW;
    // The hash keys stored in shared memory.
    __shared__ /*volatile*/ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The hash values stored in shared memory.
    __shared__ Value_type s_vals[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id( );
    const int lane_id = utils::lane_id( );
    // Constants.
    const int lane_id_div_num_threads = lane_id / NUM_THREADS_PER_ROW;
    const int lane_id_mod_num_threads = lane_id % NUM_THREADS_PER_ROW;
    // First threads load the row IDs of A needed by the CTA...
    int a_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_map<int, Value_type, SMEM_SIZE, 4, WARP_SIZE> map(&s_keys[warp_id * SMEM_SIZE],
            &g_keys[a_row_id * gmem_size],
            &s_vals[warp_id * SMEM_SIZE],
            &g_vals[a_row_id * gmem_size],
            gmem_size );

    // Loop over rows of A.
    for ( ; a_row_id < A_num_rows ; a_row_id = get_work( wk_work_queue, warp_id ) )
    {
        int c_row_id = a_row_id;

        if (Aq1 != NULL)
        {
            a_row_id = Aq1[a_row_id];
        }

        // Clear the map.
        map.clear();
        // Load the range of the row.
        int a_col_tmp = -1;

        if ( lane_id < 2 )
        {
            a_col_tmp = utils::Ld<utils::LD_NC>::load( &A_rows[a_row_id + lane_id] );
        }

        int a_col_it  = utils::shfl( a_col_tmp, 0 );
        int a_col_end = utils::shfl( a_col_tmp, 1 );

        // Iterate over the columns of A.
        for ( a_col_it += lane_id ; utils::any(a_col_it < a_col_end) ; a_col_it += WARP_SIZE )
        {
            // Is it an active thread.
            const bool is_active = a_col_it < a_col_end;
            // Columns of A maps to rows of B. Each thread of the warp loads its A-col/B-row ID.
            int b_row_id(-1);
            Value_type a_value = amgx::types::util<Value_type>::get_zero();

            if ( is_active )
            {
                b_row_id = utils::Ld<utils::LD_NC>::load( &A_cols[a_col_it] );
                a_value  = utils::Ld<utils::LD_NC>::load( &A_vals[a_col_it] );

                //b_row_id is actually column of A
                if (Aq2 != NULL)
                {
                    b_row_id = Aq2[b_row_id];
                }

                if (Bq1 != NULL)
                {
                    b_row_id = Bq1[b_row_id];
                }
            }

            const int num_rows = __popc( utils::ballot(is_active) );

            // Uniform loop: threads collaborate to load other elements.
            for ( int k = 0 ; k < num_rows ; k += NUM_LOADED_ROWS )
            {
                int local_k = k + lane_id_div_num_threads;
                // Is it an active thread.
                bool is_active_k = local_k < num_rows;
                // Threads in the warp proceeds columns of B in the range [bColIt, bColEnd).
                const int uniform_b_row_id = utils::shfl( b_row_id, k + lane_id_div_num_threads );
                // The value of A.
                const Value_type uniform_a_value = utils::shfl( a_value, k + lane_id_div_num_threads );
                // Load the range of the row of B.
                int b_col_tmp = -1;

                if ( is_active_k && lane_id_mod_num_threads < 2 )
                {
                    b_col_tmp = utils::Ld<utils::LD_NC>::load( &B_rows[uniform_b_row_id + lane_id_mod_num_threads] );
                }

                int b_col_it  = utils::shfl( b_col_tmp, lane_id_div_num_threads * NUM_THREADS_PER_ROW + 0 );
                int b_col_end = utils::shfl( b_col_tmp, lane_id_div_num_threads * NUM_THREADS_PER_ROW + 1 );

                // Iterate over the range of columns of B.
                for ( b_col_it += lane_id_mod_num_threads ; utils::any(b_col_it < b_col_end) ; b_col_it += NUM_THREADS_PER_ROW )
                {
                    int b_col_id(-1);
                    Value_type b_value = amgx::types::util<Value_type>::get_zero();

                    if ( b_col_it < b_col_end )
                    {
                        b_col_id = utils::Ld<utils::LD_NC>::load( &B_cols[b_col_it] );
                        b_value  = utils::Ld<utils::LD_NC>::load( &B_vals[b_col_it] );

                        //b_col_id is actually column of B
                        if (Bq2 != NULL)
                        {
                            b_col_id = Bq2[b_col_id];
                        }
                    }

                    map.insert( b_col_id, uniform_a_value * b_value, wk_status );
                }
            }
        }

        // Store the results.
        int c_col_tmp = -1;

        if ( lane_id < 2 )
        {
            c_col_tmp = utils::Ld<utils::LD_NC>::load( &C_rows[c_row_id + lane_id] );
        }

        int c_col_it  = utils::shfl( c_col_tmp, 0 );
        int c_col_end = utils::shfl( c_col_tmp, 1 );
        // Store the results.
        int count = c_col_end - c_col_it;

        if ( count == 0 )
        {
            continue;
        }

        map.store( count, &C_cols[c_col_it], &C_vals[c_col_it] );
    }

}


template< typename Value_type, int CTA_SIZE, int SMEM_SIZE, int WARP_SIZE >
__global__ __launch_bounds__( CTA_SIZE, 6 )
void
compute_values_RAP_ext_kernel( const int RAP_int_num_rows,
                               const int *__restrict RAP_int_rows,
                               const int *__restrict RAP_int_cols,
                               const Value_type *__restrict RAP_int_vals,
                               int **RAP_ext_row_ptrs,
                               int **RAP_ext_col_ptrs,
                               Value_type **RAP_ext_val_ptrs,
                               int *__restrict RAP_rows,
                               int *__restrict RAP_cols,
                               Value_type *__restrict RAP_vals,
                               int **flagArray_ptrs,
                               const int gmem_size,
                               int *g_keys,
                               Value_type *g_vals,
                               int *wk_work_queue,
                               int num_neighbors,
                               int *wk_status )
{
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // The hash keys stored in shared memory.
    __shared__ /*volatile*/ int s_keys[NUM_WARPS * SMEM_SIZE];
    // The hash values stored in shared memory.
    __shared__ Value_type s_vals[NUM_WARPS * SMEM_SIZE];
    // The coordinates of the thread inside the CTA/warp.
    const int warp_id = utils::warp_id( );
    const int lane_id = utils::lane_id( );
    // First threads load the row IDs of A needed by the CTA...
    int rap_int_row_id = blockIdx.x * NUM_WARPS + warp_id;
    // Create local storage for the set.
    Hash_map<int, Value_type, SMEM_SIZE, 4, WARP_SIZE> map( &s_keys[warp_id * SMEM_SIZE],
            &g_keys[rap_int_row_id * gmem_size],
            &s_vals[warp_id * SMEM_SIZE],
            &g_vals[rap_int_row_id * gmem_size],
            gmem_size );

    // Loop over rows of RAP_ext
    for ( ; rap_int_row_id < RAP_int_num_rows ; rap_int_row_id = get_work( wk_work_queue, warp_id ) )
    {
        // Clear the map.
        map.clear();
        // ---------------------------------
        // First process RAP_int
        // ---------------------------------
        sparse_add_process_row_values<Value_type, SMEM_SIZE, 4, WARP_SIZE>(rap_int_row_id, RAP_int_rows, RAP_int_cols, RAP_int_vals, lane_id, map, wk_status);

        // ---------------------------------
        // Then process RAP_ext
        // ---------------------------------

        for (int i = 0; i < num_neighbors; i++)
        {
            int flag = flagArray_ptrs[i][rap_int_row_id];

            if (flag != -1)
            {
                int *RAP_ext_rows = RAP_ext_row_ptrs[i];
                int *RAP_ext_cols = RAP_ext_col_ptrs[i];
                Value_type *RAP_ext_vals = RAP_ext_val_ptrs[i];
                int pos_in_row_ext = flag;
                sparse_add_process_row_values<Value_type, SMEM_SIZE, 4, WARP_SIZE>(pos_in_row_ext, RAP_ext_rows, RAP_ext_cols, RAP_ext_vals, lane_id, map, wk_status);
            }
        }

        // Store the results.
        int rap_col_tmp = -1;

        if ( lane_id < 2 )
        {
            rap_col_tmp = utils::Ld<utils::LD_NC>::load( &RAP_rows[rap_int_row_id + lane_id] );
        }

        int rap_col_it  = utils::shfl( rap_col_tmp, 0 );
        int rap_col_end = utils::shfl( rap_col_tmp, 1 );
        // Store the results.
        int count = rap_col_end - rap_col_it;

        if ( count == 0 )
        {
            continue;
        }

        map.store( count, &RAP_cols[rap_col_it], &RAP_vals[rap_col_it] );
    }
}




///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace csr_multiply_detail

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace amgx
{

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

enum { WARP_SIZE = 32, SMEM_SIZE = 128 };

// ====================================================================================================================

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::CSR_Multiply_Detail( bool allocate_values, int grid_size, int max_warp_count, int gmem_size ) :
    Base(allocate_values, grid_size, max_warp_count, gmem_size)
{}

// ====================================================================================================================

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::count_non_zeroes( const Matrix_d &A, const Matrix_d &B, Matrix_d &C, IVector *Aq1, IVector *Bq1, IVector *Aq2, IVector *Bq2 )
{
    const int GRID_SIZE = 1024;
    const int CTA_SIZE  = 256;
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Reset work queue.
    int work_offset = GRID_SIZE * NUM_WARPS;
    CUDA_SAFE_CALL( hipMemcpy( this->m_work_queue, &work_offset, sizeof(int), hipMemcpyHostToDevice ) );
    
    // Compute non-zero elements.
    switch ( this->m_num_threads_per_row_count )
    {
        case 2:
            csr_multiply_detail::count_non_zeroes_kernel< 2, CTA_SIZE, SMEM_SIZE, WARP_SIZE, true> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                NULL,
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                this->m_status );
            break;

        case 4:
            csr_multiply_detail::count_non_zeroes_kernel< 4, CTA_SIZE, SMEM_SIZE, WARP_SIZE, true> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                NULL,
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                this->m_status );
            break;

        case 8:
            csr_multiply_detail::count_non_zeroes_kernel< 8, CTA_SIZE, SMEM_SIZE, WARP_SIZE, true> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                NULL,
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                this->m_status );
            break;

        case 16:
            csr_multiply_detail::count_non_zeroes_kernel<16, CTA_SIZE, SMEM_SIZE, WARP_SIZE, true> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                NULL,
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                this->m_status );
            break;

        default:
            csr_multiply_detail::count_non_zeroes_kernel<CTA_SIZE, SMEM_SIZE, WARP_SIZE, true> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                NULL,
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                this->m_status );
    }

    cudaCheckError();
    //CUDA_SAFE_CALL( hipGetLastError() );

}


template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::count_non_zeroes_RAP_sparse_add( Matrix_d &RAP, const Matrix_d &RAP_int, std::vector<IVector> &RAP_ext_row_offsets, std::vector<IVector> &RAP_ext_col_indices, std::vector<MVector> &RAP_ext_values, std::vector<IVector> &RAP_ext_row_ids)

{
    const int GRID_SIZE = 1024;
    const int CTA_SIZE  = 256;
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Reset work queue.
    int work_offset = GRID_SIZE * NUM_WARPS;
    CUDA_SAFE_CALL( hipMemcpy( this->m_work_queue, &work_offset, sizeof(int), hipMemcpyHostToDevice ) );
    // This is num_owned_coarse_rows
    int RAP_size = RAP.get_num_rows();
    int RAP_int_size = RAP_int.row_offsets.size() - 1;

    if (RAP_int_size < RAP_size)
    {
        FatalError("RAP_int has less rows than RAP, need to modify sparse RAP add to handle that case\n", AMGX_ERR_NOT_IMPLEMENTED);
    }

    // Create a device vector of the raw pointers to the array
    // RAP_ext_row_ids_ptrs
    // RAP_ext_row_offsets_ptrs
    // RAP_ext_col_indices_ptrs
    // RAP_ext_values_ptrs
    int num_neighbors = RAP_ext_row_offsets.size();
    std::vector<IVector> flagArray(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        flagArray[i].resize(RAP_size);
        thrust_wrapper::fill<AMGX_device>(flagArray[i].begin(), flagArray[i].end(), -1);
    }

    cudaCheckError();
    std::vector<int *> flagArray_ptrs_h(num_neighbors);
    std::vector<int *> RAP_ext_row_offsets_ptrs_h(num_neighbors);
    std::vector<int *> RAP_ext_col_indices_ptrs_h(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        flagArray_ptrs_h[i] = amgx::thrust::raw_pointer_cast(&flagArray[i][0]);
        RAP_ext_row_offsets_ptrs_h[i] = amgx::thrust::raw_pointer_cast(&RAP_ext_row_offsets[i][0]);
        RAP_ext_col_indices_ptrs_h[i] = amgx::thrust::raw_pointer_cast(&RAP_ext_col_indices[i][0]);
    }

    device_vector_alloc<int *> flagArray_ptrs = flagArray_ptrs_h;
    device_vector_alloc<int *> RAP_ext_row_offsets_ptrs = RAP_ext_row_offsets_ptrs_h;
    device_vector_alloc<int *> RAP_ext_col_indices_ptrs = RAP_ext_col_indices_ptrs_h;

    for (int i = 0; i < num_neighbors; i++)
    {
        int size = RAP_ext_row_ids[i].size();

        if (size != 0)
        {
            int num_blocks = std::min(4096, (size + 127) / 128);
            //write the position in RAP_ext_row_ids
            csr_multiply_detail::flag_halo_rows <<< num_blocks, 128>>>(
                RAP_ext_row_ids[i].raw(),
                size,
                flagArray[i].raw(),
                i,
                RAP.manager->global_id());
        }
    }

    csr_multiply_detail::count_non_zeroes_RAP_ext_kernel<CTA_SIZE, SMEM_SIZE, WARP_SIZE, true> <<< GRID_SIZE, CTA_SIZE>>>(
        RAP_size,
        RAP_int.row_offsets.raw(),
        RAP_int.col_indices.raw(),
        amgx::thrust::raw_pointer_cast(&RAP_ext_row_offsets_ptrs[0]),
        amgx::thrust::raw_pointer_cast(&RAP_ext_col_indices_ptrs[0]),
        RAP.row_offsets.raw(),
        (int *) NULL,
        amgx::thrust::raw_pointer_cast(&flagArray_ptrs[0]),
        this->m_gmem_size,
        this->m_keys,
        this->m_work_queue,
        this->m_status,
        num_neighbors,
        RAP.manager->global_id() );
    cudaCheckError();
    //CUDA_SAFE_CALL( hipGetLastError() );
}

// ====================================================================================================================

template< int CTA_SIZE, bool COUNT_ONLY, typename Diag_traits, typename Matrix >
static void
count_non_zeroes_ilu1_dispatch( const Matrix &A, Matrix &B, int num_threads_per_row_count, int gmem_size, int *keys, int *work_queue, int *status )
{
    const int GRID_SIZE = 1024;

    switch ( num_threads_per_row_count )
    {
        case 2:
            csr_multiply_detail::count_non_zeroes_ilu1_kernel< 2, CTA_SIZE, SMEM_SIZE, WARP_SIZE, COUNT_ONLY, Diag_traits> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                amgx::thrust::raw_pointer_cast( &A.getMatrixColoring().getRowColors()[0] ),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                gmem_size,
                keys,
                work_queue,
                status );
            break;

        case 4:
            csr_multiply_detail::count_non_zeroes_ilu1_kernel< 4, CTA_SIZE, SMEM_SIZE, WARP_SIZE, COUNT_ONLY, Diag_traits> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                amgx::thrust::raw_pointer_cast( &A.getMatrixColoring().getRowColors()[0] ),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                gmem_size,
                keys,
                work_queue,
                status );
            break;

        case 8:
            csr_multiply_detail::count_non_zeroes_ilu1_kernel< 8, CTA_SIZE, SMEM_SIZE, WARP_SIZE, COUNT_ONLY, Diag_traits> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                amgx::thrust::raw_pointer_cast( &A.getMatrixColoring().getRowColors()[0] ),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                gmem_size,
                keys,
                work_queue,
                status );
            break;

        case 16:
            csr_multiply_detail::count_non_zeroes_ilu1_kernel<16, CTA_SIZE, SMEM_SIZE, WARP_SIZE, COUNT_ONLY, Diag_traits> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                amgx::thrust::raw_pointer_cast( &A.getMatrixColoring().getRowColors()[0] ),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                gmem_size,
                keys,
                work_queue,
                status );
            break;

        default:
            csr_multiply_detail::count_non_zeroes_ilu1_kernel<CTA_SIZE, SMEM_SIZE, WARP_SIZE, COUNT_ONLY, Diag_traits> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                amgx::thrust::raw_pointer_cast( &A.getMatrixColoring().getRowColors()[0] ),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                gmem_size,
                keys,
                work_queue,
                status );
    }

    cudaCheckError();
}

// ====================================================================================================================

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::count_non_zeroes_ilu1( const Matrix_d &A, Matrix_d &B )
{
    const int GRID_SIZE = 1024;

    const int CTA_SIZE  = 256;
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Reset work queue.
    int work_offset = GRID_SIZE * NUM_WARPS;
    CUDA_SAFE_CALL( hipMemcpy( this->m_work_queue, &work_offset, sizeof(int), hipMemcpyHostToDevice ) );

    // Count the number of non zeroes.
    if ( A.hasProps(DIAG) )
        count_non_zeroes_ilu1_dispatch<CTA_SIZE, true, csr_multiply_detail::With_external_diag, Matrix_d>(
            A,
            B,
            this->m_num_threads_per_row_count,
            this->m_gmem_size,
            this->m_keys,
            this->m_work_queue,
            this->m_status );
    else
        count_non_zeroes_ilu1_dispatch<CTA_SIZE, true, csr_multiply_detail::Without_external_diag, Matrix_d>(
            A,
            B,
            this->m_num_threads_per_row_count,
            this->m_gmem_size,
            this->m_keys,
            this->m_work_queue,
            this->m_status );

    // Compute non-zero elements.
    CUDA_SAFE_CALL( hipGetLastError() );
}

// ====================================================================================================================

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::compute_offsets( Matrix_d &C )
{
    amgx::thrust::device_ptr<int> offsets_begin(C.row_offsets.raw());
    amgx::thrust::device_ptr<int> offsets_end  (C.row_offsets.raw() + C.get_num_rows() + 1);
    thrust_wrapper::exclusive_scan<AMGX_device>( offsets_begin, offsets_end, offsets_begin );
    cudaCheckError();
}

// ====================================================================================================================

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::compute_sparsity( const Matrix_d &A, const Matrix_d &B, Matrix_d &C )
{
    const int GRID_SIZE = 1024;
    const int CTA_SIZE  = 256;
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Reset the work queue.
    int work_offset = GRID_SIZE * NUM_WARPS;
    CUDA_SAFE_CALL( hipMemcpy( this->m_work_queue, &work_offset, sizeof(int), hipMemcpyHostToDevice ) );

    // Compute the values.
    switch ( this->m_num_threads_per_row_count )
    {
        case 2:
            csr_multiply_detail::count_non_zeroes_kernel< 2, CTA_SIZE, SMEM_SIZE, WARP_SIZE, false> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                NULL,
                NULL,
                NULL,
                NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                NULL );
            break;

        case 4:
            csr_multiply_detail::count_non_zeroes_kernel< 4, CTA_SIZE, SMEM_SIZE, WARP_SIZE, false> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                NULL,
                NULL,
                NULL,
                NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                NULL );
            break;

        case 8:
            csr_multiply_detail::count_non_zeroes_kernel< 8, CTA_SIZE, SMEM_SIZE, WARP_SIZE, false> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                NULL,
                NULL,
                NULL,
                NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                NULL );
            break;

        case 16:
            csr_multiply_detail::count_non_zeroes_kernel<16, CTA_SIZE, SMEM_SIZE, WARP_SIZE, false> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                NULL,
                NULL,
                NULL,
                NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                NULL );
            break;

        default:
            csr_multiply_detail::count_non_zeroes_kernel<CTA_SIZE, SMEM_SIZE, WARP_SIZE, false> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                NULL,
                NULL,
                NULL,
                NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_work_queue,
                NULL );
    }

    cudaCheckError();
    //CUDA_SAFE_CALL( hipGetLastError() );
}

// ====================================================================================================================

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::compute_sparsity_ilu1( const Matrix_d &A, Matrix_d &B )
{
    const int GRID_SIZE = 1024;
    const int CTA_SIZE  = 256;
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Reset work queue.
    int work_offset = GRID_SIZE * NUM_WARPS;
    CUDA_SAFE_CALL( hipMemcpy( this->m_work_queue, &work_offset, sizeof(int), hipMemcpyHostToDevice ) );

    // Count the number of non zeroes.
    if ( A.hasProps(DIAG) )
        count_non_zeroes_ilu1_dispatch<CTA_SIZE, false, csr_multiply_detail::With_external_diag, Matrix_d>(
            A,
            B,
            this->m_num_threads_per_row_count,
            this->m_gmem_size,
            this->m_keys,
            this->m_work_queue,
            NULL );
    else
        count_non_zeroes_ilu1_dispatch<CTA_SIZE, false, csr_multiply_detail::Without_external_diag, Matrix_d>(
            A,
            B,
            this->m_num_threads_per_row_count,
            this->m_gmem_size,
            this->m_keys,
            this->m_work_queue,
            NULL );

    // Make sure it worked properly.
    CUDA_SAFE_CALL( hipGetLastError() );
}

// ====================================================================================================================

template <int CTA_SIZE>
__global__ 
void calc_max_nnz_per_row_of_C( 
                       const int A_num_rows,
                       const int *__restrict A_rows,
                       const int *__restrict A_cols,
                       const int *__restrict B_rows,
                       int *__restrict C_row_max)
{
    int a_row_id = blockIdx.x*blockDim.x + threadIdx.x;

    int expected_max_row_nnz = 0;

    if(a_row_id < A_num_rows)
    {
        for (int a_col_it = A_rows[a_row_id]; a_col_it < A_rows[a_row_id + 1]; ++a_col_it)
        {
            int a_col_id = A_cols[a_col_it];
            expected_max_row_nnz += B_rows[a_col_id+1]-B_rows[a_col_id];
        }
    }

    using BR = hipcub::BlockReduce<int, CTA_SIZE>;

    __shared__ typename BR::TempStorage max_s;
    int max_nnz_block = BR(max_s).Reduce(expected_max_row_nnz, hipcub::Max());

    if(threadIdx.x == 0)
    {
        C_row_max[blockIdx.x] = max_nnz_block;
    }
}

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
bool CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::count_non_zeroes_opt(
        const Matrix_d &A, const Matrix_d &B, Matrix_d &C, int num_threads)
{
    constexpr int cta_size = 128;

    // At least for large matrices it may be optimal to determine the maximum
    // expected hash size by investigation of the matrices

    int grid_size = A.get_num_rows() / cta_size + 1;

    IVector C_row_max_block(grid_size);
    calc_max_nnz_per_row_of_C<cta_size><<<grid_size, cta_size>>>(
                    A.get_num_rows(),
                    A.row_offsets.raw(),
                    A.col_indices.raw(),
                    B.row_offsets.raw(),
                    C_row_max_block.raw());

    int max_nnz = thrust_wrapper::reduce<AMGX_device>(
        C_row_max_block.raw(), 
        C_row_max_block.raw() + C_row_max_block.size(), 
        0, amgx::thrust::maximum<int>());

#define CNZ_OPT(group_size, hash_size) \
    csr_multiply_detail::count_non_zeroes_kernel_opt<group_size, cta_size, hash_size> \
        <<<grid_size, cta_size>>>( \
        A.get_num_rows(), \
        A.row_offsets.raw(), \
        A.col_indices.raw(), \
        B.row_offsets.raw(), \
        B.col_indices.raw(), \
        C.row_offsets.raw());

    // Operation is group per row, where group size is determined by num_threads
    switch ( num_threads )
    {
        // 16 threads per group
        case 16:
            {
                int ngroups = cta_size / 16;
                int grid_size = A.get_num_rows() / ngroups + 1;

                if(max_nnz < 256)
                { 
                    CNZ_OPT(16, 256); 
                } 
                else if(max_nnz < 512) 
                { 
                    CNZ_OPT(16, 512); 
                } 
                else if(max_nnz < 1024)
                { 
                    CNZ_OPT(16, 1024); 
                }
                else 
                {
                    return false;
                }
            }
            break;

        // 32 threads per group
        case 32:
            {
                int ngroups = cta_size / 32;
                int grid_size = A.get_num_rows() / ngroups + 1;

                if(max_nnz < 256)
                { 
                    CNZ_OPT(32, 256); 
                } 
                else if(max_nnz < 512) 
                { 
                    CNZ_OPT(32, 512); 
                } 
                else if(max_nnz < 1024)
                { 
                    CNZ_OPT(32, 1024); 
                }
                else
                {
                    return false;
                }
            }
            break;

        default:
            FatalError("count_non_zeros_opt only implemented for group size = 8, 16, 32\n", AMGX_ERR_NOT_IMPLEMENTED);
    }

    cudaCheckError();

    return true;
}


template <AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I>
template <int hash_size, int group_size>
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::cvk_opt(const Matrix_d &A, const Matrix_d &B, Matrix_d &C)
{
    typedef typename IndPrecisionMap<I>::Type Index_type;

    constexpr int cta_size = 128;
    constexpr int ngroups = cta_size / group_size;

    const int grid_size = A.get_num_rows() / ngroups + 1;

    hipDeviceProp_t deviceProps = getDeviceProperties();
    size_t max_shmem_size = deviceProps.sharedMemPerMultiprocessor;

    constexpr int shmem_size =
        (sizeof(Value_type)+sizeof(Index_type))*ngroups*hash_size + group_size; 

    if(shmem_size > max_shmem_size) 
    { 
        FatalError("In compute_values_opt the requested hash size is larger than max.\n", 
                AMGX_ERR_NOT_IMPLEMENTED); 
    } 

    hipFuncSetAttribute(reinterpret_cast<const void*>(csr_multiply_detail::compute_values_kernel_opt 
            <group_size), cta_size, hash_size, Value_type>, 
            hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size); 

    csr_multiply_detail::compute_values_kernel_opt 
        <group_size, cta_size, hash_size> 
        <<<grid_size, cta_size, shmem_size>>>( 
                A.get_num_rows(), 
                A.row_offsets.raw(), 
                A.col_indices.raw(), 
                A.values.raw(), 
                B.row_offsets.raw(), 
                B.col_indices.raw(), 
                B.values.raw(), 
                C.row_offsets.raw(), 
                C.col_indices.raw(), 
                C.values.raw()); 
}


template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::compute_values_opt( const Matrix_d &A, const Matrix_d &B, Matrix_d &C, int num_threads, int max_nnz )
{
    int C_nrows = C.get_num_rows();
    int C_nnz = C.get_num_nz();
    int C_max_nnz_per_row = max_nnz;

    // The aim is to minimise the hash size while reducing the impact of the linear
    // probing. It might actually be more optimal to just use as large tables as 
    // possible, reducing the linear probing cost and maximising the C write cost?
    float C_max_nnz_log2 = log2(static_cast<float>(C_max_nnz_per_row));
    float C_max_nnz_log2_ceil = ceil(C_max_nnz_log2);
    int C_rounded_max = static_cast<int>(2.0*pow(2.0, C_max_nnz_log2_ceil));

    // Operation is group per row, where group size is determined by num_threads
    switch ( num_threads )
    {
        case 16: // 16 threads per group
            {
                switch(C_rounded_max)
                {
                    case 2: 
                    case 4:
                    case 8: 
                    case 16:
                    case 32: cvk_opt<32, 8>(A, B, C); break;
                    case 64: cvk_opt<64, 8>(A, B, C); break;
                    case 128: cvk_opt<128, 8>(A, B, C); break;
                    case 256: cvk_opt<256, 8>(A, B, C); break;
                    case 512: cvk_opt<512, 16>(A, B, C); break;
                    default: 
                       FatalError("In compute_values_opt the requested hash size is too large.\n", AMGX_ERR_NOT_IMPLEMENTED);
                }
            }
            break;
        case 32: // Warp per group
            {
                switch(C_rounded_max)
                {
                    case 2: 
                    case 4:
                    case 8: 
                    case 16:
                    case 32: cvk_opt<32, 32>(A, B, C); break;
                    case 64: cvk_opt<64, 32>(A, B, C); break;
                    case 128: cvk_opt<128, 32>(A, B, C); break;
                    case 256: cvk_opt<256, 32>(A, B, C); break;
                    case 512: cvk_opt<512, 32>(A, B, C); break;
                    case 1024: cvk_opt<1024, 32>(A, B, C); break;
                    default: 
                       FatalError("In compute_values_opt the requested hash size is too large.\n", AMGX_ERR_NOT_IMPLEMENTED);
                }
            }
            break;
        default:
            FatalError("compute_values_opt only implemented for group size = 16, 32\n", AMGX_ERR_NOT_IMPLEMENTED);
    }

    hipDeviceSynchronize();

    cudaCheckError();
}

template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::compute_values( const Matrix_d &A, const Matrix_d &B, Matrix_d &C, int num_threads, IVector *Aq1, IVector *Bq1, IVector *Aq2, IVector *Bq2  )
{
    const int GRID_SIZE = 1024;
    const int CTA_SIZE  = 128;
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Reset the work queue.
    int work_offset = GRID_SIZE * NUM_WARPS;
    CUDA_SAFE_CALL( hipMemcpy( this->m_work_queue, &work_offset, sizeof(int), hipMemcpyHostToDevice ) );
    // Compute the values.
    int *status = NULL;

    if ( num_threads != this->m_num_threads_per_row_compute )
    {
        status = this->m_status;
    }

    		

    switch ( num_threads )
    {
        case 2:
            csr_multiply_detail::compute_values_kernel< 2, Value_type, CTA_SIZE, SMEM_SIZE, WARP_SIZE> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                A.values.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                B.values.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                C.values.raw(),
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_vals,
                this->m_work_queue,
                status );
            break;

        case 4:
            csr_multiply_detail::compute_values_kernel< 4, Value_type, CTA_SIZE, SMEM_SIZE, WARP_SIZE> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                A.values.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                B.values.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                C.values.raw(),
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_vals,
                this->m_work_queue,
                status );
            break;

        case 8:
            csr_multiply_detail::compute_values_kernel< 8, Value_type, CTA_SIZE, SMEM_SIZE, WARP_SIZE> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                A.values.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                B.values.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                C.values.raw(),
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_vals,
                this->m_work_queue,
                status );
            break;

        case 16:
            csr_multiply_detail::compute_values_kernel<16, Value_type, CTA_SIZE, SMEM_SIZE, WARP_SIZE> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                A.values.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                B.values.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                C.values.raw(),
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_vals,
                this->m_work_queue,
                status );
            break;

        default:
            csr_multiply_detail::compute_values_kernel<Value_type, CTA_SIZE, SMEM_SIZE, WARP_SIZE> <<< GRID_SIZE, CTA_SIZE>>>(
                A.get_num_rows(),
                A.row_offsets.raw(),
                A.col_indices.raw(),
                A.values.raw(),
                B.row_offsets.raw(),
                B.col_indices.raw(),
                B.values.raw(),
                C.row_offsets.raw(),
                C.col_indices.raw(),
                C.values.raw(),
                (Aq1 != NULL) ? Aq1->raw() : NULL,
                (Bq1 != NULL) ? Bq1->raw() : NULL,
                (Aq2 != NULL) ? Aq2->raw() : NULL,
                (Bq2 != NULL) ? Bq2->raw() : NULL,
                this->m_gmem_size,
                this->m_keys,
                this->m_vals,
                this->m_work_queue,
                status );
    }

    cudaCheckError();
    //CUDA_SAFE_CALL( hipGetLastError() );
}


template< AMGX_VecPrecision V, AMGX_MatPrecision M, AMGX_IndPrecision I >
void CSR_Multiply_Detail<TemplateConfig<AMGX_device, V, M, I> >::compute_values_RAP_sparse_add( Matrix_d &RAP, const Matrix_d &RAP_int, std::vector<IVector> &RAP_ext_row_offsets, std::vector<IVector> &RAP_ext_col_indices, std::vector<MVector> &RAP_ext_values, std::vector<IVector> &RAP_ext_row_ids, int num_threads)
{
    const int GRID_SIZE = 1024;
    const int CTA_SIZE  = 128;
    const int NUM_WARPS = CTA_SIZE / WARP_SIZE;
    // Reset the work queue.
    int work_offset = GRID_SIZE * NUM_WARPS;
    CUDA_SAFE_CALL( hipMemcpy( this->m_work_queue, &work_offset, sizeof(int), hipMemcpyHostToDevice ) );
    // Compute the values.
    int *status = NULL;

    if ( num_threads != this->m_num_threads_per_row_compute )
    {
        status = this->m_status;
    }

    // This is num_owned_coarse_rows
    int RAP_size = RAP.get_num_rows();
    int RAP_int_size = RAP_int.row_offsets.size() - 1;

    if (RAP_int_size < RAP_size)
    {
        FatalError("RAP_int has less rows than RAP, need to modify sparse RAP add to handle that case\n", AMGX_ERR_NOT_IMPLEMENTED);
    }

    //TODO: Optimize: reuse arrays from count nonzeros
    int num_neighbors = RAP_ext_row_offsets.size();
    std::vector<IVector> flagArray(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        flagArray[i].resize(RAP_size);
        thrust_wrapper::fill<AMGX_device>(flagArray[i].begin(), flagArray[i].end(), -1);
    }

    cudaCheckError();
    std::vector<int *> flagArray_ptrs_h(num_neighbors);
    std::vector<int *> RAP_ext_row_offsets_ptrs_h(num_neighbors);
    std::vector<int *> RAP_ext_col_indices_ptrs_h(num_neighbors);
    std::vector<Value_type *> RAP_ext_values_ptrs_h(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        flagArray_ptrs_h[i] = amgx::thrust::raw_pointer_cast(&flagArray[i][0]);
        RAP_ext_row_offsets_ptrs_h[i] = amgx::thrust::raw_pointer_cast(&RAP_ext_row_offsets[i][0]);
        RAP_ext_col_indices_ptrs_h[i] = amgx::thrust::raw_pointer_cast(&RAP_ext_col_indices[i][0]);
        RAP_ext_values_ptrs_h[i] = amgx::thrust::raw_pointer_cast(&RAP_ext_values[i][0]);
    }

    device_vector_alloc<int *> flagArray_ptrs = flagArray_ptrs_h;
    device_vector_alloc<int *> RAP_ext_row_offsets_ptrs = RAP_ext_row_offsets_ptrs_h;
    device_vector_alloc<int *> RAP_ext_col_indices_ptrs = RAP_ext_col_indices_ptrs_h;
    device_vector_alloc<Value_type *> RAP_ext_values_ptrs = RAP_ext_values_ptrs_h;

    for (int i = 0; i < num_neighbors; i++)
    {
        int size = RAP_ext_row_ids[i].size();

        if (size != 0)
        {
            int num_blocks = std::min(4096, (size + 127) / 128);
            //write the position in RAP_ext_row_ids
            csr_multiply_detail::flag_halo_rows <<< num_blocks, 128>>>(
                RAP_ext_row_ids[i].raw(),
                size,
                flagArray[i].raw(),
                i,
                RAP.manager->global_id());
        }
    }

    cudaCheckError();
    //CUDA_SAFE_CALL( hipGetLastError() );
    csr_multiply_detail::compute_values_RAP_ext_kernel< Value_type, CTA_SIZE, SMEM_SIZE, WARP_SIZE> <<< GRID_SIZE, CTA_SIZE>>>(
        RAP_size,
        RAP_int.row_offsets.raw(),
        RAP_int.col_indices.raw(),
        RAP_int.values.raw(),
        amgx::thrust::raw_pointer_cast(&RAP_ext_row_offsets_ptrs[0]),
        amgx::thrust::raw_pointer_cast(&RAP_ext_col_indices_ptrs[0]),
        amgx::thrust::raw_pointer_cast(&RAP_ext_values_ptrs[0]),
        RAP.row_offsets.raw(),
        RAP.col_indices.raw(),
        RAP.values.raw(),
        amgx::thrust::raw_pointer_cast(&flagArray_ptrs[0]),
        this->m_gmem_size,
        this->m_keys,
        this->m_vals,
        this->m_work_queue,
        num_neighbors,
        status );
    cudaCheckError();
    //CUDA_SAFE_CALL( hipGetLastError() );
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

#define AMGX_CASE_LINE(CASE) template class CSR_Multiply_Detail<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace amgx

