#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2013 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include <cycles/cg_flex_cycle.h>
#include <blas.h>
#include <multiply.h>

#include <amgx_types/util.h>

namespace amgx
{

template<class T_Config>
struct DispatchAuxCGF
{
    static const AMGX_VecPrecision vecPrec = T_Config::vecPrec;
    static const AMGX_MatPrecision matPrec = T_Config::matPrec;
    static const AMGX_IndPrecision indPrec = T_Config::indPrec;
    static void dispatch_aux( AMG<vecPrec, matPrec, indPrec> *amg, AMG_Level<T_Config> *level, Vector<T_Config> &b, Vector<T_Config> &x )
    {
        typedef typename Vector<T_Config>::value_type ValueTypeB;
        int N = (int)b.size();
        //create temperary vectors
        Vector<T_Config> y(N);
        Vector<T_Config> z(N);
        Vector<T_Config> r(N);
        Vector<T_Config> d(N);
        Vector<T_Config> p(N);
        y.tag = 9989 * 100 + 1;
        z.tag = 9989 * 100 + 2;
        r.tag = 9989 * 100 + 3;
        p.tag = 9989 * 100 + 4;
        y.set_block_dimy(level->getA().get_block_dimy());
        y.set_block_dimx(1);
        z.set_block_dimy(level->getA().get_block_dimy());
        z.set_block_dimx(1);
        r.set_block_dimy(level->getA().get_block_dimy());
        r.set_block_dimx(1);
        d.set_block_dimy(level->getA().get_block_dimy());
        d.set_block_dimx(1);
        p.set_block_dimy(level->getA().get_block_dimy());
        p.set_block_dimx(1);

        //TODO account for X being 0's
        //not doing this optimization at the moment
        if (level->isInitCycle())
        {
            fill(x, types::util<ValueTypeB>::get_zero());
            level->unsetInitCycle();
        }

        // y = Ax
        multiply(level->getA(), x, y);
        // r = b - A*x
        axpby(b, y, r, types::util<ValueTypeB>::get_one(), types::util<ValueTypeB>::get_minus_one());
        // z = M*r
        level->setInitCycle();
        CG_Flex_Cycle<T_Config> cycle_init( amg, level, r, z );
        // p = z
        copy(z, p);
        int k = 0;

        while (true)
        {
            // y = Ap
            multiply(level->getA(), p, y);
            // rz = <r^H, z>
            ValueTypeB rz = dotc(r, z);
            // alpha = <r,z>/<y,p>
            ValueTypeB alpha =  rz / dotc(y, p);
            // x = x + alpha * p
            axpy(p, x, alpha);

            if (++k == amg->getCycleIters())
            {
                break;
            }

            //d=r
            copy(r, d);
            // r = r - alpha * y
            axpy(y, r, alpha * types::util<ValueTypeB>::get_minus_one());
            //TODO:  if norm(r)<tolerance break
            //d=r-d
            axpby(r, d, d, types::util<ValueTypeB>::get_one(), types::util<ValueTypeB>::get_minus_one());
            // z = M*r
            level->setInitCycle();
            CG_Flex_Cycle<T_Config> cycle( amg, level, r, z );
            // zd = <z, d>
            ValueTypeB zd = dotc(z, d);
            // beta <- <z_{i+1},d}>/<r,z>
            ValueTypeB beta = zd / rz;
            // p += z + beta*p
            axpby(z, p, p, types::util<ValueTypeB>::get_one(), beta);
        }
    }
};

template<AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec >
void CG_Flex_CycleDispatcher<t_vecPrec, t_matPrec, t_indPrec>::dispatch( AMG_Class *amg, AMG_Level<TConfig_h> *level, Vector<TConfig_h> &b, Vector<TConfig_h> &x ) const
{
    DispatchAuxCGF<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::dispatch_aux( amg, level, b, x );
}

template<AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec >
void CG_Flex_CycleDispatcher<t_vecPrec, t_matPrec, t_indPrec>::dispatch( AMG_Class *amg, AMG_Level<TConfig_d> *level, Vector<TConfig_d> &b, Vector<TConfig_d> &x ) const
{
    DispatchAuxCGF<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::dispatch_aux( amg, level, b, x );
}

/****************************************
 * Explict instantiations
 ***************************************/
template class CG_Flex_CycleDispatcher<AMGX_vecDouble, AMGX_matDouble, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecFloat, AMGX_matFloat, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecDouble, AMGX_matFloat, AMGX_indInt>;

template class CG_Flex_CycleDispatcher<AMGX_vecComplex, AMGX_matComplex, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecDoubleComplex, AMGX_matComplex, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecDoubleComplex, AMGX_matDoubleComplex, AMGX_indInt>;

} // namespace amgx
