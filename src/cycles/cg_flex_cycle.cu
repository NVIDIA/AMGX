#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cycles/cg_flex_cycle.h>
#include <blas.h>
#include <multiply.h>

#include <amgx_types/util.h>

namespace amgx
{

template<class T_Config>
struct DispatchAuxCGF
{
    static const AMGX_VecPrecision vecPrec = T_Config::vecPrec;
    static const AMGX_MatPrecision matPrec = T_Config::matPrec;
    static const AMGX_IndPrecision indPrec = T_Config::indPrec;
    static void dispatch_aux( AMG<vecPrec, matPrec, indPrec> *amg, AMG_Level<T_Config> *level, Vector<T_Config> &b, Vector<T_Config> &x )
    {
        typedef typename Vector<T_Config>::value_type ValueTypeB;
        int N = (int)b.size();
        //create temperary vectors
        Vector<T_Config> y(N);
        Vector<T_Config> z(N);
        Vector<T_Config> r(N);
        Vector<T_Config> d(N);
        Vector<T_Config> p(N);
        y.tag = 9989 * 100 + 1;
        z.tag = 9989 * 100 + 2;
        r.tag = 9989 * 100 + 3;
        p.tag = 9989 * 100 + 4;
        y.set_block_dimy(level->getA().get_block_dimy());
        y.set_block_dimx(1);
        z.set_block_dimy(level->getA().get_block_dimy());
        z.set_block_dimx(1);
        r.set_block_dimy(level->getA().get_block_dimy());
        r.set_block_dimx(1);
        d.set_block_dimy(level->getA().get_block_dimy());
        d.set_block_dimx(1);
        p.set_block_dimy(level->getA().get_block_dimy());
        p.set_block_dimx(1);

        //TODO account for X being 0's
        //not doing this optimization at the moment
        if (level->isInitCycle())
        {
            fill(x, types::util<ValueTypeB>::get_zero());
            level->unsetInitCycle();
        }

        // y = Ax
        multiply(level->getA(), x, y);
        // r = b - A*x
        axpby(b, y, r, types::util<ValueTypeB>::get_one(), types::util<ValueTypeB>::get_minus_one());
        // z = M*r
        level->setInitCycle();
        CG_Flex_Cycle<T_Config> cycle_init( amg, level, r, z );
        // p = z
        copy(z, p);
        int k = 0;

        while (true)
        {
            // y = Ap
            multiply(level->getA(), p, y);
            // rz = <r^H, z>
            ValueTypeB rz = dotc(r, z);
            // alpha = <r,z>/<y,p>
            ValueTypeB alpha =  rz / dotc(y, p);
            // x = x + alpha * p
            axpy(p, x, alpha);

            if (++k == amg->getCycleIters())
            {
                break;
            }

            //d=r
            copy(r, d);
            // r = r - alpha * y
            axpy(y, r, alpha * types::util<ValueTypeB>::get_minus_one());
            //TODO:  if norm(r)<tolerance break
            //d=r-d
            axpby(r, d, d, types::util<ValueTypeB>::get_one(), types::util<ValueTypeB>::get_minus_one());
            // z = M*r
            level->setInitCycle();
            CG_Flex_Cycle<T_Config> cycle( amg, level, r, z );
            // zd = <z, d>
            ValueTypeB zd = dotc(z, d);
            // beta <- <z_{i+1},d}>/<r,z>
            ValueTypeB beta = zd / rz;
            // p += z + beta*p
            axpby(z, p, p, types::util<ValueTypeB>::get_one(), beta);
        }
    }
};

template<AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec >
void CG_Flex_CycleDispatcher<t_vecPrec, t_matPrec, t_indPrec>::dispatch( AMG_Class *amg, AMG_Level<TConfig_h> *level, Vector<TConfig_h> &b, Vector<TConfig_h> &x ) const
{
    DispatchAuxCGF<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::dispatch_aux( amg, level, b, x );
}

template<AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec >
void CG_Flex_CycleDispatcher<t_vecPrec, t_matPrec, t_indPrec>::dispatch( AMG_Class *amg, AMG_Level<TConfig_d> *level, Vector<TConfig_d> &b, Vector<TConfig_d> &x ) const
{
    DispatchAuxCGF<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::dispatch_aux( amg, level, b, x );
}

/****************************************
 * Explict instantiations
 ***************************************/
template class CG_Flex_CycleDispatcher<AMGX_vecDouble, AMGX_matDouble, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecFloat, AMGX_matFloat, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecDouble, AMGX_matFloat, AMGX_indInt>;

template class CG_Flex_CycleDispatcher<AMGX_vecComplex, AMGX_matComplex, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecDoubleComplex, AMGX_matComplex, AMGX_indInt>;
template class CG_Flex_CycleDispatcher<AMGX_vecDoubleComplex, AMGX_matDoubleComplex, AMGX_indInt>;

} // namespace amgx
