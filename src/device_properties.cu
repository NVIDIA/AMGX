// SPDX-FileCopyrightText: 2011 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include <device_properties.h>
namespace amgx
{
static hipDeviceProp_t deviceProps;
static bool initialized=false;

hipDeviceProp_t getDeviceProperties()
{
    if(!initialized) {
        int dev;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&deviceProps, dev);
        initialized=true;
    }
    return deviceProps;
}

// Return the number of Streaming Multiprocessors on the current device
int getSMCount()
{
    auto devProp = getDeviceProperties();
    return devProp.multiProcessorCount;
}

}
