#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <distributed/distributed_manager.h>
#include <distributed/comms_mpi_gpudirect.h>
#include <distributed/comms_mpi_hostbuffer_stream.h>
#include <distributed/comms_visitors.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/unique.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust_wrapper.h>
#include <basic_types.h>
#include <error.h>
#include <util.h>
#include <types.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <assert.h>
#include "hip/hip_runtime.h"
#include "reorder_partition.h"

#include "amgx_types/util.h"

#include <algorithm>
#include <iostream> //debug only:

struct is_my_part : public thrust::unary_function<int, bool>
{
    const int _my_part;
    is_my_part(int my_part) : _my_part(my_part) { }

    __host__ __device__
    bool operator()(const int part)
    {
        return (part == _my_part);
    }
};

using namespace std;
namespace amgx
{

static int insertDiagonals = 1;

template <typename index_type>
static __device__ __forceinline__
index_type internal_index(index_type i, index_type j, index_type k, index_type nx, index_type ny, index_type nz)
{
    return k * (nx * ny) + j * nx + i;
}

template <typename index_type>
static __device__ __forceinline__
int64_t get_global_offset(index_type p, index_type q, index_type r, index_type P, index_type Q, index_type R, index_type num_rows)
{
    int rank_id = r * (P * Q) + q * P + p;
    return ((int64_t) rank_id) * ((int64_t) num_rows);
}

template <typename index_type>
__global__
void poisson7pt_count_row_len(index_type *row_len, index_type nx, index_type ny, index_type nz, index_type p, index_type q, index_type r, index_type P, index_type Q, index_type R, index_type num_rows)
{
    for (int tidx = threadIdx.x + blockIdx.x * blockDim.x; tidx < num_rows ; tidx += blockDim.x * gridDim.x)
    {
        /* compute p,q,r from P,Q,R and myid */
        int i = tidx % nx; // Position in x direction
        int j = (( tidx - i) / nx) % ny; // Position in y
        int k = ( tidx - i - nx * j) / ( nx * ny ); // Position in z
        int substract = ((i == 0) && (p == 0));
        substract += ((i == nx - 1) && (p == P - 1));
        substract += ((j == 0) && (q == 0));
        substract += ((j == ny - 1) && (q == Q - 1));
        substract += ((k == 0) && (r == 0));
        substract += ((k == nz - 1) && (r == R - 1));
        // Store 7 in position (num_rows+1), such that row_len[num_rows+1] = 0
        //substract = (tidx == num_rows+1) ? 7 : substract;
        row_len[tidx] = 7 - substract;
    }
}

template <typename index_type, typename mat_value_type>
__global__
void poisson7pt_set_col_values(const index_type *__restrict__ row_offsets, index_type *__restrict__ col_indices, mat_value_type *__restrict__  values, index_type *__restrict__ diag, int64_t *__restrict__ local_to_global, index_type nx, index_type ny, index_type nz, index_type p, index_type q, index_type r, index_type P, index_type Q, index_type R, index_type num_rows)
{
    for (int row = threadIdx.x + blockIdx.x * blockDim.x; row < num_rows ; row += blockDim.x * gridDim.x)
    {
        /* compute p,q,r from P,Q,R and myid */
        int i = row % nx; // Position in x direction
        int j = (( row - i) / nx) % ny; // Position in y
        int k = ( row - i - nx * j) / ( nx * ny ); // Position in z
        int halo_offset = num_rows;
        int pos = row_offsets[row];
        // Diagonal element
        diag[row] = pos;
        col_indices[pos] = row;
        values[pos++] = types::util<mat_value_type>::get_one() * 6.;

        // ----------------------------
        // Neighbor at position i-1
        // ----------------------------
        if (i)
        {
            // Has a i-1 neighbor, which is an internal node at position (i-1,j,k)
            col_indices[pos] = internal_index(i - 1, j, k, nx, ny, nz);
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
        }
        else if (p)
        {
            // Has a i-1 neighbor, which is a halo node
            int halo_index = halo_offset + k * ny + j;
            col_indices[pos] = halo_index;
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
            int64_t global_offset = get_global_offset(p - 1, q, r, P, Q, R, num_rows);
            local_to_global[halo_index - num_rows] = global_offset + internal_index(nx - 1, j, k, nx, ny, nz);
        }

        if (p)
        {
            halo_offset += ny * nz;
        }

        // ----------------------------
        // Neighbor at position i+1
        // ----------------------------
        if (i < nx - 1)
        {
            // Has i+1 neighbor, which is an internal node at position (i+1,j,k)
            col_indices[pos] = internal_index(i + 1, j, k, nx, ny, nz);
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
        }
        else
        {
            if (p < P - 1)
            {
                // Has i+1 neighbor, which is a halo node
                int halo_index = halo_offset + k * ny + j;
                col_indices[pos] = halo_index;
                values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
                int64_t global_offset = get_global_offset(p + 1, q, r, P, Q, R, num_rows);
                local_to_global[halo_index - num_rows] = global_offset + internal_index(0, j, k, nx, ny, nz);
            }
        }

        if (p < P - 1)
        {
            halo_offset += ny * nz;
        }

        // ----------------------------
        // Neighbor at position j-1
        // ----------------------------
        if (j)
        {
            // Has a j-1 neighbor, which is an internal node at position (i,j-1,k)
            col_indices[pos] = internal_index(i, j - 1, k, nx, ny, nz);
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
        }
        else if (q)
        {
            // Has a j-1 neighbor, which is a halo node
            int halo_index = halo_offset + k * nx + i;
            col_indices[pos] = halo_index;
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
            int64_t global_offset = get_global_offset(p, q - 1, r, P, Q, R, num_rows);
            local_to_global[halo_index - num_rows] = global_offset + internal_index(i, ny - 1, k, nx, ny, nz);
        }

        if (q)
        {
            halo_offset += nx * nz;
        }

        // ----------------------------
        // Neighbor at position j+1
        // ----------------------------
        if (j < ny - 1)
        {
            // Has a j+1 neighbor, which is an internal node at position (i,j+1,k)
            col_indices[pos] = internal_index(i, j + 1, k, nx, ny, nz);
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
        }
        else
        {
            if (q < Q - 1)
            {
                // Has a j+1 neighbor, which is a halo node
                int halo_index = halo_offset + k * nx + i;
                col_indices[pos] = halo_index;
                values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
                int64_t global_offset = get_global_offset(p, q + 1, r, P, Q, R, num_rows);
                local_to_global[halo_index - num_rows] = global_offset + internal_index(i, 0, k, nx, ny, nz);
            }
        }

        if (q < Q - 1)
        {
            halo_offset += nx * nz;
        }

        // ----------------------------
        // Neighbor at position k-1
        // ----------------------------
        if (k)
        {
            // Has a k-1 neighbor, which is an internal node at position (i,j,k-1)
            col_indices[pos] = internal_index(i, j, k - 1, nx, ny, nz);
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
        }
        else if (r)
        {
            // Has a k-1 neighbor, which is a halo node
            int halo_index = halo_offset + j * nx + i;
            col_indices[pos] = halo_index;
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
            int64_t global_offset = get_global_offset(p, q, r - 1, P, Q, R, num_rows);
            local_to_global[halo_index - num_rows] = global_offset + internal_index(i, j, nz - 1, nx, ny, nz);
        }

        if (r)
        {
            halo_offset += nx * ny;
        }

        // ----------------------------
        // Neighbor at position k+1
        // ----------------------------
        if (k < nz - 1)
        {
            // Has a k+1 neighbor, which is an internal node at position (i,j,k+1)
            col_indices[pos] = internal_index(i, j, k + 1, nx, ny, nz);
            values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
        }
        else
        {
            if (r < R - 1)
            {
                // Has a k+1 neighbor, which is a halo node
                int halo_index = halo_offset + j * nx + i;
                col_indices[pos] = halo_index;
                values[pos++] = types::util<mat_value_type>::invert(types::util<mat_value_type>::get_one());
                int64_t global_offset = get_global_offset(p, q, r + 1, P, Q, R, num_rows);
                local_to_global[halo_index - num_rows] = global_offset + internal_index(i, j, 0, nx, ny, nz);
            }
        }

        if (r < R - 1)
        {
            halo_offset += nx * ny;
        }
    }
}

__global__ void flag_halo_ids_kernel(INDEX_TYPE *flags, INDEX_TYPE *ids, INDEX_TYPE offset, INDEX_TYPE size, INDEX_TYPE upper)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        flags[ids[idx] - offset] = 1;
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void read_halo_ids_kernel(INDEX_TYPE *flags, INDEX_TYPE *ids, INDEX_TYPE offset, INDEX_TYPE size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        ids[idx] = flags[ids[idx] - offset];
        idx += blockDim.x * gridDim.x;
    }
}

template<class T>
__global__ void reorder_vector_values(T *dst, const T *src, const INDEX_TYPE *map, INDEX_TYPE blocksize, INDEX_TYPE num_rows)
{
    int row = blockIdx.x * (blockDim.x / blocksize) + threadIdx.x / blocksize; //vectorised by block size
    int vec_id = threadIdx.x % blocksize;

    if (threadIdx.x >= (blockDim.x / blocksize)*blocksize ) { return; }

    while (row < num_rows)
    {
        dst[map[row]*blocksize + vec_id] = src[row * blocksize + vec_id];
        row += gridDim.x * (blockDim.x / blocksize);
    }
}

template<class T>
__global__ void inverse_reorder_vector_values(T *dst, T *src, INDEX_TYPE *map, INDEX_TYPE blocksize, INDEX_TYPE num_rows)
{
    int row = blockIdx.x * (blockDim.x / blocksize) + threadIdx.x / blocksize;
    int vec_id = threadIdx.x % blocksize;

    if (threadIdx.x >= (blockDim.x / blocksize)*blocksize ) { return; }

    while (row < num_rows)
    {
        dst[row * blocksize + vec_id] = src[map[row] * blocksize + vec_id];
        row += gridDim.x * (blockDim.x / blocksize);
    }
}

__global__ void remove_boundary_kernel(INDEX_TYPE *flags, INDEX_TYPE *maps, INDEX_TYPE size)
{
    int element = blockIdx.x * blockDim.x + threadIdx.x;

    while (element < size)
    {
        flags[maps[element]] = 0; //this won't be a problem, because we are overwriting the same thing
        element += blockDim.x * gridDim.x;
    }
}

__global__ void get_unassigned_kernel(INDEX_TYPE *unassigned_flags, INDEX_TYPE *map, INDEX_TYPE *output, INDEX_TYPE part_size, INDEX_TYPE uf_size )
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < part_size)
    {
        if (map[idx] < uf_size)
        {
            if (unassigned_flags[map[idx]] == 0)
            {
                unassigned_flags[map[idx]] = 1;
                output[idx] = 1;
            }
        }

        idx += blockDim.x * gridDim.x;
    }
}

__global__ void set_unassigned_kernel(INDEX_TYPE *part_assigned_flags, INDEX_TYPE *part_num, INDEX_TYPE *map, INDEX_TYPE *renum, INDEX_TYPE part_size, INDEX_TYPE max_element, INDEX_TYPE renum_size /*, INDEX_TYPE rank*/)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < part_size)
    {
        if (map[idx] < renum_size)
        {
            if (part_assigned_flags[idx] == 1)
            {
                renum[map[idx]] = max_element + part_num[idx];
            }

            //also update the B2L map
            map[idx] = renum[map[idx]];
        }

        idx += blockDim.x * gridDim.x;
    }
}

__global__ void renumber_b2l_maps(INDEX_TYPE *map, INDEX_TYPE *renum, INDEX_TYPE part_size, INDEX_TYPE renum_size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < part_size)
    {
        if (map[idx] < renum_size)
        {
            //update the B2L map
            map[idx] = renum[map[idx]];
            idx += blockDim.x * gridDim.x;
        }
    }
}

__global__ void calc_inverse_renumbering(INDEX_TYPE *renum, INDEX_TYPE *irenum, INDEX_TYPE max_element)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < max_element)
    {
        if (renum[idx] < 0 || renum[idx] >= max_element) { printf("Renumbering error: %d %d\n", renum[idx], max_element); }

        irenum[renum[idx]] = idx;
        idx += blockDim.x * gridDim.x;
    }
}

__global__ void create_halo_mapping(INDEX_TYPE *mapping, INDEX_TYPE *node_list, int64_t base_index, INDEX_TYPE map_offset, INDEX_TYPE size)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    while (row < size)
    {
        int idx = node_list[row] - base_index;
        mapping[idx] = map_offset + row;
        row += blockDim.x * gridDim.x;
    }
}

__global__ void apply_h2l2b_mapping(INDEX_TYPE *mapping, INDEX_TYPE *node_list, int64_t base_index, INDEX_TYPE *b2l_map, INDEX_TYPE size)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    while (row < size)
    {
        int idx = node_list[row] - base_index;
        mapping[idx] = b2l_map[row];
        row += blockDim.x * gridDim.x;
    }
}

template <int coop>
__global__ void map_col_indices_and_count_rowlen(INDEX_TYPE *row_offsets, INDEX_TYPE *col_indices, INDEX_TYPE *row_length,
        INDEX_TYPE *mapping, INDEX_TYPE num_rows, INDEX_TYPE insert_diagonal)
{
    extern __shared__ volatile int reduction[];
    int row = blockIdx.x * blockDim.x / coop + threadIdx.x / coop;
    int coopIdx = threadIdx.x % coop;

    while (row < num_rows)
    {
        int valid = 0;

        for (int idx = row_offsets[row] + coopIdx; idx < row_offsets[row + 1]; idx += coop) //this may look horrible, but I expect low branch divergence, because col indices in a row usually belong to the same partition (or at most one more)
        {
            int colIdx = col_indices[idx];
            int new_col_idx = mapping[colIdx];

            if (new_col_idx >= 0)
            {
                valid++;
                col_indices[idx] = new_col_idx;
            }
            else
            {
                col_indices[idx] = -1;
            }
        }

        reduction[threadIdx.x] = valid;

        for (int s = 2; s > 0; s >>= 1)
        {
            if (coopIdx < s)
            {
                reduction[threadIdx.x] += reduction[threadIdx.x + s];
            }

            __syncthreads();
        }

        if (coopIdx == 0)
        {
            row_length[row] = reduction[threadIdx.x] + insert_diagonal;
        }

        row += gridDim.x * blockDim.x / coop;
    }
}


__global__ void renumber_P_col_indices(INDEX_TYPE *__restrict__ col_indices, const INDEX_TYPE *__restrict__ renum, INDEX_TYPE num_owned_coarse_pts, INDEX_TYPE num_owned_fine_pts)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < num_owned_fine_pts )
    {
        INDEX_TYPE col_id = col_indices[idx];

        if (col_id < num_owned_coarse_pts)
        {
            col_indices[idx] = renum[col_id];
        }

        idx += blockDim.x * gridDim.x;
    }
}

template <int coop, class T>
__global__ void reorder_R_matrix(const INDEX_TYPE *old_rows, const INDEX_TYPE *old_cols, const T *old_vals, const INDEX_TYPE *rows, INDEX_TYPE *cols, T *vals, const INDEX_TYPE *renumbering, INDEX_TYPE bsize, INDEX_TYPE num_rows, INDEX_TYPE num_owned_rows)
{
    int row = blockIdx.x * blockDim.x / coop  + threadIdx.x / coop;
    int coopIdx = threadIdx.x % coop;

    while (row < num_rows)
    {
        INDEX_TYPE src_base = old_rows[row];
        INDEX_TYPE dst_base = row < num_owned_rows ? rows[renumbering[row]] : src_base;

        for (int i = coopIdx; i < old_rows[row + 1]*bsize - src_base * bsize; i += coop)
        {
            vals[dst_base * bsize + i] = old_vals[src_base * bsize + i];
        }

        for (int i = coopIdx; i < old_rows[row + 1] - src_base; i += coop)
        {
            cols[dst_base + i] = old_cols[src_base + i];
        }

        row += blockDim.x * gridDim.x / coop;
    }
}

template <int coop, class T>
__global__ void reorder_whole_matrix(INDEX_TYPE *old_rows, INDEX_TYPE *old_cols, T *old_vals, INDEX_TYPE *rows, INDEX_TYPE *cols, T *vals, INDEX_TYPE *renumbering, INDEX_TYPE bsize, INDEX_TYPE num_rows, INDEX_TYPE insert_diagonal)
{
    int row = blockIdx.x * blockDim.x / coop  + threadIdx.x / coop;
    int coopIdx = threadIdx.x % coop;

    while (row < num_rows)
    {
        INDEX_TYPE src_base = old_rows[row];
        INDEX_TYPE dst_base = rows[renumbering[row]];

        if (insert_diagonal)
        {
            if (coopIdx == 0) { cols[dst_base] = renumbering[row]; }

            for (int i = coopIdx; i < bsize; i += coop)
            {
                vals[dst_base * bsize + i] = old_vals[(old_rows[num_rows] + row) * bsize + i];
            }

            dst_base++;
        }

        for (int i = coopIdx; i < old_rows[row + 1]*bsize - src_base * bsize; i += coop)
        {
            vals[dst_base * bsize + i] = old_vals[src_base * bsize + i];
        }

        for (int i = coopIdx; i < old_rows[row + 1] - src_base; i += coop)
        {
            cols[dst_base + i] = old_cols[src_base + i];
        }

        row += blockDim.x * gridDim.x / coop;
    }
}

template <int coop, class T>
__global__ void replace_values_matrix(const T *src_vals_h, const T *src_diag_h, const INDEX_TYPE *old_rows, const INDEX_TYPE *rows, T *vals, const INDEX_TYPE *renumbering, INDEX_TYPE bsize, INDEX_TYPE num_rows)
{
    int row = blockIdx.x * blockDim.x / coop  + threadIdx.x / coop;
    int coopIdx = threadIdx.x % coop;

    while (row < num_rows)
    {
        INDEX_TYPE src_base = old_rows[row];
        INDEX_TYPE dst_base = rows[renumbering[row]];

        for (int i = coopIdx; i < bsize; i += coop)
        {
            vals[dst_base * bsize + i] = src_diag_h[row * bsize + i];
        }

        dst_base++;

        for (int i = coopIdx; i < old_rows[row + 1]*bsize - src_base * bsize; i += coop)
        {
            vals[dst_base * bsize + i] = src_vals_h[src_base * bsize + i];
        }

        row += blockDim.x * gridDim.x / coop;
    }
}

template <int coop, class T>
__global__ void replace_values_matrix(const T *src_vals_h, const INDEX_TYPE *old_rows, const INDEX_TYPE *rows, T *vals, const INDEX_TYPE *renumbering, INDEX_TYPE bsize, INDEX_TYPE num_rows)
{
    int row = blockIdx.x * blockDim.x / coop  + threadIdx.x / coop;
    int coopIdx = threadIdx.x % coop;

    while (row < num_rows)
    {
        INDEX_TYPE src_base = old_rows[row];
        INDEX_TYPE dst_base = rows[renumbering[row]];

        for (int i = coopIdx; i < old_rows[row + 1]*bsize - src_base * bsize; i += coop)
        {
            vals[dst_base * bsize + i] = src_vals_h[src_base * bsize + i];
        }

        row += blockDim.x * gridDim.x / coop;
    }
}

//TODO: optimize by vectorizing
template <class T>
__global__ void reorder_whole_halo_matrix(INDEX_TYPE *old_rows, INDEX_TYPE *old_cols, T *old_vals, INDEX_TYPE *rows, INDEX_TYPE *cols, T *vals,
        INDEX_TYPE *renumbering, INDEX_TYPE bsize, INDEX_TYPE num_rows, INDEX_TYPE insert_diagonal,
        INDEX_TYPE global_offset, INDEX_TYPE local_offset, INDEX_TYPE halo_rows)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    while (row < num_rows)
    {
        INDEX_TYPE src_base = old_rows[row];
        INDEX_TYPE dst = rows[row];

        if (insert_diagonal)
        {
            cols[dst] = global_offset + row;

            for (int j = 0; j < bsize; j++) { vals[dst * bsize + j] = old_vals[(old_rows[halo_rows - local_offset] + local_offset + row) * bsize + j]; }

            dst++;
        }

        for (int i = 0; i < old_rows[row + 1] - src_base; i++)
        {
            INDEX_TYPE colIdx = old_cols[src_base + i];

            if (colIdx >= 0)
            {
                cols[dst] = colIdx;

                for (int j = 0; j < bsize; j++) { vals[dst * bsize + j] = old_vals[(src_base + i) * bsize + j]; }

                dst++;
            }
        }

        row += blockDim.x * gridDim.x;
    }
}

__global__ void calc_rowlen_reorder(INDEX_TYPE *row_offsets, INDEX_TYPE *row_len, INDEX_TYPE *map, INDEX_TYPE size, INDEX_TYPE insert_diag)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while (idx < size)
    {
        row_len[map[idx]] = row_offsets[idx + 1] - row_offsets[idx] + insert_diag;
        idx += blockDim.x * gridDim.x;
    }
}

template <class TConfig >
inline void DistributedManagerBase<TConfig>::set_initialized(IVector &row_offsets)
{
    // For P and R sizes the sizes are fixed at creation
    if(m_fixed_view_size)
    {
        return;
    }

    if (neighbors.size() > 0)
    {
        //distributed: cache num_rows/num_nz for different views
        _num_rows_interior = _num_interior_nodes;
        _num_nz_interior = row_offsets[_num_rows_interior];
        _num_rows_owned = _num_interior_nodes + _num_boundary_nodes;
        _num_nz_owned = row_offsets[_num_rows_owned];
        _num_rows_full = halo_offsets[neighbors.size()];

        if (_num_rows_full >= row_offsets.size())
        {
            _num_nz_full = row_offsets[row_offsets.size() - 1];
        }
        else
        {
            _num_nz_full = row_offsets[_num_rows_full];
        }

        _num_rows_all = halo_offsets[halo_offsets.size() - 1];
        _num_nz_all = _num_nz_full;
    }
    else
    {
        _num_rows_interior = _num_interior_nodes;
        _num_nz_interior = row_offsets[_num_rows_interior];
        _num_rows_owned = _num_interior_nodes;
        _num_nz_owned = row_offsets[_num_rows_owned];
        _num_rows_full = _num_rows_owned;
        _num_nz_full = _num_nz_owned;
        _num_rows_all = _num_rows_owned;
        _num_nz_all = _num_nz_owned;
    }
}

template <class TConfig>
inline DistributedManagerBase<TConfig>::DistributedManagerBase(Matrix<TConfig> &a) :
    m_fine_level_comms(NULL), A(&a), m_pinned_buffer_size(0), m_pinned_buffer(NULL), _num_interior_nodes(0), _num_boundary_nodes(0), _comms(NULL), has_B2L(false),
    neighbors(_neighbors), B2L_maps(_B2L_maps), L2H_maps(_L2H_maps),  B2L_rings(_B2L_rings),
    halo_rows_ref_count(0), halo_btl_ref_count(0), halo_ranges(_halo_ranges), halo_ranges_h(_halo_ranges_h), part_offsets(_part_offsets), part_offsets_h(_part_offsets_h), halo_rows(NULL), halo_btl(NULL), m_fixed_view_size(false)
{
    hipEventCreate(&comm_event);
    hipStreamCreateWithFlags(&m_int_stream, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&m_bdy_stream, hipStreamNonBlocking);
    this->createComms(A->getResources());
    int my_id = this->getComms()->get_global_id();
    int num_parts = this->getComms()->get_num_partitions();
    this->set_global_id(my_id);
    this->set_num_partitions(num_parts);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::generatePoisson7pt(int nx, int ny, int nz, int P, int Q, int R)
{
    int my_id = this->getComms()->get_global_id();
    int p, q, r;

    if (nx < P  || ny < Q || nz < R)
    {
        FatalError("(nx < P) or (ny < Q) or (nz < R) not supported\n", AMGX_ERR_NOT_IMPLEMENTED);
    }

    /* compute p,q,r from P,Q,R and myid */
    p = my_id % P; // Position in x direction
    q = (( my_id - p) / P) % Q; // Position in y
    r = ( my_id - p - P * q) / ( P * Q ); // Position in z
    // Create A.row_indices, A.col_indices, A.values, A.diag
    int num_rows = nx * ny * nz;
    int num_nonzeros = num_rows * 7; // Ignoring any boundary, 7 nnz per row
    int num_substract = 0;

    if (p == 0) { num_substract += ny * nz; }

    if (p == P - 1) { num_substract += ny * nz; }

    if (q == 0) { num_substract += nx * nz; }

    if (q == Q - 1) { num_substract += nx * nz; }

    if (r == 0) { num_substract += nx * ny; }

    if (r == R - 1) { num_substract += nx * ny; }

    num_nonzeros -= num_substract;
    int num_halo_nodes = 2 * (ny * nz + nx * nz + nx * ny) - num_substract;
    this->local_to_global_map.resize(num_halo_nodes);
    this->A->set_initialized(0);
    this->A->resize(0, 0, 0, 1, 1, 1);
    this->A->addProps(CSR);
    this->A->resize(num_rows, num_rows + num_halo_nodes, num_nonzeros, 1, 1, 1);
    const int cta_size = 128;
    const int grid_size = std::min( 4096, (num_rows + cta_size - 1) / cta_size );
    poisson7pt_count_row_len <<< grid_size, cta_size>>>(this->A->row_offsets.raw(), nx, ny, nz, p, q, r, P, Q, R, num_rows);
    thrust_wrapper::exclusive_scan(this->A->row_offsets.begin(), this->A->row_offsets.end(), this->A->row_offsets.begin());
    cudaCheckError();
    // Now set nonzeros columns and values
    // TODO: vectorize this
    const int grid_size2 = std::min( 4096, (num_rows + cta_size - 1) / cta_size );
    poisson7pt_set_col_values <<< grid_size2, cta_size>>>
    (this->A->row_offsets.raw(),
     this->A->col_indices.raw(),
     this->A->values.raw(),
     this->A->diag.raw(),
     this->local_to_global_map.raw(),
     nx, ny, nz,
     p, q, r,
     P, Q, R,
     num_rows);
    cudaCheckError();
    // fill parts_offsets_h
    // All ranks have same number of nodes
    int num_ranks = P * Q * R;
    this->part_offsets_h.resize(num_ranks + 1);
    this->part_offsets_h[0] = (int64_t) 0;

    for (int i = 1; i < num_ranks + 1; i++)
    {
        this->part_offsets_h[i] = this->part_offsets_h[i - 1] + (int64_t) num_rows;
    }

    // Device to host copy
    this->part_offsets = this->part_offsets_h;
    this->num_rows_global = P * Q * R * nx * ny * nz;
//  this->A->set_initialized(1);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
template <typename t_colIndex>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::loadDistributed_SetOffsets(
    int num_ranks, int num_rows_global, const t_colIndex* partition_offsets)
{
    // fill part offsets internal data structures
    this->part_offsets_h.resize(num_ranks + 1);

    for (int i = 0; i <= num_ranks; i++)
    {
        this->part_offsets_h[i] = partition_offsets[i];
    }
    // copy to device
    this->part_offsets = this->part_offsets_h;
    // set num of global rows
    this->num_rows_global = num_rows_global;
    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
template <typename t_colIndex>
map<t_colIndex, int> DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::loadDistributed_LocalToGlobal(int num_rows, I64Vector_h &off_diag_cols)
{
    // sort global column indices
    thrust::sort(off_diag_cols.begin(), off_diag_cols.end());
    // find unique columns and set local <-> global mappings
    // 1) Removed unneeded vector 2) Create map on host first, upload later (less thrust calls)
    I64Vector_h local_to_global_h;
    map<t_colIndex, int> global_to_local;        // temporary

    if (off_diag_cols.size() > 0)
    {
        global_to_local[off_diag_cols[0]] = num_rows;
        local_to_global_h.push_back(off_diag_cols[0]);
    }

    for (int i = 1; i < off_diag_cols.size(); i++)
    {
        if (off_diag_cols[i] != off_diag_cols[i - 1])
        {
            global_to_local[off_diag_cols[i]] = num_rows + local_to_global_h.size();
            local_to_global_h.push_back(off_diag_cols[i]);
        }
    }
    // Upload finished map in one piece
    this->local_to_global_map.resize(local_to_global_h.size());
    thrust::copy(local_to_global_h.begin(), local_to_global_h.end(), this->local_to_global_map.begin());
    return global_to_local;
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::loadDistributed_InitLocalMatrix(
    IVector_h local_col_indices,
    int num_rows,
    int num_nonzeros,
    const int block_dimx,
    const int block_dimy,
    const int *row_offsets,
    const mat_value_type *values,
    const void *diag)
{
    // init local matrix
    this->A->set_initialized(0);
    this->A->resize(0, 0, 0, 1, 1, 1);
    this->A->addProps(CSR);

    if (diag)
    {
        this->A->addProps(DIAG);
    }

    this->A->resize(num_rows, num_rows + this->local_to_global_map.size(), num_nonzeros, block_dimx, block_dimy, 1);
    cudaCheckError();
    // set local matrix
    thrust::copy(row_offsets, row_offsets + num_rows + 1, this->A->row_offsets.begin());
    this->A->col_indices = local_col_indices;

    thrust::copy(values, values + num_nonzeros * block_dimx * block_dimy, this->A->values.begin());
    cudaCheckError();

    // setup diagonal
    if (diag)
    {
        hipMemcpy(this->A->values.raw() + this->A->diagOffset()*this->A->get_block_size(), diag, sizeof(mat_value_type) * num_rows * block_dimx * block_dimy, hipMemcpyDefault);
    }
    else
    {
        this->A->computeDiagonal();
    }
    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
template <typename t_colIndex>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::loadDistributedMatrixPartitionVec(
    int num_rows,
    int num_nonzeros,
    const int block_dimx,
    const int block_dimy,
    const int *row_offsets,
    const t_colIndex *col_indices,
    const mat_value_type *values,
    int num_ranks,
    int num_rows_global,
    const void *diag,
    const int *partition)
{
    // fetch my rank
    int my_id = this->getComms()->get_global_id();
    // setup partition vector
    IVector_h partitionVec(num_rows_global);

    if (partition == NULL)
    {
        IVector_h rowCounts(num_ranks);
        this->getComms()->all_gather(num_rows, rowCounts, 1);

        int p = 0;
        for (int i = 0; i < num_ranks; ++i)
        {
            for (int j = 0; j < rowCounts[i]; ++j)
            {
                partitionVec[p++] = i;
            }
        }
    }
    else
    {
        // use existing partition info
        for (int i = 0; i < num_rows_global; i++)
        {
            partitionVec[i] = partition[i];
        }
    }

    // compute partition offsets (based on number of elements per partition). Will be modified when calculating partition map.
    t_colIndex *partition_offsets = (t_colIndex *)calloc(num_ranks + 1, sizeof(t_colIndex));

    for (int i = 0; i < num_rows_global; i++)
    {
        int pvi = partitionVec[i];
        partition_offsets[pvi + 1]++;
    }

    thrust::inclusive_scan(partition_offsets, partition_offsets + num_ranks + 1, partition_offsets);

    loadDistributed_SetOffsets(num_ranks, num_rows_global, partition_offsets);

    // compute partition map (which tells you how the global elements are mapped into the partitions)
    t_colIndex *partition_map = (t_colIndex *)calloc(num_rows_global, sizeof(t_colIndex));

    for (int i = 0; i < num_rows_global; i++)
    {
        int     pvi = partitionVec[i];
        t_colIndex poi = partition_offsets[pvi];
        partition_map[poi] = i;
        partition_offsets[pvi]++;
    }
    free(partition_offsets);

    // compute the inverse partition map
    t_colIndex *ipartition_map = (t_colIndex *)calloc(num_rows_global, sizeof(t_colIndex));

    for (int i = 0; i < num_rows_global; i++)
    {
        ipartition_map[partition_map[i]] = i;
    }
    free(partition_map);

    int h_cidx_allocated = 0;
    const t_colIndex *h_col_indices_global = (const t_colIndex *)this->getHostPointerForData(col_indices, num_nonzeros * sizeof(t_colIndex), &h_cidx_allocated);
    // gather all off-diag columns
    I64Vector_h off_diag_cols;

    for (int i = 0; i < num_nonzeros; i++)
    {
        if (partitionVec[h_col_indices_global[i]] != my_id)
        {
            off_diag_cols.push_back(ipartition_map[h_col_indices_global[i]]);
        }
    }

    auto global_to_local = loadDistributed_LocalToGlobal<t_colIndex>(num_rows, off_diag_cols);

    // set 1, then scan to compute local row indices
    IVector_h my_indices(num_rows_global);

    for (int i = 0; i < num_nonzeros; i++)
    {
        if (partitionVec[h_col_indices_global[i]] == my_id)     // find my local columns and set to 1
        {
            my_indices[ipartition_map[h_col_indices_global[i]]] = 1;
        }
    }

    thrust::exclusive_scan(my_indices.begin(), my_indices.end(), my_indices.begin());
    // remap colums to local
    IVector_h local_col_indices(num_nonzeros);

    for (int i = 0; i < num_nonzeros; i++)
    {
        if (partitionVec[h_col_indices_global[i]] != my_id)
        {
            // off-diag
            local_col_indices[i] = global_to_local[ipartition_map[h_col_indices_global[i]]];
        }
        else
        {
            // diag
            local_col_indices[i] = my_indices[ipartition_map[h_col_indices_global[i]]];
        }
    }
    free(ipartition_map);

    loadDistributed_InitLocalMatrix(local_col_indices, num_rows, num_nonzeros, block_dimx, block_dimy, row_offsets, values, diag);

    cudaCheckError();

    // don't free possibly allocated pinned buffer, since it could be used later. if it would not - it would be deallocated automatically
    /*if (h_cidx_allocated)
    {
      free((void*)h_col_indices_global);
    }*/
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
template <typename t_colIndex>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::loadDistributedMatrixPartitionOffsets(
    int num_rows,
    int num_nonzeros,
    const int block_dimx,
    const int block_dimy,
    const int *row_offsets,
    const t_colIndex *col_indices,
    const mat_value_type *values,
    int num_ranks,
    int num_rows_global,
    const void *diag,
    const t_colIndex *partition_offsets)
{
    // fetch my rank
    int my_id = this->getComms()->get_global_id();
    // sanity check, cheap to perform, and helps prevent harder-to-debug errors later on
    if (!std::is_sorted(partition_offsets, partition_offsets + num_ranks + 1)) {
        FatalError("Partition offsets are not sorted.", AMGX_ERR_BAD_PARAMETERS);
    }
    loadDistributed_SetOffsets(num_ranks, num_rows_global, partition_offsets);

    // Create predicate to determine if a column is in the local diagonal block
    t_colIndex my_first_col = this->part_offsets_h[my_id];
    t_colIndex one_past_my_last_col = this->part_offsets_h[my_id + 1];
    auto in_local_diagonal_block = [my_first_col, one_past_my_last_col](const t_colIndex col_index) {
        return col_index >= my_first_col && col_index < one_past_my_last_col;
    };

    int h_cidx_allocated = 0;
    const t_colIndex *h_col_indices_global = (const t_colIndex *)this->getHostPointerForData(col_indices, num_nonzeros * sizeof(t_colIndex), &h_cidx_allocated);
    // gather all off-diag columns
    I64Vector_h off_diag_cols;
    for (int i = 0; i < num_nonzeros; i++)
    {
        if (!in_local_diagonal_block(h_col_indices_global[i]))
        {
            off_diag_cols.push_back(h_col_indices_global[i]);
        }
    }
    auto global_to_local = loadDistributed_LocalToGlobal<t_colIndex>(num_rows, off_diag_cols);
    // set 1, then scan to compute local row indices
    // "coordinate-shift" columns so they lie in much smaller range of my diagonal indices
    int diagonal_size = this->part_offsets_h[my_id  + 1] - this->part_offsets_h[my_id];
    IVector_h my_indices(diagonal_size);
    for (int i = 0; i < num_nonzeros; i++)
    {
        t_colIndex col_index = h_col_indices_global[i];
        if (in_local_diagonal_block(h_col_indices_global[i]))     // find my local columns and set to 1
        {
            // olumns that are on *my* diag partition cannot have an index from 0..num_rows_global
            // instead, part_offsets_h[my_id] <= col_index < part_offsets[my_id+1]
            col_index -= this->part_offsets_h[my_id];
            my_indices[col_index] = 1;
        }
    }
    thrust::exclusive_scan(my_indices.begin(), my_indices.end(), my_indices.begin());

    // remap colums to local
    IVector_h local_col_indices(num_nonzeros);
    for (int i = 0; i < num_nonzeros; i++)
    {
        t_colIndex col_index = h_col_indices_global[i];
        if (!in_local_diagonal_block(col_index))
        {
            // off-diag
            local_col_indices[i] = global_to_local[col_index];
        }
        else
        {
            // diag
            col_index -= this->part_offsets_h[my_id];
            local_col_indices[i] = my_indices[col_index];
        }
    }
    loadDistributed_InitLocalMatrix(local_col_indices, num_rows, num_nonzeros, block_dimx, block_dimy, row_offsets, values, diag);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
template <typename t_colIndex>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::loadDistributedMatrix(
    int num_rows,
    int num_nonzeros,
    const int block_dimx,
    const int block_dimy,
    const int *row_offsets,
    const t_colIndex *col_indices,
    const mat_value_type *values,
    int num_ranks,
    int num_rows_global,
    const void *diag,
    const MatrixDistribution &dist)
{
    using PI = MatrixDistribution::PartitionInformation;
    switch (dist.getPartitionInformationStyle()) {
        case PI::PartitionVec:
            loadDistributedMatrixPartitionVec(num_rows, num_nonzeros, block_dimx, block_dimy, 
                row_offsets, col_indices, values, num_ranks, num_rows_global, diag, (const int*) dist.getPartitionData());
            break;
        case PI::PartitionOffsets:
            loadDistributedMatrixPartitionOffsets(num_rows, num_nonzeros, block_dimx, block_dimy, 
                row_offsets, col_indices, values, num_ranks, num_rows_global, diag, (const t_colIndex*) dist.getPartitionData());
            break;
        default:
            FatalError("Unsupported partitioning data format used with loadDistributedMatrix", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::renumberMatrixOneRing(int update_neighbours)
{
    FatalError("Distributed classical AMG not implemented on host", AMGX_ERR_NOT_IMPLEMENTED);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::renumberMatrixOneRing(int update_neighbours)
{
    // Step 1: Using halo_ranges, flag neighbors and at the same time, flag halo_nodes (flag_halo_nodes_local)
    int my_id = this->global_id();
    int num_parts = this->get_num_partitions();
    this->set_base_index(this->part_offsets_h[my_id]);
    this->set_index_range(this->part_offsets_h[my_id + 1] - this->part_offsets_h[my_id]);
    DistributedArranger<TConfig_d> *prep = new DistributedArranger<TConfig_d>;

    // Create/update list of neighbors
    if (update_neighbours)
    {
        typedef typename TConfig::template setVecPrec<AMGX_vecInt64>::Type i64vec_value_type;
        typedef Vector<i64vec_value_type> I64Vector;
        typedef typename Matrix<TConfig>::MVector MVector;
        std::vector<IVector> halo_row_offsets(this->neighbors.size());
        std::vector<I64Vector> halo_global_indices(this->neighbors.size());
        std::vector<MVector> halo_values(this->neighbors.size());
        prep->create_halo_rows_global_indices(*(this->A), halo_row_offsets, halo_global_indices, halo_values);
        prep->update_neighbors_list(*(this->A), this->neighbors, this->halo_ranges_h, this->halo_ranges, this->part_offsets_h, this->part_offsets, halo_row_offsets, halo_global_indices);
    }
    else
    {
        prep->create_neighbors_v2(*(this->A));
    }

    this->getComms()->set_neighbors(this->neighbors.size());
    // Create B2L_maps and L2H_maps
    prep->create_boundary_lists_v3(*(this->A));
    // halo_offsets
    int neighbors = this->A->manager->num_neighbors();
    int A_num_rows, offset;
    this->A->getOffsetAndSizeForView(OWNED, &offset, &A_num_rows);
    this->halo_offsets.resize(neighbors + 1, 0);
    this->halo_offsets[0] = A_num_rows;

    for (int i = 0; i < neighbors; i++)
    {
        this->halo_offsets[i + 1] = this->halo_offsets[i] + this->B2L_maps[i].size();
    }

    this->getComms()->exchange_vectors(this->A->manager->B2L_maps, *(this->A), 0);
    // Initialize B2L_rings
    int num_neighbors = this->neighbors.size();
    this->B2L_rings.resize(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        this->B2L_rings[i].resize(2);
        this->B2L_rings[i][0] = 0;
        this->B2L_rings[i][1] = this->B2L_maps[i].size();
    }

    prep->initialize_B2L_maps_offsets(*(this->A), 1);
    delete prep;
    //Use the exchanged halo row matrices and the boundary/halo index lists to renumber the matrix
    // Step 5: renumber all owned rows and columns
    this->reorder_matrix_owned();
    // Step 6: renumber local_to_global_map
    int num_owned_rows = this->A->manager->halo_offsets[0];
    int size_one_ring;
    this->A->getOffsetAndSizeForView(FULL, &offset, &size_one_ring);
    I64Vector_d global_col_indices(size_one_ring);
    thrust::sequence(global_col_indices.begin(), global_col_indices.begin() + num_owned_rows, this->base_index() );
    cudaCheckError();
    global_col_indices.dirtybit = 1;
    this->exchange_halo(global_col_indices, global_col_indices.tag);
    thrust_wrapper::copy(global_col_indices.begin() + num_owned_rows, global_col_indices.begin() + size_one_ring, this->local_to_global_map.begin(), this->get_int_stream(), true);
    cudaCheckError();
}


template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::renumber_P_R(Matrix_h &P, Matrix_h &R, Matrix_h &A_fine)
{
    FatalError("Distributed classical AMG not implemented on host", AMGX_ERR_NOT_IMPLEMENTED);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::renumber_P_R(Matrix_d &P, Matrix_d &R, Matrix_d &A_fine)
{
    int cta_size = 256;
    int num_owned_fine_pts = A_fine.manager->halo_offsets[0];
    int num_owned_coarse_pts, offset;
    // matrix Ac
    this->A->getOffsetAndSizeForView(OWNED, &offset, &num_owned_coarse_pts);
    // Renumber the owned col indices of P (not the halo columns ,since P.manager was created assunming some other numbering)
    int nnz_owned_fine_pts = P.row_offsets[num_owned_fine_pts];
    int num_blocks_fine = min(4096, (nnz_owned_fine_pts + cta_size - 1) / cta_size);

    if (num_blocks_fine > 0)
    {
        renumber_P_col_indices <<< num_blocks_fine, cta_size>>>(P.col_indices.raw(), this->renumbering.raw(), num_owned_coarse_pts, nnz_owned_fine_pts);
        cudaCheckError();
    }

    // Renumber the B2L_maps of P
    for (int i = 0; i < P.manager->neighbors.size(); i++)
    {
        thrust::copy(thrust::make_permutation_iterator(this->renumbering.begin(), P.manager->B2L_maps[i].begin()),
                     thrust::make_permutation_iterator(this->renumbering.begin(), P.manager->B2L_maps[i].end()),
                     P.manager->B2L_maps[i].begin());
    }

    cudaCheckError();
    // Don't renumber the L2H_maps or the halo
    // Renumber the local_to_global_map of matrix P (since neighbors renumbered their owned rows)
    // Swap owned rows of R
    IVector new_row_offsets(R.row_offsets.size());
    int insert = 0;
    // Only renumber the owned rows
    int num_blocks_owned = min(4096, (num_owned_coarse_pts + cta_size - 1) / cta_size);

    if (num_blocks_owned > 0)
    {
        calc_rowlen_reorder <<< num_blocks_owned, cta_size >>>(R.row_offsets.raw(), new_row_offsets.raw(), this->renumbering.raw(), num_owned_coarse_pts, insert);
        cudaCheckError();
    }

    thrust_wrapper::exclusive_scan(new_row_offsets.begin(), new_row_offsets.begin() + num_owned_coarse_pts + 1, new_row_offsets.begin());
    cudaCheckError();
    // Copy the row_offsets for halo rows
    thrust::copy(R.row_offsets.begin() + num_owned_coarse_pts, R.row_offsets.end(), new_row_offsets.begin() + num_owned_coarse_pts);
    cudaCheckError();
    // Reorder the rows of R (no need to reorder the column indices)
    int new_nnz = new_row_offsets[new_row_offsets.size() - 1];
    int halo_offset = new_row_offsets[num_owned_coarse_pts];
    typedef typename MatPrecisionMap<t_matPrec>::Type ValueTypeA;
    VVector new_values(new_nnz * R.get_block_size(), types::util< ValueTypeA >::get_zero());
    IVector new_col_indices(new_nnz, 0);
    int num_blocks_total = min(4096, (R.get_num_rows() + cta_size - 1) / cta_size);

    if (num_blocks_total > 0)
    {
        reorder_R_matrix <32> <<< num_blocks_total, 512>>>(R.row_offsets.raw(), R.col_indices.raw(), R.values.raw(), new_row_offsets.raw(), new_col_indices.raw(), new_values.raw(), this->renumbering.raw(), R.get_block_size(), R.get_num_rows(), num_owned_coarse_pts);
        cudaCheckError();
    }

    R.col_indices.swap(new_col_indices);
    R.row_offsets.swap(new_row_offsets);
    R.values.swap(new_values);

    // Renumber the local_to_global_map (since neighbors have changed their owned numbering)
    if (P.manager->neighbors.size() != 0)
    {
        int size_one_ring = P.manager->halo_offsets[P.manager->neighbors.size()];
        I64Vector_d global_col_indices(size_one_ring);
        thrust::sequence(global_col_indices.begin(), global_col_indices.begin() + num_owned_coarse_pts, this->base_index());
        cudaCheckError();
        global_col_indices.dirtybit = 1;
        P.manager->exchange_halo(global_col_indices, global_col_indices.tag);
        thrust_wrapper::copy(global_col_indices.begin() + num_owned_coarse_pts, global_col_indices.begin() + size_one_ring, P.manager->local_to_global_map.begin(), this->get_int_stream(), true);
        cudaCheckError();
    }

    DistributedArranger<TConfig_d> *prep = new DistributedArranger<TConfig_d>;
    prep->initialize_B2L_maps_offsets(P, 1);
    delete prep;
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::createOneRingHaloRows()
{
    // Input:
    // A matrix with 1-ring B2L_maps, 1-ring halo_offsets
    // Outputs:
    // A matrix with: 1-ring rows,
    //                2-ring B2L_maps,
    //                2-ring halo_offsets
    //                2-ring neighbors
    // Implement here:
    // Look at function create_B2L_from_maps, which calls create_rings, create_halo_btl, create_halo_rows and comms->exchange_matrix_halo
    DistributedArranger<TConfig_d> *prep = new DistributedArranger<TConfig_d>;
    prep->create_one_ring_halo_rows(*(this->A));
    // I believe this can be removed since we don't use masked SpMV anymore
    prep->createRowsLists(*(this->A), false);
    delete prep;
    // this is not necessary anymore becasue we don't use latency hiding
    // however in future we might want to get back to this in case we want to use latency hiding
    //this->reorder_matrix();
}

template <class TConfig>
inline DistributedManagerBase<TConfig>::DistributedManagerBase(
    Matrix<TConfig> &a,
    INDEX_TYPE allocated_halo_depth,
    INDEX_TYPE num_import_rings,
    int num_neighbors,
    const VecInt_t *neighbors_) : m_fine_level_comms(NULL), A(&a), m_pinned_buffer_size(0), m_pinned_buffer(NULL), _num_interior_nodes(0), _num_boundary_nodes(0), _comms(NULL), has_B2L(false), neighbors(_neighbors), halo_rows_ref_count(0), halo_rows(NULL), halo_btl_ref_count(0), halo_btl(NULL), halo_ranges(_halo_ranges), halo_ranges_h(_halo_ranges_h), part_offsets(_part_offsets), part_offsets_h(_part_offsets_h),
    B2L_maps(_B2L_maps),  L2H_maps(_L2H_maps), B2L_rings(_B2L_rings), m_fixed_view_size(false)
{
    hipStreamCreateWithFlags(&m_int_stream, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&m_bdy_stream, hipStreamNonBlocking);

    if (num_import_rings != 1)
    {
        FatalError("num_rings > 1 not supported in fine_level consolidation", AMGX_ERR_NOT_IMPLEMENTED);
    }

    if (allocated_halo_depth != 1)
    {
        FatalError("allocated_halo_depth > 1 not supported in fine_level consolidation", AMGX_ERR_NOT_IMPLEMENTED);
    }

    this->set_num_halo_rings(num_import_rings);
    neighbors.resize(num_neighbors);
    hipMemcpy(neighbors.raw(), neighbors_, num_neighbors * sizeof(VecInt_t), hipMemcpyDefault);
    cudaCheckError();
}

template <class TConfig>
inline void DistributedManagerBase<TConfig>::cacheMaps(const VecInt_t *b2l_maps, const VecInt_t *b2l_ptrs, const VecInt_t *l2h_maps, const VecInt_t *l2h_ptrs)
{
    int num_neighbors = this->neighbors.size();
    this->cached_B2L_maps.resize(num_neighbors);
    this->cached_L2H_maps.resize(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        int size = b2l_ptrs[i + 1] - b2l_ptrs[i];
        this->cached_B2L_maps[i].resize(size);
        int count = 0;

        for (int j = b2l_ptrs[i]; j < b2l_ptrs[i + 1]; j++)
        {
            this->cached_B2L_maps[i][count] = b2l_maps[j];
            count++;
        }

        size = l2h_ptrs[i + 1] - l2h_ptrs[i];
        this->cached_L2H_maps[i].resize(size);
        count = 0;

        for (int j = l2h_ptrs[i]; j < l2h_ptrs[i + 1]; j++)
        {
            this->cached_L2H_maps[i][count] = l2h_maps[j];
            count++;
        }
    }
}

template <class TConfig>
inline void DistributedManagerBase<TConfig>::cacheMapsOneRing()
{
    int num_neighbors = this->neighbors.size();
    this->cached_B2L_maps.resize(num_neighbors);
    this->cached_L2H_maps.resize(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        this->cached_B2L_maps[i] = this->B2L_maps[i];
        this->cached_L2H_maps[i] = this->L2H_maps[i];
    }
}

template <class TConfig>
inline void DistributedManagerBase<TConfig>::cacheMapsOneRing(const VecInt_t **b2l_maps, const VecInt_t *b2l_sizes, const VecInt_t **l2h_maps, const VecInt_t *l2h_sizes)
{
    int num_neighbors = this->neighbors.size();
    this->cached_B2L_maps.resize(num_neighbors);
    this->cached_L2H_maps.resize(num_neighbors);
    // buffering in the case of GPU data. This shouldn't much affect performance
    std::vector<VecInt_t *> b2l_buffer, l2h_buffer;
    std::vector<VecInt_t> b2l_sizes_buffer, l2h_sizes_buffer;
    b2l_buffer.resize(num_neighbors);
    l2h_buffer.resize(num_neighbors);
    b2l_sizes_buffer.resize(num_neighbors);
    l2h_sizes_buffer.resize(num_neighbors);
    hipMemcpy(&(b2l_sizes_buffer[0]), b2l_sizes, sizeof(VecInt_t) * num_neighbors, hipMemcpyDefault);
    hipMemcpy(&(l2h_sizes_buffer[0]), l2h_sizes, sizeof(VecInt_t) * num_neighbors, hipMemcpyDefault);
    hipMemcpy(&(b2l_buffer[0]), b2l_maps, sizeof(VecInt_t *) * num_neighbors, hipMemcpyDefault);
    hipMemcpy(&(l2h_buffer[0]), l2h_maps, sizeof(VecInt_t *) * num_neighbors, hipMemcpyDefault);

    // caching all of the maps
    for (int i = 0; i < num_neighbors; i++)
    {
        int size = b2l_sizes_buffer[i];
        this->cached_B2L_maps[i].resize(size);
        hipMemcpy(&(this->cached_B2L_maps[i][0]), b2l_buffer[i], sizeof(VecInt_t) * size, hipMemcpyDefault);
        cudaCheckError();
        size = l2h_sizes_buffer[i];
        this->cached_L2H_maps[i].resize(size);
        hipMemcpy(&(this->cached_L2H_maps[i][0]), l2h_buffer[i], sizeof(VecInt_t) * size, hipMemcpyDefault);
        cudaCheckError();
    }
}

template <class TConfig>
void DistributedManagerBase<TConfig>::uploadMatrix(int n, int nnz, int block_dimx, int block_dimy, const int *row_ptrs, const int *col_indices, const void *data, const void *diag, Matrix<TConfig> &in_A)
{
    this->A->manager->initializeUploadReorderAll(n, nnz, block_dimx, block_dimy, row_ptrs, col_indices, data, diag, *(this->A));
}

template <class TConfig>
void DistributedManagerBase<TConfig>::checkPinnedBuffer(size_t size)
{
    if ((m_pinned_buffer_size < size) && (m_pinned_buffer != NULL))
    {
        hipHostFree(m_pinned_buffer);
        m_pinned_buffer = NULL;
        m_pinned_buffer_size = 0;
    }

    if (m_pinned_buffer == NULL)
    {
        m_pinned_buffer_size = (size_t)(size * 1.1);
        hipHostMalloc(&m_pinned_buffer, m_pinned_buffer_size);
    }
}


template <class TConfig>
DistributedManagerBase<TConfig>::~DistributedManagerBase()
{
    if (m_pinned_buffer != NULL)
    {
        hipHostFree(m_pinned_buffer);
    }

    destroyComms();
    // from childrens:
    hipStreamDestroy(this->m_int_stream);
    hipStreamDestroy(this->m_bdy_stream);

    if (!this->halo_rows_ref_count && this->halo_rows != NULL)
    {
        delete this->halo_rows;
        this->halo_rows = NULL;
    }

    if (!this->halo_btl_ref_count && this->halo_btl != NULL)
    {
        delete this->halo_btl;
        this->halo_btl = NULL;
    }
}

// if pointer is host pointer - returns data. If it is device pointer - copies it to the m_pinned_buffer and returns pointer to m_pinned_buffer
template <class TConfig>
void *DistributedManagerBase<TConfig>::getHostPointerForData(void *ptr, size_t size, int *allocated)
{
    hipError_t rc;
    hipPointerAttribute_t att;
    void *ptr_h;
    cudaCheckError();
    /* WARNING: We may accept the following types of allocation for ptr:
       1. malloc                    [host memory]
       2. hipMalloc                [device memory]
       3. malloc + hipHostRegister [AMGX_pin_memory/AMGX_unpin_memory host memory]
       4. hipHostAlloc             [pinned host memory form the beginning]

       The correct way to conver these cases is the following:
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2, 3 or 4.
       }
       else{
         //you are in case 1.
       }
       The following pattern of checks should be implemented
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2 or 4.
       }
       else{
         st = hipHostGetDevicePointer(ptr_on_device, ptr, 0);
         if (st == hipSuccess){
           //you are in case 3.
         }
         else{
           //you are in case 1.
         }
       }
       The above pattern will be used whenever we need to process input data.

       Obs.: parameter size is in bytes and
             parameter allocated indicates whether memory was allocated
             and needs to be release later on. */
    /*
    // original implementation
    hipPointerGetAttributes(&att, ptr);
    if (att.hostPointer == NULL)
    {
      checkPinnedBuffer(size);
      hipMemcpy(m_pinned_buffer, ptr, size, hipMemcpyDefault);
      return m_pinned_buffer;
    }
    else {
      return ptr;
    }
    */
    *allocated = 0;
    // get pointer to values on the device
    rc = hipPointerGetAttributes(&att, ptr);

    if (rc == hipSuccess)
    {
        //you are in case 2 or 4 from the above comment.
        if (att.hostPointer == NULL)
        {
            //you are in case 2
            checkPinnedBuffer(size);
            rc = hipMemcpy(m_pinned_buffer, ptr, size, hipMemcpyDefault);

            if (rc != hipSuccess)
            {
                FatalError("Could not copy into the temporary (host) storage. Try pinning the memory to avoid the hipMemcpy.", AMGX_ERR_BAD_PARAMETERS);
            }

            ptr_h = m_pinned_buffer;
            *allocated = 1;
        }
        else
        {
            //you are in case 4
            ptr_h = ptr;
        }
    }
    else
    {
        //you are in case 1 or 3 from the above comment
        ptr_h = ptr;
    }

    hipGetLastError(); //to reset last error

    /* check for null pointers */
    if (ptr_h == NULL)
    {
        FatalError("Result of (host) allocation of required temporary storage is NULL. Try pinning the memory to reduce storage requirements.", AMGX_ERR_BAD_PARAMETERS);
    }

    return ptr_h;
}

// if pointer is host pointer - returns data. If it is device pointer - copies it to the m_pinned_buffer and returns pointer to m_pinned_buffer
template <class TConfig>
const void *DistributedManagerBase<TConfig>::getHostPointerForData(const void *ptr, size_t size, int *allocated)
{
    hipError_t rc;
    hipPointerAttribute_t att;
    void *ptr_h;
    cudaCheckError();
    /* WARNING: We may accept the following types of allocation for ptr:
       1. malloc                    [host memory]
       2. hipMalloc                [device memory]
       3. malloc + hipHostRegister [AMGX_pin_memory/AMGX_unpin_memory host memory]
       4. hipHostAlloc             [pinned host memory form the beginning]

       The correct way to conver these cases is the following:
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2, 3 or 4.
       }
       else{
         //you are in case 1.
       }
       The following pattern of checks should be implemented
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2 or 4.
       }
       else{
         st = hipHostGetDevicePointer(ptr_on_device, ptr, 0);
         if (st == hipSuccess){
           //you are in case 3.
         }
         else{
           //you are in case 1.
         }
       }
       The above pattern will be used whenever we need to process input data.

       Obs.: parameter size is in bytes and
             parameter allocated indicates whether memory was allocated
             and needs to be release later on. */
    *allocated = 0;
    // get pointer to values on the device
    rc = hipPointerGetAttributes(&att, ptr);

    if (rc == hipSuccess)
    {
        //you are in case 2 or 4 from the above comment.
        if (att.hostPointer == NULL)
        {
            //you are in case 2
            checkPinnedBuffer(size);
            rc = hipMemcpy(m_pinned_buffer, ptr, size, hipMemcpyDefault);

            if (rc != hipSuccess)
            {
                FatalError("Could not copy into the temporary (host) storage. Try pinning the memory to avoid the hipMemcpy.", AMGX_ERR_BAD_PARAMETERS);
            }

            ptr_h = m_pinned_buffer;
            *allocated = 1;
            hipGetLastError(); //to reset last error
            return ptr_h;
        }
        else
        {
            //you are in case 4
            hipGetLastError(); //to reset last error
            return ptr;
        }
    }
    else
    {
        hipGetLastError(); //to reset last error
        //you are in case 1 or 3 from the above comment
        return ptr;
    }
}


template <class TConfig>
void *DistributedManagerBase<TConfig>::getDevicePointerForData(void *ptr, size_t size, int *allocated)
{
    hipError_t rc;
    hipPointerAttribute_t att;
    void *ptr_d;
    cudaCheckError();
    /* WARNING: We may accept the following types of allocation for ptr:
       1. malloc                    [host memory]
       2. hipMalloc                [device memory]
       3. malloc + hipHostRegister [AMGX_pin_memory/AMGX_unpin_memory host memory]
       4. hipHostAlloc             [pinned host memory form the beginning]

       The correct way to conver these cases is the following:
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2, 3 or 4.
       }
       else{
         //you are in case 1.
       }
       The following pattern of checks should be implemented
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2 or 4.
       }
       else{
         st = hipHostGetDevicePointer(ptr_on_device, ptr, 0);
         if (st == hipSuccess){
           //you are in case 3.
         }
         else{
           //you are in case 1.
         }
       }
       The above pattern will be used whenever we need to process input data.

       Obs.: parameter size is in bytes and
             parameter allocated indicates whether memory was allocated
             and needs to be release later on. */
    *allocated = 0;
    // get pointer to values on the device
    rc = hipPointerGetAttributes(&att, ptr);

    if (rc == hipSuccess)
    {
        //you are in case 2 or 4 from the above comment.
        ptr_d = (void *)att.devicePointer;
    }
    else
    {
        //you are in case 1 or 3 from the above comment
        rc = hipHostGetDevicePointer(&ptr_d, ptr, 0);

        if (rc != hipSuccess)
        {
            //you are in case 1
            rc = hipMalloc(&ptr_d, size);

            if (rc != hipSuccess)
            {
                FatalError("Could not allocate required temporary storage. Try pinning the memory to reduce storage requirements.", AMGX_ERR_BAD_PARAMETERS);
            }

            rc = hipMemcpy(ptr_d, ptr, size, hipMemcpyDefault);

            if (rc != hipSuccess)
            {
                FatalError("Could not copy into the temporary storage. Try pinning the memory to avoid the hipMemcpy.", AMGX_ERR_BAD_PARAMETERS);
            }

            *allocated = 1;
        }
    }

    /* check for null pointers */
    if (ptr_d == NULL)
    {
        FatalError("Result of allocation of required temporary storage is NULL. Try pinning the memory to reduce storage requirements.", AMGX_ERR_BAD_PARAMETERS);
    }

    hipGetLastError(); //to reset last error
    return ptr_d;
}

template <class TConfig>
const void *DistributedManagerBase<TConfig>::getDevicePointerForData(const void *ptr, size_t size, int *allocated)
{
    hipError_t rc;
    hipPointerAttribute_t att;
    void *ptr_d;
    cudaCheckError();
    /* WARNING: We may accept the following types of allocation for ptr:
       1. malloc                    [host memory]
       2. hipMalloc                [device memory]
       3. malloc + hipHostRegister [AMGX_pin_memory/AMGX_unpin_memory host memory]
       4. hipHostAlloc             [pinned host memory form the beginning]

       The correct way to conver these cases is the following:
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2, 3 or 4.
       }
       else{
         //you are in case 1.
       }
       The following pattern of checks should be implemented
       hipPointerAttribute_t att;
       hipError_t st = hipPointerGetAttributes(&att, ptr);
       if (st == hipSuccess) {
         //you are in case 2 or 4.
       }
       else{
         st = hipHostGetDevicePointer(ptr_on_device, ptr, 0);
         if (st == hipSuccess){
           //you are in case 3.
         }
         else{
           //you are in case 1.
         }
       }
       The above pattern will be used whenever we need to process input data.

       Obs.: parameter size is in bytes and
             parameter allocated indicates whether memory was allocated
             and needs to be release later on. */
    *allocated = 0;
    // get pointer to values on the device
    rc = hipPointerGetAttributes(&att, ptr);

    if (rc == hipSuccess)
    {
        //you are in case 2 or 4 from the above comment.
        hipGetLastError(); //to reset last error
        return (const void *)att.devicePointer;
    }
    else
    {
        //you are in case 1 or 3 from the above comment
        rc = hipHostGetDevicePointer(&ptr_d, (void *)ptr, 0);

        if (rc != hipSuccess)
        {
            //you are in case 1
            rc = hipMalloc(&ptr_d, size);

            if (rc != hipSuccess)
            {
                FatalError("Could not allocate required temporary storage. Try pinning the memory to reduce storage requirements.", AMGX_ERR_BAD_PARAMETERS);
            }

            rc = hipMemcpy(ptr_d, ptr, size, hipMemcpyDefault);

            if (rc != hipSuccess)
            {
                FatalError("Could not copy into the temporary storage. Try pinning the memory to avoid the hipMemcpy.", AMGX_ERR_BAD_PARAMETERS);
            }

            *allocated = 1;
            hipGetLastError(); //to reset last error
            return (const void *)ptr_d;
        }
    }

    /* check for null pointers */
    if (ptr_d == NULL)
    {
        FatalError("Result of allocation of required temporary storage is NULL. Try pinning the memory to reduce storage requirements.", AMGX_ERR_BAD_PARAMETERS);
    }

    // shouldn't get there
    hipGetLastError(); //to reset last error
    return NULL;
}

template <class TConfig>
void initializeMatrixCopyAll(int n, int nnz, int block_dimx, int block_dimy, const int *row_ptrs, const int *col_indices, const void *data, const void *diag, Matrix<TConfig> *A)
{
    typedef typename TConfig::MatPrec  mat_value_type;
    A->resize( n, n, nnz, block_dimx, block_dimy );
    //Upload the entire matrix
    hipMemcpy( A->row_offsets.raw(), row_ptrs, (n + 1) * sizeof(int), hipMemcpyDefault );
    cudaCheckError();
    hipMemcpy( A->col_indices.raw(), col_indices, (nnz) * sizeof(int), hipMemcpyDefault );
    cudaCheckError();
    hipMemcpy( A->values.raw(), (mat_value_type *)data, (nnz * block_dimx * block_dimy) * sizeof(mat_value_type), hipMemcpyDefault );
    cudaCheckError();

    if (diag)
    {
        hipMemcpy( A->values.raw() + A->diagOffset()*A->get_block_size(), (mat_value_type *)diag, (n * block_dimx * block_dimy) * sizeof(mat_value_type), hipMemcpyDefault );
    }
    else
    {
        A->computeDiagonal();
    }

    cudaCheckError();
}

template <class TConfig>
void DistributedManagerBase<TConfig>::updateMapsReorder()
{
    int my_id = this->getComms()->get_global_id();
    DistributedComms<TConfig> *comms_tmp = this->getComms();
    DistributedComms<TConfig> **comms_ = &comms_tmp;
    // Copy B2L_maps in their final place
    int num_neighbors = this->neighbors.size();
    B2L_maps.resize(num_neighbors);
    L2H_maps.resize(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        B2L_maps[i] = this->cached_B2L_maps[i];
        L2H_maps[i] = this->cached_L2H_maps[i];
    }

    //Create a DistributedArranger object to map further halo rings and to construct halo row matrices and exchange them (if halo_coloring != LAST)
    DistributedArranger<TConfig> *prep = new DistributedArranger<TConfig>;
    prep->create_B2L_from_maps( (*(this->A)), my_id, this->num_halo_rings(), neighbors,
                                B2L_maps, L2H_maps, B2L_rings, comms_, &halo_rows, &halo_btl);
    DistributedManagerBaseInit(my_id, 0, this->A->get_num_rows(), *(this->A), comms_, NULL, NULL);
    //Use the exchanged halo row matrices and the boundary/halo index lists to renumber the matrix
    this->reorder_matrix();
    prep->initialize_B2L_maps_offsets(*(this->A), this->num_halo_rings());
    delete prep;
}

template <class TConfig>
void DistributedManagerBase<TConfig>::initializeUploadReorderAll(int n, int nnz, int block_dimx, int block_dimy, const int *row_ptrs, const int *col_indices, const void *data, const void *diag, Matrix<TConfig> &in_A)
{
    this->A = &in_A;
    initializeMatrixCopyAll<TConfig>(n, nnz, block_dimx, block_dimy, row_ptrs, col_indices, data, diag, this->A);
    this->updateMapsReorder();
}

template <class TConfig>
void DistributedManagerBase<TConfig>::destroyComms()
{
    if ( (this->_comms) != NULL )
    {
        if (this->_comms->decr_ref_count())
        {
            delete (this->_comms);
            this->_comms = NULL;
        }
    }

    if ( (this->m_fine_level_comms) != NULL)
    {
        if (this->m_fine_level_comms->decr_ref_count())
        {
            delete (this->m_fine_level_comms);
            this->m_fine_level_comms = NULL;
        }
    }
}

template <class TConfig>
void DistributedManagerBase<TConfig>::initComms(Resources *rsrc)
{
    this->createComms(rsrc);
    int my_id = this->getComms()->get_global_id();
    int num_parts = this->getComms()->get_num_partitions();
    this->set_global_id(my_id);
    this->set_num_partitions(num_parts);
}


template <class TConfig>
void DistributedManagerBase<TConfig>::createComms(Resources *rsrc)
{
    // create communicator
#ifdef AMGX_WITH_MPI
    destroyComms();
    if (rsrc == NULL)
        FatalError("Resources should not be NULL", AMGX_ERR_INTERNAL);

    MPI_Comm *mpi_comm = rsrc->getMpiComm();
    AMG_Config *cfg = rsrc->getResourcesConfig();
    std::string comm_value, comm_scope;
    cfg->getParameter<std::string>("communicator", comm_value, "default", comm_scope);
    int rank = -1;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    if (comm_value == "MPI_DIRECT")
    {
        _comms = new CommsMPIDirect<TConfig>(*cfg, comm_scope, mpi_comm);
        std::string comm_log("Using CUDA-Aware MPI (GPU Direct) communicator...\n");
        amgx_distributed_output(comm_log.c_str(), comm_log.length());
    }
    else if (comm_value == "MPI")
    {
        _comms =  new CommsMPIHostBufferStream<TConfig>(*cfg, comm_scope, mpi_comm);
        std::string comm_log("Using Normal MPI (Hostbuffer) communicator...\n");
        amgx_distributed_output(comm_log.c_str(), comm_log.length());
    }
    else 
    { 
        FatalError("Bad communicator value", AMGX_ERR_BAD_PARAMETERS); 
    }

#endif
}

template <class TConfig>
void DistributedManagerBase<TConfig>::malloc_export_maps(VecInt_t ***b2l_maps_e, VecInt_t **b2l_maps_sizes_e, VecInt_t ***l2h_maps_e, VecInt_t **l2h_maps_sizes_e)
{
    *b2l_maps_e = (VecInt_t **) malloc(sizeof(VecInt_t *)*this->num_neighbors());
    *l2h_maps_e = (VecInt_t **) malloc(sizeof(VecInt_t *)*this->num_neighbors());
    *b2l_maps_sizes_e = (VecInt_t *) malloc(sizeof(VecInt_t) * (this->num_neighbors()));
    *l2h_maps_sizes_e = (VecInt_t *) malloc(sizeof(VecInt_t) * (this->num_neighbors()));

    for (int i = 0; i < this->num_neighbors(); i++)
    {
        (*b2l_maps_sizes_e)[i] = B2L_maps[i].size();
        (*l2h_maps_sizes_e)[i] = L2H_maps[i].size();
        (*b2l_maps_e)[i] =  (VecInt_t *) malloc(sizeof(VecInt_t) * ( (*b2l_maps_sizes_e)[i]) );

        if (L2H_maps[i].size() != 0)
        {
            (*l2h_maps_e)[i] =  (VecInt_t *) malloc(sizeof(VecInt_t) * ( (*l2h_maps_sizes_e)[i]) );
            thrust::copy(L2H_maps[i].begin(), L2H_maps[i].end(), (*l2h_maps_e)[i]);
        }

        thrust::copy(B2L_maps[i].begin(), B2L_maps[i].end(), (*b2l_maps_e)[i]);
    }

    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::createRenumbering(IVector &renumbering)
{
    int num_neighbors = this->neighbors.size();
    // still renumber if the number of neighbors = 0, to support non-symmetric matrices
    // if (num_neighbors == 0) return;
    /*
      EXAMPLE
      Example matrix, partition 1 arrives with state:
      A.row_offsets = [0 4 11 15 20]
      A.col_indices = [4 0 1 2
                       4 5 0 1 2 3 7
                       0 1 2 3
                       1 2 3 6 7]
      num_neighbors=2; neighbors = [0 2]
      B2L_rings[[0 2 4][0 2 4]] B2L_maps[[0 1| 2 3][1 3| 0 2]]
      L2H_maps (and halo_lists) [[4 5][6 7]]
    */
    int size = 0;

    if (this->L2H_maps.size())
    {
        size = thrust_wrapper::reduce(this->A->col_indices.begin(), this->A->col_indices.end(), int(0), thrust::maximum<int>()) + 1; //Sufficient to do reduction on lth maps
        cudaCheckError();
    }
    else
    {
        size = this->A->get_num_rows();
    }

    int rings = (this->B2L_rings.size() > 0) ? this->B2L_rings[0].size() - 1 : 0;
    //initial size to size+1 so we have the total size after a scan
    renumbering.resize(size + 1);
    int global_size = size;
    //
    // Step 1 - in the main matrix, separate interior and boundary nodes (1/0 in flagArray), renumber interior ones with an exclusive scan
    //
    IVector flagArray(size + 1);
    thrust::fill(flagArray.begin(), flagArray.begin() + size + 1, 1);
    cudaCheckError();

    //sets 1 for interior nodes, 0 for boundary node
    for (int i = 0; i < num_neighbors; i++ )
    {
        int size = this->B2L_rings[i][1];
        int num_blocks = min(4096, (size + 127) / 128);

        if (size > 0)
        {
            remove_boundary_kernel <<< num_blocks, 128>>>(flagArray.raw(), this->B2L_maps[i].raw(), size);
        }

        //If there are any L2H maps
        if (this->L2H_maps.size() && this->L2H_maps[i].size())
        {
            int size = this->L2H_maps[i].size();
            int num_blocks = min(4096, (size + 127) / 128);
            remove_boundary_kernel <<< num_blocks, 128>>>(flagArray.raw(), this->L2H_maps[i].raw(), size);
        }

        cudaCheckError();
    }

    //gets the renumbering of interior nodes
    thrust_wrapper::exclusive_scan(flagArray.begin(), flagArray.begin() + size + 1, renumbering.begin());
    cudaCheckError();
    /*
     EXAMPLE
     After removing 1-ring boundary nodes and halo nodes from flagArray: [0 0 1 0 0 0 0 0]
     After exclusive scan, which gives renumbering for interior nodes (only node #2)
     renumbering: [0 0 0 1 1 1 1 1]
     */
    //
    // Step 2 - Renumber nodes that are in the boundary, stepping through each B2L map, and renumbering ones that have not been renumbered yet
    //
    //what is the biggest B2L size
    INDEX_TYPE max_size = 0;

    for (int i = 0; i < num_neighbors; i++)
    {
        max_size = max_size > this->B2L_rings[i][1] ? max_size : this->B2L_rings[i][1];

        if (this->L2H_maps.size())
        {
            max_size = max_size > this->L2H_maps[i].size() ? max_size : this->L2H_maps[i].size();
        }
    }

    //allocate work vectors (should be pretty small) that are used to renumber boundary nodes
    IVector boundary_renum_flags(max_size);
    IVector boundary_renum(max_size);
    //the number of renumbered nodes so far
    int max_element = renumbering[size];
    this->_num_interior_nodes = max_element;
    this->_num_boundary_nodes = this->A->get_num_rows() - max_element;
    renumbering.resize(size);

    /*
     EXAMPLE
     size = 8
     max_size = 2, max_element = 1, num_interior_nodes=1, num_boundary_nodes = 4-1 = 3
     */

    for (int i = 0; i < num_neighbors; i++)
    {
        //find nodes that are part of the current boundary and they haven't been renumbered yet
        thrust::fill(boundary_renum_flags.begin(), boundary_renum_flags.begin() + max_size, 0);
        int size = this->B2L_rings[i][1];
        int num_blocks = min(4096, (size + 191) / 192);

        if (size > 0)
            get_unassigned_kernel <<< num_blocks, 192>>>(flagArray.raw(),
                    this->B2L_maps[i].raw(),
                    boundary_renum_flags.raw(), size, global_size /*,rank*/);

        //calculate the local renumbering (within this boundary region) of these nodes
        thrust_wrapper::exclusive_scan(boundary_renum_flags.begin(), boundary_renum_flags.begin() + max_size, boundary_renum.begin());

        //apply renumbering to the big numbering table
        if (size > 0)
            set_unassigned_kernel <<< num_blocks, 192>>>(boundary_renum_flags.raw(),
                    boundary_renum.raw(),
                    this->B2L_maps[i].raw(),
                    renumbering.raw(),
                    size, max_element, global_size /*,rank*/);

        //update the number of renumbered nodes
        max_element += boundary_renum[max_size - 1] + boundary_renum_flags[max_size - 1];
        /*
         EXAMPLE
         for neighbor 0 (ID 0)
         boundary_renum_flags = [0 0], size = 2, flagArray [0 0 1 0 0 0 0 0]
         get_unassigned_kernel's output: boundary_renum_flags = [1 1] flagArray [1 1 1 0 0 0 0 0]
         after exclusive scan: boundary_renum [0 1]
         set_unassigned_kernel updates these arrays and renumbers B2L map:
         renumbering = [1 2 0 1 1 1 1 1] B2L_maps[0] = [1 2| 2 3] (note that after element 3 in renumbering and after element 2 we have invalid/not yet updated values)
         max_element = 3

         for neighbor 1 (ID 2)
         get_unassigned_kernels's output: boundary_renum_flags [0 1] flagArray [1 1 1 1 0 0 0 0]
         after exclusive scan boundary_renum [0 0]
         set_unassigned_kernel renumbering [1 2 0 3 1 1 1 1] B2L_maps[1] = [2 3| 0 2]
         max_element = 4
         */
    }

    cudaCheckError();

    //Get renumbering for halo indices
    if (this->L2H_maps.size())
    {
        //TODO: simplify this, we don't need to check whether it has already been renumbered, there is no overlap between halos
        for (int i = 0; i < num_neighbors; i++)
        {
            //find nodes that are part of the current boundary and they haven't been renumbered yet
            thrust::fill(boundary_renum_flags.begin(), boundary_renum_flags.begin() + max_size, 0);
            int size = this->L2H_maps[i].size();
            int num_blocks = min(4096, (size + 191) / 192);

            if (size > 0)
                get_unassigned_kernel <<< num_blocks, 192>>>(flagArray.raw(),
                        this->L2H_maps[i].raw(),
                        boundary_renum_flags.raw(), size, global_size /*,rank*/);

            //calculate the local renumbering (within this boundary region) of these nodes
            thrust_wrapper::exclusive_scan(boundary_renum_flags.begin(), boundary_renum_flags.begin() + max_size, boundary_renum.begin());

            //apply renumbering to the big numbering table
            if (size > 0)
                set_unassigned_kernel <<< num_blocks, 192>>>(boundary_renum_flags.raw(),
                        boundary_renum.raw(),
                        this->L2H_maps[i].raw(),
                        renumbering.raw(),
                        size, max_element, global_size /*,rank*/);

            //update the number of renumbered nodes
            max_element += boundary_renum[max_size - 1] + boundary_renum_flags[max_size - 1];
            /*
             EXAMPLE
             for neighbor 0 (ID 0)
             boundary_renum_flags = [0 0], size = 2, flagArray [1 1 1 1 0 0 0 0]
             get_unassigned_kernel's output: boundary_renum_flags = [1 1] flagArray [1 1 1 1 1 1 0 0]
             after exclusive scan: boundary_renum [0 1]
             set_unassigned_kernel updates these arrays and renumbers B2L map:
             renumbering = [1 2 0 3 4 5 1 1] L2H_maps[0] = [4 5]
             max_element = 6

             for neighbor 1 (ID 2)
             get_unassigned_kernels's output: boundary_renum_flags [1 1] flagArray [1 1 1 1 1 1 1 1]
             after exclusive scan boundary_renum [0 1]
             set_unassigned_kernel renumbering = [1 2 0 3 4 5 6 7] L2H_maps[1] = [6 7]
             max_element = 8
             */
        }

        cudaCheckError();
    }

    //apply renumbering to further halo rings too
    if (rings > 1)
    {
        for (int i = 0; i < num_neighbors; i++)
        {
            int size = this->B2L_rings[i][this->B2L_rings[i].size() - 1] - this->B2L_rings[i][1];
            int num_blocks = min(4096, (size + 127) / 128);
            renumber_b2l_maps <<< num_blocks, 128>>>(this->B2L_maps[i].raw() + this->B2L_rings[i][1], renumbering.raw(), size, global_size /*, rank*/);
        }

        cudaCheckError();
    }

    /*
     EXAMPLE
     renumbers further boundary rings as listed in B2L_maps, since they have not been replaced yet with their renumbered values
     B2L_maps [[1 2| 0 3][2 3| 1 0]]
     */
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::reorder_matrix_owned()
{
    int num_neighbors = this->neighbors.size();
    int size = this->A->get_num_rows();
    int num_blocks = min(4096, (size + 511) / 512);
    int rings = (this->B2L_rings.size() > 0) ? this->B2L_rings[0].size() - 1 : 0;
    this->set_num_halo_rings(rings);
    int diag = this->A->hasProps(DIAG);

    if (diag)
    {
        FatalError("External diag not supported in classical path", AMGX_ERR_NOT_IMPLEMENTED);
    }

//
// Step 1 & 2 - create renumbering
//
    this->createRenumbering(this->renumbering);
    //now we have the full renumbering table in renum, calculate the inverse
    this->inverse_renumbering.resize(this->renumbering.size());

    if (this->renumbering.size() > 1)
    {
        calc_inverse_renumbering <<< min(4096, ((int)this->renumbering.size() + 511) / 512), 512 >>> (this->renumbering.raw(), this->inverse_renumbering.raw(), this->renumbering.size());
        cudaCheckError();
    }

//
// Step 4 - calculate number/offset of nodes in the halos from the neighbors, ring by ring
//
    this->halo_offsets.resize(num_neighbors + 1);
    this->halo_offsets[0] = size;

    for (int i = 0; i < num_neighbors; i++)
    {
        this->halo_offsets[i + 1] = this->halo_offsets[i] + this->L2H_maps[i].size();
    }

    this->set_num_halo_rows(this->halo_offsets[this->halo_offsets.size() - 1] - size);
    int nh = this->num_halo_rows();
    int total_rows = size + nh;
    cudaCheckError();
//
// Step 6 - renumber halo matrices and calculate row length (to eventually append to the big matrix)
//
    int insert = 0;
    //recalculate row_offsets
    IVector new_row_offsets(size + 1);

    if (num_blocks > 0)
    {
        calc_rowlen_reorder <<< num_blocks, 512>>>(this->A->row_offsets.raw(), new_row_offsets.raw(), this->renumbering.raw(), size, insert);
        cudaCheckError();
    }

    thrust::copy(thrust::make_permutation_iterator(this->renumbering.begin(), this->A->col_indices.begin()),
                 thrust::make_permutation_iterator(this->renumbering.begin(), this->A->col_indices.end()),
                 this->A->col_indices.begin());
    cudaCheckError();
    //row_offsets array created by exclusive scan of row sizes
    thrust_wrapper::exclusive_scan(new_row_offsets.begin(), new_row_offsets.begin() + size + 1, new_row_offsets.begin());
    cudaCheckError();
//
// Step 7 - consolidate column indices and values
//
    int new_nnz = new_row_offsets[new_row_offsets.size() - 1];
    typedef typename MatPrecisionMap<t_matPrec>::Type ValueTypeA;
    VVector new_values((new_nnz + 1 )* this->A->get_block_size(), types::util<ValueTypeA>::get_zero());
    IVector new_col_indices(new_nnz, 0);

    //reorder based on row permutation
    if (num_blocks > 0)
    {
        reorder_whole_matrix <32> <<< num_blocks, 512>>>(this->A->row_offsets.raw(), this->A->col_indices.raw(), this->A->values.raw(), new_row_offsets.raw(), new_col_indices.raw(), new_values.raw(), this->renumbering.raw(), this->A->get_block_size(), size, insert);
        cudaCheckError();
    }

    //create and append halo rows size
    //create an identity matrix in CSR format
    int nnz = this->A->get_num_nz();
    IVector identity_csr_rows(nh + 1);
    IVector identity_csr_cols(nh);
    VVector identity_csr_vals(nh, types::util<ValueTypeA>::get_one()); //needs to be changed to MVector, but this definition is messed up in the header file (should fix later)
    thrust::sequence(identity_csr_rows.begin(), identity_csr_rows.end());
    thrust::sequence(identity_csr_cols.begin(), identity_csr_cols.end());
    /*for example, 2x2 identity_csr matrix is created:
      identity_csr_rows = {   0,   1,   2  }
      identity_csr_cols = {   0,   1 }
      identity_csr_vals = { 1.0, 1.0 } */
    //shift identity tmatrix by size = this->A->get_num_rows();
    thrust::transform(identity_csr_rows.begin(), identity_csr_rows.end(), thrust::constant_iterator<INDEX_TYPE>(nnz), identity_csr_rows.begin(), thrust::plus<INDEX_TYPE>());
    thrust::transform(identity_csr_cols.begin(), identity_csr_cols.end(), thrust::constant_iterator<INDEX_TYPE>(size), identity_csr_cols.begin(), thrust::plus<INDEX_TYPE>());
    /*for example, 2x2 identity_csr matrix is created:
      identity_csr_rows = {   0,   1,   2  }
      identity_csr_cols = {size, size+1 }
      identity_csr_vals = { 1.0, 1.0 } */
    /* WARNING: you must be very careful with the view you are setting (cuurently the view coming here by default is ALL = FULL). If
                - classical path is selected then the createOneRingHaloRows -> create_one_ring_halo_rows -> append_halo_rows
                routine will be called. It will overwrite the halo rows setup here (and will use view OWNED, which will ignore the
                halo rows setup here, to determine how the new halo rows should be placed).
                - aggregation path is selected then the extra rows setup here will be used in the R*A*P product, where (in order to match
                dimensions of R and P) it is assumed that (the local partition) matrix A is square, therefore it must be padded by identity
                rows at the bottom to compensate for the "extra" columns that are outside of the main square part. The old routines for the
                aggregation path do this padding at the end of the reorder_matrix routine below.  */
    //ViewType v = this->A->currentView();
    //this->A->setView(ALL);
    //Approach 1: use existing routine to append the identity matrix to the existing one
    //            (seems like too much overhead, also need identity matrix per neighbor)
    //DistributedArranger<TConfig_d> *prep = new DistributedArranger<TConfig_d>;
    //prep->append_halo_rows(this->A, identity_csr_rows, identity_csr_cols, identity_csr_vals);
    //delete prep;
    //Approach 2: custom for this routine
    new_row_offsets.resize(total_rows + 1);
    new_col_indices.resize(nnz + nh);
    new_values.resize(nnz + nh + 1); //extra 1 element stores zero at the end (to follow the original design)
    //new_values[nnz]=-1;        //marker to track the last element
    thrust::copy(identity_csr_rows.begin(), identity_csr_rows.end(), new_row_offsets.begin() + size );
    thrust::copy(identity_csr_cols.begin(), identity_csr_cols.end(), new_col_indices.begin() + nnz);
    thrust::copy(new_values.begin() + nnz,    new_values.begin() + nnz + 1, new_values.begin() + nnz + nh);
    thrust::copy(identity_csr_vals.begin(), identity_csr_vals.end(),  new_values.begin() + nnz);
    /* WARNING: see above. */
    this->A->set_num_cols(total_rows);
    this->A->set_num_rows(total_rows);
    this->A->col_indices.swap(new_col_indices);
    new_row_offsets.resize(total_rows + 1);
    this->A->row_offsets.swap(new_row_offsets);
    new_row_offsets.swap(this->old_row_offsets);
    this->A->values.swap(new_values);
    this->A->m_seq_offsets.resize(total_rows + 1);
    thrust::sequence(this->A->m_seq_offsets.begin(), this->A->m_seq_offsets.end());
    cudaCheckError();
    //TODO: only do this if AMG_Config matrix_halo_exchange!=2
    this->A->delProps(COO);
    if (!insert)
    {
        this->A->computeDiagonal();
    }

    this->set_initialized(this->A->row_offsets);
    this->A->setView(OWNED);
}



template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::reorder_matrix()
{
    int num_neighbors = this->neighbors.size();

    if (num_neighbors == 0) { return; }

    int size = this->A->get_num_rows();
    int num_blocks = min(4096, (size + 511) / 512);
    int rings = this->B2L_rings[0].size() - 1;
    this->set_num_halo_rings(rings);
    int diag = this->A->hasProps(DIAG);
    std::vector<Matrix<TConfig_d> > &halo_rows = *this->halo_rows;
    std::vector<DistributedManager<TConfig_d> > &halo_btl = *this->halo_btl;
    /*
     EXAMPLE
     The example matrix, on partition 1 arrives at this point with the following state:
     num_rings=2
     A.num_rows = 4; A.num_nz = 20
     A.row_offsets = [0 4 11 15 20]
     A.col_indices = [4 0 1 2
                    4 5 0 1 2 3 7
                    0 1 2 3
                    1 2 3 6 7]
     num_neighbors=2; neighbors = [0 2]
     B2L_rings[[0 2 4][0 2 4]] B2L_maps[[0 1| 2 3][1 3| 0 2]]
     L2H_maps (and halo_lists) [[4 5][6 7]]

     With the exchange halo rows:
     halo_btl[0] (received from neighbor ID 0)
     global_id = 0; base_index=0; index_range=6; B2L_rings[0] = [0 2 4]; B2L_maps[0] = [2 3| 0 1] L2H_maps = [4 5]
     halo_rows[0].row_offsets = [0 5 13 17 21]
     halo_rows[0].col_indices = [1 2 3 4 5
                      0 1 2 3 4 5 6 7
                      0 1 3 6
                      0 1 2 3]

     halo_btl[1] (received from neighbor ID 2)
     global_id = 2; base_index=0; index_range=8; B2L_rings[0] = [0 2 4]; B2L_maps[0] = [1 2| 0 3] L2H_maps = [6 7]
     halo_rows[1].row_offsets = [0 4 11 16 20]
     halo_rows[1].col_indices = [7 1 2 3
                      5 6 7 0 1 2 3
                      4 5 0 2 3
                      0 1 2 3]
     */
//
// Step 1 & 2 - create renumbering
//
    this->createRenumbering(this->renumbering);
    cudaCheckError();
    /*
     EXAMPLE
     this->renumbering = [1 2 0 3 4 5 6 7]
     B2L_maps = [[1 2| 0 3][2 3| 1 0]]
     L2H_maps = [[4 5][6 7]]
     */
//
// Step 3 - given a full renumbering of owned nodes, calculate inverse renumbering
//
    //now we have the full renumbering table in renum, calculate the inverse
    this->inverse_renumbering.resize(this->renumbering.size());
    calc_inverse_renumbering <<< min(4096, ((int)this->renumbering.size() + 511) / 512), 512 >>> (this->renumbering.raw(), this->inverse_renumbering.raw(), this->renumbering.size());
    cudaCheckError();
    /*
     EXAMPLE
     this->inverse_renumbering = [2 0 1 3 4 5 6 7]
     */
//
// Step 4 - calculate number/offset of nodes in the halos from the neighbors, ring by ring
//
    this->halo_offsets.resize(rings * num_neighbors + 1, 0);

    for (int ring = 0; ring < rings; ring++)
    {
        for (int i = 0; i < num_neighbors; i++)
        {
            this->halo_offsets[ring * num_neighbors + i] = halo_btl[i].B2L_rings[0][ring + 1] - halo_btl[i].B2L_rings[0][ring];
        }
    }

    thrust::exclusive_scan(this->halo_offsets.begin(), this->halo_offsets.end(), this->halo_offsets.begin(), size);
    cudaCheckError();
    this->set_num_halo_rows(this->halo_offsets[this->halo_offsets.size() - 1] - size);
    int total_rows = size + this->num_halo_rows();

    if (total_rows < this->renumbering.size()) { FatalError("total rows < renumbering.size(), send/recv maps should cover all matrix halo columns", AMGX_ERR_NOT_IMPLEMENTED); }

    if (total_rows > this->renumbering.size())
    {
        this->A->getResources()->warning("# owned nodes + # halo nodes > matrix columns: send/recv maps have some unreferences halo indices, they are not directly connected to our partition and therefore we won't compute them, please use 2-ring comms maps if you want to specify 2nd ring neighbors");
    }

    cudaCheckError();
    /*
     EXAMPLE
     halo_offsets [2 2 2 2]
     after exclusive scan: 4 + [0 2 4 6 8] = [4 6 8 10 12]
     num_halo_rows = 8, total_rows = 12
     */
//
// Step 5 - create big mapping table of all halo indices we received (this may use a little too much memory)
//
    //count number of fine rows of neighbors
    thrust::host_vector<INDEX_TYPE> neighbor_rows(num_neighbors + 1);
    int max_num_rows = 0;

    for (int i = 0; i < num_neighbors; i++)
    {
        neighbor_rows[i] = halo_btl[i].index_range();
        max_num_rows = max_num_rows > halo_rows[i].get_num_rows() ? max_num_rows : halo_rows[i].get_num_rows();
    }

    thrust::exclusive_scan(neighbor_rows.begin(), neighbor_rows.end(), neighbor_rows.begin());
    cudaCheckError();
    int total_rows_of_neighbors = neighbor_rows[num_neighbors];
    /*
     EXAMPLE
     neigbor_rows = [0 6 14]
     total_rows_of_neighbors = 14
     */
    IVector halo_mapping(total_rows_of_neighbors);
    thrust::fill(halo_mapping.begin(), halo_mapping.end(), -1);
    cudaCheckError();

    //ring by ring, neighbor by neighbor assign sequentially increasing numbers for halo nodes
    for (int ring = 0; ring < rings; ring++)
    {
        for (int i = 0; i < num_neighbors; i++)
        {
            int size = halo_btl[i].B2L_rings[0][ring + 1] - halo_btl[i].B2L_rings[0][ring];
            int num_blocks = min(4096, (size + 127) / 128);
            //This renumbering has to result in the same renumbering that comes out of L2H renumbering
            create_halo_mapping <<< num_blocks, 128>>>(halo_mapping.raw() + neighbor_rows[i],
                    halo_btl[i].B2L_maps[0].raw() + halo_btl[i].B2L_rings[0][ring],
                    halo_btl[i].base_index(), this->halo_offsets[ring * num_neighbors + i], size);
            cudaCheckError();
            /*
             EXAMPLE
             ring 0 neighbor 0 - halo_btl[0].B2L_maps[0] = [2 3| 0 1] halo_btl[0].L2H_maps = [4 5]
            halo_mapping = [-1 -1 4 5 -1 -1 |-1 -1 -1 -1 -1 -1 -1 -1]
            ring 0 neighbor 1 - halo_btl[1].B2L_maps[0] = [1 2| 0 3] halo_btl[1].L2H_maps = [6 7]
            halo_mapping = [-1 -1 4 5 -1 -1 |-1 6 7 -1 -1 -1 -1 -1]
            ring 1 neighbor 0 - halo_btl[0].B2L_maps[0] = [2 3| 0 1] halo_btl[0].L2H_maps = [4 5]
            halo_mapping = [8 9 4 5 -1 -1 |-1 6 7 -1 -1 -1 -1 -1]
            ring 1 neighbor 1 - halo_btl[1].B2L_maps[0] = [1 2| 0 3] halo_btl[1].L2H_maps = [6 7]
            halo_mapping = [8 9 4 5 -1 -1 |10 6 7 11 -1 -1 -1 -1]

             */
        }
    }

    cudaCheckError();

    for (int i = 0; i < num_neighbors; i++)
    {
        int size = halo_btl[i].L2H_maps[0].size();
        int num_blocks = min(4096, (size + 127) / 128);
        //Map the column indices of the halo rows that point back to boundary nodes
        apply_h2l2b_mapping <<< num_blocks, 128>>>(halo_mapping.raw() + neighbor_rows[i],
                halo_btl[i].L2H_maps[0].raw(),
                halo_btl[i].base_index(), this->B2L_maps[i].raw(), size);
        cudaCheckError();
        /*
         EXAMPLE
         neighbor 0 - mapping back to our own (boundary) indices
         halo_mapping = [8 9 4 5 1 2 |10 6 7 11 -1 -1 -1 -1]
         neighbor 1 - mapping back to our own (boundary) indices
        halo_mapping = [8 9 4 5 1 2 |10 6 7 11 -1 -1 2 3]
         */
    }

    cudaCheckError();
    /*
     EXAMPLE
     neighbor_rows = [0 6 14]
     halo_mapping = [8 9 4 5 1 2 |10 6 7 11 -1 -1 2 3]
     The first part (0-6) of halo_mapping gives a local index for all the indices that we want to know about in halo_btl[0]
     The second part (7-14) gives local indices for halo_btl[1], that is both halo ring there, and the column indices representing vertices in this partition's boundary.
     Note that it does not give indices (-1) for vertices 5 and 6 in neighbor 1 (ID 2), which are column indices connecting it to neighbor 0, hence the two halo regions are not connected
     */
//
// Step 6 - renumber halo matrices and calculate row length (to eventually append to the big matrix)
//
    int insert = 0;

    if (this->A->hasProps(DIAG) && insertDiagonals) { insert = 1; }

    diag = diag && !insertDiagonals;
    //recalculate row_offsets
    IVector new_row_offsets(size + this->num_halo_rows() + 1);
    calc_rowlen_reorder <<< num_blocks, 512>>>(this->A->row_offsets.raw(), new_row_offsets.raw(), this->renumbering.raw(), size, insert);
    cudaCheckError();
    IVector neighbor_rows_d(num_neighbors + 1);
    thrust::copy(neighbor_rows.begin(), neighbor_rows.end(), neighbor_rows_d.begin());
    cudaCheckError();
    /*
     EXAMPLE
     get row length according to renumbering
     new_row_offsets = [4 4 7 5 0 0 0 0 0 0 0 0 0]
     */
    //map column indices of my own matrix
    /*map_col_indices<4><<<num_blocks, 512>>>(this->A->row_offsets.raw(),
                                         this->A->col_indices.raw(),
                                         this->renumbering.raw(),
                                         this->halo_ranges.raw(),
                                         halo_mapping.raw(),
                                         neighbor_rows_d.raw(),
                                         this->base_index(), num_neighbors, size);*/
    thrust::copy(thrust::make_permutation_iterator(this->renumbering.begin(), this->A->col_indices.begin()),
                 thrust::make_permutation_iterator(this->renumbering.begin(), this->A->col_indices.end()),
                 this->A->col_indices.begin());
    cudaCheckError();
    /*
     EXAMPLE
     use this->renumbering = [1 2 0 3 4 5 6 7]
     to map old column indices to new column indices (i.e. according to interior - boundary - halo separation), but do not reshuffle them into their place yet
     A.col_indices = [4 0 1 2
                  4 5 0 1 2 3 7
                  0 1 2 3
                  1 2 3 6 7]
    becomes
    A.col_indices = [4 1 2 0
                  4 5 1 2 0 3 7
                  1 2 0 3
                  2 0 3 6 7]

     */
    cudaCheckError();
    IVector temp_row_len(max_num_rows);

    for (int i = 0; i < num_neighbors; i++)
    {
        //map column indices of halo matrices and count of nonzeros we will keep
        int size = halo_rows[i].get_num_rows();
        int num_blocks = min(4096, (size + 127) / 128);
        map_col_indices_and_count_rowlen<4> <<< num_blocks, 128, 128 * sizeof(INDEX_TYPE)>>>(
            halo_rows[i].row_offsets.raw(),
            halo_rows[i].col_indices.raw(),
            temp_row_len.raw(),
            halo_mapping.raw() + neighbor_rows[i],
            size, insert);
        cudaCheckError();

        //number of nonzeros per row copied into big row sizes array
        for (int ring = 0; ring < rings; ring++)
        {
            thrust::copy(temp_row_len.begin() + halo_btl[i].B2L_rings[0][ring], temp_row_len.begin() + halo_btl[i].B2L_rings[0][ring + 1], new_row_offsets.begin() + this->halo_offsets[ring * num_neighbors + i]);
        }

        cudaCheckError();
        /*
         EXAMPLE
         halo_mapping = [8 9 4 5 1 2 |10 6 7 11 -1 -1 2 3]
         look at halo row matrices, and halo_mapping, count column indices that do not map to -1 and map them to their new, local index
         halo_rows[0].col_indices = [1 2 3 4 5
                    0 1 2 3 4 5 6 7
                    0 1 3 6
                    0 1 2 3]
        becomes
        halo_rows[0].col_indices = [9 4 5 1 2
                    8 9 4 5 1 2 -1 -1
                    8 9 5 -1
                    8 9 4 5]
        with temp_row_len = [5 6 3 4] copied into new_row_offsets: [4 4 7 5| 5 6| 0 0| 3 4| 0 0 0]

        halo_rows[1].col_indices = [7 1 2 3
                    5 6 7 0 1 2 3
                    4 5 0 2 3
                    0 1 2 3]
        becomes
        halo_rows[1].col_indices = [3 6 7 11
                    -1 2 3 10 6 7 11
                    -1 -1 10 7 11
                    10 6 7 11]
        with temp_row_len = [4 6 3 4] copied into new_row_offsets: [4 4 7 5| 5 6| 4 6| 3 4| 3 4 0]
         */
    }

    cudaCheckError();
    //row_offsets array created by exclusive scan of row sizes
    thrust_wrapper::exclusive_scan(new_row_offsets.begin(), new_row_offsets.begin() + size + this->num_halo_rows() + 1, new_row_offsets.begin());
    cudaCheckError();
    /*
     EXAMPLE
     Exclusive scan to get new_row_offsets array:
     new_row_offsets = [0 4 8 15 20| 25 31| 35 41| 44 48| 51 55]
     */
//
// Step 7 - consolidate column indices and values
//
    int new_nnz = new_row_offsets[new_row_offsets.size() - 1];
    typedef typename MatPrecisionMap<t_matPrec>::Type ValueTypeA;
    VVector new_values((new_nnz + 1 + diag * (total_rows - 1))* this->A->get_block_size(), types::util<ValueTypeA>::get_zero());
    IVector new_col_indices(new_nnz, 0);
    //reorder based on row permutation
    reorder_whole_matrix <32> <<< num_blocks, 512>>>(this->A->row_offsets.raw(), this->A->col_indices.raw(), this->A->values.raw(), new_row_offsets.raw(), new_col_indices.raw(), new_values.raw(), this->renumbering.raw(), this->A->get_block_size(), size, insert);
    cudaCheckError();

    if (diag)
    {
        //reorder based on row permutation
        reorder_vector_values <<< num_blocks, 512>>>(new_values.raw() + new_row_offsets[total_rows]*this->A->get_block_size(),
                this->A->values.raw() + this->A->row_offsets[size]*this->A->get_block_size(),
                this->renumbering.raw(),
                this->A->get_block_size(), size);
        cudaCheckError();
    }

    int cumulative_num_rows = size;

    for (int i = 0; i < num_neighbors; i++)
    {
        for (int ring = 0; ring < rings; ring++)
        {
            int num_rows = halo_btl[i].B2L_rings[0][ring + 1] - halo_btl[i].B2L_rings[0][ring];
            int num_blocks = min(4096, (num_rows + 127) / 128);
            //copy in nonzeros that we are keeping
            //TODO: access pattern - should be implemented with warp-wide scans to decide which nonzeros we are keeping and where the rest is going
            reorder_whole_halo_matrix <<< num_blocks, 128>>>(halo_rows[i].row_offsets.raw() + halo_btl[i].B2L_rings[0][ring],
                    halo_rows[i].col_indices.raw(), halo_rows[i].values.raw(),
                    new_row_offsets.raw() + this->halo_offsets[ring * num_neighbors + i],
                    new_col_indices.raw(), new_values.raw(), NULL, this->A->get_block_size(), num_rows,
                    insert, this->halo_offsets[ring * num_neighbors + i], halo_btl[i].B2L_rings[0][ring], halo_btl[i].B2L_rings[0][rings]);

            if (diag)
            {
                thrust::copy(halo_rows[i].values.begin() + (halo_rows[i].row_offsets[halo_rows[i].get_num_rows()] + halo_btl[i].B2L_rings[0][ring])*this->A->get_block_size(),
                             halo_rows[i].values.begin() + (halo_rows[i].row_offsets[halo_rows[i].get_num_rows()] + halo_btl[i].B2L_rings[0][ring + 1])*this->A->get_block_size(),
                             new_values.begin() + (new_row_offsets[total_rows] + cumulative_num_rows)*this->A->get_block_size());
                cumulative_num_rows += num_rows;
            }
        }
    }

    cudaCheckError();
    /*
     EXAMPLE
     copy everything in place, dropping -1 column indices in the halo and reordering the owned rows
     new_row_offsets = [0 4 8 15 20| 25 31| 35 41| 44 48| 51 55]
     new_col_indices = [1 2 0 3
                  4 1 2 0
                  4 5 1 2 0 3 7
                  2 0 3 6 7 -end of owned
                  9 4 5 1 2
                  8 9 4 5 1 2 - end of neighbor 0 ring 0
                  3 6 7 11
                  2 3 10 6 7 11 - end of neighbor 1 ring 0
                  8 9 5
                  8 9 4 5 - end of neighbor 0 ring 1
                  10 7 11
                  10 6 7 11] - end of neighbor 1 ring 1

     */
    this->A->set_num_cols(total_rows);
    this->A->set_num_rows(size);
    this->A->col_indices.swap(new_col_indices);
    new_row_offsets.resize(total_rows + 1);
    this->A->row_offsets.swap(new_row_offsets);
    new_row_offsets.swap(this->old_row_offsets);
    this->A->values.swap(new_values);
    this->A->m_seq_offsets.resize(total_rows + 1);
    thrust::sequence(this->A->m_seq_offsets.begin(), this->A->m_seq_offsets.end());

    if (insert)
    {
        this->A->delProps(DIAG);
        this->A->diag.resize(total_rows);
        thrust::copy(this->A->row_offsets.begin(), this->A->row_offsets.end() - 1, this->A->diag.begin());
    }

    cudaCheckError();
    delete this->halo_rows;
    delete this->halo_btl;
    //set halo_rows and halo_btl to NULL to avoid a potential double free situation in the future
    this->halo_rows = NULL;
    this->halo_btl = NULL;
    this->A->delProps(COO);
    this->A->set_initialized(1);

    //TODO: only do this if AMG_Config matrix_halo_exchange!=2
    if (!insert) { this->A->computeDiagonal(); }

    this->A->setView(OWNED);
}

//function object (functor) for thrust calls (it is a unary operator to add a constant)
template<typename T>
class add_constant_op
{
        const T c;
    public:
        add_constant_op(T _c) : c(_c) {}
        __host__ __device__ T operator()(const T &x) const
        {
            return x + c;
        }
};

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::obtain_shift_l2g_reordering(index_type n, I64Vector_d &l2g, IVector_d &p, IVector_d &q)
{
    /* WARNING: Exchange halo of the inverse_reordering, which is implicitly based on the local_to_global_map (l2g).
                Notice that it is implicit in the exchange_halo routine, since you are getting exactly the vector
                halo elements, which are exactly the elements you need. They however must be shifted by the partition
                starting points (starting global row indices, which are containe din array part_offsets).
                This allows us to avoid constructing the global vector for inverse permutation,
                as is done in reference MATLAB code. */
    //Recall that part_offsets provide the starting point (global row index) of every partition, in other words,
    //they contain the prefix sum of number of rows assigned to each partition. Also, notice that part_offsets and
    //part_offsets_h have the same values on device and host, respectively. See below few lines for details:
    index_type tag = 1 * 133 + 3 * 7 + 0; //some random number for the tag
    index_type l = p.size();
    q.resize(l);
    thrust::copy     (p.begin(), p.end(),     q.begin());
    thrust::transform(q.begin(), q.end(),     q.begin(), add_constant_op<index_type>(this->part_offsets[this->global_id()]));
    this->exchange_halo(q, tag);
    thrust::sequence (q.begin(), q.begin() + n);
    thrust::transform(q.begin(), q.begin() + n, q.begin(), add_constant_op<index_type>(this->part_offsets[this->global_id()]));
    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::unpack_partition(index_type *Bp, index_type *Bc, mat_value_type *Bv)
{
    index_type l, n, nnz, offset;
    index_type     *ir;
    index_type     *Ap;
    index_type     *Ac;
    mat_value_type *Av;
    IVector q;
    //some initializations
    this->A->getOffsetAndSizeForView(OWNED, &offset, &n);
    this->A->getNnzForView(OWNED, &nnz);
    l  = this->inverse_renumbering.size();
    ir = this->inverse_renumbering.raw();
    Ap = this->A->row_offsets.raw();
    Ac = this->A->col_indices.raw();
    Av = this->A->values.raw();
    //(i) reorder the matrix back (into mixed interior-boundary nodes)
    //applies to rows and columns (out-of-place)
    reorder_partition<index_type, mat_value_type, true, true>
    (n, nnz, Ap, Ac, Av, Bp, Bc, Bv, l, ir);
    cudaCheckError();
    //obtain reordering q that combines the shift of the diagonal block with the off-diagonal block indices conversion from local to global
    this->obtain_shift_l2g_reordering(n, this->local_to_global_map, this->inverse_renumbering, q);
    cudaCheckError();
    //(ii) reorder the matrix back (shift the diagonal block and convert off-diagonal block column indices from local to global)
    //applies columns only (in-place)
    reorder_partition<index_type, mat_value_type, false, true>
    (n, nnz, Bp, Bc, Bv, Bp, Bc, Bv, q.size(), q.raw());
    cudaCheckError();
}

template <class TConfig>
void DistributedManagerBase<TConfig>::createNeighToDestPartMap(IVector_h &neigh_to_part, IVector_h &neighbors, IVector_h &destination_part, int num_neighbors)
{
    neigh_to_part.resize(num_neighbors);

    for (int i = 0; i < num_neighbors; i++)
    {
        neigh_to_part[i] = destination_part[neighbors[i]];
    }
}

template <class TConfig>
void DistributedManagerBase<TConfig>::read_halo_ids(int size, IVector_d &scratch, IVector_d &halo_aggregates, VecInt_t min_index_coarse_halo)
{
    int block_size = 128;
    const int num_blocks = min( AMGX_GRID_MAX_SIZE, (size - 1) / block_size + 1);
    read_halo_ids_kernel <<< num_blocks, block_size>>>(scratch.raw(), halo_aggregates.raw(), min_index_coarse_halo, size);
    cudaCheckError();
}

template <class TConfig>
void DistributedManagerBase<TConfig>::read_halo_ids(int size, IVector_h &scratch, IVector_h &halo_aggregates, VecInt_t min_index_coarse_halo)
{
    FatalError("read_halo_ids not implemented on host yet", AMGX_ERR_NOT_IMPLEMENTED);
}


template <class TConfig>
void DistributedManagerBase<TConfig>::flag_halo_ids(int size, IVector_d &scratch, IVector_d &halo_aggregates, VecInt_t min_index_coarse_halo,  int max_index, int min_index)
{
    int block_size = 128;
    const int num_blocks = min( AMGX_GRID_MAX_SIZE, (size - 1) / block_size + 1);
    flag_halo_ids_kernel <<< num_blocks, block_size>>>(scratch.raw(), halo_aggregates.raw(), min_index_coarse_halo, size, max_index - min_index + 1);
    cudaCheckError();
}

template <class TConfig>
void DistributedManagerBase<TConfig>::flag_halo_ids(int size, IVector_h &scratch, IVector_h &halo_aggregates, VecInt_t min_index_coarse_halo,  int max_index, int min_index)
{
    FatalError("flag_halo_ids not implemented on host yet", AMGX_ERR_NOT_IMPLEMENTED);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::replaceMatrixCoefficients(int n, int nnz, const mat_value_type *data_pinned, const mat_value_type *diag_pinned)
{
    //matrix parameters
    //int num_nnz  = this->A->get_num_nz();
    int num_rows   = this->halo_offsets[0];
    int total_rows = num_rows + this->num_halo_rows();
    int block_size = this->A->get_block_size();
    mat_value_type *data_hd = NULL;
    mat_value_type *diag_hd = NULL;
    int data_alloc = 0;
    int diag_alloc = 0;
    //cuda parameters
    int num_blocks = min(4096, (num_rows + 127) / 128);

    /* WARNING: the number of non-zero elements (nnz) in the array data_pinned and A->values (num_nnz) might be different at this point.
       1. If the matrix has CSR property and therefore diagonal is included in the matrix this values will be the same.
       2. If the matrix has DIAG property and therefore diagonal is originally stored separately, and later appended to the array
          of values, and subsequently inserted into the matrix than num_nnz = nnz + n. We have to account for this fact when replacing the
          coefficients (and use nnz not num_nnz).
       obs.: see calls to computeDiagonal (matrix.cu), AMGX_matrix_upload and AMGX_replace_coefficients (amgx_c.cu), and
          uploadMatrix and replaceMatrixCoefficients[No|With]Cons (distributed_manager.cu) for details. */

    /* check early exit */
    if (this->neighbors.size() == 0 || this->renumbering.size() == 0)
    {
        return;
    }

    cudaCheckError();
    /* allocate if data and diag if they are not pinned */
    data_hd = (mat_value_type *) this->getDevicePointerForData((void *)data_pinned, nnz * block_size * sizeof(mat_value_type), &data_alloc);

    if (diag_pinned != NULL)
    {
        diag_hd = (mat_value_type *) this->getDevicePointerForData((void *)diag_pinned, num_rows * block_size * sizeof(mat_value_type), &diag_alloc);
    }

    /* replace the values (reordering them if needed) */
    if (insertDiagonals && diag_pinned != NULL)
    {
        replace_values_matrix <32> <<< num_blocks, 512>>>(data_hd, diag_hd, this->old_row_offsets.raw(), this->A->row_offsets.raw(), this->A->values.raw(), this->renumbering.raw(), block_size, num_rows);
    }
    else
    {
        replace_values_matrix <32> <<< num_blocks, 512>>>(data_hd, this->old_row_offsets.raw(), this->A->row_offsets.raw(), this->A->values.raw(), this->renumbering.raw(), block_size, num_rows);

        if (diag_pinned != NULL)
        {
            reorder_vector_values <<< num_blocks, 512>>>(this->A->values.raw() + this->A->row_offsets[total_rows]*block_size, diag_hd, this->renumbering.raw(), block_size, num_rows);
        }
    }

    cudaCheckError();

    /* free memory (if needed) */
    if (data_alloc) { hipFree(data_hd); }
    if (diag_alloc) { hipFree(diag_hd); }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::transformAndUploadVector(VVector_v &v, const void *data, int n, int block_dim)
{
    v.resize(n * block_dim);
    cudaCheckError();
    // Upload on host
    hipMemcpy(v.raw(), (value_type *)data, n * block_dim * sizeof(value_type), hipMemcpyDefault);
    cudaCheckError();
    // Permute based on renumbering vector
    transformVector(v);
    int tag = 0;
    // Exchange halos
    this->exchange_halo(v, tag);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::transformVector(VVector_v &v)
{
    if (this->neighbors.size() == 0) { return; }
    else if (this->renumbering.size() == 0)
    {
        v.resize(this->halo_offsets[this->neighbors.size()]*v.get_block_size());
        return;
    }

    if (v.get_block_size() != this->A->get_block_dimx()) { printf("Blocksize mismatch!\n"); }

    if (v.size() < this->halo_offsets[this->neighbors.size()]*v.get_block_size())
    {
        VVector_v temp(this->halo_offsets[this->neighbors.size()]*v.get_block_size(), types::util<value_type>::get_zero());
        temp.set_block_dimx(v.get_block_dimx());
        temp.set_block_dimy(v.get_block_dimy());

        if (v.size() < this->halo_offsets[0]*this->A->get_block_dimx())
        {
            FatalError("Unknown size of input vector - smaller than the number of rows owned by this partition", AMGX_ERR_NOT_IMPLEMENTED);
        }

        //reorder based on row permutation
        int size = this->halo_offsets[0];
        int num_blocks = min(4096, (size + 511) / 512);
        reorder_vector_values <<< num_blocks, 512>>>(temp.raw(), v.raw(), this->renumbering.raw(), v.get_block_size(), size);
        v.swap(temp);
    }
    else
    {
        VVector_v temp(this->halo_offsets[0]*v.get_block_size());
        int size = this->halo_offsets[0];
        int num_blocks = min(4096, (size + 511) / 512);
        reorder_vector_values <<< num_blocks, 512>>>(temp.raw(), v.raw(), this->renumbering.raw(), v.get_block_size(), size);
        thrust::copy(temp.begin(), temp.end(), v.begin());
    }

    cudaCheckError();
    v.set_transformed();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::revertVector(VVector_v &v)
{
    if (this->neighbors.size() == 0 || this->renumbering.size() == 0) { return; }

    if (v.get_block_size() != this->A->get_block_dimx()) { printf("Blocksize mismatch!\n"); }

    VVector_v temp(this->halo_offsets[0]*this->A->get_block_dimx());

    if (v.size() < this->halo_offsets[0]*v.get_block_size())
    {
        FatalError("Unknown size of input vector - smaller than the number of rows owned by this partition", AMGX_ERR_NOT_IMPLEMENTED);
    }

    //reorder based on row permutation
    int size = this->halo_offsets[0];
    int num_blocks = min(4096, (size + 511) / 512);
    inverse_reorder_vector_values <<< num_blocks, 512>>>(temp.raw(), v.raw(), this->renumbering.raw(), v.get_block_size(), size);
    cudaCheckError();
    v.resize(this->halo_offsets[0]*this->A->get_block_dimx());
    thrust::copy(temp.begin(), temp.end(), v.begin());
    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::revertAndDownloadVector(VVector_v &v_in, const void *data, int n, int block_dimy)
{
    if ( n == 0 )
    {
        FatalError("Cannot download if size = 0", AMGX_ERR_NOT_IMPLEMENTED);
    }

    if (data == NULL )
    {
        FatalError("Cannot download to a NULL pointer", AMGX_ERR_NOT_IMPLEMENTED);
    }

    if (v_in.size() == 0 )
    {
        FatalError("Cannot download an empty vector", AMGX_ERR_NOT_IMPLEMENTED);
    }

    VVector_v v_out;
    revertVector(v_in, v_out);
    hipMemcpy((value_type *)data, v_out.raw(), n * block_dimy * sizeof(value_type), hipMemcpyDefault);
    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::revertVector(VVector_v &v_in, VVector_v &v_out)
{
    if (this->neighbors.size() == 0 || this->renumbering.size() == 0) { return;}

    if (v_in.get_block_size() != this->A->get_block_dimx()) { printf("Blocksize mismatch!\n"); }

    if (v_in.size() < this->halo_offsets[0]*v_in.get_block_size())
    {
        FatalError("Unknown size of input vector - smaller than the number of rows owned by this partition", AMGX_ERR_NOT_IMPLEMENTED);
    }

    int size = this->halo_offsets[0];

    if (v_out.size() != size * this->A->get_block_dimx())
    {
        v_out.resize(size * this->A->get_block_dimx());
    }

    //reorder based on row permutation
    int num_blocks = min(4096, (size + 511) / 512);
    inverse_reorder_vector_values <<< num_blocks, 512>>>(v_out.raw(), v_in.raw(), this->renumbering.raw(), v_in.get_block_size(), size);
    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::transformAndUploadVector(VVector_v &v, const void *data, int n, int block_dim)
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}


template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::transformVector(VVector_v &v)
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::revertAndDownloadVector(VVector_v &v, const void *data, int n, int block_dim)
{
    FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::revertVector(VVector_v &v_in, VVector_v &v_out)
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::revertVector(VVector_v &v)
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::reorder_matrix()
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::reorder_matrix_owned()
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::obtain_shift_l2g_reordering(index_type n, I64Vector_h &l2g, IVector_h &p, IVector_h &q)
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::unpack_partition(index_type *Bp, index_type *Bc, mat_value_type *Bv)
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::generatePoisson7pt(int nx, int ny, int nz, int P, int Q, int R)
{
    FatalError("GeneratePoisson7pt only implemented on devices", AMGX_ERR_NOT_IMPLEMENTED);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
template <typename t_colIndex>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::loadDistributedMatrix(
    int num_rows, int num_nonzeros, const int block_dimx, const int block_dimy, const int *row_offsets, 
    const t_colIndex *col_indices, const mat_value_type *values, int num_ranks, int num_rows_global, const void *diag, const MatrixDistribution &dist)
{
    FatalError("loadDistributedMatrix only implemented on devices", AMGX_ERR_NOT_IMPLEMENTED);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::createOneRingHaloRows()
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::replaceMatrixCoefficients(int n, int nnz, const mat_value_type *data, const mat_value_type *diag)
{
    FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void DistributedManager<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::createRenumbering(IVector &renumbering)
{
    if (this->neighbors.size() > 0)
    {
        FatalError("Distributed solve only supported on devices", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template < class TConfig >
void DistributedManagerBase<TConfig>::get_unassigned(IVector_h &flagArray, IVector_h &B2L_map, IVector_h &partition_flags, int size, int fa_size/*, int rank*/)
{
    for (int i = 0; i < size; i++)
    {
        if (B2L_map[i] < fa_size)
        {
            if (flagArray[B2L_map[i]] == 0)
            {
                flagArray[B2L_map[i]] = 1;
                partition_flags[i] = 1;
            }
        }
    }
}

template < class TConfig >
void DistributedManagerBase<TConfig>::set_unassigned(IVector_h &partition_flags, IVector_h &partition_renum, IVector_h &B2L_map, IVector_h &renumbering, int size, int max_element, int renum_size/*, int rank*/)
{
    for (int i = 0; i < size; i++)
    {
        if (B2L_map[i] < renum_size)
        {
            if (partition_flags[i] == 1)
            {
                renumbering[B2L_map[i]] = max_element + partition_renum[i];
            }

            B2L_map[i] = renumbering[B2L_map[i]];
        }
    }
}

/* print manager for target rank to a file or stdout */
template<class TConfig>
void DistributedManagerBase<TConfig>::print(char *f, char *s, int trank)
{
    DistributedManagerBase<TConfig> *m = this;
    int rank = 0;
    int level = 0;
    char filename[1024];
    FILE *fid = NULL;
    int i, j, k, t1, t2;
#ifdef AMGX_WITH_MPI
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif

    //check target rank
    if (rank == trank)
    {
        //check whether to output to stdout or a file
        if (f == NULL)
        {
            fid = stdout;
        }
        else
        {
            level = m->A->amg_level_index;
#ifdef _WIN32
            _snprintf_s(filename, 1024, 1024, "%s_r%d_l%d.mtx", f, rank, level);
#else
            snprintf(filename, 1024, "%s_r%d_l%d.mtx", f, rank, level);
#endif
            fid = fopen(filename, "w");
        }

        hipDeviceSynchronize();
        cudaCheckError();
        fprintf(fid, "%s\n", s);
        //--- communication info ---
        //compare neighbors
        t1 = m->neighbors.size();
        fprintf(fid, "neighbors %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->neighbors[i];
            fprintf(fid, "%d\n", k);
        }

        //compare B2L_rings
        t1 = B2L_rings.size();
        fprintf(fid, "B2L_rings %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            t2 = m->B2L_rings[i].size();
            fprintf(fid, "B2L_rings-%d [%d]\n", i, t2);

            for (j = 0; j < t2; j++)
            {
                k = m->B2L_rings[i][j];
                fprintf(fid, "%d\n", k);
            }
        }

        //compare B2L_maps
        t1 = B2L_maps.size();
        fprintf(fid, "B2L_maps %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            t2 = m->B2L_maps[i].size();
            fprintf(fid, "B2L_maps-%d [%d]\n", i, t2);

            for (j = 0; j < t2; j++)
            {
                k = m->B2L_maps[i][j];
                fprintf(fid, "%d\n", k);
            }
        }

        //compare L2H_maps
        t1 = L2H_maps.size();
        fprintf(fid, "L2H_maps %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            t2 = m->L2H_maps[i].size();
            fprintf(fid, "L2H_maps-%d [%d]\n", i, t2);

            for (j = 0; j < t2; j++)
            {
                k = m->L2H_maps[i][j];
                fprintf(fid, "%d\n", k);
            }
        }

        //--- matrix info ---
        fprintf(fid, "num_rows_global=%ld\n", num_rows_global);
        fprintf(fid, "_num_rows_interior=%d\n", m->_num_rows_interior);
        fprintf(fid, "_num_rows_owned=%d\n", m->_num_rows_owned);
        fprintf(fid, "_num_rows_full=%d\n", m->_num_rows_full);
        fprintf(fid, "_num_rows_all=%d\n", m->_num_rows_all);
        fprintf(fid, "_num_nz_interior=%d\n", m->_num_nz_interior);
        fprintf(fid, "_num_nz_owned=%d\n", m->_num_nz_owned);
        fprintf(fid, "_num_nz_full=%d\n", m->_num_nz_full);
        fprintf(fid, "_num_nz_all=%d\n", m->_num_nz_all);
        //compare # halo rows and halo offsets
        fprintf(fid, "# halo rings %d and rows %d\n", m->num_halo_rings(), m->num_halo_rows());
        t1 = m->halo_offsets.size();
        fprintf(fid, "halo_offsets %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->halo_offsets[i];
            fprintf(fid, "%d\n", k);
        }

        //compare halo ranges
        t1 = m->halo_ranges.size();
        fprintf(fid, "halo_ranges %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->halo_ranges[i];
            fprintf(fid, "%d\n", k);
        }

        //compare halo ranges (host)
        t1 = m->halo_ranges_h.size();
        fprintf(fid, "halo_ranges_h %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->halo_ranges_h[i];
            fprintf(fid, "%d\n", k);
        }

        //compare part offsets
        t1 = m->part_offsets.size();
        fprintf(fid, "part_offsets %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->part_offsets[i];
            fprintf(fid, "%d\n", k);
        }

        //compare part offsets (host)
        t1 = m->part_offsets_h.size();
        fprintf(fid, "part_offsets_h %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->part_offsets_h[i];
            fprintf(fid, "%d\n", k);
        }

        //compare interior row list
        t1 = m->interior_rows_list.size();
        fprintf(fid, "interior_rows_list %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->interior_rows_list[i];
            fprintf(fid, "%d\n", k);
        }

        //compare boundary row list
        t1 = m->boundary_rows_list.size();
        fprintf(fid, "boundary_rows_list %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->boundary_rows_list[i];
            fprintf(fid, "%d\n", k);
        }

        //compare halo1 row list
        t1 = m->halo1_rows_list.size();
        fprintf(fid, "halo1_rows_list %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->halo1_rows_list[i];
            fprintf(fid, "%d\n", k);
        }

        fprintf(fid, "pointers halo_rows=%p and halo_btl=%p\n", m->halo_rows, m->halo_btl);
        //--- packing info ---
        //compare local to global map
        t1 = m->local_to_global_map.size();
        fprintf(fid, "local_to_global_map %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->local_to_global_map[i];
            fprintf(fid, "%d\n", k);
        }

        //compare renumbering
        t1 = m->renumbering.size();
        fprintf(fid, "renumbering %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->renumbering[i];
            fprintf(fid, "%d\n", k);
        }

        //compare inverse renumbering
        t1 = m->inverse_renumbering.size();
        fprintf(fid, "inverse_renumbering %d\n", t1);

        for (i = 0; i < t1; i++)
        {
            k = m->inverse_renumbering[i];
            fprintf(fid, "%d\n", k);
        }

        //--- GPU related and miscellaneous info
        //streams
        fprintf(fid, "streams i=%p, b=%p\n", m->get_int_stream(), m->get_bdy_stream());
        //miscellaneous info
        int64_t bi = m->base_index();       //inlined function
        int np = m->get_num_partitions();   //inlined function
        fprintf(fid, "gid=%d,bi=%ld,np=%d,ir=%d,in=%d,bn=%d\n", m->global_id(), bi, np, m->index_range(), m->num_interior_nodes(), m->num_boundary_nodes());
        hipDeviceSynchronize();
        hipGetLastError();

        if (fid != stdout)
        {
            fclose(fid);
        }
    }
}

/* print manager for target rank to a file or stdout (for all ranks) */
template<class TConfig>
void DistributedManagerBase<TConfig>::printToFile(char *f, char *s)
{
    DistributedManagerBase<TConfig> *m = this;
    int rank = 0;
#ifdef AMGX_WITH_MPI
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
#endif
    //notice that print will be called with different (target) rank on different ranks/processes
    m->print(f, s, rank);
}

/* compare two managers */
template<class TConfig>
int DistributedManagerBase<TConfig>::compare(DistributedManagerBase<TConfig> *m2)
{
    DistributedManagerBase<TConfig> *m1 = this;
    int i, j, t1, t2;
    //compare neighbors
    t1 = m1->neighbors.size();
    t2 = m2->neighbors.size();

    if (t1 != t2)
    {
        return 1;
    }

    for (i = 0; i < t1; i++)
    {
        if (m1->neighbors[i] != m2->neighbors[i])
        {
            return 2;
        }
    }

    //compare B2L_rings
    for (i = 0; i < (m1->neighbors.size()); i++)
    {
        t1 = m1->B2L_rings[i].size();
        t2 = m2->B2L_rings[i].size();

        if (t1 != t2)
        {
            return 3;
        }

        for (j = 0; j < t1; j++)
        {
            if (m1->B2L_rings[i][j] != m2->B2L_rings[i][j])
            {
                return 4;
            }
        }
    }

    //compare B2L_maps
    t1 = m1->B2L_maps.size();
    t2 = m2->B2L_maps.size();

    if (t1 != t2)
    {
        return 5;
    }

    for (i = 0; i < t1; i++)
    {
        if (m1->B2L_maps[i] != m2->B2L_maps[i])
        {
            return 6;
        }
    }

    //compare L2H_maps
    t1 = m1->L2H_maps.size();
    t2 = m2->L2H_maps.size();

    if (t1 != t2)
    {
        return 7;
    }

    for (i = 0; i < t1; i++)
    {
        if (m1->L2H_maps[i] != m2->L2H_maps[i])
        {
            return 8;
        }
    }

    return 0;
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
DistributedManager< TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::~DistributedManager< TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >()
{
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
DistributedManager< TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::~DistributedManager< TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >()
{
}

/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class DistributedManager<TemplateMode<CASE>::Type >;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

#define AMGX_CASE_LINE(CASE) template void DistributedManager<TemplateMode<CASE>::Type>::loadDistributedMatrix( \
    int, int, const int, const int, const int*, const int *col_indices, const mat_value_type*, int, int, const void*, const MatrixDistribution &dist); \
    template void DistributedManager<TemplateMode<CASE>::Type>::loadDistributedMatrix( \
    int, int, const int, const int, const int*, const int64_t *col_indices, const mat_value_type*, int, int, const void*, const MatrixDistribution &dist);
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

#define AMGX_CASE_LINE(CASE) template class DistributedManagerBase<TemplateMode<CASE>::Type >;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

} // namespace amgx

