#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2011 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include <matrix_io.h>
#include "misc.h"
#include "util.h"
#include <string>
#include <iostream>

#ifdef _WIN32
#pragma warning (push)
#pragma warning (disable : 4244 4267 4521)
#endif
#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#ifdef _WIN32
#pragma warning (pop)
#endif

#include <types.h>
#include <iomanip>
#include <map>
#include <basic_types.h>
#include <matrix.h>
#include <amgx_timer.h>

#include "amgx_types/util.h"
#include "amgx_types/io.h"

namespace amgx
{

template<class T_Config>
typename MatrixIO<T_Config>::readerMap &MatrixIO<T_Config>::getReaderMap()
{
    static readerMap readers_map;
    return readers_map;
}

template<class T_Config>
void MatrixIO<T_Config>::registerReader(std::string key, readerFunc func)
{
    readerMap &readers_map = getReaderMap();
    typename readerMap::const_iterator iter = readers_map.find(key);

    if (iter != readers_map.end())
    {
        std::string err = "Reader '" + key + "' is already registered";
        FatalError(err, AMGX_ERR_CORE);
    }

    readers_map[key] = func;
}

template<class T_Config>
void MatrixIO<T_Config>::unregisterReaders()
{
    readerMap &readers_map = getReaderMap();
    readers_map.clear();
}

template<class T_Config>
typename MatrixIO<T_Config>::writerMap &MatrixIO<T_Config>::getWriterMap()
{
    static writerMap writer_map;
    return writer_map;
}

template<class T_Config>
void MatrixIO<T_Config>::registerWriter(std::string key, writerFunc func)
{
    writerMap &writer_map = getWriterMap();
    typename writerMap::const_iterator iter = writer_map.find(key);

    if (iter != writer_map.end())
    {
        std::string err = "Reader '" + key + "' is already registered";
        FatalError(err, AMGX_ERR_CORE);
    }

    writer_map[key] = func;
}

template<class T_Config>
void MatrixIO<T_Config>::unregisterWriters()
{
    writerMap &writer_map = getWriterMap();
    writer_map.clear();
}

template<class T_Config>
bool MatrixIO<T_Config>::writeSystemMatrixMarket(const char *fname, const Matrix<T_Config> *pA, const VVector *pb, const VVector *px)
{
    typedef typename T_Config::MatPrec ValueTypeA;
    typedef typename T_Config::VecPrec ValueTypeB;

    if (!fname)
    {
        FatalError( "Bad filename", AMGX_ERR_BAD_PARAMETERS);
    }

    if (!pA)
    {
        FatalError( "MatrixMarket should contain matrix", AMGX_ERR_BAD_PARAMETERS);
    }

    std::ofstream fout;
    std::string err = "Writing system to file " + std::string(fname) + "\n";
    amgx_output(err.c_str(), err.length());
    fout.open(fname);

    if (!fout)
    {
        FatalError( "Cannot open file for writing!", AMGX_ERR_BAD_PARAMETERS);
    }

    const Matrix<T_Config> &A = *pA;
    bool is_mtx = true;
    bool is_rhs = pb != NULL && pb->size() > 0;
    bool is_soln = px != NULL && px->size() > 0;
    fout << "%%MatrixMarket";

    if (is_mtx)
    {
        fout << " matrix coordinate ";

        if (types::util<typename Matrix<T_Config>::value_type>::is_real)
        {
            fout << "real ";
        }
        else
        {
            fout << "complex ";
        }

        fout << "general";
    }
    else
    {
        if (types::util<typename Matrix<T_Config>::value_type>::is_real)
        {
            fout << "real ";
        }
        else
        {
            fout << "complex ";
        }
    }

    fout << std::endl;
    fout << "%%NVAMG " << A.get_block_dimx() << " " << A.get_block_dimy() << " ";

    if (A.hasProps(DIAG) && is_mtx) { fout << "diagonal "; }

    // Note: we don't actaully have any guarantee that columns are sorted here
    // if (is_mtx) { fout << "sorted "; }

    if (is_rhs) { fout << "rhs "; }

    if (is_soln) { fout << "solution"; }

    fout << std::endl;
    fout << A.get_num_rows()*A.get_block_dimx()  << " " << A.get_num_cols()*A.get_block_dimy() << " " << A.get_num_nz()*A.get_block_size() <<  std::endl;
    // rules are simple: If there is csr property - write csr and coo (if exists). Else write coo.
    fout << std::setprecision(std::numeric_limits<ValueTypeA>::digits10 + 1) << std::scientific;

    if (is_mtx)
    {
        if (A.hasProps(COO))
        {
            for (int i = 0; i < A.get_num_nz(); i++)
            {
                for (int kx = 0; kx < A.get_block_dimx(); kx++)
                    for (int ky = 0; ky < A.get_block_dimy(); ky++)
                    {
                        fout << A.row_indices[i]*A.get_block_dimx() + kx + 1 << " " << A.col_indices[i]*A.get_block_dimy() + ky + 1 << " " << A.values[i * A.get_block_size() + kx * A.get_block_dimy() + ky] << std::endl;
                    }
            }
        }
        else if (A.hasProps(CSR))
        {
            typename Matrix<T_Config>::IVector_h A_row_offsets;
            typename Matrix<T_Config>::IVector_h A_col_indices;
            typename Matrix<T_Config>::MVector_h A_values;

            A_row_offsets.resize(A.row_offsets.size());
            A_row_offsets = A.row_offsets;

            A_col_indices.resize(A.col_indices.size());
            A_col_indices = A.col_indices;

            A_values.resize(A.values.size());
            A_values = A.values;

            int interval = max(10, A.get_num_rows() / 100);

            for (int i = 0; i < A.get_num_rows(); i++)
            {
                if (i%interval == 0)
                {
                    float perc = ceil( (double(i)/A.get_num_rows()) * 100 * 100 ) / 100.0;
                    std::cout << perc << " % done " << std::endl;
                }
                for (int j = A_row_offsets[i]; j < A_row_offsets[i + 1]; j++)
                {
                    int c = A_col_indices[j];

                    //      typename Matrix::value_type v=A.values[j];
                    for (int kx = 0; kx < A.get_block_dimx(); kx++)
                        for (int ky = 0; ky < A.get_block_dimy(); ky++)
                        {
                            fout << i *A.get_block_dimx() + kx + 1 << " " << c *A.get_block_dimy() + ky + 1 << " " << A_values[j * A.get_block_size() + kx * A.get_block_dimy() + ky] << std::endl;
                        }
                }
            }
        }

        if (A.hasProps(DIAG))
        {
            for (int i = 0; i < A.get_num_rows(); i++)
            {
                for (int k = 0; k < A.get_block_size(); k++)
                {
                    fout << A.values[A.diag[i]*A.get_block_size() + k] << " ";
                }

                fout << std::endl;
            }
        }
    } // End of writing matrix

    fout << std::setprecision(std::numeric_limits<ValueTypeB>::digits10 + 1) << std::scientific;

    //write rhs
    if (is_rhs)
    {
        const VVector &b = *pb;
        fout << b.size() << std::endl;

        for (int i = 0; i < b.size(); i++)
        {
            fout << b[i] << std::endl;
        }
    }

    // write initial guess if we have it
    if (is_soln)
    {
        const VVector &x = *px;
        fout << x.size() << std::endl;

        for (int i = 0; i < x.size(); i++)
        {
            fout << x[i] << std::endl;
        }
    }

    fout.close();
    err = "Done writing system to file!\n";
    amgx_output(err.c_str(), err.length());
    return true;
}


template<class T_Config>
bool MatrixIO<T_Config>::writeSystemBinary(const char *fname, const Matrix<T_Config> *pA, const VVector *pb, const VVector *px)
{
    typedef typename T_Config::template setMemSpace<AMGX_host>::Type TConfig_h;
    typedef typename TConfig_h::template setVecPrec<AMGX_vecInt>::Type ivec_value_type_h;
    typedef Vector<ivec_value_type_h> IVector_h;
    typedef Vector<TConfig_h> VVector_h;
    typedef typename Matrix<TConfig_h>::MVector MVector_h;
    typedef typename Matrix<T_Config>::index_type IndexType;
    typedef typename Matrix<T_Config>::value_type ValueTypeA;
    typedef typename Vector<T_Config>::value_type ValueTypeB; // change back to matrix type later
    typedef typename types::util<ValueTypeA>::uptype UpValueType;

    if (!fname)
    {
        FatalError( "Bad filename", AMGX_ERR_BAD_PARAMETERS);
    }

    if (!pA)
    {
        FatalError( "MatrixMarket should contain matrix", AMGX_ERR_BAD_PARAMETERS);
    }

    FILE *fout;
    const char header [] = "%%NVAMGBinary\n";
    std::string err = "Writing system to file " + std::string(fname) + "\n";
    amgx_output(err.c_str(), err.length());
    fout = fopen(fname, "wb");

    if (!fout)
    {
        FatalError( "Cannot open output file!11", AMGX_ERR_BAD_PARAMETERS);
    }

    bool is_mtx = true;
    bool is_rhs = pb != NULL && pb->size() > 0;
    bool is_soln = px != NULL && px->size() > 0;
    const Matrix<T_Config> &A = *pA;
    uint32_t matrix_format = 42;

    if (A.hasProps(CSR))
    {
        matrix_format = 0;
    }
    else if (A.hasProps(COO))
    {
        matrix_format = 1;
    }
    else
    {
        FatalError("Unsupported matrix format", AMGX_ERR_BAD_PARAMETERS);
    }

    if (types::util<ValueTypeA>::is_complex)
    {
        matrix_format += COMPLEX;
    }

    const int system_header_size = 9;
    uint32_t system_flags [] = { (uint32_t)(is_mtx), (uint32_t)(is_rhs), (uint32_t)(is_soln), matrix_format, (uint32_t)(A.hasProps(DIAG)),
                                 (uint32_t)(A.get_block_dimx()), (uint32_t)(A.get_block_dimy()), (uint32_t)(A.get_num_rows()), (uint32_t)(A.get_num_nz())
                               };
    fwrite(header, sizeof(char), strlen(header), fout);
    fwrite(system_flags, sizeof(uint32_t), system_header_size, fout);
    std::vector< ValueTypeA > tempVA(A.values.size());
    std::vector< UpValueType > tempv(A.values.size());
    thrust::copy(A.values.begin(), A.values.end(), tempVA.begin());

    if (is_mtx)
    {
        if (A.hasProps(CSR))
        {
            IVector_h t_int = A.row_offsets;
            fwrite(t_int.raw(), sizeof(int), A.get_num_rows() + 1, fout); //assuming int as an index
            t_int = A.col_indices;
            fwrite(t_int.raw(), sizeof(int), A.get_num_nz(), fout); //assuming int as an index

            for (int k = 0; k < A.values.size(); k++)
            {
                types::util<ValueTypeA>::to_uptype(tempVA[k], tempv[k]);
            }

            fwrite(&tempv[0], sizeof(UpValueType), A.get_block_dimx() * A.get_block_dimy() * (A.get_num_nz() + (A.hasProps(DIAG) ? A.get_num_rows() : 0) ), fout); // including diag in the end if exists.
        }
        else
        {
            FatalError("Unsupported matrix format for now", AMGX_ERR_IO);
        }
    } // End of writing matrix

    VVector_h tvec;

    //write rhs
    if (is_rhs)
    {
        if (pb->size() != A.get_num_rows()*A.get_block_dimy())
        {
            FatalError("rhs vector and matrix dimension does not match", AMGX_ERR_BAD_PARAMETERS);
        }

        std::vector< ValueTypeB > tempvB(pb->size());
        thrust::copy(&(*pb)[0], &(*pb)[pb->size()-1], tempvB.begin());

        tempv.resize(A.get_num_rows()*A.get_block_dimy());

        for (int k = 0; k < pb->size(); k++)
        {
            types::util<ValueTypeB>::to_uptype(tempvB[k], tempv[k]);
        }

        fwrite(&tempv[0], sizeof(UpValueType), pb->size(), fout);
    }

    // write initial guess if we have it
    if (is_soln)
    {
        if (px->size() != A.get_num_rows()*A.get_block_dimx())
        {
            FatalError("solution vector and matrix dimension does not match", AMGX_ERR_BAD_PARAMETERS);
        }

        std::vector< ValueTypeB > tempvB(px->size());
        thrust::copy(&(*px)[0], &(*px)[px->size()-1], tempvB.begin());

        tempv.resize(A.get_num_rows()*A.get_block_dimy());

        for (int k = 0; k < px->size(); k++)
        {
            types::util<ValueTypeB>::to_uptype(tempvB[k], tempv[k]);
        }

        fwrite(&tempv[0], sizeof(UpValueType), px->size(), fout);
    }

    fclose(fout);
    err = "Done writing system to file!\n";
    amgx_output(err.c_str(), err.length());
    return true;
}


template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::readSystem(const char *fname
        , Matrix<T_Config> &A
        , VVector &b
        , VVector &x
        , const AMG_Config &cfg
        , unsigned int props
        , const IVector_h &rank_rows // row indices for given rank
                                         )
{
    AMGX_CPU_PROFILER( "MatrixIO::read_sytem " );

    try
    {
        readerMap &readers_map = getReaderMap();
        //open file
        std::string err;

        if (io_config::hasProps(io_config::SIZE, props))
        {
            err = "Reading matrix dimensions in file: " + std::string(fname) + "\n";
        }
        else if (io_config::hasProps(io_config::PRINT, props))
        {
            err = "Reading matrix in file: " + std::string(fname) + "\n";
        }

        amgx_output(err.c_str(), err.length());
        std::ifstream fin(fname);

        if (!fin)
        {
            err = "Error opening file '" + std::string(fname) + "'\n";
            FatalError(err.c_str(), AMGX_ERR_IO);
        }

        // Extract the file format from the file
        std::string fformat;
        fin >> fformat;

        if (fformat.substr(0, 2) != "%%")
        {
            err = "Invalid header line in file " + std::string(fname) + " First line should begin with: %%MatrixFormat\n";
            FatalError(err.c_str(), AMGX_ERR_IO);
        }
        else
        {
            fformat = fformat.substr(2, fformat.size());
        }

        typename readerMap::const_iterator iter = readers_map.find(fformat);

        if (iter == readers_map.end())
        {
            err = "Could not find a reader for matrix of type '" + fformat + "'\n";
            FatalError(err.c_str(), AMGX_ERR_IO);
        }

        //call reader
        A.set_initialized(0);
        (iter->second)(fin
                       , fname
                       , A
                       , b
                       , x
                       , cfg
                       , props
                       , rank_rows
                      );
        A.computeDiagonal();
        A.set_initialized(1);
        fin.close();
    }
    catch (amgx_exception e)
    {
        std::string err = "Error while reading matrix: ";
        amgx_output(err.c_str(), err.length());
        amgx_output(e.what(), strlen(e.what()));
        return AMGX_ERR_IO;
    }

    return AMGX_OK;
}

template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::writeSystem (const char *filename, const Matrix<T_Config> *A, const VVector *b, const VVector *x)
{
    std::string format;

    try
    {
        AMG_Config *cfg = NULL;

        if (A)
        {
            cfg = A->getResources()->getResourcesConfig();
        }

        if (b)
        {
            cfg = b->getResources()->getResourcesConfig();
        }

        if (!cfg)
        {
            FatalError("Couldn't get resources from matrix or vector", AMGX_ERR_BAD_PARAMETERS);
        }

        format = cfg->AMG_Config::template getParameter<std::string>("matrix_writer", "default");
    }
    catch (amgx_exception e)
    {
        std::string err = "Error while writing matrix: ";
        amgx_output(err.c_str(), err.length());
        amgx_output(e.what(), strlen(e.what()));
        return AMGX_ERR_IO;
    }

    // call to actual writeMatrixWithFormat:
    return writeSystemWithFormat (filename, format.c_str(), A, b, x);
}

template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::writeSystemWithFormat (const char *filename, const char *format, const Matrix<T_Config> *A, const VVector *b, const VVector *x)
{
    AMGX_CPU_PROFILER( "MatrixIO::sytem " );

    try
    {
        writerMap &writers_map = getWriterMap();
        typename writerMap::const_iterator iter = writers_map.find(format);

        if (iter == writers_map.end())
        {
            std::string err;
            err = "Could not find a writer: '" + std::string(format) + "'\n";
            FatalError(err.c_str(), AMGX_ERR_IO);
        }

        if ( !(iter->second)( filename, A, b, x ) )
        {
            return AMGX_ERR_IO;
        }
    }
    catch (amgx_exception e)
    {
        std::string err = "Error while writing matrix: ";
        amgx_output(err.c_str(), err.length());
        amgx_output(e.what(), strlen(e.what()));
        return AMGX_ERR_IO;
    }

    return AMGX_OK;
}



template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::readSystem(const char *fname
        , Matrix<T_Config> &A
        , const AMG_Config &cfg
        , unsigned int props
        , const IVector_h &rank_rows // row indices for given rank
                                         )
{
    VVector b = VVector(0);
    VVector x = VVector(0);
    return readSystem(fname, A, b, x, cfg, props, rank_rows);
}

template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::readSystem(const char *fname
        , Matrix<T_Config> &A
        , VVector &b
        , const AMG_Config &cfg
        , unsigned int props
        , const IVector_h &rank_rows // row indices for given rank
                                         )
{
    VVector v = VVector(0);

    if (io_config::hasProps(io_config::RHS, props))
    {
        return readSystem(fname, A, b, v, cfg, props, rank_rows);
    }
    else
    {
        return readSystem(fname, A, v, b, cfg, props, rank_rows);
    }
}

template<class T_Config>
std::string MatrixIO<T_Config>::readSystemFormat(const char *fname)
{
    readerMap &readers_map = getReaderMap();
    //open file
    std::string out = "Reading matrix format in file: " + std::string(fname) + "\n";
    amgx_output(out.c_str(), out.length());
    std::ifstream fin(fname);

    if (!fin)
    {
        out = "Error opening file: " + std::string(fname) + "\n";
        FatalError(out.c_str(), AMGX_ERR_IO);
    }

    // Extract the file format from the file
    std::string fformat;
    fin >> fformat;

    if (fformat.substr(0, 2) != "%%")
    {
        out = "Invalid header line in file " + std::string( fname ) + " First line should begin with: %%MatrixFormat\n";
        FatalError(out.c_str(), AMGX_ERR_IO);
    }
    else
    {
        fformat = fformat.substr(2, fformat.size());
    }

    return fformat;
}

/*template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::readGeometry( AuxData* obj, const char* fname)
{
  std::string err;
  err = "Reading matrix in file: " + std::string(fname) + "\n";
  amgx_output(err.c_str(), err.length());

  std::ifstream fin(fname);
  if(!fin) {
    err = "Error opening file '" + std::string(fname) + "'\n";
      FatalError(err.c_str(), AMGX_ERR_IO);
  }

  int n,dimension;
  fin >> n >> dimension;

  MVector_h hgeo_x;
  MVector_h hgeo_y;
  MVector* geo_x = new MVector;
  MVector* geo_y = new MVector;
  hgeo_x.resize(n);
  hgeo_y.resize(n);

  if (dimension == 3)
  {
    MVector_h hgeo_z;
    MVector* geo_z = new MVector;
    hgeo_z.resize(n);
    for(int i = 0;i < n;i ++)
        fin >> hgeo_x[i] >> hgeo_y[i] >> hgeo_z[i];
    *geo_z = hgeo_z;
    obj->setParameterPtr< MVector > ("geo.z", geo_z);
  }
  else if (dimension == 2)
  {
    for(int i = 0;i < n;i ++)
        fin >> hgeo_x[i] >> hgeo_y[i];
  }

  obj->setParameter<int>("dim", dimension);
  obj->setParameter<int>("geo_size",(int)(hgeo_x.size()));
  *geo_x = hgeo_x;
  *geo_y = hgeo_y;
  obj->setParameterPtr< MVector > ("geo.x", geo_x);
  obj->setParameterPtr< MVector > ("geo.y", geo_y);

return AMGX_OK;
}

template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::readColoring( AuxData* obj, const char* fname)
{
  std::string err;
  err = "Reading matrix in file: " + std::string(fname) + "\n";
  amgx_output(err.c_str(), err.length());

  std::ifstream fin(fname);
  if(!fin) {
    err = "Error opening file '" + std::string(fname) + "'\n";
      FatalError(err.c_str(), AMGX_ERR_IO);
  }

  int num_rows, num_colors;
  fin >> num_rows >> num_colors;

  typedef TemplateConfig<AMGX_host, T_Config::vecPrec, T_Config::matPrec, T_Config::indPrec> TConfig_h;
  typedef typename Matrix<TConfig_h>::IVector IVector_h;
  IVector_h* row_coloring = new IVector_h;

  row_coloring->resize(num_rows);

  for(int i = 0;i < num_rows;i ++)
      fin >> (*row_coloring)[i];

  obj->setParameter<int>("coloring_size", num_rows);
  obj->setParameter<int>("colors_num", num_colors);
  obj->setParameterPtr< IVector_h > ("coloring", row_coloring);

return AMGX_OK;
}


template<class T_Config>
AMGX_ERROR MatrixIO<T_Config>::readGeometry( AuxData* obj, int n,int dimension )
{
  typedef typename Matrix<T_Config>::MVector VVector;
  MVector_h geo_x;
  MVector_h geo_y;
  geo_x.resize(n);
  geo_y.resize(n);

  int num_one_dim;
  if (dimension == 3)
  {
    MVector_h geo_z;
    geo_z.resize(n);
    num_one_dim = (int) cbrt((double)n);
    for (int i = 0;i < num_one_dim;i++)
        for (int j = 0;j < num_one_dim;j++)
            for (int k = 0;k < num_one_dim;k++)
            {
                geo_x[i + j*num_one_dim + k*num_one_dim*num_one_dim] = 1.0*i/(num_one_dim-1);
                geo_y[i + j*num_one_dim + k*num_one_dim*num_one_dim] = 1.0*j/(num_one_dim-1);
                geo_z[i + j*num_one_dim + k*num_one_dim*num_one_dim] = 1.0*k/(num_one_dim-1);
            }
    VVector *dgeo_z = new VVector;
    *dgeo_z = geo_z;
    obj->setParameterPtr< VVector > ("geo.z", dgeo_z);
  }
  else if (dimension == 2)
  {
    num_one_dim = (int) sqrt((double)n);
    for (int i = 0;i < num_one_dim;i++)
        for (int j = 0;j < num_one_dim;j++)
        {
            geo_x[i + j*num_one_dim] = 1.0*i/(num_one_dim-1);
            geo_y[i + j*num_one_dim] = 1.0*j/(num_one_dim-1);
            //(*geo_z)[i + j*num_one_dim] = 0;
        }
  }

  VVector *dgeo_y = new VVector;
  VVector *dgeo_x = new VVector;
  *dgeo_y = geo_y;
  *dgeo_x = geo_x;
  obj->setParameter<int>("dim", dimension);
  obj->setParameter<int>("geo_size",(int)(n));
  obj->setParameterPtr< VVector > ("geo.x", dgeo_x);
  obj->setParameterPtr< VVector > ("geo.y", dgeo_y);

  return AMGX_OK;
}*/
/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class MatrixIO<TemplateMode<CASE>::Type >;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
AMGX_FORCOMPLEX_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE
//AMGX_FORCOMPLEX_BUILDS_DEVICE(AMGX_CASE_LINE)
//  template class MatrixIO<Matrix_d>;
//  template class MatrixIO<Matrix_h>;
} // end namespace amgx
