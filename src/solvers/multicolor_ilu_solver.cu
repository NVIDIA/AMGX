#include "hip/hip_runtime.h"
/* Copyright (c) 2013-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <string.h>
#include <cutil.h>
#include <miscmath.h>
#include <amgx_cusparse.h>
#include <thrust/copy.h>
#include <solvers/multicolor_ilu_solver.h>
#include <solvers/block_common_solver.h>
#include <csr_multiply.h>
#include <gaussian_elimination.h>
#include <basic_types.h>
#include <util.h>
#include <texture.h>
#include <ld_functions.h>
#include <matrix_io.h>
#include <permute.h>
#include <thrust/logical.h>
#include <sm_utils.inl>
#include <algorithm>

// TODO: Have 2 groups of 16 threads collaborate
// TODO: Add support for outside diagonal
// TODO: Add support for unsorted rows

#define EXPERIMENTAL_LU_FACTORS
#define EXPERIMENTAL_LU_FORWARD
#define EXPERIMENTAL_LU_BACKWARD

namespace amgx
{

namespace multicolor_ilu_solver
{

// -----------
// Kernels
// -----------

#ifdef EXPERIMENTAL_LU_FORWARD

template<typename IndexType, typename ValueTypeA, typename ValueTypeB, int CtaSize, int bsize, bool ROW_MAJOR, bool hasDiag>
__global__
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
__launch_bounds__( CtaSize, 16 )
#elif defined(__CUDA_ARCH__)
__launch_bounds__( CtaSize, 16 )
#endif
void LU_forward_4x4_kernel_warp( const IndexType *LU_row_offsets,
                                 const IndexType *LU_smaller_color_offsets,
                                 const IndexType *LU_column_indices,
                                 const ValueTypeA *LU_nonzero_values,
                                 const IndexType *A_row_offsets,
                                 const IndexType *A_column_indices,
                                 const ValueTypeA *A_nonzero_values,
                                 const IndexType *A_dia_indices,
                                 const ValueTypeB *x,
                                 const ValueTypeB *b,
                                 ValueTypeB *delta,
                                 const int *sorted_rows_by_color,
                                 const int num_rows_per_color,
                                 const int current_color,
                                 bool xIsZero )
{
    const int nHalfWarps = CtaSize / 16; // Number of half warps per Cta
    const int warpId = utils::warp_id();
    const int laneId = utils::lane_id();
    const int halfWarpId = threadIdx.x / 16;
    const int halfLaneId = threadIdx.x % 16;
    const int halfLaneId_div_4 = halfLaneId / 4;
    const int halfLaneId_mod_4 = halfLaneId % 4;
    const int upperHalf = 16 * (laneId / 16);
    // Shared memory needed to exchange X and delta.
    __shared__ volatile ValueTypeB s_mem[CtaSize];
    // Each thread keeps its own pointer to shared memory to avoid some extra computations.
    volatile ValueTypeB *my_s_mem = &s_mem[16 * halfWarpId];

    // Iterate over the rows of the matrix. One warp per row.
    for ( int aRowIt = blockIdx.x * nHalfWarps + halfWarpId ; aRowIt < num_rows_per_color ; aRowIt += gridDim.x * nHalfWarps )
    {
        int aRowId = sorted_rows_by_color[aRowIt];
        // Load one block of B.
        ValueTypeB my_bmAx(0);

        unsigned int active_mask = utils::activemask();

        if ( ROW_MAJOR )
        {
            if ( halfLaneId_mod_4 == 0 )
            {
                my_bmAx = __cachingLoad(&b[4 * aRowId + halfLaneId_div_4]);
            }
        }
        else
        {
            if ( halfLaneId_div_4 == 0 )
            {
                my_bmAx = __cachingLoad(&b[4 * aRowId + halfLaneId_mod_4]);
            }
        }

        // Don't do anything if X is zero.
        if ( !xIsZero )
        {
            int aColBegin = A_row_offsets[aRowId  ];
            int aColEnd   = A_row_offsets[aRowId + 1];
            int aColMax = aColEnd;

            if ( hasDiag )
            {
                ++aColMax;
            }

            // Each warp load column indices of 32 nonzero blocks
            for ( ; utils::any( aColBegin < aColMax, active_mask ) ; aColBegin += 16 )
            {
                int aColIt = aColBegin + halfLaneId;
                // Get the ID of the column.
                int aColId = -1;

                if ( aColIt < aColEnd )
                {
                    aColId = A_column_indices[aColIt];
                }

                if ( hasDiag && aColIt == aColEnd )
                {
                    aColId = aRowId;
                }

                // Count the number of active columns.
                int vote =  utils::ballot(aColId != -1, active_mask);
                // The number of iterations.
                int nCols = max( __popc( vote & 0x0000ffff ), __popc( vote & 0xffff0000 ) );

                // Loop over columns. We compute 8 columns per iteration.
                for ( int k = 0 ; k < nCols ; k += 4 )
                {
                    int my_k = k + halfLaneId_div_4;
                    // Load 8 blocks of X.
                    int waColId = utils::shfl( aColId, upperHalf + my_k, warpSize, active_mask );
                    ValueTypeB my_x(0);

                    if ( waColId != -1 )
                    {
                        my_x = __cachingLoad(&x[4 * waColId + halfLaneId_mod_4]);
                    }

                    my_s_mem[halfLaneId] = my_x;
                    // Load 8 blocks of A.
#pragma unroll
                    for ( int i = 0 ; i < 4 ; ++i )
                    {
                        int w_aColTmp = aColBegin + k + i, w_aColIt = -1;

                        if ( w_aColTmp < aColEnd )
                        {
                            w_aColIt = w_aColTmp;
                        }

                        if ( hasDiag && w_aColTmp == aColEnd )
                        {
                            w_aColIt = A_dia_indices[aRowId];
                        }

                        ValueTypeA my_val(0);

                        if ( w_aColIt != -1 )
                        {
                            my_val = A_nonzero_values[16 * w_aColIt + halfLaneId];
                        }

                        if ( ROW_MAJOR )
                        {
                            my_bmAx -= my_val * my_s_mem[4 * i + halfLaneId_mod_4];
                        }
                        else
                        {
                            my_bmAx -= my_val * my_s_mem[4 * i + halfLaneId_div_4];
                        }
                    }
                } // Loop over k
            } // Loop over aColIt
        } // if xIsZero

        // Contribution from each nonzero column that has color less than yours
        if ( current_color != 0 )
        {
            // TODO: Use constant or texture here
            int aColBegin = LU_row_offsets[aRowId];
            int aColEnd   = LU_smaller_color_offsets[aRowId];

            // Each warp load column indices of 32 nonzero blocks
            for ( ; utils::any( aColBegin < aColEnd, active_mask ) ; aColBegin += 16 )
            {
                int aColIt = aColBegin + halfLaneId;
                int aColId = -1;

                if ( aColIt < aColEnd )
                {
                    aColId = LU_column_indices[aColIt];
                }

                // Count the number of active columns.
                int vote =  utils::ballot(aColId != -1, active_mask);
                // The number of iterations.
                int nCols = max( __popc( vote & 0x0000ffff ), __popc( vote & 0xffff0000 ) );

                for ( int k = 0 ; k < nCols ; k += 4 )
                {
                    int my_k = k + halfLaneId_div_4;
                    // Load 8 blocks of X.
                    int waColId = utils::shfl( aColId, upperHalf + my_k, warpSize, active_mask );
                    ValueTypeB my_delta(0);

                    if ( waColId != -1 )
                    {
                        my_delta = delta[4 * waColId + halfLaneId_mod_4];
                    }

                    my_s_mem[halfLaneId] = my_delta;
                    utils::syncwarp(); // making sure smem write propagated
                    // Update b-Ax.
#pragma unroll
                    for ( int i = 0 ; i < 4 ; ++i )
                    {
                        int w_aColTmp = aColBegin + k + i, w_aColIt = -1;

                        if ( w_aColTmp < aColEnd )
                        {
                            w_aColIt = w_aColTmp;
                        }

                        ValueTypeA my_val(0);

                        if ( w_aColIt != -1 )
                        {
                            my_val = LU_nonzero_values[16 * w_aColIt + halfLaneId];
                        }

                        if ( ROW_MAJOR )
                        {
                            my_bmAx -= my_val * my_s_mem[4 * i + halfLaneId_mod_4];
                        }
                        else
                        {
                            my_bmAx -= my_val * my_s_mem[4 * i + halfLaneId_div_4];
                        }
                    }
                } // Loop over k
            } // Loop over aColIt
        } // If current_color != 0

        // Reduce bmAx terms.
        if ( ROW_MAJOR )
        {
            my_bmAx += utils::shfl_xor( my_bmAx, 1, warpSize, active_mask );
            my_bmAx += utils::shfl_xor( my_bmAx, 2, warpSize, active_mask );
        }
        else
        {
            my_bmAx += utils::shfl_xor( my_bmAx, 4, warpSize, active_mask );
            my_bmAx += utils::shfl_xor( my_bmAx, 8, warpSize, active_mask );
        }

        // Store the results.
        if ( ROW_MAJOR )
        {
            if ( halfLaneId_mod_4 == 0 )
            {
                delta[4 * aRowId + halfLaneId_div_4] = my_bmAx;
            }
        }
        else
        {
            if ( halfLaneId_div_4 == 0 )
            {
                delta[4 * aRowId + halfLaneId_mod_4] = my_bmAx;
            }
        }
    }
}

#else
template<typename IndexType, typename ValueTypeA, typename ValueTypeB, int blockrows_per_cta, int blockrows_per_warp, int bsize, bool ROW_MAJOR>
__global__
void LU_forward_4x4_kernel(const IndexType *LU_row_offsets, const IndexType *LU_smaller_color_offsets, const IndexType *LU_column_indices, const ValueTypeA *LU_nonzero_values,  const IndexType *A_row_offsets, const IndexType *A_column_indices, const ValueTypeA *A_nonzero_values,
                           const ValueTypeB *x, const ValueTypeB *b,  ValueTypeB *delta, const int *sorted_rows_by_color,
                           const int num_rows_per_color, const int current_color, bool xIsZero)


{
    int warp_id = threadIdx.x / 32;
    int warp_thread_id = threadIdx.x & 31;

    // padding row blocks to fit in a single warp
    if ( warp_thread_id >= blockrows_per_warp * bsize ) { return; }

    // new thread id with padding
    int tid = warp_id * blockrows_per_warp * bsize + warp_thread_id;
    // Here we use one thread per row (not block row)
    int cta_blockrow_id = (tid) / bsize;
    int blockrow_id = blockIdx.x * blockrows_per_cta + cta_blockrow_id;
    const int vec_entry_index = tid - cta_blockrow_id * bsize;
    volatile __shared__ ValueTypeB s_delta_temp[ bsize * blockrows_per_cta];
    int offset, s_offset, i;
    ValueTypeB bmAx, temp[bsize];

    while (blockrow_id < num_rows_per_color &&  cta_blockrow_id < blockrows_per_cta)
    {
        i = sorted_rows_by_color[blockrow_id];
        // Load RHS and x
        offset = i * bsize + vec_entry_index;
        bmAx = b[offset];

        if (!xIsZero)
        {
            int jmin = A_row_offsets[i];
            int jmax = A_row_offsets[i + 1];

            //TODO: Assumes inside diagonal
            for (int jind = jmin; jind < jmax; jind++)
            {
                IndexType jcol = A_column_indices[jind];
                offset = jcol * bsize + vec_entry_index;
                s_delta_temp[tid] = x[offset];

                // Load nonzero_values
                if (ROW_MAJOR)
                {
                    offset = jind * bsize * bsize + vec_entry_index * bsize;
                    loadAsVector<bsize>(A_nonzero_values + offset, temp);
                }
                else
                {
                    offset = jind * bsize * bsize + vec_entry_index;
#pragma unroll
                    for (int m = 0; m < bsize; m++)
                    {
                        temp[m] = A_nonzero_values[offset + bsize * m];
                    }
                }

                // Do matrix multiply
                s_offset = cta_blockrow_id * bsize;
#pragma unroll
                for (int m = 0; m < bsize; m++)
                {
                    bmAx -= temp[m] * s_delta_temp[s_offset++];
                }
            }
        }

        // Contribution from each nonzero column that has color less than yours
        if (current_color != 0)
        {
            int jmin = LU_row_offsets[i];
            int jmax = LU_smaller_color_offsets[i];

            for (int jind = jmin; jind < jmax; jind++)
            {
                IndexType jcol = LU_column_indices[jind];
                offset = jcol * bsize + vec_entry_index;
                s_delta_temp[tid] = ld_cg(delta + offset);

                // Load nonzero_values
                if (ROW_MAJOR)
                {
                    offset = jind * bsize * bsize + vec_entry_index * bsize;
                    loadAsVector<bsize>(LU_nonzero_values + offset, temp);
                }
                else
                {
                    offset = jind * bsize * bsize + vec_entry_index;
#pragma unroll
                    for (int m = 0; m < bsize; m++)
                    {
                        temp[m] = LU_nonzero_values[offset + bsize * m];
                    }
                }

                // Do matrix multiply
                s_offset = cta_blockrow_id * bsize;
#pragma unroll
                for (int m = 0; m < bsize; m++)
                {
                    bmAx -= temp[m] * s_delta_temp[s_offset++];
                }
            }
        }

        delta[i * bsize + vec_entry_index] = bmAx;
        blockrow_id += blockrows_per_cta * gridDim.x;
    }
}
#endif

#ifdef EXPERIMENTAL_LU_BACKWARD

template< typename IndexType, typename ValueTypeA, typename ValueTypeB, int CtaSize, bool ROW_MAJOR >
__global__
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
__launch_bounds__( CtaSize, 16 )
#elif defined(__CUDA_ARCH__)
__launch_bounds__( CtaSize, 16 )
#endif
void LU_backward_4x4_kernel_warp( const IndexType *row_offsets,
                                  const IndexType *larger_color_offsets,
                                  const IndexType *column_indices,
                                  const IndexType *dia_indices,
                                  const ValueTypeA *nonzero_values,
                                  const ValueTypeB *delta,
                                  ValueTypeB *Delta,
                                  ValueTypeB *x,
                                  const int *sorted_rows_by_color,
                                  const int num_rows_per_color,
                                  const int current_color,
                                  const int num_colors,
                                  const ValueTypeB weight,
                                  bool xIsZero )
{
    const int nHalfWarps = CtaSize / 16; // Number of half warps per CTA.
    const int warpId = utils::warp_id();
    const int laneId = utils::lane_id();
    const int halfWarpId = threadIdx.x / 16;
    const int halfLaneId = threadIdx.x % 16;
    const int halfLaneId_div_4 = halfLaneId / 4;
    const int halfLaneId_mod_4 = halfLaneId % 4;
    const int upperHalf = 16 * (laneId / 16);
    // Shared memory needed to exchange X and delta.
    __shared__ volatile ValueTypeB s_mem[CtaSize];
    // Each thread keeps its own pointer to shared memory to avoid some extra computations.
    volatile ValueTypeB *my_s_mem = &s_mem[16 * halfWarpId];

    // Iterate over the rows of the matrix. One warp per two rows.
    for ( int aRowIt = blockIdx.x * nHalfWarps + halfWarpId ; aRowIt < num_rows_per_color ; aRowIt += gridDim.x * nHalfWarps )
    {
        int aRowId = sorted_rows_by_color[aRowIt];
        unsigned int active_mask = utils::activemask();
        // Load one block of B.
        ValueTypeB my_bmAx(0);

        if ( ROW_MAJOR )
        {
            if ( halfLaneId_mod_4 == 0 )
            {
                my_bmAx = delta[4 * aRowId + halfLaneId_div_4];
            }
        }
        else
        {
            if ( halfLaneId_div_4 == 0 )
            {
                my_bmAx = delta[4 * aRowId + halfLaneId_mod_4];
            }
        }

        // Don't do anything if the color is not the interesting one.
        if ( current_color != num_colors - 1 )
        {
            // The range of the rows.
            int aColBegin = larger_color_offsets[aRowId], aColEnd = row_offsets[aRowId + 1];

            // Each warp load column indices of 16 nonzero blocks
            for ( ; utils::any( aColBegin < aColEnd, active_mask ) ; aColBegin += 16 )
            {
                int aColIt = aColBegin + halfLaneId;
                // Get the ID of the column.
                int aColId = -1;

                if ( aColIt < aColEnd )
                {
                    aColId = column_indices[aColIt];
                }

                // Loop over columns. We compute 8 columns per iteration.
                for ( int k = 0 ; k < 16 ; k += 4 )
                {
                    int my_k = k + halfLaneId_div_4;
                    // Exchange column indices.
                    int waColId = utils::shfl( aColId, upperHalf + my_k, warpSize, active_mask );
                    // Load 8 blocks of X if needed.
                    ValueTypeB *my_ptr = Delta;

                    if ( xIsZero )
                    {
                        my_ptr = x;
                    }

                    ValueTypeB my_x(0);

                    if ( waColId != -1 )
                    {
                        my_x = my_ptr[4 * waColId + halfLaneId_mod_4];
                    }

                    my_s_mem[halfLaneId] = my_x;
                    utils::syncwarp();
                    // Load 8 blocks of A.
#pragma unroll
                    for ( int i = 0 ; i < 4 ; ++i )
                    {
                        int w_aColTmp = aColBegin + k + i, w_aColIt = -1;

                        if ( w_aColTmp < aColEnd )
                        {
                            w_aColIt = w_aColTmp;
                        }

                        ValueTypeA my_val(0);

                        if ( w_aColIt != -1 )
                        {
                            my_val = nonzero_values[16 * w_aColIt + halfLaneId];
                        }

                        if ( ROW_MAJOR )
                        {
                            my_bmAx -= my_val * my_s_mem[4 * i + halfLaneId_mod_4];
                        }
                        else
                        {
                            my_bmAx -= my_val * my_s_mem[4 * i + halfLaneId_div_4];
                        }
                    }
                } // Loop over k
            } // Loop over aColIt

            // Reduce bmAx terms.
            if ( ROW_MAJOR )
            {
                my_bmAx += utils::shfl_xor( my_bmAx, 1, warpSize, active_mask );
                my_bmAx += utils::shfl_xor( my_bmAx, 2, warpSize, active_mask );
            }
            else
            {
                my_bmAx += utils::shfl_xor( my_bmAx, 4, warpSize, active_mask );
                my_bmAx += utils::shfl_xor( my_bmAx, 8, warpSize, active_mask );
            }
        } // if current_color != num_colors-1

        // Update the shared terms.
        if ( ROW_MAJOR )
        {
            if ( halfLaneId_mod_4 == 0 )
            {
                my_s_mem[halfLaneId_div_4] = my_bmAx;
            }
        }
        else
        {
            if ( halfLaneId_div_4 == 0 )
            {
                my_s_mem[halfLaneId_mod_4] = my_bmAx;
            }
        }

        // Update the diagonal term.
        int w_aColIt = dia_indices[aRowId];
        ValueTypeA my_val(0);
        utils::syncwarp();

        if ( w_aColIt != -1 )
        {
            my_val = nonzero_values[16 * w_aColIt + halfLaneId];
        }

        if ( ROW_MAJOR )
        {
            my_bmAx = my_val * my_s_mem[halfLaneId_mod_4];
        }
        else
        {
            my_bmAx = my_val * my_s_mem[halfLaneId_div_4];
        }

        // Regroup results.
        if ( ROW_MAJOR )
        {
            my_bmAx += utils::shfl_xor( my_bmAx, 1 );
            my_bmAx += utils::shfl_xor( my_bmAx, 2 );
        }
        else
        {
            my_bmAx += utils::shfl_xor( my_bmAx, 4 );
            my_bmAx += utils::shfl_xor( my_bmAx, 8 );
        }

        // Store the results.
        if ( ROW_MAJOR )
        {
            ValueTypeB my_x(0);

            if ( !xIsZero && halfLaneId_mod_4 == 0 )
            {
                my_x = x[4 * aRowId + halfLaneId_div_4];
            }

            my_x += weight * my_bmAx;

            if ( !xIsZero && halfLaneId_mod_4 == 0 )
            {
                Delta[4 * aRowId + halfLaneId_div_4] = my_bmAx;
            }

            if ( halfLaneId_mod_4 == 0 )
            {
                x[4 * aRowId + halfLaneId_div_4] = my_x;
            }
        }
        else
        {
            ValueTypeB my_x(0);

            if ( !xIsZero && halfLaneId_div_4 == 0 )
            {
                my_x = x[4 * aRowId + halfLaneId_mod_4];
            }

            my_x += weight * my_bmAx;

            if ( !xIsZero && halfLaneId_div_4 == 0 )
            {
                Delta[4 * aRowId + halfLaneId_mod_4] = my_bmAx;
            }

            if ( halfLaneId_div_4 == 0 )
            {
                x[4 * aRowId + halfLaneId_mod_4] = my_x;
            }
        }
    }
}

#else

template<typename IndexType, typename ValueTypeA, typename ValueTypeB, int blockrows_per_cta, int blockrows_per_warp, int bsize, bool ROW_MAJOR>
__global__
void LU_backward_4x4_kernel(const IndexType *row_offsets, const IndexType *larger_color_offsets, const IndexType *column_indices, const IndexType *dia_indices, const ValueTypeA *nonzero_values,
                            const ValueTypeB *delta,  ValueTypeB *Delta, ValueTypeB *x, const int *sorted_rows_by_color,
                            const int num_rows_per_color, const int current_color, const int num_colors, const ValueTypeB weight, bool xIsZero)


{
    int warp_id = threadIdx.x / 32;
    int warp_thread_id = threadIdx.x & 31;

    // padding row blocks to fit in a single warp
    if ( warp_thread_id >= blockrows_per_warp * bsize ) { return; }

    // new thread id with padding
    int tid = warp_id * blockrows_per_warp * bsize + warp_thread_id;
    // Here we use one thread per row (not block row)
    int cta_blockrow_id = (tid) / bsize;
    int blockrow_id = blockIdx.x * blockrows_per_cta + cta_blockrow_id;
    const int vec_entry_index = tid - cta_blockrow_id * bsize;
    volatile __shared__ ValueTypeB s_x_temp[ bsize * blockrows_per_cta];
    int offset, s_offset, i;
    ValueTypeB bmAx, temp[bsize];

    while (blockrow_id < num_rows_per_color &&  cta_blockrow_id < blockrows_per_cta)
    {
        i = sorted_rows_by_color[blockrow_id];
        // Load RHS and x
        offset = i * bsize + vec_entry_index;
        bmAx = delta[offset];

        // Contribution from each nonzero column that has color less than yours
        if (current_color != num_colors)
        {
            int jmin = larger_color_offsets[i];
            int jmax = row_offsets[i + 1];

            for (int jind = jmin; jind < jmax; jind++)
            {
                IndexType jcol = column_indices[jind];
                offset = jcol * bsize + vec_entry_index;

                if (xIsZero)
                {
                    s_x_temp[tid] = ld_cg(x + offset);
                }
                else
                {
                    s_x_temp[tid] = ld_cg(Delta + offset);
                }

                // Load nonzero_values
                if (ROW_MAJOR)
                {
                    offset = jind * bsize * bsize + vec_entry_index * bsize;
                    loadAsVector<bsize>(nonzero_values + offset, temp);
                }
                else
                {
                    offset = jind * bsize * bsize + vec_entry_index;
#pragma unroll
                    for (int m = 0; m < bsize; m++)
                    {
                        temp[m] = nonzero_values[offset + bsize * m];
                    }
                }

                // Do matrix multiply
                s_offset = cta_blockrow_id * bsize;
#pragma unroll
                for (int m = 0; m < bsize; m++)
                {
                    bmAx -= temp[m] * s_x_temp[s_offset++];
                }
            }
        }

        s_x_temp[tid] = bmAx;
        bmAx = 0.;

        // Load diagonals (which store the inverse)
        if (ROW_MAJOR)
        {
            offset = dia_indices[i] * bsize * bsize + vec_entry_index * bsize;
            loadAsVector<bsize>(nonzero_values + offset, temp);
        }
        else
        {
            offset = dia_indices[i] * bsize * bsize + vec_entry_index;
#pragma unroll
            for (int m = 0; m < bsize; m++)
            {
                temp[m] = nonzero_values[offset + bsize * m];
            }
        }

        // Do matrix-vector multiply
        s_offset = cta_blockrow_id * bsize;
#pragma unroll
        for (int m = 0; m < bsize; m++)
        {
            bmAx += temp[m] * s_x_temp[s_offset++];
        }

        offset = i * bsize + vec_entry_index;

        if (xIsZero)
        {
            x[offset] = weight * bmAx;
        }
        else
        {
            Delta[offset] = bmAx;
            x[offset] += weight * bmAx ;
        }

        blockrow_id += blockrows_per_cta * gridDim.x;
    }
}

#endif

// Assumptions:
// CtaSize must be multiple of 32
// SMemSize should be larger than the maximum number of columns in the matrix
// Matrix B is superset of matrix A

template< int CtaSize, int SMemSize>
__global__ __launch_bounds__( CtaSize )
void
computeAtoLUmapping_kernel( int A_nRows,
                            const int *__restrict A_row_offsets,
                            const int *__restrict A_col_indices,
                            const int *__restrict B_row_offsets,
                            const int *__restrict B_col_indices,
                            int *__restrict AtoBmapping,
                            int *wk_returnValue )
{
    const int nWarps = CtaSize / 32; // Number of warps per Cta
    const int warpId = utils::warp_id();
    const int laneId = utils::lane_id();
    // Rows are stored in SMEM. Linear storage.
    __shared__ volatile int s_colInd[nWarps][SMemSize];
    // The row this warp is responsible for
    int aRowId = blockIdx.x * nWarps + warpId;

    // Loop over rows of A.
    for ( ; aRowId < A_nRows ; aRowId += nWarps * gridDim.x )
    {
        // Insert all the column indices of matrix B in the shared memory table
        int bColBeg = B_row_offsets[aRowId];
        int bColEnd = B_row_offsets[aRowId + 1];
        // The number of columns.
        const int nCols = bColEnd - bColBeg;

        //TODO: Add fallback for cases where number of nonzeros exceed SMemSize
        if ( nCols > SMemSize )
        {
            wk_returnValue[0] = 1;
            return;
        }

        // Fill-in the local table.
        const int NUM_STEPS = SMemSize / 32;
#pragma unroll
        for ( int step = 0, k = laneId ; step < NUM_STEPS ; ++step, k += 32 )
        {
            int bColIt = bColBeg + k;
            int bColId = -1;

            if ( bColIt < bColEnd )
            {
                bColId = B_col_indices[bColIt];
            }

            s_colInd[warpId][k] = bColId;
        }

        // Now load column indices of current row of A
        int aColIt  = A_row_offsets[aRowId];
        int aColEnd = A_row_offsets[aRowId + 1];

        for ( aColIt += laneId ; utils::any(aColIt < aColEnd) ; aColIt += 32 )
        {
            // The column.
            int aColId = -1;

            if ( aColIt < aColEnd )
            {
                aColId = A_col_indices[aColIt];
            }

            // Each thread searches for its column id, and gets the corresponding bColIt
            // TODO: Try binary search or using hash table
            int foundOffset = -1;

            if ( aColId == -1 )
            {
                foundOffset = -2;
            }

            for ( int i = 0 ; i < nCols && utils::any(foundOffset == -1) ; ++i )
                if ( foundOffset == -1 && s_colInd[warpId][i] == aColId )
                {
                    foundOffset = i;
                }

            // Store the result.
            if ( aColIt < aColEnd )
            {
                AtoBmapping[aColIt] = bColBeg + foundOffset;
            }
        }
    } // if RowId < A_nRows;
}

template< int CtaSize, int SMemSize>
__global__ __launch_bounds__( CtaSize )
void
computeAtoLUmappingExtDiag_kernel( int A_nRows,
                                   const int *__restrict A_row_offsets,
                                   const int *__restrict A_col_indices,
                                   const int *__restrict A_dia_indices,
                                   const int *__restrict B_row_offsets,
                                   const int *__restrict B_col_indices,
                                   int *__restrict AtoBmapping,
                                   int *wk_returnValue )
{
    const int nWarps = CtaSize / 32; // Number of warps per Cta
    const int warpId = utils::warp_id();
    const int laneId = utils::lane_id();
    // Rows are stored in SMEM. Linear storage.
    __shared__ volatile int s_colInd[nWarps][SMemSize];
    // The row this warp is responsible for
    int aRowId = blockIdx.x * nWarps + warpId;

    // Loop over rows of A.
    for ( ; aRowId < A_nRows ; aRowId += nWarps * gridDim.x )
    {
        // Insert all the column indices of matrix B in the shared memory table
        int bColBeg = B_row_offsets[aRowId];
        int bColEnd = B_row_offsets[aRowId + 1];
        // The number of columns.
        const int nCols = bColEnd - bColBeg;

        //TODO: Add fallback for cases where number of nonzeros exceed SMemSize
        if ( nCols > SMemSize )
        {
            wk_returnValue[0] = 1;
            return;
        }

        // Fill-in the local table.
        const int NUM_STEPS = SMemSize / 32;
#pragma unroll
        for ( int step = 0, k = laneId ; step < NUM_STEPS ; ++step, k += 32 )
        {
            int bColIt = bColBeg + k;
            int bColId = -1;

            if ( bColIt < bColEnd )
            {
                bColId = B_col_indices[bColIt];
            }

            s_colInd[warpId][k] = bColId;
        }

        // Now load column indices of current row of A
        int aColIt  = A_row_offsets[aRowId];
        int aColEnd = A_row_offsets[aRowId + 1];

        for ( aColIt += laneId ; utils::any(aColIt <= aColEnd) ; aColIt += 32 )
        {
            // The column.
            int aColId = -1;

            if ( aColIt < aColEnd )
            {
                aColId = A_col_indices[aColIt];
            }

            if ( aColIt == aColEnd )
            {
                aColId = aRowId;
            }

            // Each thread searches for its column id, and gets the corresponding bColIt
            // TODO: Try binary search or using hash table
            int foundOffset = -1;

            if ( aColId == -1 )
            {
                foundOffset = -2;
            }

            for ( int i = 0 ; i < nCols && utils::any(foundOffset == -1) ; ++i )
                if ( foundOffset == -1 && s_colInd[warpId][i] == aColId )
                {
                    foundOffset = i;
                }

            // Store the result.
            int aDst = -1;

            if ( aColIt < aColEnd )
            {
                aDst = aColIt;
            }

            if ( aColIt == aColEnd )
            {
                aDst = A_dia_indices[aRowId];
            }

            if ( aDst != -1 )
            {
                AtoBmapping[aDst] = bColBeg + foundOffset;
            }
        }
    }
}

#ifdef EXPERIMENTAL_LU_FACTORS

template< typename ValueTypeA, int CtaSize, int SMemSize, bool ROW_MAJOR >
__global__
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
__launch_bounds__( CtaSize, 12 )
#elif defined(__CUDA_ARCH__)
__launch_bounds__( CtaSize, 12 )
#endif
void
compute_LU_factors_4x4_kernel_warp( int A_nRows,
                                    const int *__restrict A_row_offsets,
                                    const int *__restrict A_col_indices,
                                    const int *__restrict A_dia_indices,
                                    ValueTypeA *__restrict A_nonzero_values,
                                    const int *__restrict A_smaller_color_offsets,
                                    const int *__restrict A_larger_color_offsets,
                                    const int *sorted_rows_by_color,
                                    const int num_rows_per_color,
                                    const int current_color,
                                    int *wk_returnValue )
{
    const int nWarps = CtaSize / 32; // Number of warps per Cta
    const int warpId = utils::warp_id();
    const int laneId = utils::lane_id();
    // Lane ID in the 2 16-wide segments.
    const int lane_id_div_16 = laneId / 16;
    const int lane_id_mod_16 = laneId % 16;
    // Coordinates inside a 4x4 block of the matrix.
    const int idx_i = lane_id_mod_16 / 4;
    const int idx_j = lane_id_mod_16 % 4;
    int globalWarpId = blockIdx.x * nWarps + warpId;
    // Shared memory to store the blocks to process
    __shared__ volatile ValueTypeA s_C_mtx[nWarps][32];
    __shared__ volatile ValueTypeA s_F_mtx[nWarps][16];
    // Shared memory to store the proposed column to load
    __shared__ volatile int s_aColSrc[nWarps][32];
    // Shared memory to store the column indices of the current row
    __shared__ volatile int s_keys[nWarps][SMemSize];

    while (globalWarpId < num_rows_per_color)
    {
        int storedRowId[2];
        int I = 0;

        for (; I < 2 && globalWarpId < num_rows_per_color ; I++)
        {
            int aRowId = sorted_rows_by_color[globalWarpId];
            storedRowId[I] = aRowId;
            int aColBeg = A_row_offsets[aRowId + 0];
            int aColEnd = A_row_offsets[aRowId + 1];
            int aColSmaller = A_smaller_color_offsets[aRowId];
            // The number of columns.
            const int nCols = aColEnd - aColBeg;

            //TODO: Add fallback for cases where number of nonzeros exceed SMemSize
            if ( nCols > SMemSize )
            {
                wk_returnValue[0] = 1;
                return;
            }

            // Fill-in the local table.
            const int NUM_STEPS = SMemSize / 32;

#pragma unroll
            for ( int step = 0, k = laneId ; step < NUM_STEPS ; ++step, k += 32 )
            {
                int aColIt = aColBeg + k;
                int aColId = -1;

                if ( aColIt < aColEnd )
                {
                    aColId = A_col_indices[aColIt];
                }

                s_keys[warpId][k] = aColId;
            }

            // Now load all column indices of neighbours that have colors smaller than yours
            for ( int aColIt = aColBeg; aColIt < aColSmaller ; aColIt++)
            {
                unsigned int active_mask = utils::activemask();
                // Read the row to process, should be a broadcast
                int waRowId = s_keys[warpId][aColIt - aColBeg];
                // Compute multiplicative factor, load C_jj in first half, C_ij in second half
                int aColIdx = aColIt;

                if ( lane_id_div_16 == 0 )
                {
                    aColIdx = A_dia_indices[waRowId];
                }

                s_C_mtx[warpId][laneId] = A_nonzero_values[16 * aColIdx + lane_id_mod_16];
                // Threads 0-15 perform the matrix product
                ValueTypeA tmp(0);

                if (ROW_MAJOR)
                {

#pragma unroll
                    for ( int m = 0 ; m < 4 ; ++m )
                    {
                        tmp += s_C_mtx[warpId][16 + 4 * idx_i + m] * s_C_mtx[warpId][4 * m + idx_j];
                    }
                }
                else
                {

#pragma unroll
                    for ( int m = 0 ; m < 4 ; ++m )
                    {
                        tmp += s_C_mtx[warpId][16 + 4 * m + idx_j] * s_C_mtx[warpId][4 * idx_i + m];
                    }
                }

                if ( lane_id_div_16 == 0 )
                {
                    s_F_mtx[warpId][laneId] = tmp;
                    A_nonzero_values[16 * aColIt + laneId] = tmp;
                }

                int waColIt  = ld_cg(A_larger_color_offsets + waRowId);
                int waColEnd = ld_cg(A_row_offsets + waRowId + 1);

                // Load the first 32 columns of waRowId
                for (waColIt += laneId ; utils::any(waColIt < waColEnd, active_mask ); waColIt += 32 )
                {
                    // Each thread loads its column id
                    int waColId = -1;

                    if ( waColIt < waColEnd )
                    {
                        waColId = A_col_indices[waColIt];
                    }

                    // Find the right column.
                    int found_aColIt = -1;

#pragma unroll 4
                    for ( int i = 0, num_keys = aColEnd - aColBeg ; i < num_keys ; ++i )
                        if ( s_keys[warpId][i] == waColId )
                        {
                            found_aColIt = i;
                        }

                    if ( found_aColIt != -1 )
                    {
                        found_aColIt += aColBeg;
                    }

                    // Store all the columns that have been found
                    const int pred = found_aColIt != -1;
                    int vote = utils::ballot( pred, active_mask );
                    const int idst = __popc(vote & utils::lane_mask_lt());

                    if (pred)
                    {
                        s_aColSrc[warpId][idst] = laneId;
                    }
                    utils::syncwarp(active_mask);

                    const int n_cols = __popc( vote );

                    // Process all columns that have been found
                    for ( int k = 0 ; k < n_cols ; k += 2 )
                    {
                        const int my_k = k + lane_id_div_16;
                        // Where to get columns from.
                        int a_col_it = -1, w_col_it = -1;
                        // Load column to load
                        a_col_it = utils::shfl(found_aColIt, s_aColSrc[warpId][my_k], warpSize, active_mask);
                        w_col_it = utils::shfl(waColIt,      s_aColSrc[warpId][my_k], warpSize, active_mask);

                        if ( my_k >= n_cols )
                        {
                            a_col_it = -1;
                            w_col_it = -1;
                        }

                        ValueTypeA my_C(0);

                        if ( w_col_it != -1 )
                        {
                            my_C = A_nonzero_values[16 * w_col_it + lane_id_mod_16];
                        }

                        s_C_mtx[warpId][laneId] = my_C;
                        // Run the matrix-matrix product.
                        ValueTypeA tmp(0);
                        utils::syncwarp( active_mask );

                        if (ROW_MAJOR)
                        {
#pragma unroll
                            for ( int m = 0 ; m < 4 ; ++m )
                            {
                                tmp += s_F_mtx[warpId][4 * idx_i + m] * s_C_mtx[warpId][16 * lane_id_div_16 + 4 * m + idx_j];
                            }
                        }
                        else
                        {
#pragma unroll
                            for ( int m = 0 ; m < 4 ; ++m )
                            {
                                tmp += s_F_mtx[warpId][4 * m + idx_j] * s_C_mtx[warpId][16 * lane_id_div_16 + 4 * idx_i + m];
                            }
                        }

                        if ( a_col_it != -1 )
                        {
                            A_nonzero_values[16 * a_col_it + lane_id_mod_16] -= tmp;
                        }
                    } // Loop over columns that have a match (for k=0;k<n_cols)
                } // Loop over the columns of waRowId

                //}  // Loop j=0;j<32
            } // Loop over the columns of aRowId

            globalWarpId += nWarps * gridDim.x;
        } // end of loop over I

        // Now compute the inverse of the block C_jj
        if ( lane_id_div_16 == 0 || I == 2 )
        {
            const int offset = 16 * A_dia_indices[storedRowId[lane_id_div_16]] + lane_id_mod_16;
            s_C_mtx[warpId][laneId] = A_nonzero_values[offset];
            utils::syncwarp(utils::activemask());

            if (ROW_MAJOR)
            {
                compute_block_inverse_row_major4x4_formula2<int, ValueTypeA, 4, true>( s_C_mtx[warpId], 16 * lane_id_div_16, offset, idx_i, idx_j, A_nonzero_values );
            }
            else
            {
                compute_block_inverse_col_major4x4_formula2<int, ValueTypeA, 4, true>( s_C_mtx[warpId], 16 * lane_id_div_16, offset, idx_i, idx_j, A_nonzero_values );
            }
        } // End of if statement
    } // End of while loop
}

#else

template< typename ValueTypeA, int CtaSize, int SMemSize, bool ROW_MAJOR>
__global__ __launch_bounds__( CtaSize )
void
computeLUFactors_4x4_kernel( int A_nRows,
                             const int *__restrict A_row_offsets,
                             const int *__restrict A_col_indices,
                             const int *__restrict A_dia_indices,
                             ValueTypeA *__restrict A_nonzero_values,
                             const int *__restrict A_smaller_color_offsets,
                             const int *__restrict A_larger_color_offsets,
                             const int *sorted_rows_by_color,
                             const int num_rows_per_color,
                             const int current_color,
                             int *wk_returnValue )
{
    const int nWarps = CtaSize / 32; // Number of warps per Cta
    const int warpId = utils::warp_id();
    const int laneId = utils::lane_id();
    int lane_mask_lt = utils::lane_mask_lt();

    // Lane ID in the 2 16-wide segments.
    const int lane_id_div_16 = laneId / 16;
    const int lane_id_mod_16 = laneId % 16;
    // Coordinates inside a 4x4 block of the matrix.
    const int idx_i = lane_id_mod_16 / 4;
    const int idx_j = lane_id_mod_16 % 4;
    int globalWarpId = blockIdx.x * nWarps + warpId;
    // Shared memory to store the blocks to process
    __shared__ volatile ValueTypeA s_C_mtx[nWarps][32];
    // Shared memory to store the proposed column to load
    __shared__ volatile int s_aColItToLoad[nWarps][32];
    __shared__ volatile int s_waColItToLoad[nWarps][32];
    // Shared memory to store the proposed column to load
    __shared__ volatile unsigned s_aColIds[nWarps][32];
    // The size of the hash table (one per warp - shared memory).
    __shared__ volatile int s_size[nWarps][2];
    // Shared memory to store the column indices of the current row
    __shared__ volatile int s_keys[nWarps][SMemSize];

    while (globalWarpId < num_rows_per_color)
    {
        int aRowId = sorted_rows_by_color[globalWarpId];
        // Insert all the column indices in shared memory
        // TODO: Use texture here
        int aColBeg = A_row_offsets[aRowId];
        int aColEnd = A_row_offsets[aRowId + 1];
        int aColIt  = aColBeg;

        // Check if number of nonzeros will fit in shared memory
        if ( (aColEnd - aColBeg) > SMemSize )
        {
            wk_returnValue[0] = 1;
            return;
        }

        // Load the all the column indices of row into shared memory
        for ( aColIt += laneId ; utils::any( aColIt < aColEnd ) ; aColIt += 32 )
        {
            int aColId = aColIt < aColEnd ? (int) A_col_indices[aColIt] : -1;
            s_keys[warpId][aColIt - aColBeg] = aColId;
        }

        // Now load all column indices of neighbours that have colors smaller than yours
        aColIt  = aColBeg;
        int aColSmaller = A_smaller_color_offsets[aRowId];

        for ( ; utils::any( (aColIt + laneId) < aColSmaller ) ; aColIt += 32 )
        {
            int aColId = (aColIt + laneId) < aColSmaller ? (int) A_col_indices[aColIt + laneId] : -1;
            // Each thread pushes its column
            s_aColIds[warpId][laneId] = aColId;

            // Have warp collaborate to load each row
            for ( int j = 0; j < 32; j++)
            {
                // Check if row to load is valid
                if ( ( aColIt + j ) >= aColSmaller ) { break; }

                // Read the row to process, should be a broadcast
                int waRowId = s_aColIds[warpId][j];

                // Compute multiplicative factor, load C_jj in first half, C_ij in second half
                if (lane_id_div_16 == 0)
                {
                    s_C_mtx[warpId][laneId] = A_nonzero_values[ 16 * A_dia_indices[waRowId] + lane_id_mod_16 ];
                }
                else
                {
                    s_C_mtx[warpId][laneId] = A_nonzero_values[ 16 * (aColIt + j) + lane_id_mod_16 ];
                }

                // Threads 0-15 perform the matrix product
                utils::syncwarp();
                if (lane_id_div_16 == 0)
                {
                    ValueTypeA tmp(0);

                    if (ROW_MAJOR)
                    {
#pragma unroll
                        for ( int m = 0 ; m < 4 ; ++m )
                        {
                            tmp += s_C_mtx[warpId][16 + 4 * idx_i + m] * s_C_mtx[warpId][4 * m + idx_j];
                        }
                    }
                    else
                    {
#pragma unroll
                        for ( int m = 0 ; m < 4 ; ++m )
                        {
                            tmp += s_C_mtx[warpId][16 + 4 * m + idx_j] * s_C_mtx[warpId][4 * idx_i + m];
                        }
                    }

                    s_C_mtx[warpId][laneId] = tmp;
                    A_nonzero_values[16 * (aColIt + j) + laneId] = tmp;
                }

                int waColIt  = A_larger_color_offsets[waRowId];
                int waColEnd = A_row_offsets[waRowId + 1];

                //// Load the first 32 columns of waRowId
                for (waColIt += laneId ; utils::any(waColIt < waColEnd ); waColIt += 32 )
                {
                    // Each thread loads its column id
                    int waColId = waColIt < waColEnd ? A_col_indices[waColIt] : int (-1);
                    // TODO: Try binary search if columns are ordered
                    int found_aColIt = -1;

                    //TODO: if invalid waColId, don't search
                    for (int i = 0 ; utils::any(found_aColIt == -1) && i < aColEnd - aColBeg ; i++)
                    {
                        if (s_keys[warpId][i] == waColId) { found_aColIt = aColBeg + i; }
                    }

                    // Store all the columns that have been found
                    const int pred = found_aColIt != -1;
                    const int vote = utils::ballot( pred );
                    const int idst = __popc(vote & lane_mask_lt);

                    if (pred)
                    {
                        s_aColItToLoad [warpId][idst] = found_aColIt;
                        s_waColItToLoad[warpId][idst] = waColIt;
                    }

                    const int n_cols = __popc( vote );

                    // Process all columns that have been found
                    for ( int k = 0 ; k < n_cols ; k++ )
                    {
                        // Load column to load
                        const int a_col_it = k < n_cols ? s_aColItToLoad [warpId][k] : -1;
                        const int w_col_it = k < n_cols ? s_waColItToLoad[warpId][k] : -1;

                        if (lane_id_div_16 == 1)
                        {
                            s_C_mtx[warpId][laneId] = A_nonzero_values[16 * w_col_it + lane_id_mod_16];
                            // Run the matrix-matrix product.
                            ValueTypeA tmp(0);
                            utils::syncwarp(utils::activemask());

                            if (ROW_MAJOR)
                            {

#pragma unroll
                                for ( int m = 0 ; m < 4 ; ++m )
                                {
                                    tmp += s_C_mtx[warpId][4 * idx_i + m] * s_C_mtx[warpId][16 + 4 * m + idx_j];
                                }
                            }
                            else
                            {

#pragma unroll
                                for ( int m = 0 ; m < 4 ; ++m )
                                {
                                    tmp += s_C_mtx[warpId][4 * m + idx_j] * s_C_mtx[warpId][16 + 4 * idx_i + m];
                                }
                            }

                            A_nonzero_values[16 * a_col_it + lane_id_mod_16] -= tmp;
                        }
                    } // Loop over columns that have a match (for k=0;k<n_cols)
                } // Loop over the columns of waRowId
            }  // Loop j=0;j<32
        } // Loop over the columns of aRowId

        // TODO: Have one warp deal with two rows
        // Now compute the inverse of the block C_jj
        if (lane_id_div_16 == 0)
        {
            const int offset = 16 * A_dia_indices[aRowId] + lane_id_mod_16;
            s_C_mtx[warpId][laneId] = A_nonzero_values[offset];
            utils::syncwarp(utils::activemask());

            if (ROW_MAJOR)
            {
                compute_block_inverse_row_major<int, ValueTypeA, 0, 4, 16>
                (s_C_mtx[warpId], 0, offset, idx_i, idx_j, A_nonzero_values);
            }
            else
            {
                compute_block_inverse_col_major<int, ValueTypeA, 0, 4, 16>
                (s_C_mtx[warpId], 0, offset, idx_i, idx_j, A_nonzero_values);
            }
        }

        globalWarpId += nWarps * gridDim.x;
    } // if RowId < A_nRows;
}
#endif
// ----------
// Methods
// ----------

// Constructor
template<class T_Config>
MulticolorILUSolver_Base<T_Config>::MulticolorILUSolver_Base( AMG_Config &cfg, const std::string &cfg_scope) : Solver<T_Config>( cfg, cfg_scope)
{
    m_sparsity_level = cfg.AMG_Config::getParameter<int>("ilu_sparsity_level", cfg_scope);
    m_weight = cfg.AMG_Config::getParameter<double>("relaxation_factor", cfg_scope);
    this->m_reorder_cols_by_color_desired = (cfg.AMG_Config::getParameter<int>("reorder_cols_by_color", cfg_scope) != 0);
    this->m_insert_diagonal_desired = (cfg.AMG_Config::getParameter<int>("insert_diag_while_reordering", cfg_scope) != 0);

    if (cfg.AMG_Config::getParameter<int>("use_bsrxmv", cfg_scope))
    {
        this->m_use_bsrxmv = 1;
    }
    else
    {
        this->m_use_bsrxmv = 0;
    }

    if (m_weight == ValueTypeB(0.))
    {
        m_weight = 1.;
        amgx_printf("Warning, setting weight to 1 instead of estimating largest_eigen_value in Multicolor DILU smoother\n");
    }
}

// Destructor
template<class T_Config>
MulticolorILUSolver_Base<T_Config>::~MulticolorILUSolver_Base()
{
    m_LU.set_initialized(0);
    m_A_to_LU_mapping.clear();
    m_A_to_LU_mapping.shrink_to_fit();
    m_LU.resize(0, 0, 0, 1);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::computeAtoLUmapping()
{
    FatalError("Haven't implemented Multicolor ILU smoother for host format", AMGX_ERR_NOT_SUPPORTED_TARGET);
}


template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::computeAtoLUmapping()
{
    Matrix<TConfig_d> &m_A = *this->m_explicit_A;
    const int CtaSize = 128; // Number of threads per CTA
    const int SMemSize = 128;  // per warp
    const int nWarps = CtaSize / 32;
    int GridSize = std::min( AMGX_GRID_MAX_SIZE, ( this->m_explicit_A->get_num_rows( ) + nWarps - 1 ) / nWarps );
    // Global memory workspaces
    device_vector_alloc<int> returnValue(1);
    returnValue[0] = 0;

    if (this->m_explicit_A->hasProps(DIAG))
    {
        computeAtoLUmappingExtDiag_kernel<CtaSize, SMemSize> <<< GridSize, CtaSize >>> (
            m_A.get_num_rows( ),
            thrust::raw_pointer_cast( &m_A.row_offsets[0] ),
            thrust::raw_pointer_cast( &m_A.col_indices[0] ),
            thrust::raw_pointer_cast( &m_A.diag[0] ),
            thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
            thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
            thrust::raw_pointer_cast( &this->m_A_to_LU_mapping[0] ),
            thrust::raw_pointer_cast( &returnValue[0] ));
    }
    else
    {
        computeAtoLUmapping_kernel<CtaSize, SMemSize> <<< GridSize, CtaSize >>> (
            m_A.get_num_rows( ),
            thrust::raw_pointer_cast( &m_A.row_offsets[0] ),
            thrust::raw_pointer_cast( &m_A.col_indices[0] ),
            thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
            thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
            thrust::raw_pointer_cast( &this->m_A_to_LU_mapping[0] ),
            thrust::raw_pointer_cast( &returnValue[0] ));
    }

    cudaCheckError();

    // fallback path that allows 1024 nonzeros per row
    if (returnValue[0] == 1)
    {
        returnValue[0] = 0;
        const int SMemSize2 = 1024 ;  // per warp

        if (this->m_explicit_A->hasProps(DIAG))
        {
            computeAtoLUmappingExtDiag_kernel<CtaSize, SMemSize2> <<< GridSize, CtaSize >>> (
                m_A.get_num_rows( ),
                thrust::raw_pointer_cast( &m_A.row_offsets[0] ),
                thrust::raw_pointer_cast( &m_A.col_indices[0] ),
                thrust::raw_pointer_cast( &m_A.diag[0] ),
                thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
                thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
                thrust::raw_pointer_cast( &this->m_A_to_LU_mapping[0] ),
                thrust::raw_pointer_cast( &returnValue[0] ));
        }
        else
        {
            computeAtoLUmapping_kernel<CtaSize, SMemSize2> <<< GridSize, CtaSize >>> (
                m_A.get_num_rows( ),
                thrust::raw_pointer_cast( &m_A.row_offsets[0] ),
                thrust::raw_pointer_cast( &m_A.col_indices[0] ),
                thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
                thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
                thrust::raw_pointer_cast( &this->m_A_to_LU_mapping[0] ),
                thrust::raw_pointer_cast( &returnValue[0] ));
        }

        cudaCheckError();
    }

    if (returnValue[0] == 1)
    {
        FatalError( "Number of nonzeros per row exceeds allocated shared memory", AMGX_ERR_NO_MEMORY);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::fillLUValuesWithAValues()
{
    FatalError("Haven't implemented Multicolor ILU smoother for host format", AMGX_ERR_NOT_SUPPORTED_TARGET);
}


template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::fillLUValuesWithAValues()
{
    if (this->m_sparsity_level == 0)
    {
        this->m_LU.values = this->m_explicit_A->values;
    }
    else
    {
        // TODO: Should probably store the inverse mapping of AtoLUmapping instead
        //       This will allow to use unpermuteVector and have coalesced writes
        //       instead of coalesced reads
        thrust::fill(this->m_LU.values.begin(), this->m_LU.values.end(), 0.);
        cudaCheckError();

        if (this->m_explicit_A->hasProps(DIAG))
        {
            amgx::permuteVector(this->m_explicit_A->values, this->m_LU.values, this->m_A_to_LU_mapping, (this->m_explicit_A->get_num_nz() + this->m_explicit_A->get_num_rows())*this->m_explicit_A->get_block_size());
        }
        else
        {
            amgx::permuteVector(this->m_explicit_A->values, this->m_LU.values, this->m_A_to_LU_mapping, this->m_explicit_A->get_num_nz()*this->m_explicit_A->get_block_size());
        }
    }
}


template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::computeLUSparsityPattern()
{
    FatalError("Haven't implemented Multicolor ILU smoother for host format", AMGX_ERR_NOT_SUPPORTED_TARGET);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::computeLUSparsityPattern()
{
    // ILU0
    if (this->m_sparsity_level == 0)
    {
        // Copy everything except the values
        this->m_LU.copy_structure(*this->m_explicit_A);
    }
    // ILU1
    else if (this->m_sparsity_level == 1)
    {
        this->sparsity_wk = CSR_Multiply<TConfig_d>::csr_workspace_create( *this->m_cfg, "default" );
        CSR_Multiply<TConfig_d>::csr_sparsity_ilu1( *this->m_explicit_A, this->m_LU, this->sparsity_wk );
        CSR_Multiply<TConfig_d>::csr_workspace_delete( this->sparsity_wk );

        if (this->m_use_bsrxmv)
        {
            this->m_LU.set_initialized(0);
            this->m_LU.computeDiagonal();
            this->m_LU.set_initialized(1);
        }

        this->m_LU.setMatrixColoring(&(this->m_explicit_A->getMatrixColoring()));
    }
    else
    {
        FatalError("Haven't implemented Multicolor ILU smoother for this sparsity level. ", AMGX_ERR_NOT_IMPLEMENTED);
    }
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::computeLUFactors()
{
    FatalError("Haven't implemented Multicolor ILU smoother for host format", AMGX_ERR_NOT_SUPPORTED_TARGET);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::computeLUFactors()
{
    const int CtaSize = 128; // Number of threads per CTA
    const int SMemSize = 128;
    const int nWarps = CtaSize / 32;
    device_vector_alloc<int> returnValue(1);
    returnValue[0] = 0;
    int num_colors = this->m_LU.getMatrixColoring().getNumColors();
    const IndexType *LU_sorted_rows_by_color_ptr = this->m_LU.getMatrixColoring().getSortedRowsByColor().raw();

    for (int i = 0; i < num_colors; i++)
    {
        const IndexType color_offset = this->m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i];
        const IndexType num_rows_per_color = this->m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i + 1] - color_offset;
#ifdef EXPERIMENTAL_LU_FACTORS
        int GridSize = std::min( 2048, ( num_rows_per_color + nWarps - 1 ) / nWarps );

        if ( GridSize == 0 )
        {
            continue;    // if perfect coloring (color 0 has no vertices)
        }

        if ( this->m_LU.get_block_dimx() == 4 && this->m_LU.get_block_dimy() == 4 )
        {
            if ( this->m_explicit_A->getBlockFormat() == ROW_MAJOR )
            {
                compute_LU_factors_4x4_kernel_warp<ValueTypeA, CtaSize, SMemSize, true> <<< GridSize, CtaSize>>>(
                    this->m_LU.get_num_rows( ),
                    thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.diag[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.values[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_smaller_color_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_larger_color_offsets[0] ),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    thrust::raw_pointer_cast( &returnValue[0] ) );
            }
            else
            {
                compute_LU_factors_4x4_kernel_warp<ValueTypeA, CtaSize, SMemSize, false> <<< GridSize, CtaSize>>>(
                    this->m_LU.get_num_rows( ),
                    thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.diag[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.values[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_smaller_color_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_larger_color_offsets[0] ),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    thrust::raw_pointer_cast( &returnValue[0] ) );
            }

            cudaCheckError();
        }
        else
        {
            FatalError("Unsupported block size for Multicolor ILU solver, computeLUFactors", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
        }

#else
        int GridSize = std::min( AMGX_GRID_MAX_SIZE, ( num_rows_per_color + nWarps - 1 ) / nWarps );

        if ( GridSize == 0 )
        {
            continue;    // if perfect coloring (color 0 has no vertices)
        }

        if ( this->m_LU.get_block_dimx() == 4 && this->m_LU.get_block_dimy() == 4 )
        {
            //computeLUFactors_4x4_kernel<ValueTypeA,CtaSize,SMemSize> <<< GridSize, CtaSize>>> (
            if (this->m_explicit_A->getBlockFormat() == ROW_MAJOR)
            {
                computeLUFactors_4x4_kernel<ValueTypeA, CtaSize, SMemSize, true> <<< GridSize, CtaSize>>> (
                    this->m_LU.get_num_rows( ),
                    thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.diag[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.values[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_smaller_color_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_larger_color_offsets[0] ),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    thrust::raw_pointer_cast( &returnValue[0] ) );
            }
            else
            {
                computeLUFactors_4x4_kernel<ValueTypeA, CtaSize, SMemSize, false> <<< GridSize, CtaSize>>> (
                    this->m_LU.get_num_rows( ),
                    thrust::raw_pointer_cast( &this->m_LU.row_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.col_indices[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.diag[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.values[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_smaller_color_offsets[0] ),
                    thrust::raw_pointer_cast( &this->m_LU.m_larger_color_offsets[0] ),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    thrust::raw_pointer_cast( &returnValue[0] ) );
            }

            cudaCheckError();
        }
        else
        {
            FatalError("Unsupported block size for Multicolor ILU solver, computeLUFactors", AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE);
        }

#endif
    }

    // Check returnValue flag
    if ( returnValue[0] == 1 )
    {
        FatalError( "Number of nonzeros per row exceeds allocated shared memory", AMGX_ERR_NO_MEMORY);
    }
}


// Solver pre-setup
template<class T_Config>
void
MulticolorILUSolver_Base<T_Config>::pre_setup()
{
    // Check if matrix is colored
    if (this->m_explicit_A->getColoringLevel() < m_sparsity_level + 1)
    {
        FatalError("Matrix must be colored with coloring_level > sparsity_level for the multicolorILUsolver", AMGX_ERR_CONFIGURATION);
    }

    // Compute extended sparsity pattern based on coloring and matrix A
    computeLUSparsityPattern();

    if (this->m_LU.hasProps(DIAG))
    {
        FatalError("Multicolor ILU smoother does not support outside diagonal. Try setting reorder_cols_by_color=1 and insert_diag_while_reordering=1 in the multicolor_ilu solver scope in configuration file", AMGX_ERR_NOT_IMPLEMENTED);
    }

    if (m_sparsity_level == 0 && !this->m_LU.getColsReorderedByColor())
    {
        FatalError("Multicolor ILU smoother requires matrix to be reordered by color with ILU0 solver. Try setting reorder_cols_by_color=1 and insert_diag_while_reordering=1 in the multicolor_ilu solver scope in configuration file", AMGX_ERR_NOT_IMPLEMENTED);
    }

    // Reorder the columns of LU by color
    if (m_sparsity_level != 0)
    {
        // Reorder columns of LU by color
        m_LU.reorderColumnsByColor(false);

        // Compute mapping between entries in A and entries in LU
        if (this->m_explicit_A->hasProps(DIAG))
        {
            m_A_to_LU_mapping.resize(this->m_explicit_A->get_num_nz() + this->m_explicit_A->get_num_rows());
        }
        else
        {
            m_A_to_LU_mapping.resize(this->m_explicit_A->get_num_nz());
        }

        computeAtoLUmapping();
    }

    int N = this->m_LU.get_num_rows() * this->m_LU.get_block_dimy();
    m_delta.resize(N);
    m_Delta.resize(N);
    m_Delta.set_block_dimy(this->m_explicit_A->get_block_dimy());
    m_Delta.set_block_dimx(1);
    m_delta.set_block_dimy(this->m_explicit_A->get_block_dimy());
    m_delta.set_block_dimx(1);
}

template<class T_Config>
void
MulticolorILUSolver_Base<T_Config>::printSolverParameters() const
{
    std::cout << "relaxation_factor = " << this->m_weight << std::endl;
    std::cout << "use_bsrxmv = " << this->m_use_bsrxmv << std::endl;
    std::cout << "ilu_sparsity_level = " << this->m_sparsity_level <<  std::endl;
}


// Solver setup
template<class T_Config>
void
MulticolorILUSolver_Base<T_Config>::solver_setup(bool reuse_matrix_structure)
{
    this->m_explicit_A = dynamic_cast<Matrix<T_Config>*>(this->m_A);

    if (!this->m_explicit_A)
    {
        FatalError("MulticolorILUSolver only works with explicit matrices", AMGX_ERR_INTERNAL);
    }

    if (this->m_explicit_A->getColoringLevel() < 1)
    {
        FatalError("Matrix must be colored to use multicolor ilu solver. Try setting: coloring_level=1 or coloring_level=2 in the configuration file", AMGX_ERR_NOT_IMPLEMENTED);
    }

    if (!reuse_matrix_structure)
    {
        this->pre_setup();
    }

    // Fill LU sparsity pattern
    fillLUValuesWithAValues();
    // Compute LU factors in place (update LU.values)
    computeLUFactors();
}

//
template<class T_Config>
void
MulticolorILUSolver_Base<T_Config>::solve_init( VVector &b, VVector &x, bool xIsZero )
{}

// Solve one iteration
template<class T_Config>
bool
MulticolorILUSolver_Base<T_Config>::solve_iteration( VVector &b, VVector &x, bool xIsZero )
{
    if ( !m_use_bsrxmv && (this->m_LU.get_block_dimx() == 4 && this->m_LU.get_block_dimy() == 4) )
    {
        smooth_4x4(b, x, xIsZero);
    }
    else
    {
        smooth_bxb(b, x, xIsZero);
    }

    // Do we converge ?
    return this->converged(b, x);
}

template<class T_Config>
void
MulticolorILUSolver_Base<T_Config>::solve_finalize( VVector &b, VVector &x )
{}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::smooth_4x4(const VVector &b, VVector &x, bool xIsZero)
{
    FatalError("Haven't implemented Multicolor DILU smoother for host format", AMGX_ERR_NOT_SUPPORTED_TARGET);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::smooth_4x4(const VVector &b, VVector &x, bool xIsZero)
{
    Matrix<TConfig_d> &m_LU = this->m_LU;
    Matrix<TConfig_d> &m_A = *this->m_explicit_A;
    int N = m_LU.get_num_rows() * m_LU.get_block_dimy();
    cudaCheckError();

    if (!m_LU.getColsReorderedByColor())
    {
        FatalError("ILU solver currently only works if columns are reordered by color. Try setting reordering_cols_by_color=1 in the multicolor_ilu solver scope in the configuration file", AMGX_ERR_NOT_IMPLEMENTED);
    }

    // ---------------------------------------------------------
    // Solving Lower triangular system, with identity diagonal
    // ---------------------------------------------------------
    const IndexType *LU_sorted_rows_by_color_ptr = m_LU.getMatrixColoring().getSortedRowsByColor().raw();
    int num_colors = this->m_LU.getMatrixColoring().getNumColors();

    for (int i = 0; i < num_colors; i++)
    {
        const IndexType color_offset = m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i];
        const IndexType num_rows_per_color = m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i + 1] - color_offset;
#ifdef EXPERIMENTAL_LU_FORWARD
        const int CtaSize = 128; // Number of threads per CTA
        const int nHalfWarps = CtaSize / 16;
        int GridSize = std::min( 2048, ( num_rows_per_color + nHalfWarps - 1 ) / nHalfWarps );

        if ( GridSize == 0 )
        {
            continue;    // if perfect coloring (color 0 has no vertices)
        }

        if ( this->m_explicit_A->getBlockFormat() == ROW_MAJOR )
        {
            if (m_A.hasProps(DIAG))
            {
                LU_forward_4x4_kernel_warp<IndexType, ValueTypeA, ValueTypeB, CtaSize, 4, true, true> <<< GridSize, CtaSize>>>(
                    m_LU.row_offsets.raw(),
                    m_LU.m_smaller_color_offsets.raw(),
                    m_LU.col_indices.raw(),
                    m_LU.values.raw(),
                    m_A.row_offsets.raw(),
                    m_A.col_indices.raw(),
                    m_A.values.raw(),
                    m_A.diag.raw(),
                    x.raw(),
                    b.raw(),
                    this->m_delta.raw(),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    xIsZero );
            }
            else
            {
                LU_forward_4x4_kernel_warp<IndexType, ValueTypeA, ValueTypeB, CtaSize, 4, true, false> <<< GridSize, CtaSize>>>(
                    m_LU.row_offsets.raw(),
                    m_LU.m_smaller_color_offsets.raw(),
                    m_LU.col_indices.raw(),
                    m_LU.values.raw(),
                    m_A.row_offsets.raw(),
                    m_A.col_indices.raw(),
                    m_A.values.raw(),
                    m_A.diag.raw(),
                    x.raw(),
                    b.raw(),
                    this->m_delta.raw(),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    xIsZero );
            }
        }
        else
        {
            // COL_MAJOR
            if (m_A.hasProps(DIAG))
            {
                LU_forward_4x4_kernel_warp<IndexType, ValueTypeA, ValueTypeB, CtaSize, 4, false, true> <<< GridSize, CtaSize>>>(
                    m_LU.row_offsets.raw(),
                    m_LU.m_smaller_color_offsets.raw(),
                    m_LU.col_indices.raw(),
                    m_LU.values.raw(),
                    m_A.row_offsets.raw(),
                    m_A.col_indices.raw(),
                    m_A.values.raw(),
                    m_A.diag.raw(),
                    x.raw(),
                    b.raw(),
                    this->m_delta.raw(),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    xIsZero );
            }
            else
            {
                LU_forward_4x4_kernel_warp<IndexType, ValueTypeA, ValueTypeB, CtaSize, 4, false, false> <<< GridSize, CtaSize>>>(
                    m_LU.row_offsets.raw(),
                    m_LU.m_smaller_color_offsets.raw(),
                    m_LU.col_indices.raw(),
                    m_LU.values.raw(),
                    m_A.row_offsets.raw(),
                    m_A.col_indices.raw(),
                    m_A.values.raw(),
                    m_A.diag.raw(),
                    x.raw(),
                    b.raw(),
                    this->m_delta.raw(),
                    LU_sorted_rows_by_color_ptr + color_offset,
                    num_rows_per_color,
                    i,
                    xIsZero );
            }
        }

#else
        const int CtaSize = 128;
        const int blockrows_per_cta = CtaSize / 4;
        const int GridSize = min( AMGX_GRID_MAX_SIZE, (int) (num_rows_per_color + blockrows_per_cta - 1) / blockrows_per_cta);

        if ( GridSize == 0 )
        {
            continue;    // if perfect coloring (color 0 has no vertices)
        }

        if (this->m_explicit_A->hasProps(DIAG))
        {
            FatalError("this implementation of LU forward solve does not support A with external diagonal", AMGX_ERR_NOT_IMPLEMENTED);
        }

        if (this->m_explicit_A->getBlockFormat() == ROW_MAJOR)
        {
            LU_forward_4x4_kernel<IndexType, ValueTypeA, ValueTypeB, blockrows_per_cta, 8, 4, true> <<< GridSize, CtaSize>>>
            (m_LU.row_offsets.raw(),
             m_LU.m_smaller_color_offsets.raw(),
             m_LU.col_indices.raw(),
             m_LU.values.raw(),
             m_A.row_offsets.raw(),
             m_A.col_indices.raw(),
             m_A.values.raw(),
             x.raw(),
             b.raw(),
             delta.raw(),
             LU_sorted_rows_by_color_ptr + color_offset,
             num_rows_per_color,
             i,
             xIsZero);
        }
        else
        {
            LU_forward_4x4_kernel<IndexType, ValueTypeA, ValueTypeB, blockrows_per_cta, 8, 4, false> <<< GridSize, CtaSize>>>
            (m_LU.row_offsets.raw(),
             m_LU.m_smaller_color_offsets.raw(),
             m_LU.col_indices.raw(),
             m_LU.values.raw(),
             m_A.row_offsets.raw(),
             m_A.col_indices.raw(),
             m_A.values.raw(),
             x.raw(),
             b.raw(),
             delta.raw(),
             LU_sorted_rows_by_color_ptr + color_offset,
             num_rows_per_color,
             i,
             xIsZero);
        }

#endif
        cudaCheckError();
    }

    // --------------------
    // Backward Sweep
    // --------------------
    for (int i = num_colors - 1; i >= 0; i--)
    {
        const IndexType color_offset = m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i];
        const IndexType num_rows_per_color = m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i + 1] - color_offset;
#ifdef EXPERIMENTAL_LU_BACKWARD
        const int CtaSize = 128; // Number of threads per CTA
        const int nHalfWarps = CtaSize / 16;
        int GridSize = std::min( 2048, ( num_rows_per_color + nHalfWarps - 1 ) / nHalfWarps );

        if ( GridSize == 0 )
        {
            continue;    // if perfect coloring (color 0 has no vertices)
        }

        if (this->m_explicit_A->getBlockFormat() == ROW_MAJOR)
        {
            LU_backward_4x4_kernel_warp<IndexType, ValueTypeA, ValueTypeB, CtaSize, true> <<< GridSize, CtaSize>>>(
                m_LU.row_offsets.raw(),
                m_LU.m_larger_color_offsets.raw(),
                m_LU.col_indices.raw(),
                m_LU.diag.raw(),
                m_LU.values.raw(),
                this->m_delta.raw(),
                this->m_Delta.raw(),
                x.raw(),
                LU_sorted_rows_by_color_ptr + color_offset,
                num_rows_per_color,
                i,
                num_colors,
                this->m_weight,
                xIsZero);
        }
        else
        {
            LU_backward_4x4_kernel_warp<IndexType, ValueTypeA, ValueTypeB, CtaSize, false> <<< GridSize, CtaSize>>>(
                m_LU.row_offsets.raw(),
                m_LU.m_larger_color_offsets.raw(),
                m_LU.col_indices.raw(),
                m_LU.diag.raw(),
                m_LU.values.raw(),
                this->m_delta.raw(),
                this->m_Delta.raw(),
                x.raw(),
                LU_sorted_rows_by_color_ptr + color_offset,
                num_rows_per_color,
                i,
                num_colors,
                this->m_weight,
                xIsZero);
        }

#else
        const int CtaSize = 128;
        const int blockrows_per_cta = CtaSize / 4;
        const int GridSize = min( AMGX_GRID_MAX_SIZE, (int) (num_rows_per_color + blockrows_per_cta - 1) / blockrows_per_cta);

        if ( GridSize == 0 )
        {
            continue;    // if perfect coloring (color 0 has no vertices)
        }

        if (this->m_explicit_A->getBlockFormat() == ROW_MAJOR)
        {
            LU_backward_4x4_kernel<IndexType, ValueTypeA, ValueTypeB, blockrows_per_cta, 8, 4, true> <<< GridSize, CtaSize>>>
            (m_LU.row_offsets.raw(),
             m_LU.m_larger_color_offsets.raw(),
             m_LU.col_indices.raw(),
             m_LU.diag.raw(),
             m_LU.values.raw(),
             this->m_delta.raw(),
             this->m_Delta.raw(),
             x.raw(),
             LU_sorted_rows_by_color_ptr + color_offset,
             num_rows_per_color,
             i,
             num_colors,
             this->m_weight,
             xIsZero);
        }
        else
        {
            LU_backward_4x4_kernel<IndexType, ValueTypeA, ValueTypeB, blockrows_per_cta, 8, 4, false> <<< GridSize, CtaSize>>>
            (m_LU.row_offsets.raw(),
             m_LU.m_larger_color_offsets.raw(),
             m_LU.col_indices.raw(),
             m_LU.diag.raw(),
             m_LU.values.raw(),
             this->m_delta.raw(),
             this->m_Delta.raw(),
             x.raw(),
             LU_sorted_rows_by_color_ptr + color_offset,
             num_rows_per_color,
             i,
             num_colors,
             this->m_weight,
             xIsZero);
        }

#endif
    }

    cudaCheckError();
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_host, t_vecPrec, t_matPrec, t_indPrec> >::smooth_bxb(const VVector &b, VVector &x, bool xIsZero)
{
    FatalError("Haven't implemented Multicolor DILU smoother for host format", AMGX_ERR_NOT_SUPPORTED_TARGET);
}

template <AMGX_VecPrecision t_vecPrec, AMGX_MatPrecision t_matPrec, AMGX_IndPrecision t_indPrec>
void MulticolorILUSolver<TemplateConfig<AMGX_device, t_vecPrec, t_matPrec, t_indPrec> >::smooth_bxb(const VVector &b, VVector &x, bool xIsZero)
{
    Matrix<TConfig_d> &m_LU = this->m_LU;
    Matrix<TConfig_d> &m_A = *this->m_explicit_A;
    int N = m_LU.get_num_rows() * m_LU.get_block_dimy();

    if (!m_LU.getColsReorderedByColor())
    {
        FatalError("ILU solver currently only works if columns are reordered by color. Try setting reorder_cols_by_color=1 in the multicolor_ilu solver scope in the configuration file", AMGX_ERR_NOT_IMPLEMENTED);
    }

    if (this->m_explicit_A->getBlockFormat() == COL_MAJOR)
    {
        FatalError("ILU solver for arbitrary block sizes only works with ROW_MAJOR matrices", AMGX_ERR_NOT_IMPLEMENTED);
    }

    // ---------------------------------------------------------
    // Solving Lower triangular system, with identity diagonal
    // ---------------------------------------------------------
    const IndexType *LU_sorted_rows_by_color_ptr = m_LU.getMatrixColoring().getSortedRowsByColor().raw();
    int num_colors = this->m_LU.getMatrixColoring().getNumColors();
    //delta = b;
    thrust::copy(b.begin(), b.end(), this->m_delta.begin());
    //delta = delta - Ax;
    Cusparse::bsrmv((ValueTypeA) - 1.0, m_A, x, (ValueTypeA)1.0, this->m_delta);
    cudaCheckError();
    // Setting Delta to zero
    thrust::fill(this->m_Delta.begin(), this->m_Delta.end(), (ValueTypeB)0.0f);
    cudaCheckError();
    bool skipped_end = false;

    for (int i = 0; i < num_colors; i++)
    {
        const IndexType color_offset = m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i];
        const IndexType num_rows_per_color = m_LU.getMatrixColoring().getOffsetsRowsPerColor()[i + 1] - color_offset;

        if (num_rows_per_color == 0) { continue; } // if perfect coloring (color 0 has no vertices)

        if (skipped_end)
        {
            // delta = delta - LU*Delta smaller colors
            Cusparse::bsrmv(Cusparse::SMALLER_COLORS, i, (ValueTypeA) - 1.0f, m_LU, this->m_delta, (ValueTypeA)1.0f, this->m_delta);
        }

        if (num_rows_per_color > 0)
        {
            skipped_end = true;
        }
    }

    cudaCheckError();
    skipped_end = false;

    // --------------------
    // Backward Sweep
    // --------------------
    for (int i = num_colors - 1; i >= 0; i--)
    {
        // delta = delta - LU*Delta larger colors
        Cusparse::bsrmv(Cusparse::LARGER_COLORS, i, (ValueTypeA) - 1.0f, m_LU, this->m_Delta, (ValueTypeA)1.0f, this->m_delta);
        // Multiple by inverse stored on diagonal
        Cusparse::bsrmv(Cusparse::DIAG_COL, i, (ValueTypeA) 1.0f, m_LU, this->m_delta, 0.0f, this->m_Delta);
    }

    cudaCheckError();
    axpy(this->m_Delta, x, this->m_weight, 0, x.size());
    cudaCheckError();
}



/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class MulticolorILUSolver_Base<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

#define AMGX_CASE_LINE(CASE) template class MulticolorILUSolver<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE

}
} // namespace amgx
