#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2011 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

// This test is intended to check that the distributed matrix upload API
// accepts host, host registered, device, device managed pointers

#include "unit_test.h"
#include "amgx_c.h"
#include "amgxP_c.h"

namespace amgx

{

void preamble(
    AMGX_matrix_handle& A,
    AMGX_distribution_handle& dist,
    AMGX_resources_handle& rsrc,
    int& nrows,
    int& nnz,
    std::vector<int>& rows,
    std::vector<int>& cols)
{
    AMGX_finalize();

    nrows = 10;
    nnz = nrows*nrows;

    rows.resize(nrows+1);
    cols.resize(nnz);

    rows[0] = 0;
    for(int i = 0; i < nrows; ++i)
    {
        rows[i+1] = (i+1)*nrows;
        for(int j = 0; j < nrows; ++j)
        {
            cols[i*nrows + j] = j;
        }
    }

    std::string config_string;
    config_string="config_version=2, ";
    config_string+="solver(slv)=PCG, ";
    config_string+="slv:preconditioner(amg)=NOSOLVER, ";
    config_string+="slv:print_solve_stats=1, ";
    config_string+="slv:obtain_timings=1, ";
    config_string+="slv:max_iters=100, ";
    config_string+="slv:monitor_residual=1, ";
    config_string+="slv:convergence=ABSOLUTE, ";
    config_string+="slv:tolerance=1e-07, ";
    config_string+="slv:norm=L2";

    int dev = 0;
    hipSetDevice(dev);

    AMGX_initialize();

    AMGX_config_handle cfg;
    AMGX_config_create(&cfg, config_string.c_str());
    AMGX_config_add_parameters(&cfg, "exception_handling=1");

    AMGX_resources_create_simple(&rsrc, cfg);

    AMGX_Mode mode = AMGX_mode_dDDI;
    AMGX_matrix_create(&A, rsrc, mode);

    AMGX_distribution_create(&dist, cfg);
    AMGX_distribution_set_32bit_colindices(dist, true);
    AMGX_distribution_set_partition_data(dist, AMGX_DIST_PARTITION_VECTOR, NULL);
}

void cleanup(AMGX_matrix_handle& A, AMGX_resources_handle& rsrc)
{
    AMGX_matrix_destroy(A);
    AMGX_resources_destroy(rsrc);
    AMGX_finalize();
}

DECLARE_UNITTEST_BEGIN(CAPIUploadCudaMalloc);

void run()
{
    AMGX_matrix_handle A;
    AMGX_distribution_handle dist;
    AMGX_resources_handle rsrc;
    int nrows; 
    int nnz;

    std::vector<int> rows_h;
    std::vector<int> cols_h;
    preamble(A, dist, rsrc, nrows, nnz, rows_h, cols_h);

    int* rows;
    hipMalloc(&rows, sizeof(int)*(nrows+1));
    hipMemcpy(rows, rows_h.data(), sizeof(int)*(nrows+1), hipMemcpyDefault);

    int* cols;
    hipMalloc(&cols, sizeof(int)*nnz);
    hipMemcpy(cols, cols_h.data(), sizeof(int)*nnz, hipMemcpyDefault);

    double* vals;
    hipMalloc(&vals, sizeof(double)*nnz);

    UNITTEST_ASSERT_EQUAL(AMGX_matrix_upload_distributed(A, nrows, nrows, nnz, 1, 1, rows, cols, vals, NULL, dist), AMGX_RC_OK);

    hipFree(rows);
    hipFree(cols);

    cleanup(A, rsrc);
}

DECLARE_UNITTEST_END(CAPIUploadCudaMalloc);

DECLARE_UNITTEST_BEGIN(CAPIUploadCudaMallocManaged);

void run()
{
    AMGX_matrix_handle A;
    AMGX_distribution_handle dist;
    AMGX_resources_handle rsrc;
    int nrows;
    int nnz;

    std::vector<int> rows_h;
    std::vector<int> cols_h;
    preamble(A, dist, rsrc, nrows, nnz, rows_h, cols_h);

    int* rows;
    hipMallocManaged(&rows, sizeof(int)*(nrows+1));
    hipMemcpy(rows, rows_h.data(), sizeof(int)*(nrows+1), hipMemcpyDefault);

    int* cols;
    hipMallocManaged(&cols, sizeof(int)*nnz);
    hipMemcpy(cols, cols_h.data(), sizeof(int)*nnz, hipMemcpyDefault);

    double* vals;
    hipMallocManaged(&vals, sizeof(double)*nnz);

    UNITTEST_ASSERT_EQUAL(AMGX_matrix_upload_distributed(A, nrows, nrows, nnz, 1, 1, rows, cols, vals, NULL, dist), AMGX_RC_OK);

    hipFree(rows);
    hipFree(cols);
    cleanup(A, rsrc);
}

DECLARE_UNITTEST_END(CAPIUploadCudaMallocManaged);

DECLARE_UNITTEST_BEGIN(CAPIUploadNew);

void run()
{
    AMGX_matrix_handle A;
    AMGX_distribution_handle dist;
    AMGX_resources_handle rsrc;
    int nrows;
    int nnz;

    std::vector<int> rows_h;
    std::vector<int> cols_h;
    preamble(A, dist, rsrc, nrows, nnz, rows_h, cols_h);

    double* vals = new double[nnz];

    UNITTEST_ASSERT_EQUAL(AMGX_matrix_upload_distributed(A, nrows, nrows, nnz, 1, 1, rows_h.data(), cols_h.data(), vals, NULL, dist), AMGX_RC_OK);
    cleanup(A, rsrc);
}

DECLARE_UNITTEST_END(CAPIUploadNew);

DECLARE_UNITTEST_BEGIN(CAPIUploadCudaHostRegister);

void run()
{
    AMGX_matrix_handle A;
    AMGX_distribution_handle dist;
    AMGX_resources_handle rsrc;
    int nrows; 
    int nnz;

    std::vector<int> rows_h;
    std::vector<int> cols_h;
    preamble(A, dist, rsrc, nrows, nnz, rows_h, cols_h);

    hipHostRegister(rows_h.data(), sizeof(int)*(nrows+1), hipHostRegisterDefault);

    hipHostRegister(cols_h.data(), sizeof(int)*nnz, hipHostRegisterDefault);

    double* vals = new double[nnz];
    hipHostRegister(&vals, sizeof(double)*nnz, hipHostRegisterDefault);

    UNITTEST_ASSERT_EQUAL(AMGX_matrix_upload_distributed(A, nrows, nrows, nnz, 1, 1, rows_h.data(), cols_h.data(), vals, NULL, dist), AMGX_RC_OK);
    cleanup(A, rsrc);
}

DECLARE_UNITTEST_END(CAPIUploadCudaHostRegister);

DECLARE_UNITTEST_BEGIN(CAPIUploadCudaMallocHost);

void run()
{
    AMGX_matrix_handle A;
    AMGX_distribution_handle dist;
    AMGX_resources_handle rsrc;
    int nrows; 
    int nnz;

    std::vector<int> rows_h;
    std::vector<int> cols_h;
    preamble(A, dist, rsrc, nrows, nnz, rows_h, cols_h);

    int* rows;
    hipHostMalloc(&rows, sizeof(int)*(nrows+1));
    memcpy(rows, rows_h.data(), sizeof(int)*(nrows+1));

    int* cols;
    hipHostMalloc(&cols, sizeof(int)*nnz);
    memcpy(cols, cols_h.data(), sizeof(int)*nnz);

    double* vals;
    hipHostMalloc(&vals, sizeof(double)*nnz);

    UNITTEST_ASSERT_EQUAL(AMGX_matrix_upload_distributed(A, nrows, nrows, nnz, 1, 1, rows, cols, vals, NULL, dist), AMGX_RC_OK);

    hipHostFree(rows);
    hipHostFree(cols);
    cleanup(A, rsrc);
}

DECLARE_UNITTEST_END(CAPIUploadCudaMallocHost);

// or you can specify several desired configs
CAPIUploadCudaMallocHost <TemplateMode<AMGX_mode_dDDI>::Type>  CAPIUploadCudaMallocHost_dDDI;
CAPIUploadCudaHostRegister <TemplateMode<AMGX_mode_dDDI>::Type>  CAPIUploadCudaHostRegister_dDDI;
CAPIUploadNew <TemplateMode<AMGX_mode_dDDI>::Type>  CAPIUploadNew_dDDI;
CAPIUploadCudaMallocManaged <TemplateMode<AMGX_mode_dDDI>::Type>  CAPIUploadCudaMallocManaged_dDDI;
CAPIUploadCudaMalloc <TemplateMode<AMGX_mode_dDDI>::Type>  CAPIUploadCudaMalloc_dDDI;

} //namespace amgx
