#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "unit_test.h"
#include "core.h"
#include "amg_config.h"
#include "test_utils.h"
#include "util.h"
#include "cutil.h"
#include "amg_solver.h"
#include "resources.h"

#include "aggregation/coarseAgenerators/coarse_A_generator.h"
#include "aggregation/selectors/agg_selector.h"
#include "matrix_coloring/matrix_coloring.h"
#include "matrix_coloring/min_max.h"
#include "solvers/solver.h"

#include "classical/selectors/selector.h"
#include "classical/interpolators/interpolator.h"
#include "classical/strength/strength.h"

#include <cusp/print.h>
#include <cusp/gallery/poisson.h>

namespace amgx

{

DECLARE_UNITTEST_BEGIN(LargeMatricesSupport);

typedef typename TConfig_h::template setVecPrec<(AMGX_VecPrecision)AMGX_GET_MODE_VAL(AMGX_MatPrecision, TConfig::mode)>::Type vvec_h;
typedef typename TConfig::template setVecPrec<AMGX_vecInt>::Type ivec;
typedef typename TConfig_h::template setVecPrec<AMGX_vecInt>::Type ivec_h;

// setup restriction on HOST
void fillRowOffsetsAndColIndices(const int num_aggregates,
                                 Vector<ivec_h> aggregates,
                                 const int R_num_cols,
                                 Vector<ivec_h> &R_row_offsets,
                                 Vector<ivec_h> &R_col_indices)
{
    for (int i = 0; i < num_aggregates + 1; i++)
    {
        R_row_offsets[i] = 0;
    }

    // Count number of neighbors for each row
    for (int i = 0; i < R_num_cols; i++)
    {
        int I = aggregates[i];
        R_row_offsets[I]++;
    }

    R_row_offsets[num_aggregates] = R_num_cols;

    for (int i = num_aggregates - 1; i >= 0; i--)
    {
        R_row_offsets[i] = R_row_offsets[i + 1] - R_row_offsets[i];
    }

    /* Set column indices. */
    for (int i = 0; i < R_num_cols; i++)
    {
        int I = aggregates[i];
        int Ip = R_row_offsets[I]++;
        R_col_indices[Ip] = i;
    }

    /* Reset r[i] to start of row memory. */
    for (int i = num_aggregates - 1; i > 0; i--)
    {
        R_row_offsets[i] = R_row_offsets[i - 1];
    }

    R_row_offsets[0] = 0;
}

void test_coarsers(Matrix<T_Config> &A, AMG_Config &cfg, const std::string &cfg_scope)
{
    Matrix<T_Config> Ac;
    int num_aggregates = A.get_num_rows();
    Vector<ivec_h> h_aggregates;
    h_aggregates.resize( A.get_num_rows() );

    for ( int i = 0; i < h_aggregates.size(); i++ )
    {
        h_aggregates[i] = i;
    }

    Vector<ivec_h> h_R_row_offsets;
    Vector<ivec_h> h_R_col_indices;
    h_R_row_offsets.resize( num_aggregates + 1 );
    h_R_col_indices.resize( A.get_num_rows() );
    fillRowOffsetsAndColIndices( num_aggregates, h_aggregates, A.get_num_rows(), h_R_row_offsets, h_R_col_indices );
    Vector<ivec> aggregates = h_aggregates;
    Vector<ivec> R_row_offsets = h_R_row_offsets;
    Vector<ivec> R_col_indices = h_R_col_indices;
    cudaCheckError();
    typename aggregation::CoarseAGeneratorFactory<T_Config>::Iterator iter = aggregation::CoarseAGeneratorFactory<T_Config>::getIterator();
    aggregation::CoarseAGenerator<TConfig> *generator;

    while (!aggregation::CoarseAGeneratorFactory<T_Config>::isIteratorLast(iter))
    {
        generator = NULL;
        generator = iter->second->create(cfg, cfg_scope);
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        UNITTEST_ASSERT_TRUE_DESC("Generator is not created\n", generator != NULL);
        UNITTEST_ASSERT_EXCEPTION_START;
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        generator->computeAOperator(A, Ac, aggregates, R_row_offsets, R_col_indices, num_aggregates);
        UNITTEST_ASSERT_TRUE_DESC("Coarser matrix contains nans\n", !containsNan<ValueTypeA>(Ac.values.raw(), Ac.values.size()));
        UNITTEST_ASSERT_EXCEPTION_END_NOT_IMPLEMENTED;

        if (generator != NULL) { delete generator; }

        ++iter;
        UNITTEST_ASSERT_TRUE(true);
    }
}

void test_selectors(Matrix<T_Config> &A, AMG_Config &cfg, const std::string &cfg_scope)
{
    typename aggregation::SelectorFactory<T_Config>::Iterator iter = aggregation::SelectorFactory<T_Config>::getIterator();
    aggregation::Selector<TConfig> *selector;
    IVector vec, vec1;
    int num;

    while (!aggregation::SelectorFactory<T_Config>::isIteratorLast(iter))
    {
        selector = NULL;
        PrintOnFail("processing: %s\n", iter->first.c_str());
        selector = iter->second->create(cfg, cfg_scope);
        PrintOnFail("Selector creation\n");
        UNITTEST_ASSERT_TRUE(selector != NULL);
        UNITTEST_ASSERT_EXCEPTION_START;
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        selector->setAggregates(A, vec, vec1, num);
        UNITTEST_ASSERT_EXCEPTION_END_NOT_IMPLEMENTED;

        if (selector != NULL) { delete selector; }

        ++iter;
        UNITTEST_ASSERT_TRUE(true);
    }
}

void test_matrix_coloring(Matrix<T_Config> &A, AMG_Config &cfg, const std::string &cfg_scope)
{
    MatrixColoring<TConfig> *color;
    typename MatrixColoringFactory<T_Config>::Iterator iter = MatrixColoringFactory<T_Config>::getIterator();

    while (!MatrixColoringFactory<T_Config>::isIteratorLast(iter))
    {
        color = NULL;
        UNITTEST_ASSERT_EXCEPTION_START;
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        color = iter->second->create(cfg, cfg_scope);
        UNITTEST_ASSERT_TRUE(color != NULL);
        A.colorMatrix(cfg, cfg_scope);
        int num_colors = A.getMatrixColoring().getNumColors();
        UNITTEST_ASSERT_TRUE(num_colors != 0);
        UNITTEST_ASSERT_EXCEPTION_END_NOT_IMPLEMENTED;

        if (color != NULL) { delete color; }

        ++iter;
        UNITTEST_ASSERT_TRUE(true);
    }
}

void test_solvers(Matrix<T_Config> &A, AMG_Config &cfg, const std::string &cfg_scope)
{
    Vector<T_Config> b (A.get_num_rows()*A.get_block_dimy()), x (A.get_num_rows()*A.get_block_dimy());
    cusp::blas::fill(b, 1);
    Solver<TConfig> *solver;
    typename SolverFactory<T_Config>::Iterator iter = SolverFactory<T_Config>::getIterator();

    while (!SolverFactory<T_Config>::isIteratorLast(iter))
    {
        solver = NULL;
        thrust::fill(x.begin(), x.end(), static_cast<ValueTypeB>(1.0));
        UNITTEST_ASSERT_EXCEPTION_START;
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        solver = iter->second->create(cfg, cfg_scope);

        if (solver != NULL)
        {
            solver->setup(A, false);
            solver->set_max_iters(1);
            solver->solve(b, x, false);
            UNITTEST_ASSERT_TRUE_DESC("Smoother result contains nans\n", !containsNan<ValueTypeB>(x.raw(), x.size()));
        }

        UNITTEST_ASSERT_EXCEPTION_END_NOT_IMPLEMENTED;

        if (solver != NULL) { delete solver; }

        ++iter;
        UNITTEST_ASSERT_TRUE(true);
    }
}

void generatePoissonForTest(Matrix<TConfig > &Aout, int block_size, bool diag_prop, int points, int x, int y, int z = 1)
{
    Matrix<TConfig_h > Ac;
    {
        Matrix<TConfig_h > A;
        A.set_initialized(0);
        A.addProps(CSR);
        MatrixCusp<TConfig_h, cusp::csr_format> wA(&A);

        switch (points)
        {
            case 5:
                cusp::gallery::poisson5pt(wA, x, y);
                break;

            case 7:
                cusp::gallery::poisson7pt(wA, x, y, z);
                break;

            case 9:
                cusp::gallery::poisson9pt(wA, x, y);
                break;

            case 27:
                cusp::gallery::poisson27pt(wA, x, y, z);
                break;
        }

        A.set_initialized(1);
        Ac.convert( A, ( diag_prop ? DIAG : 0 ) | CSR, block_size, block_size );
        Ac.set_initialized(1);
    }
    Aout = Ac;
}

void test_levels(Resources *res, Matrix<T_Config> &A)
{
    Vector<T_Config> b (A.get_num_rows()*A.get_block_dimy()), x (A.get_num_rows()*A.get_block_dimy());
    cusp::blas::fill(b, 1);
    cusp::blas::fill(x, 1);
    int bsize = A.get_block_dimy();
    b.set_block_dimx(1);
    b.set_block_dimy(bsize);
    x.set_block_dimy(1);
    x.set_block_dimx(bsize);
    AMGX_STATUS solve_status;
    {
        AMG_Configuration cfg;
        AMGX_ERROR err = AMGX_OK;
        UNITTEST_ASSERT_TRUE( cfg.parseParameterString("algorithm=CLASSICAL, smoother=MULTICOLOR_DILU, presweeps=1, postsweeps=1, matrix_coloring_scheme=MIN_MAX, determinism_flag=1, max_levels=2, max_iters=1, norm=L1") == AMGX_OK);
        AMG_Solver<TConfig> amg(res, cfg);
        err = amg.setup(A);

        if (err != AMGX_ERR_NOT_SUPPORTED_TARGET && err != AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE && err != AMGX_ERR_NOT_IMPLEMENTED)
        {
            PrintOnFail("Classical algorithm: Matrix properties: blocksize = %d, diag_prop = %d\n", A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
            UNITTEST_ASSERT_EQUAL(err, AMGX_OK);
            err = amg.solve( b, x, solve_status, true);

            if (err != AMGX_ERR_NOT_SUPPORTED_TARGET && err != AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE && err != AMGX_ERR_NOT_IMPLEMENTED)
            {
                PrintOnFail("Classical algorithm: Matrix properties: blocksize = %d, diag_prop = %d\n", A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
                UNITTEST_ASSERT_EQUAL(err, AMGX_OK);
                PrintOnFail("Classical algorithm: Matrix properties: blocksize = %d, diag_prop = %d\n", A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
                UNITTEST_ASSERT_TRUE_DESC("Level solve result contains nans\n", !containsNan<ValueTypeB>(x.raw(), x.size()));
            }
        }
    }
    cusp::blas::fill(x, 1);
    {
        AMG_Configuration cfg;
        AMGX_ERROR err = AMGX_OK;
        UNITTEST_ASSERT_TRUE( cfg.parseParameterString("algorithm=AGGREGATION, smoother=MULTICOLOR_DILU, presweeps=1, postsweeps=1, selector=SIZE_4, coarseAgenerator=LOW_DEG, matrix_coloring_scheme=MIN_MAX, determinism_flag=1, max_levels=2, max_iters=1, norm=L1") == AMGX_OK);
        AMG_Solver<TConfig> amg(res, cfg);
        err = amg.setup(A);

        if (err != AMGX_ERR_NOT_SUPPORTED_TARGET && err != AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE && err != AMGX_ERR_NOT_IMPLEMENTED)
        {
            PrintOnFail("Aggregation algorithm: Matrix properties: blocksize = %d, diag_prop = %d\n", A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
            UNITTEST_ASSERT_EQUAL(err, AMGX_OK);
            err = amg.solve( b, x, solve_status, true);

            if (err != AMGX_ERR_NOT_SUPPORTED_TARGET && err != AMGX_ERR_NOT_SUPPORTED_BLOCKSIZE && err != AMGX_ERR_NOT_IMPLEMENTED)
            {
                PrintOnFail("Aggregation algorithm: Matrix properties: blocksize = %d, diag_prop = %d\n", A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
                UNITTEST_ASSERT_EQUAL(err, AMGX_OK);
                PrintOnFail("Aggregation algorithm: Matrix properties: blocksize = %d, diag_prop = %d\n", A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
                UNITTEST_ASSERT_TRUE_DESC("Level solve result contains nans\n", !containsNan<ValueTypeB>(x.raw(), x.size()));
            }
        }
    }
}

void test_strength(Matrix<T_Config> &A, AMG_Config &cfg, const std::string &cfg_scope, StrengthFactory<TConfig> **good )
{
    //allocate necessary memory
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecInt>::Type> IVector;
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecBool>::Type> BVector;
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecFloat>::Type> FVector;
    FVector weights(A.get_num_rows(), 0.0);
    BVector s_con(A.get_num_nz(), false);
    IVector cf_map(A.get_num_rows(), 0);
    IVector scratch(A.get_num_rows(), 0); //scratch memory of size num_rows
    //compute strong connections and weights
    double max_row_sum = cfg.getParameter<double>("max_row_sum", cfg_scope);
    Strength<T_Config> *strength;
    typename StrengthFactory<T_Config>::Iterator iter = StrengthFactory<T_Config>::getIterator();

    while (!StrengthFactory<T_Config>::isIteratorLast(iter))
    {
        strength = NULL;
        UNITTEST_ASSERT_EXCEPTION_START;
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        strength = iter->second->create(cfg, cfg_scope);
        UNITTEST_ASSERT_TRUE(strength != NULL);

        if (strength != NULL)
        {
            strength->computeStrongConnectionsAndWeights(A, s_con, weights, max_row_sum);
            UNITTEST_ASSERT_TRUE_DESC("Strength result contains nans\n", !containsNan<float>(weights.raw(), weights.size()));
            *good = iter->second;
        }

        UNITTEST_ASSERT_EXCEPTION_END_NOT_IMPLEMENTED;

        if (strength != NULL) { delete strength; }

        ++iter;
        UNITTEST_ASSERT_TRUE(true);
    }
}

void test_selectors(Matrix<T_Config> &A, AMG_Config &cfg, const std::string &cfg_scope, StrengthFactory<TConfig> *strengthf, classical::SelectorFactory<TConfig> **good )
{
    //allocate necessary memory
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecInt>::Type> IVector;
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecBool>::Type> BVector;
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecFloat>::Type> FVector;
    FVector weights(A.get_num_rows(), 0.0);
    BVector s_con(A.get_num_nz(), false);
    IVector cf_map(A.get_num_rows(), 0);
    IVector scratch(A.get_num_rows(), 0); //scratch memory of size num_rows
    //compute strong connections and weights
    double max_row_sum = cfg.getParameter<double>("max_row_sum", cfg_scope);
    Strength<T_Config> *strength = strengthf->create(cfg, cfg_scope);
    strength->computeStrongConnectionsAndWeights(A, s_con, weights, max_row_sum);
    classical::Selector<T_Config> *selector;
    typename classical::SelectorFactory<T_Config>::Iterator iter = classical::SelectorFactory<T_Config>::getIterator();

    while (!classical::SelectorFactory<T_Config>::isIteratorLast(iter))
    {
        selector = NULL;
        UNITTEST_ASSERT_EXCEPTION_START;
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        selector = iter->second->create();
        UNITTEST_ASSERT_TRUE(strength != NULL);

        if (selector != NULL)
        {
            selector->markCoarseFinePoints(A, weights, s_con, cf_map, scratch);

            for (int i = 0; i < A.get_num_rows(); i++)
            {
                UNITTEST_ASSERT_TRUE(cf_map[i] != UNASSIGNED);
            }

            *good = iter->second;
        }

        UNITTEST_ASSERT_EXCEPTION_END_NOT_IMPLEMENTED;

        if (selector != NULL) { delete selector; }

        ++iter;
        UNITTEST_ASSERT_TRUE(true);
    }
}

void test_interpolators(Resources *res, Matrix<T_Config> &A, AMG_Config &cfg, const std::string &cfg_scope, StrengthFactory<TConfig> *strengthf, classical::SelectorFactory<TConfig> *selectorf )
{
    //allocate necessary memory
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecInt>::Type> IVector;
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecBool>::Type> BVector;
    typedef Vector<typename T_Config::template setVecPrec<AMGX_vecFloat>::Type> FVector;
    Matrix<TConfig> P;
    FVector weights(A.get_num_rows(), 0.0);
    BVector s_con(A.get_num_nz(), false);
    IVector cf_map(A.get_num_rows(), 0);
    IVector scratch(A.get_num_rows(), 0); //scratch memory of size num_rows
    //compute strong connections and weights
    double max_row_sum = cfg.getParameter<double>("max_row_sum", cfg_scope);
    Strength<T_Config> *strength = strengthf->create(cfg, cfg_scope);
    classical::Selector<T_Config> *selector = selectorf->create();
    strength->computeStrongConnectionsAndWeights(A, s_con, weights, max_row_sum);
    selector->markCoarseFinePoints(A, weights, s_con, cf_map, scratch);
    Interpolator<T_Config> *interpolator;
    typename InterpolatorFactory<T_Config>::Iterator iter = InterpolatorFactory<T_Config>::getIterator();
    AMG_Configuration scfg;
    AMG_Solver<TConfig> amg(res, scfg);

    while (!InterpolatorFactory<T_Config>::isIteratorLast(iter))
    {
        interpolator = NULL;
        UNITTEST_ASSERT_EXCEPTION_START;
        PrintOnFail("%s : Matrix properties: blocksize = %d, diag_prop = %d\n", iter->first.c_str(), A.get_block_dimy(), (A.hasProps(DIAG) ? 1 : 0));
        interpolator = iter->second->create(cfg, cfg_scope);
        UNITTEST_ASSERT_TRUE(strength != NULL);

        if (interpolator != NULL)
        {
            interpolator->generateInterpolationMatrix(A, cf_map, s_con, scratch, P, &amg);
        }

        UNITTEST_ASSERT_EXCEPTION_END_NOT_IMPLEMENTED;

        if (interpolator != NULL) { delete interpolator; }

        ++iter;
        UNITTEST_ASSERT_TRUE(true);
    }
}


void run()
{
    cudaCheckError();
    randomize( 30 );
    set_forge_ahead(true);
    int nrows_array[] = {8500000, 8500000, 8500002, 4250000}; // limited by gpu ram

    for (int bsize = 1; bsize < 1; ++bsize)
    {
        AMG_Config cfg;
        cfg.parseParameterString("determinism_flag=1");
        const std::string &cfg_scope = "default";
        Resources res;        // default resources
        {
            MatrixA A;
            generateMatrixRandomStruct<TConfig>::generateExact(A, nrows_array[bsize], true, bsize, false, 2);
            random_fill(A);
// aggregation
            test_coarsers(A, cfg, cfg_scope);
            test_selectors(A, cfg, cfg_scope);
            test_matrix_coloring(A, cfg, cfg_scope);
            A.colorMatrix(cfg, cfg_scope);
            test_solvers(A, cfg, cfg_scope);
// classical
//TODO: if strength cannot process matrix
            StrengthFactory<TConfig> *good_strength = NULL;
            test_strength(A, cfg, cfg_scope, &good_strength);

            if (good_strength != NULL)
            {
                classical::SelectorFactory<TConfig> *good_selector = NULL;
                test_selectors(A, cfg, cfg_scope, good_strength, &good_selector);

                if (good_selector != NULL)
                {
                    test_interpolators(&res, A, cfg, cfg_scope, good_strength, good_selector );
                }
            }

// levels
            test_levels(&res, A);
        }
        {
            MatrixA A;
            generateMatrixRandomStruct<TConfig>::generateExact(A, nrows_array[bsize], false, bsize, false, 2);
            random_fill(A);
// aggregation
            test_coarsers(A, cfg, cfg_scope);
            test_selectors(A, cfg, cfg_scope);
            test_matrix_coloring(A, cfg, cfg_scope);
            test_solvers(A, cfg, cfg_scope);
// classical
//TODO: if strength cannot process matrix
            StrengthFactory<TConfig> *good_strength = NULL;
            test_strength(A, cfg, cfg_scope, &good_strength);

            if (good_strength != NULL)
            {
                classical::SelectorFactory<TConfig> *good_selector = NULL;
                test_selectors(A, cfg, cfg_scope, good_strength, &good_selector);

                if (good_selector != NULL)
                {
                    test_interpolators(&res, A, cfg, cfg_scope, good_strength, good_selector );
                }
            }

// levels
            test_levels(&res, A);
        }
    }
}

DECLARE_UNITTEST_END(LargeMatricesSupport);


#define AMGX_CASE_LINE(CASE) LargeMatricesSupport <TemplateMode<CASE>::Type>  LargeMatricesSupport_##CASE;
AMGX_FORALL_BUILDS_DEVICE(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE


} //namespace amgx
