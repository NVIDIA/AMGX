#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "unit_test.h"
#include "amg_solver.h"
#include <matrix_io.h>
#include "test_utils.h"
#include <multiply.h>
#include<blas.h>
#include <csr_multiply.h>
#include "util.h"
#include "time.h"
#include <sstream>

namespace amgx
{

// parameter is used as test name
DECLARE_UNITTEST_BEGIN(NestedSolvers);

void run_case(Resources *res, MatrixA &A_hd, VVector &b_hd, VVector &x_hd, const std::string &parameter_string, AMGX_ERROR &err)
{
    // this is incorrect resources/obects usage, might generate memory leaks, but for the sake of simplicity let it be
    AMG_Configuration cfg;
    UNITTEST_ASSERT_TRUE(cfg.parseParameterString(parameter_string.c_str()) == AMGX_OK);
    AMG_Solver<TConfig> amg(res, cfg);
    err = amg.setup(A_hd);
}


void run()
{
    int points = 27;
    int nx, ny, nz;
    nx = ny = nz = 20;
    // Create system
    Matrix_h A;
    Vector_h b;
    Vector_h x;
    A.set_initialized(0);
    A.addProps(CSR);
    MatrixCusp<TConfig_h, cusp::csr_format> wA(&A);

    switch (points)
    {
        case 5:
            cusp::gallery::poisson5pt(wA, nx, ny);
            break;

        case 7:
            cusp::gallery::poisson7pt(wA, nx, ny, nz);
            break;

        case 9:
            cusp::gallery::poisson9pt(wA, nx, ny);
            break;

        case 27:
            cusp::gallery::poisson27pt(wA, nx, ny, nz);
            break;

        default:
            printf("Error invalid number of poisson points specified, valid numbers are 5, 7, 9, 27\n");
    }

    A.computeDiagonal();
    A.set_initialized(1);
    int bsize = A.get_block_dimy();
    int n_rows = A.get_num_rows() * bsize;
    b.set_block_dimx(1);
    b.set_block_dimy(bsize);
    x.set_block_dimy(1);
    x.set_block_dimx(bsize);
    // Fill b
    b.resize(n_rows);
    thrust::fill(b.begin(), b.end(), 1.);
    // Initialize x
    x.resize(n_rows);
    thrust::fill(x.begin(), x.end(), 0.);
    // Copy to device if necessary
    MatrixA A_hd;
    VVector x_ini_hd, x_fin_inside_hd, x_fin_outside_hd, b_hd, r_hd;
    r_hd.resize(n_rows, 0.);
    r_hd.set_block_dimy(1);
    r_hd.set_block_dimx(bsize);
    A_hd = A;
    b_hd = b;
    x_ini_hd = x;
    AMGX_ERROR err;
    // Set parameters
    std::string error_string;
    std::stringstream parameter_string;
    Resources res;        // default resources
    // Should work fine
    parameter_string << "config_version=2, solver(s1)=FGMRES, s1:preconditioner(jacobi)=BLOCK_JACOBI, jacobi:max_iters=1, s1:max_iters=" << n_rows << ",s1:norm=L2, determinism_flag=1, s1:tolerance=1e-14, s1:gmres_n_restart=" << n_rows << ", s1:convergence=RELATIVE_INI_CORE, s1:monitor_residual=1";
    run_case(&res, A_hd, b_hd, x_ini_hd, parameter_string.str(), err);
    UNITTEST_ASSERT_TRUE(err == AMGX_OK);
    parameter_string.str("");
    parameter_string << " solver=FGMRES, preconditioner=FGMRES, max_iters=100";
    UNITTEST_ASSERT_EXCEPTION_START;
    run_case(&res, A_hd, b_hd, x_ini_hd, parameter_string.str(), err);
    UNITTEST_ASSERT_EXCEPTION_END_AMGX_ERR(AMGX_ERR_BAD_PARAMETERS);
    parameter_string.str("");
    parameter_string << " solver=AMG, smoother=FGMRES, max_iters=100";
    UNITTEST_ASSERT_EXCEPTION_START;
    run_case(&res, A_hd, b_hd, x_ini_hd, parameter_string.str(), err);
    UNITTEST_ASSERT_EXCEPTION_END_AMGX_ERR(AMGX_ERR_BAD_PARAMETERS);
    parameter_string.str("");
    parameter_string << " solver=AMG, coarse_solver=AMG, max_iters=100";
    UNITTEST_ASSERT_EXCEPTION_START
    run_case(&res, A_hd, b_hd, x_ini_hd, parameter_string.str(), err);
    UNITTEST_ASSERT_EXCEPTION_END_AMGX_ERR(AMGX_ERR_BAD_PARAMETERS);
}

DECLARE_UNITTEST_END(NestedSolvers);

NestedSolvers <TemplateMode<AMGX_mode_dDDI>::Type>  NestedSolvers_instance_mode_dDDI;


} //namespace amgx
