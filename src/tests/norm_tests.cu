#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "unit_test.h"
#include "vector.h"
#include "matrix.h"
#include "norm.h"
#include "ctime"

namespace amgx
{

DECLARE_UNITTEST_BEGIN(NormTests);


void test_get_norm(Vector_h &block_nrm, const Vector_h &vec, const NormType norm_type, int bdim = 1, int offset = 0)
{
    UNITTEST_ASSERT_TRUE_DESC("Only L1 and L2 are supported in this unit test", norm_type == L1 || norm_type == L2);
    block_nrm.resize(bdim, ValueTypeB(0));
    std::vector <typename Vector_h::value_type> norm(bdim, 0.l);

    if (norm_type == L1)
    {
        for (int i = 0; i < (vec.size() / bdim); i++)
            for (int j = 0; j < bdim; j++)
            {
                norm[j] += std::fabs(vec[(offset + i) * bdim + j]);
            }

        for (int j = 0; j < bdim; j++)
        {
            block_nrm[j] = norm[j];
        }
    }
    else if (norm_type == L2)
    {
        for (int i = 0; i < (vec.size() / bdim); i++)
            for (int j = 0; j < bdim; j++)
            {
                norm[j] += vec[(offset + i) * bdim + j] * vec[(offset + i) * bdim + j];
            }

        for (int j = 0; j < bdim; j++)
        {
            block_nrm[j] = sqrt(norm[j]);
        }
    }
}

void check_norm(const int size, const int bdim, const NormType norm_type)
{
    Matrix_h A;
    //Workaround to test large vector sizes:
    A.set_initialized(0);
    A.set_block_dimx(bdim);
    A.set_block_dimy(bdim);
    A.set_num_nz(size);
    A.set_num_rows(size);
    A.set_num_cols(size);
    A.set_initialized(1);
    //Matrix_h A(size,size,size, bdim, bdim, 0);
    //generateMatrixRandomStruct<TConfig_h>::generateExact(A, size_vec, true , bdim, false);
    int offset = 0;
    Vector_h vec(size);
    vec.set_block_dimx(bdim);
    fillRandom<Vector_h>::fill(vec);
    Matrix<TConfig> A_try(A);
    Vector<TConfig> vec_try(vec);
    Vector_h norm_ref(bdim), norm_try(bdim);
    test_get_norm(norm_ref, vec, norm_type, bdim, offset);
    get_norm( A_try, vec_try, bdim, norm_type, norm_try );
    this->PrintOnFail(": error in checking norm %s, blocksize %d, size %d\n", norm_type == L1 ? "L1" : "L2", bdim, size);
    // summing on gpu and host might produce different numbers due to order of summation for L1, tuning numbers a little bit
    UNITTEST_ASSERT_EQUAL_TOL(norm_ref, norm_try, getTolerance<typename Vector_h::value_type>::get()*(norm_type == L1 ? size : 1.)); 
}

void run()
{
    randomize( 10 );
    for (int bsize = 1; bsize <= 10; bsize ++)
    {
        int size = 10000 * bsize;
        check_norm(size, bsize, L1);
        check_norm(size, bsize, L2);
    }
}

DECLARE_UNITTEST_END(NormTests);

#define AMGX_CASE_LINE(CASE) NormTests <TemplateMode<CASE>::Type>  NormTests_##CASE;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE
} //namespace amgx
