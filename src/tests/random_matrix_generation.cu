#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: 2011 - 2024 NVIDIA CORPORATION. All Rights Reserved.
//
// SPDX-License-Identifier: BSD-3-Clause

#include "unit_test.h"
#include <matrix_io.h>
#include "test_utils.h"
#include "util.h"
#include "time.h"

namespace amgx

{

// parameter is used as test name
DECLARE_UNITTEST_BEGIN(RandomMatrix);

void checkMatrix(Matrix_h &A)
{
    IVector vec1, vec2;
    int bsize = A.get_block_dimx();
    UNITTEST_ASSERT_TRUE( (A.get_num_rows() <= 10000) && (A.get_num_rows() > 0) );
    UNITTEST_ASSERT_TRUE( (A.get_num_cols() <= 10000) && (A.get_num_cols() > 0) );
    UNITTEST_ASSERT_EQUAL( A.get_num_cols(), A.get_num_rows() );
    //UNITTEST_ASSERT_TRUE( A.get_num_nz() <= 10*A.get_num_rows() );
    UNITTEST_ASSERT_EQUAL( A.get_block_size(), bsize * bsize ); // only square blocks?
    UNITTEST_ASSERT_EQUAL( A.get_block_dimx(), bsize );
    UNITTEST_ASSERT_EQUAL( A.get_block_dimy(), bsize );
    UNITTEST_ASSERT_EQUAL( A.values.size(), ((A.get_num_nz() + (A.hasProps(DIAG) ? A.get_num_rows() : 1))*bsize * bsize));
    UNITTEST_ASSERT_EQUAL( A.row_offsets.size(), (A.get_num_rows() + 1) );
    UNITTEST_ASSERT_EQUAL( A.col_indices.size(), A.get_num_nz() );
    int num_rows = A.get_num_rows();
    int num_nz = A.get_num_nz();
    UNITTEST_ASSERT_EQUAL( A.row_offsets[0], 0 );
    UNITTEST_ASSERT_EQUAL( A.row_offsets[num_rows], num_nz );

    for (int r = 0; r < num_rows; r++)
    {
        int start = A.row_offsets[r];
        int end = A.row_offsets[r + 1];
        UNITTEST_ASSERT_TRUE( (end <= num_nz) && (end >= 0) );
        UNITTEST_ASSERT_TRUE( (start <= num_nz) && (start >= 0) );

        if (!A.hasProps(DIAG))
        {
            UNITTEST_ASSERT_TRUE( start != end );    // at least one diagonal non-zero
        }

        for (int j = start; j < end; j++)
        {
            UNITTEST_ASSERT_TRUE( (A.col_indices[j] >= 0) && (A.col_indices[j] < num_rows) );

            if (j != start)
            {
                UNITTEST_ASSERT_TRUE( A.col_indices[j] > A.col_indices[j - 1] );
            }

            if ((j == r) && !A.hasProps(DIAG))
            {
                UNITTEST_ASSERT_EQUAL( A.diag[r], j );
            }
        }
    }
}

void run()
{
    Matrix_h A;
    Vector_h b;
    randomize(83);
    generateMatrixRandomStruct<TConfig>::generate(A, 10000, false, max(rand() % 10, 1), false);
    checkMatrix(A);
    generateMatrixRandomStruct<TConfig>::generate(A, 10000, true, max(rand() % 10, 1), false);
    checkMatrix(A);
    generateMatrixRandomStruct<TConfig>::generate(A, 10000, false, max(rand() % 10, 1), true);
    checkMatrix(A);
    generateMatrixRandomStruct<TConfig>::generate(A, 10000, true, max(rand() % 10, 1), true);
    checkMatrix(A);
}

DECLARE_UNITTEST_END(RandomMatrix);

RandomMatrix <TemplateMode<AMGX_mode_hDDI>::Type>  RandomMatrix_hDDI;

} //namespace amgx
