#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "testframework.h"
#include <iostream>
#include <string>
#include <vector>

#include "core.h"
#include "amg_solver.h"

namespace amgx
{

bool UnitTest::amgx_intialized = false;

UnitTest::UnitTest(const char *name, AMGX_Mode mode, const char *kwords) : _name(name), _failed(false), _forge_ahead(false), _last_seed(0), _failed_bit(false), _mode(mode)
{
    if (kwords != NULL)
    {
        add_keywords(std::string(kwords));
    }

    _assert_ss.clear();
    UnitTestDriverFramework::framework().register_test(this);
}

void UnitTest::assert_finite(double a, const char *filename, int lineno)
{
    if (isnan(a))
    {
        _assert_ss << "[ASSERT] " << " in test " << _name.c_str() << " at " << filename << " lineno: " << lineno << " last rand seed: " << _last_seed << std::endl;

        if (_err_stream.str().length()) { _assert_ss <<  _err_stream.str(); }

        set_failed();

        if (!_forge_ahead)
        {
            throw UnitTestFailedException();
        }
    }

    reset_err_buffer();
}

std::string UnitTest::base_keywords()
{
    return "default";
}

std::string UnitTest::custom_launch_line()
{
    return "";
}

void UnitTest::assert_true( bool a, const char *filename, int lineno)
{
    if (!a)
    {
        _assert_ss << "[ASSERT] " << " in test " << _name.c_str() << " at " << filename << " lineno: " << lineno << " last rand seed: " << _last_seed << std::endl;

        if (_err_stream.str().length()) { _assert_ss <<  _err_stream.str(); }

        set_failed();

        if (!_forge_ahead)
        {
            throw UnitTestFailedException();
        }
    }

    reset_err_buffer();
}

void UnitTest::assert_never( const char *filename, int lineno)
{
    _assert_ss << "[ASSERT] " << " in test " << _name.c_str() << " at " << filename << " lineno: " << lineno << " last rand seed: " << _last_seed << std::endl;

    if (_err_stream.str().length()) { _assert_ss <<  _err_stream.str(); }

    set_failed();

    if (!_forge_ahead)
    {
        throw UnitTestFailedException();
    }

    reset_err_buffer();
}

int UnitTest::start_test()
{
    {
        randomize( 1 );

        try
        {
            reset_err_buffer();
            start();

            for (int i = 0; i < get_configuration().repeats; ++i)
            {
                run();
                hipDeviceSynchronize();
                cudaCheckError();
            }

            end();
            hipDeviceSynchronize();
            cudaCheckError();
        }
        catch (UnitTestFailedException &e)
        {
            set_failed();
        }
        catch (amgx_exception &e)
        {
            _assert_ss << "[EXCEPTION] in test " << _name  << " :Caught amgx exception " << e.what() << " at " << e.where();
            set_failed();
        }
        catch (thrust::system_error &e)
        {
            _assert_ss << "[EXCEPTION] in test " << _name << ": Thrust failure: " << std::string(e.what());
            set_failed();
        }
        catch (thrust::system::detail::bad_alloc &e)
        {
            _assert_ss << "[EXCEPTION] in test " << _name << ": Not enough memory for thrust call: " << std::string(e.what());
            set_failed();
        }
        catch (std::bad_alloc &e)
        {
            _assert_ss << "[EXCEPTION] in test " << _name << ": Not enough memory: " << std::string(e.what());
            set_failed();
        }
        catch (std::exception &e)
        {
            _assert_ss << "[EXCEPTION] in test " << _name << ": Unknown exception: " << std::string(e.what());
            set_failed();
        }
        catch (...)
        {
            std::stringstream ss;
            _assert_ss << "[EXCEPTION] in test " << _name << " :Caught unhandled exception";
            set_failed();
        }
    }

    if (_failed)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

void UnitTest::randomize(int seed)
{
    if (seed == -1)
    {
        if (get_configuration().random_seed == -1 )
        {
            _last_seed = time(NULL);
        }
        else
        {
            _last_seed = get_configuration().random_seed;
        }
    }
    else
    {
        _last_seed = seed;
    }

    srand(_last_seed);
}

UnitTestConfiguration &UnitTest::get_configuration()
{
    static UnitTestConfiguration cfg;
    return cfg;
}

} // end namespace
